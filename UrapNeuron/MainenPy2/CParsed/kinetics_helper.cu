#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.h"

__device__ float calc_determinant(float mat[NSTATES-1][NSTATES-1], int n) {
	//Only if this matrix is of dimension 1x1
	if (n == 1) {
		return mat[0][0];
	}

	float L[n][n];
	float U[n][n];
	float sum = 0;

	for (int i = 0; i < n; i++) {
		U[i][i] = 1;
	}

	for (int j = 0; j < n; j++) {
		for (int i = j; i < n; i++) {
			sum = 0;
			for (int k = 0; k < j; k++) {
				sum += L[i][k] * U[k][j];
			}
			L[i][j] = mat[i][j] - sum;
		}

		for (int i = j; i < n; i++) {
			sum = 0;
			for (int k = 0; k < j; k++) { sum += (L[j][k] * U[k][i]); }
			if (L[j][j] == 0) {
				//This is bad. bad things will happen. 
				return -1.0;
			}
			U[j][i] = (mat[j][i] - sum) / L[j][j];
		}
	}
	float det = 1.0;
	for (int i = 0; i < n; i++) {
		det = det * U[i][i] * L[i][i];
	}
	return det;
}


__device__ void init_state_probs(float q[NSTATES][NSTATES], float y[NSTATES]) {
	float sum = 0;
	for (int i = 0; i < NSTATES; i++) {
		y[i] = calc_prob(q, i);
		sum += y[i];
	}
	for (int i = 0; i < NSTATES; i++) {
		y[i] /= sum;
	}
}

__device__ float calc_prob(float q[NSTATES][NSTATES], int skip) {
	int i = 0;
	int j = 0;
	int temp_i_index = 0;
	int temp_j_index = 0;
	int n = NSTATES;  

	/*for the general nxn case*/
	float temp[NSTATES-1][NSTATES-1]; //dynamically generated--should be 1 less than the dimension of q
	while (i < n) {
		if (i == skip) {
			i++;
			continue;
		}
		j = 0;
		temp_j_index = 0;
		while (j < n) {
			if (j == skip) {
				j++;
				continue;
			}
			temp[temp_i_index][temp_j_index] = q[i][j];
			j += 1;
			temp_j_index += 1;
		}
		temp_i_index += 1;
		i++;
		printf("i = %d\n", i);
		printf("j = %d\n", j);
	}
	return calc_determinant(temp, n-1);
}

__device__ float rhs(float q[NSTATES][NSTATES], int index, float y[NSTATES]){
	float yout = 0;
	for (int i = 0; i < NSTATES; i++) {
		yout = yout + q[i][index] * y[i];
	}
	return yout;
}

__device__ void Cubackwards_euler(double dt, int N, int nkinStates,float y[NSTATES],float matq[NSTATES][NSTATES]){
	double h = dt / N;
  for (int i = 0; i < nkinStates; i++) {
       double w0 = y[i];
	  
       for (int j = 0; j < N; j++) {
		   double top = (w0 - y[i] - h * rhs(matq, i, y));
		   double bottom = (1 - h * matq[i][i]);
		   double dw = top / bottom;
            w0 = w0 - dw;
       }
       y[i] = w0;
  }
}
