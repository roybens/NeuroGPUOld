#include "hip/hip_runtime.h"
// Automatically generated CU for /root/neuroGPU/UrapNeuron/MainenPy3./runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)


// GGlobals:
#define celsius (6.3)
#define stoprun (0.0)
#define clamp_resist (0.001)
#define secondorder (0.0)

// NGlobals:
// Reversals:
#define ek (-77.0f)

// Declarations:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:

// Procedures:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21) {

k12 = a12_CO*exp(z12_CO*v);
k21 = a21_CO*exp(-z21_CO*v);
;
};

// Inits:


__device__ void CuInitModel_CO(float v,float &c1,float &,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO){
	float y[NSTATES], q[NSTATES][NSTATES];

k12 = a12_CO*exp(z12_CO*v)
k21 = a21_CO*exp(-z21_CO*v)

q[0][1] = k12;q[1][0] = k21;
	for (int i = 0; i < NSTATES; i++) {
		float sum = 0;
		for (int j = 0; j < NSTATES; j++) {
			if (i != j) { 
				sum += q[i][j];
			}
		}
		q[i][i] = -sum;
	}

init_state_probs(q, y)
c1 = y[0];
o = y[1];
}

// Derivs:
__device__ void CuDerivModel_CO(float dt, float v,float &c1,float &,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO){
	float y[NSTATES], q[NSTATES][NSTATES];

k12 = a12_CO*exp(z12_CO*v)
k21 = a21_CO*exp(-z21_CO*v)

q[0][1] = k12;q[1][0] = k21;
	for (int i = 0; i < NSTATES; i++) {
		float sum = 0;
		for (int j = 0; j < NSTATES; j++) {
			if (i != j) { 
				sum += q[i][j];
			}
		}
		q[i][i] = -sum;
	}
y[1] = o;
y[0] = c1;

	Cubackwards_euler(dt, 1, NSTATES, y, q);
o = y[1];
c1 = y[0];
}

// Breaks:


__device__ void CuBreakpointModel_CO(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &c1,float &,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
float gk, g;
float ik;
g=gbar_CO*o;
ik=(1e-4)*g*(v-ek);
sumCurrents+= ik;
};
;
