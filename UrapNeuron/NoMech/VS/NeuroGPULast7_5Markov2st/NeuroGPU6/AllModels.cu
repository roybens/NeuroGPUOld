#include "hip/hip_runtime.h"
// Automatically generated CU for C:\Users\rben.KECK-CENTER\Documents\GitHub\NeuroGPU\URapNeuron\Markov2st\runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)

#define _RHS1(arg) rhs[arg]
#define _MATELM1(i, j) matq[i][j]


// GGlobals
#define celsius (6.30000)
#define stoprun (0.00000)
#define clamp_resist (0.00100)
#define secondorder (0.00000)

// NGlobals:
#define _RHS1(arg) rhs[arg]
#define _MATELM1(i, j) matq[i][j]

// Reversals:
#define ek (-77.00000f)


#define TRUNC 1000
// Declarations:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

//helper functions:
__device__ void make_q(float v, float a[2][2], float z[2][2], float q[2][2]) {
	float rows = 2;
	float cols = 2;

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if (i != j) {
				q[i][j] = (a[i][j] * exp(-z[i][j] * v));
				q[j][i] = (a[j][i] * exp(z[j][i] * v));
				
			}
		}
	}
	for (int i = 0; i < rows; i++) {
		float sum = 0;
		for (int j = 0; j < cols; j++) {
			if (i != j) { 
				sum += q[i][j];
			}
		}
		q[i][i] = -sum;
	}
}

__device__ float calc_determinant(float mat[1][1], int n) {
	//Only if this matrix is of dimension 1x1
	if (n == 1) {
		return mat[0][0];
	}

	float L[n][n];
	float U[n][n];
	float sum = 0;

	for (int i = 0; i < n; i++) {
		U[i][i] = 1;
	}

	for (int j = 0; j < n; j++) {
		for (int i = j; i < n; i++) {
			sum = 0;
			for (int k = 0; k < j; k++) {
				sum += L[i][k] * U[k][j];
			}
			L[i][j] = mat[i][j] - sum;
		}

		for (int i = j; i < n; i++) {
			sum = 0;
			for (int k = 0; k < j; k++) { sum += (L[j][k] * U[k][i]); }
			if (L[j][j] == 0) {
				//This is bad. bad things will happen. 
				return -1.0;
			}
			U[j][i] = (mat[j][i] - sum) / L[j][j];
		}
	}
	float det = 1.0;
	for (int i = 0; i < n; i++) {
		det = det * U[i][i] * L[i][i];
	}
	return det;
}

/*Skip denotes skipping row i and column i */
__device__ float calc_prob(float q[2][2], int skip) {
	//float temp[1][1]; //these dimensions are not hardcoded, they should be generated 
	int i = 0;
	int j = 0;
	int temp_i_index = 0;
	int temp_j_index = 0;
	int n = 2; //generate this 

	/*kludge that works in the 2x2 case only
	if (skip == 0) {
	return q[1][1];
	}
	return q[0][0];
	*/

	/*for the general nxn case*/
	float temp[1][1]; //dynamically generated--should be 1 less than the dimension of q
	while (i < n) {
		if (i == skip) {
			i++;
			continue;
		}
		j = 0;
		temp_j_index = 0;
		while (j < n) {
			if (j == skip) {
				j++;
				continue;
			}
			temp[temp_i_index][temp_j_index] = q[i][j];
			j += 1;
			temp_j_index += 1;
		}
		temp_i_index += 1;
		i++;
		printf("i = %d\n", i);
		printf("j = %d\n", j);
	}
	return calc_determinant(temp, n-1);
}

__device__ void init_state_probs(float q[2][2], float y[2]) {
	float sum = 0;
	for (int i = 0; i < 2; i++) {
		y[i] = calc_prob(q, i);
		sum += y[i];
	}
	for (int i = 0; i < 2; i++) {
		y[i] /= sum;
	}
}


__device__ float rhs(float q[2][2], int index, float y[2]){
	float yout = 0;
	for (int i = 0; i < 2; i++) {
		yout = yout + q[i][index] * y[i];
	}
	return yout;
}

// Kinetic Code:
__device__ void Cubackwards_euler(double dt, int N, int nkinStates,float y[2],float matq[2][2]){
	double h = dt / N;
  for (int i = 0; i < nkinStates; i++) {
       double w0 = y[i];
	  
       for (int j = 0; j < N; j++) {
		   double top = (w0 - y[i] - h * rhs(matq, i, y));
		   double bottom = (1 - h * matq[i][i]);
		   double dw = top / bottom;
            w0 = w0 - dw;
       }
       y[i] = w0;
  }
}

__device__ void dydt(float y[2], float dydx[2], float matq[2][2], int n) {
	for (int i = 0; i < n; i++) {
		float sum = 0;
		for (int j = 0; j < n; j++) {
			sum += matq[i][j] * y[j];
		}
		dydx[i] = sum;
	}
}

__device__ void Cubackwards_euler2(int numLoops, double h, float y[2], int numStates, float matq[2][2]) {
	float w0[2], dydx[2];
	float top, bot, dw;

	for (int i = 0; i < numStates; i++) {
		w0[i] = y[i];
	
		for (int j = 0; j < numLoops; j++) {
				dydt(w0, dydx, matq, 2);
				for (int state = 0; state < numStates; state++) {
					top = (w0[state] - y[state]) - h * dydx[state];
					bot = 1 - h * matq[state][state];
					dw = top / bot;
					w0[state] = w0[state] - dw;
				}
			}
		}
	for (int i = 0; i < numStates; i++) {
		y[i] = w0[i];
	}
}

__device__ void scalar_multiply(float vec[2], float scalar, float out_vec[2]) {
	for (int i = 0; i < 2; i++) {
		out_vec[i] = scalar * vec[i];
	}
}

__device__ void matrix_multiply(float mat[2][2], float vec[2], float out_vec[2]) {
	float debug = mat[1][0];
	float debug2 = mat[1][1];
	for (int i = 0; i < 2; i++) {
		float sum = 0; 
		for(int j = 0; j < 2; j++) {
			sum += mat[i][j] * vec[j]; 
		}
		out_vec[i] = sum;
	}
}

__device__ void add(float a[2], float b[2], float c[2]) {
	for (int i = 0; i < 2; i++) {
		c[i] = a[i] + b[i];
	}
}
__device__ void CuRK4(float h, float y[2], float matq[2][2]) {
	float yout[2], k1[2], k2[2], k3[2], k4[2];
	float k1_temp[2], k2_temp[2], k3_temp[2];
	
	matrix_multiply(matq, y, k1_temp);
	scalar_multiply(k1_temp, h, k1);

	scalar_multiply(k1, 0.5, k1_temp);
	add(y, k1_temp, yout);
	matrix_multiply(matq, yout, k2_temp);
	scalar_multiply(k2_temp, h, k2);

	scalar_multiply(k2, 0.5, k2_temp);
	add(y, k2_temp, yout);
	matrix_multiply(matq, yout, k3);

	add(y, k3, yout);
	matrix_multiply(matq, yout, k4);

	add(k2, k3, k3_temp);
	scalar_multiply(k3_temp, 2, k2_temp);
	add(k2_temp, k4, k1_temp);
	add(k1_temp, k1, k2_temp);
	scalar_multiply(k2_temp, 1 / 6.0, k1_temp);

	add(y, k1_temp, yout);
	float sum = 0; 
	for (int i = 0; i < 2; i++) {
		sum += yout[i];
	}
	scalar_multiply(yout, 1 / sum, y); //attempt to normalize 
}

// Functions:

// Procedures:
__device__ void Curates_CO(float v,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO,float &k12,float &k21) {
      k12 = a12_CO*exp(z12_CO*v);
      k21 = a21_CO*exp(-z21_CO*v);
}


//Derivs: 
__device__ int CuDerivModel_CO(float t, float dt, float v, float &c1, float &o, float gbar_CO, float a12_CO, float a21_CO, float z12_CO, float z21_CO) {
	float y[2], q[2][2], a[2][2], z[2][2];

	a[0][1] = a12_CO;
	a[1][0] = a21_CO;
	z[0][1] = z12_CO;
	z[1][0] = z21_CO;
	//a[0][1] = 0.05; 
	//a[1][0] = 0.05;
	//z[0][1] = 0.05;
	//z[1][0] = 0.05; 

	/*Voltage Clamp*/
//	if (t > 500) {
//		v = 30; 
//	}
//	else {
//	if (t > 63)
//		v = 100;
//	else
	//	v = -100;
//	}
	printf("v = %f. t = %f", v, t);
	make_q(v, a, z, q);

	y[0] = c1;
	y[1] = o;

	//ITERATION:
	Cubackwards_euler(dt, 1, 2, y, q);
	//CuRK4(dt, y, q);
	//Cubackwards_euler2(10, dt, y, 2, q);
	printf(" probs = %f, %f\n",y[0], y[1]);
	c1 = y[0];
	o = y[1];
}

__device__ void CuInitModel_CO(float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
	float k12,k21;
	double sum = 0;
	float temp[2];
	float tempq[2][2];
	float a[2][2], z[2][2];

	
	a[0][1] = a12_CO;
	a[1][0] = a21_CO;
	z[0][1] = z12_CO;
	z[1][0] = z21_CO;
	make_q(v, a, z, tempq);

	init_state_probs(tempq, temp);
	c1 = temp[0];
	o = temp[1];
}




// Breakpoints:
__device__ void CuBreakpointModel_CO(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &c1,float &o,float gbar_CO,float a12_CO,float a21_CO,float z12_CO,float z21_CO) {
float g,gk;
float ik;
g=gbar_CO*o;
ik=(1e-4)*g*(v-ek);
sumCurrents+= ik;
;};
