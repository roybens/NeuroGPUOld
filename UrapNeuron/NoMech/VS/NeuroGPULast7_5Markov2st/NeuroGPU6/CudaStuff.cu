#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "AllModels.cu"
#include "AllModels.cuh"

__constant__ MYFTYPE cCm[NSEG];
__constant__ MYSECONDFTYPE cE[NSEG];
__constant__ MYSECONDFTYPE cF[NSEG];
__constant__ MYDTYPE cFIdxs[NSEG*LOG_N_DEPTH];
__constant__ MYDTYPE cKs[NSEG];
__constant__ MYDTYPE cSegToComp[NSEG];
__constant__ MYDTYPE cBoolModel[NSEG * N_MODELS];//One day change this to bool
__constant__ MYDTYPE cRelStarts[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelEnds[N_FATHERS];//nFathers
__constant__ MYDTYPE cFathers[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelVec[N_CALL_FOR_FATHER];//nCallForFather
__constant__ MYDTYPE cSegStartI[N_CALL_FOR_FATHER+1];//nCallForFather
__constant__ MYDTYPE cSegEndI[N_CALL_FOR_FATHER+1];//nCallForFather

__constant__ MYDTYPE cCompByLevel32[COMP_DEPTH*WARPSIZE];//CompDepth
__constant__ MYDTYPE cCompByFLevel32[COMP_DEPTH*WARPSIZE];//CompFDepth
__constant__ MYDTYPE cLRelStarts[N_L_REL];//nLRel
__constant__ MYDTYPE cLRelEnds[N_L_REL];//nLRel
__constant__ MYDTYPE cFLRelStarts[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cFLRelEnds[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cSonNoVec[NSEG];//InMat.N 


#ifdef ILP1
	#define SUPERILPMACRO(x) ; ## x ## (1);
#endif
#ifdef ILP2
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2);
#endif
#ifdef ILP3
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3);
#endif
#ifdef ILP4
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); 
#endif
#ifdef ILP5
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); 
#endif
#ifdef ILP6
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6);
#endif
#ifdef ILP7
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7);
#endif
#ifdef ILP8
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8);
#endif
#ifdef ILP9
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9);
#endif
#ifdef ILP10
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10);
#endif
#ifdef ILP11
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11);
#endif
#ifdef ILP12
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12);
#endif
#ifdef ILP13
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13);
#endif
#ifdef ILP14
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14);
#endif
#ifdef ILP15
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15);
#endif
#ifdef ILP16
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16);
#endif
#ifdef ILP17
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17);
#endif
#ifdef ILP18
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18);
#endif
#ifdef ILP19
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19);
#endif
#ifdef ILP20
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20);
#endif
#ifdef ILP21
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21);
#endif
#ifdef ILP22
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22);
#endif
#ifdef ILP23
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23);
#endif
#ifdef ILP24
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24);
#endif
#ifdef ILP25
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25);
#endif
#ifdef ILP26
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26);
#endif
#ifdef ILP27
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27);
#endif
#ifdef ILP28
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28);
#endif
#ifdef ILP29
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29);
#endif
#ifdef ILP30
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30);
#endif
#ifdef ILP31
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31);
#endif
#ifdef ILP32
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); 
#endif
#ifdef ILP33
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33);
#endif
#ifdef ILP34
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34);
#endif
#ifdef ILP35
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35);
#endif
#ifdef ILP36
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36);
#endif
#ifdef ILP37
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37);
#endif
#ifdef ILP38
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38);
#endif
#ifdef ILP39
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39);
#endif
#ifdef ILP40
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40);
#endif
#ifdef ILP41
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41);
#endif
#ifdef ILP42
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42);
#endif
#ifdef ILP43
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43);
#endif
#ifdef ILP44
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44);
#endif
#ifdef ILP45
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45);
#endif
#ifdef ILP46
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46);
#endif
#ifdef ILP47
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47);
#endif
#ifdef ILP48
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48);
#endif
#ifdef ILP49
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49);
#endif
#ifdef ILP50
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49); ## x ## (50);
#endif

__device__ void BeforeLU(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel,LRelIndex;
	MYDTYPE JumctionI;
	LRelIndex=cLRelStarts[CurLevel];
	LRelIndex=LRelIndex+cLRelEnds[CurLevel];
	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		
		for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
		//for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx]-1;
			for(i=cSegStartI[JumctionI]-1;i<cSegEndI[JumctionI];i++) {
				MYSECONDFTYPE uHPm1=uHP[i-1];
			
				uHP[i]=uHP[i]-cF[i-1]*(cE[i-1]/uHPm1); // So far same as paper parallel
				uHPm1=uHP[i-1];
				MYSECONDFTYPE bHPm1=bHP[i-1];
				bHP[i]=bHP[i]-bHPm1*cE[i-1]/uHPm1; // bH is y
			}
		}
		if(CurLevel<Depth) {
			for(LRelIndex=cFLRelStarts[CurLevel];LRelIndex<=cFLRelEnds[CurLevel];LRelIndex++){ 
				CurB=cCompByFLevel32[(LRelIndex)*WARPSIZE+PIdx]-1;//RB i inserted another  -1 into the index RB 2 i removed the-1 from the curlevel 
				CurJ=cFathers[CurB]-1;
				MYDTYPE St=cRelStarts[CurB];
				MYDTYPE En=cRelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=cRelVec[j-1]-1;
					MYSECONDFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]-=cF[t-1]*(cE[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYSECONDFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]-=bHPm1*cE[t-1]/uHPm1; 
				}
			}
		}	
	}
}

#ifdef BKSUB1
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* PX, MYSECONDFTYPE* PF,MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE LognDepth)
{
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
#define THISCOMMANDHERE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*(## VARILP -1)); MYDTYPE NextID_ ## VARILP ;
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); MYDTYPE NextID_1 ;MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); MYDTYPE NextID_2 ;
	MYDTYPE i;
	
	PX=bHP;
	PF=uHP;
	// PX[PIdx_1]=PX[PIdx_1]/PF[PIdx_1]; // PF[PIdx_1]=-cF[PIdx_1]/PF[PIdx_1];
#define THISCOMMANDHERE2(VARILP) PX[PIdx_ ## VARILP ]=PX[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ]; PF[PIdx_ ## VARILP ]=-cF[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ];
	PX[PIdx_1 ]=PX[PIdx_1 ]/PF[PIdx_1 ]; PF[PIdx_1 ]=-cF[PIdx_1 ]/PF[PIdx_1 ];PX[PIdx_2 ]=PX[PIdx_2 ]/PF[PIdx_2 ]; PF[PIdx_2 ]=-cF[PIdx_2 ]/PF[PIdx_2 ];

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		// NextID_1=cFIdxs[i*InMat.N+PIdx_1]-1; // MYFTYPE OldPXj_1=PX[PIdx_1]; // MYFTYPE OldPXNextID_1=PX[NextID_1]; // PX[PIdx_1]=OldPXj_1+OldPXNextID_1*PF[PIdx_1];
#define THISCOMMANDHERE3(VARILP) NextID_ ## VARILP =cFIdxs[i*InMat.N+PIdx_ ## VARILP ]-1; MYFTYPE OldPXj_ ## VARILP =PX[PIdx_ ## VARILP ]; MYFTYPE OldPXNextID_ ## VARILP =PX[NextID_ ## VARILP ]; PX[PIdx_ ## VARILP ]=OldPXj_ ## VARILP +OldPXNextID_ ## VARILP *PF[PIdx_ ## VARILP ];
	NextID_1 =cFIdxs[i*InMat.N+PIdx_1 ]-1; MYFTYPE OldPXj_1 =PX[PIdx_1 ]; MYFTYPE OldPXNextID_1 =PX[NextID_1 ]; PX[PIdx_1 ]=OldPXj_1 +OldPXNextID_1 *PF[PIdx_1 ];NextID_2 =cFIdxs[i*InMat.N+PIdx_2 ]-1; MYFTYPE OldPXj_2 =PX[PIdx_2 ]; MYFTYPE OldPXNextID_2 =PX[NextID_2 ]; PX[PIdx_2 ]=OldPXj_2 +OldPXNextID_2 *PF[PIdx_2 ];

		// PX[j]=PX[j]+PX[NextID]*PF[j];
		// PF[j]=PF[j]*PF[NextID];
		// MYFTYPE OldPFj_1=PF[PIdx_1]; // MYFTYPE OldPFNextID_1=PF[NextID_1]; // PF[PIdx_1]=OldPFj_1*OldPFNextID_1;
#define THISCOMMANDHERE4(VARILP) MYFTYPE OldPFj_ ## VARILP =PF[PIdx_ ## VARILP ]; MYFTYPE OldPFNextID_ ## VARILP =PF[NextID_ ## VARILP ]; PF[PIdx_ ## VARILP ]=OldPFj_ ## VARILP *OldPFNextID_ ## VARILP ;
	MYFTYPE OldPFj_1 =PF[PIdx_1 ]; MYFTYPE OldPFNextID_1 =PF[NextID_1 ]; PF[PIdx_1 ]=OldPFj_1 *OldPFNextID_1 ;MYFTYPE OldPFj_2 =PF[PIdx_2 ]; MYFTYPE OldPFNextID_2 =PF[NextID_2 ]; PF[PIdx_2 ]=OldPFj_2 *OldPFNextID_2 ;
	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP,MYSECONDFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); Out[PIdx_1 ]=0;MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); Out[PIdx_2 ]=0;

	MYDTYPE j,CurJ,CurB,t;
	MYDTYPE JumctionI;
	short CurLevel,i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex,k;
	MYFTYPE temp;
 for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
  //     Run all independent set for this level, in parallel
  // for JumctionI=find(Level==CurLevel) % in parallel
  for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
   JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx_1]-1;
   // for i=(cSegEndI(JumctionI)):-1:(cSegStartI(JumctionI)-1)
   for(i=cSegEndI[JumctionI]-1;i>=(cSegStartI[JumctionI]-2);i--) {
    // k=cKsB(i+1);
     k=cKsB[i+1];
	 
    // bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
   Out[i]=(bHP[i]-Out[k]*cF[i])/uHP[i];
   }
  }
 }
 } 
#endif

__device__ void runSimulation(HMat InMat,MYFTYPE* ParamsM, MYFTYPE* V,Stim stim,Sim sim,MYFTYPE* VHotGlobal,MYFTYPE* SMemVHot,MYFTYPE* amps,MYDTYPE offset)
{


	MYSECONDFTYPE *uHP,*bHP;
	MYDTYPE StimID=threadIdx.y;
		
#ifdef BKSUB2
    MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(InMat.N+2*WARPSIZE)*sizeof(MYFTYPE);
#endif
#ifdef BKSUB1

  MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(WARPSIZE + sim.NRecSites*(WARPSIZE))*sizeof(MYFTYPE);
  PerStimulus =ceilf(MYSECONDFTYPE(PerStimulus)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE); 
#endif
	//MYDTYPE PerStimulus;
	//PerStimulus = InMat.N+2;
	offset+=PerStimulus*threadIdx.y;
	uHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);
	//uHP[InMat.N+2] =0;
	bHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);
	//bHP[InMat.N+2] =0;
	offset = ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	//Vs = (MYFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYFTYPE);
	// Vs[PIdx_3]=V[PIdx_3];
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=(WARPSIZE)*sim.NRecSites*sizeof(MYFTYPE);
	SMemVHot[WARPSIZE]=0;
    amps = (MYFTYPE*) &smem[offset];
    offset+=WARPSIZE*sizeof(MYFTYPE);
	MYDTYPE NeuronID = blockIdx.x;
	int Nt=stim.Nt;
	MYFTYPE t=0;
	MYSECONDFTYPE *PX,*PF;
	PX=bHP;
	PF=uHP;
	#define THISCOMMANDHERE18(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1));
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1));MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1));

	 #define THISCOMMANDHERE19(VARILP) MYFTYPE isModel_ ## VARILP , dVec_ ## VARILP , Vmid_ ## VARILP ; MYFTYPE ModelStates_ ## VARILP [NSTATES]; MYFTYPE v_ ## VARILP =V[PIdx_ ## VARILP ]; MYFTYPE dv_ ## VARILP;
	MYFTYPE isModel_1 , dVec_1 , Vmid_1 ; MYFTYPE ModelStates_1 [NSTATES]; MYFTYPE v_1 =V[PIdx_1 ]; MYFTYPE dv_1;MYFTYPE isModel_2 , dVec_2 , Vmid_2 ; MYFTYPE ModelStates_2 [NSTATES]; MYFTYPE v_2 =V[PIdx_2 ]; MYFTYPE dv_2;

	 #define THISCOMMANDHERE20(VARILP) MYSECONDFTYPE sumCurrents_ ## VARILP=0 ,sumCurrentsDv_ ## VARILP=0; MYFTYPE  sumConductivity_ ## VARILP=0 ,sumConductivityDv_ ## VARILP=0 ;
	MYSECONDFTYPE sumCurrents_1=0 ,sumCurrentsDv_1=0; MYFTYPE  sumConductivity_1=0 ,sumConductivityDv_1=0 ;MYSECONDFTYPE sumCurrents_2=0 ,sumCurrentsDv_2=0; MYFTYPE  sumConductivity_2=0 ,sumConductivityDv_2=0 ;
	// end for model in init
	ModelStates_1 [0]=0;ModelStates_1 [1]=0;;ModelStates_2 [0]=0;ModelStates_2 [1]=0;;
	#define THISCOMMANDHERE21(VARILP) MYDTYPE SonNo_ ## VARILP =cSonNoVec[PIdx_ ## VARILP ];
	MYDTYPE SonNo_1 =cSonNoVec[PIdx_1 ];MYDTYPE SonNo_2 =cSonNoVec[PIdx_2 ];
	
	#define THISCOMMANDHERE22(VARILP) bHP[PIdx_ ## VARILP]=0;dv_ ## VARILP =0;
	bHP[PIdx_1]=0;dv_1 =0;bHP[PIdx_2]=0;dv_2 =0;


	#define THISCOMMANDHERE23(VARILP) MYDTYPE parentIndex_ ## VARILP ; MYDTYPE Eidx_ ## VARILP ; Eidx_ ## VARILP = InMat.N-PIdx_ ## VARILP -1;
	MYDTYPE parentIndex_1 ; MYDTYPE Eidx_1 ; Eidx_1 = InMat.N-PIdx_1 -1;MYDTYPE parentIndex_2 ; MYDTYPE Eidx_2 ; Eidx_2 = InMat.N-PIdx_2 -1;
	// RRRXXX This probably should be in ILP too!
	if(Eidx_1>InMat.N-1){
		Eidx_1=InMat.N-1;
	}
	MYDTYPE perThreadParamMSize = InMat.NComps*NPARAMS;
#define param_macro(paramInd,segmentInd) ParamsM[NeuronID*perThreadParamMSize + paramInd*InMat.NComps+cSegToComp[segmentInd] ]
	
	  if(cBoolModel[PIdx_1 +0*InMat.N]){CuInitModel_CO(v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*InMat.N]){CuInitModel_CO(v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) );}
	
		// parentIndex_3 =InMat.N-InMat.Ks[InMat.N-PIdx_3];
#define THISCOMMANDHERE24(VARILP) parentIndex_ ## VARILP =InMat.N-cKs[InMat.N-PIdx_ ## VARILP ];
	parentIndex_1 =InMat.N-cKs[InMat.N-PIdx_1 ];parentIndex_2 =InMat.N-cKs[InMat.N-PIdx_2 ];
	MYDTYPE stimLoc = stim.loc;
    MYFTYPE stimArea = stim.area;
    MYDTYPE dtCounter=0;
    MYFTYPE  dt = sim.dt;
#ifndef STIMFROMFILE
    MYFTYPE stimDel = stim.dels[StimID];//Should be constant
    MYFTYPE stimDur = stim.durs[StimID];//Should be constant
    MYFTYPE stimAmp = stim.amps[StimID];
#endif // !STIMFROMFILE
	// MYFTYPE rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#define THISCOMMANDHERE25(VARILP) MYSECONDFTYPE rhs_ ## VARILP ,D_ ## VARILP; MYFTYPE gModel_ ## VARILP ,StimCurrent_ ## VARILP;
	MYSECONDFTYPE rhs_1 ,D_1; MYFTYPE gModel_1 ,StimCurrent_1;MYSECONDFTYPE rhs_2 ,D_2; MYFTYPE gModel_2 ,StimCurrent_2;

for(int i=0;i<Nt;i++) {
        if(i==stim.dtInds[dtCounter]){
                dt = stim.durs[dtCounter];
                if (dtCounter != stim.numofdts-1){
                    dtCounter++;
                }
        }
        t+=0.5*dt;
 
       if((i%(WARPSIZE)==0)){
            if(i>0){
               for(int recInd =0;recInd<sim.NRecSites;recInd++){
					VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + (i-WARPSIZE)+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
				}
            }
			
            amps[PIdx_1]=stim.amps[threadIdx.y*Nt+i+PIdx_1];
        }
        for(int recInd =0;recInd<sim.NRecSites;recInd++){
			if(sim.RecSites[recInd]%WARPSIZE == threadIdx.x)
				//This is done by all threads why???
				SMemVHot[recInd*WARPSIZE+i%(WARPSIZE)]=v_1;//This is going to be challenging to make it general but possible.
			
		}
		 // rhs_3=0; // D_3=0;// StimCurrent_3=0;
#define THISCOMMANDHERE26(VARILP) rhs_ ## VARILP =0; D_ ## VARILP =0; sumCurrents_ ## VARILP=0; sumConductivity_ ## VARILP=0; sumCurrentsDv_ ## VARILP=0; sumConductivityDv_ ## VARILP=0;StimCurrent_ ## VARILP =0;
	rhs_1 =0; D_1 =0; sumCurrents_1=0; sumConductivity_1=0; sumCurrentsDv_1=0; sumConductivityDv_1=0;StimCurrent_1 =0;rhs_2 =0; D_2 =0; sumCurrents_2=0; sumConductivity_2=0; sumCurrentsDv_2=0; sumConductivityDv_2=0;StimCurrent_2 =0;
		// Before matrix
		// if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#ifndef STIMFROMFILE
    #define THISCOMMANDHERE27(VARILP) if(t>=stimDel && t<stimDel+stimDur && PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP  = 100*stimAmp/stimArea;}
	if(t>=stimDel && t<stimDel+stimDur && PIdx_1  == stimLoc){StimCurrent_1  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_2  == stimLoc){StimCurrent_2  = 100*stimAmp/stimArea;}
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
    
    #define THISCOMMANDHERE28(VARILP) if(PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP = 100*amps[i%WARPSIZE]/stimArea;}
	if(PIdx_1  == stimLoc){StimCurrent_1 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_2  == stimLoc){StimCurrent_2 = 100*amps[i%WARPSIZE]/stimArea;}
#endif // STIMFROMFILE
	 if(cBoolModel[PIdx_1 +0*InMat.N]){CuBreakpointModel_CO(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) );} if(cBoolModel[PIdx_2 +0*InMat.N]){CuBreakpointModel_CO(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) );}
	 if(cBoolModel[PIdx_1 +0*InMat.N]){CuBreakpointModel_CO(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) );} if(cBoolModel[PIdx_2 +0*InMat.N]){CuBreakpointModel_CO(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) );}

	// gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; 
#define THISCOMMANDHERE29(VARILP) gModel_ ## VARILP  = (sumCurrentsDv_ ## VARILP -sumCurrents_ ## VARILP )/EPS_V;  
	gModel_1  = (sumCurrentsDv_1 -sumCurrents_1 )/EPS_V;  gModel_2  = (sumCurrentsDv_2 -sumCurrents_2 )/EPS_V;  
	
#define THISCOMMANDHERE30(VARILP) rhs_ ## VARILP =StimCurrent_ ## VARILP -sumCurrents_ ## VARILP ;
	rhs_1 =StimCurrent_1 -sumCurrents_1 ;rhs_2 =StimCurrent_2 -sumCurrents_2 ;
#define THISCOMMANDHERE31(VARILP) D_ ## VARILP =gModel_ ## VARILP +cCm[PIdx_ ## VARILP ]/(dt*1000); D_ ## VARILP -=cF[InMat.N-PIdx_ ## VARILP -1];
	D_1 =gModel_1 +cCm[PIdx_1 ]/(dt*1000); D_1 -=cF[InMat.N-PIdx_1 -1];D_2 =gModel_2 +cCm[PIdx_2 ]/(dt*1000); D_2 -=cF[InMat.N-PIdx_2 -1];
   
    //this should be out and correct cKs
	if(PIdx_1==0) {
		parentIndex_1=0;
	};
#define THISCOMMANDHERE32(VARILP)  dv_ ## VARILP +=bHP[InMat.N-parentIndex_ ## VARILP -1]-bHP[InMat.N-PIdx_ ## VARILP -1]; rhs_ ## VARILP -=cF[InMat.N-PIdx_ ## VARILP -1]*dv_ ## VARILP ;
	 dv_1 +=bHP[InMat.N-parentIndex_1 -1]-bHP[InMat.N-PIdx_1 -1]; rhs_1 -=cF[InMat.N-PIdx_1 -1]*dv_1 ; dv_2 +=bHP[InMat.N-parentIndex_2 -1]-bHP[InMat.N-PIdx_2 -1]; rhs_2 -=cF[InMat.N-PIdx_2 -1]*dv_2 ;
		// D_3=sumConductivity_3+InMat.Cms[PIdx_3]/(sim.dt*1000); // D_3-=InMat.f[InMat.N-PIdx_3-1];
		// bHP[InMat.N-PIdx_3-1]=rhs_3; // uHP[InMat.N-PIdx_3-1]=D_3;
#define THISCOMMANDHERE33(VARILP) bHP[InMat.N-PIdx_ ## VARILP -1]=rhs_ ## VARILP ; uHP[InMat.N-PIdx_ ## VARILP -1]=D_ ## VARILP ;
		bHP[InMat.N-PIdx_1 -1]=rhs_1 ; uHP[InMat.N-PIdx_1 -1]=D_1 ;bHP[InMat.N-PIdx_2 -1]=rhs_2 ; uHP[InMat.N-PIdx_2 -1]=D_2 ;
		syncthreads();
		
		// if(SonNo_3==1) {bHP[InMat.N-parentIndex_3-1]+=cE[Eidx_3]*dv_3;uHP[InMat.N-parentIndex_3-1]-=cE[Eidx_3];}
		
#define THISCOMMANDHERE34(VARILP) if(cSonNoVec[PIdx_ ## VARILP] ==1) {bHP[InMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[InMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
		if(cSonNoVec[PIdx_1] ==1) {bHP[InMat.N-parentIndex_1 -1]+=cE[Eidx_1 ]*dv_1 ;uHP[InMat.N-parentIndex_1 -1]-=cE[Eidx_1 ];}if(cSonNoVec[PIdx_2] ==1) {bHP[InMat.N-parentIndex_2 -1]+=cE[Eidx_2 ]*dv_2 ;uHP[InMat.N-parentIndex_2 -1]-=cE[Eidx_2 ];}
		// if(SonNo_3==2) {bHP[InMat.N-parentIndex_3-1]+=InMat.e[Eidx_3]*dv_3;uHP[InMat.N-parentIndex_3-1]-=InMat.e[Eidx_3];}
#define THISCOMMANDHERE35(VARILP) if(cSonNoVec[PIdx_ ## VARILP] ==2) {bHP[InMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[InMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
		if(cSonNoVec[PIdx_1] ==2) {bHP[InMat.N-parentIndex_1 -1]+=cE[Eidx_1 ]*dv_1 ;uHP[InMat.N-parentIndex_1 -1]-=cE[Eidx_1 ];}if(cSonNoVec[PIdx_2] ==2) {bHP[InMat.N-parentIndex_2 -1]+=cE[Eidx_2 ]*dv_2 ;uHP[InMat.N-parentIndex_2 -1]-=cE[Eidx_2 ];}
		syncthreads();
		BeforeLU(InMat,uHP,bHP,InMat.Depth);
#ifdef BKSUB1
		BkSub(InMat, PX,PF,uHP,bHP,InMat.LognDepth);
		#define THISCOMMANDHERE36(VARILP) Vmid_ ## VARILP =PX[InMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ; 
		Vmid_1 =PX[InMat.N-PIdx_1 -1]; v_1 +=Vmid_1 ; Vmid_2 =PX[InMat.N-PIdx_2 -1]; v_2 +=Vmid_2 ; 
#endif

		
#ifdef BKSUB2
		#define THISCOMMANDHERE37(VARILP) MYFTYPE vTemp_ ## VARILP=Vs[PIdx_ ## VARILP];
		MYFTYPE vTemp_1=Vs[PIdx_1];MYFTYPE vTemp_2=Vs[PIdx_2];
		syncthreads();
		BkSub(InMat,uHP,bHP,Vs,InMat.Depth);
		syncthreads();
		PX=Vs;

#define THISCOMMANDHERE38(VARILP) Vmid_ ## VARILP =PX[InMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ;
		Vmid_1 =PX[InMat.N-PIdx_1 -1]; v_1 +=Vmid_1 ;Vmid_2 =PX[InMat.N-PIdx_2 -1]; v_2 +=Vmid_2 ;
		syncthreads(); 
#define THISCOMMANDHERE39(VARILP) Vs[PIdx_ ## VARILP ]= vTemp_ ## VARILP +Vmid_ ## VARILP ;
		Vs[PIdx_1 ]= vTemp_1 +Vmid_1 ;Vs[PIdx_2 ]= vTemp_2 +Vmid_2 ;
#endif		
            t+=0.5*dt;

        

//      if(InMat.boolModel[PIdx_1 +0*InMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],p0_1 ,p1_1 ,ModelStates_1[8],ModelStates_1[9]);} if(InMat.boolModel[PIdx_1 +1*InMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(InMat.boolModel[PIdx_1 +2*InMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],p2_1 ,p3_1 ,p4_1 ,p5_1 ,ModelStates_1[8]);} if(InMat.boolModel[PIdx_1 +3*InMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],p6_1 ,p7_1 ,p8_1 ,p9_1 ,p10_1 );} if(InMat.boolModel[PIdx_1 +4*InMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],p11_1 ,p12_1 ,p13_1 ,p14_1 ,p15_1 );} if(InMat.boolModel[PIdx_1 +5*InMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],p16_1 ,p17_1 ,p18_1 ,p19_1 ,p20_1 ,p21_1 ,p22_1 ,p23_1 ,p24_1 ,p25_1 ,p26_1 ,p27_1 );}  if(InMat.boolModel[PIdx_2 +0*InMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],p0_2 ,p1_2 ,ModelStates_2[8],ModelStates_2[9]);} if(InMat.boolModel[PIdx_2 +1*InMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(InMat.boolModel[PIdx_2 +2*InMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],p2_2 ,p3_2 ,p4_2 ,p5_2 ,ModelStates_2[8]);} if(InMat.boolModel[PIdx_2 +3*InMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],p6_2 ,p7_2 ,p8_2 ,p9_2 ,p10_2 );} if(InMat.boolModel[PIdx_2 +4*InMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],p11_2 ,p12_2 ,p13_2 ,p14_2 ,p15_2 );} if(InMat.boolModel[PIdx_2 +5*InMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],p16_2 ,p17_2 ,p18_2 ,p19_2 ,p20_2 ,p21_2 ,p22_2 ,p23_2 ,p24_2 ,p25_2 ,p26_2 ,p27_2 );}  if(InMat.boolModel[PIdx_3 +0*InMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],p0_3 ,p1_3 ,ModelStates_3[8],ModelStates_3[9]);} if(InMat.boolModel[PIdx_3 +1*InMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(InMat.boolModel[PIdx_3 +2*InMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],p2_3 ,p3_3 ,p4_3 ,p5_3 ,ModelStates_3[8]);} if(InMat.boolModel[PIdx_3 +3*InMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],p6_3 ,p7_3 ,p8_3 ,p9_3 ,p10_3 );} if(InMat.boolModel[PIdx_3 +4*InMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],p11_3 ,p12_3 ,p13_3 ,p14_3 ,p15_3 );} if(InMat.boolModel[PIdx_3 +5*InMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],p16_3 ,p17_3 ,p18_3 ,p19_3 ,p20_3 ,p21_3 ,p22_3 ,p23_3 ,p24_3 ,p25_3 ,p26_3 ,p27_3 );}
		  if(cBoolModel[PIdx_1 +0*InMat.N]){CuDerivModel_CO(dt, v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*InMat.N]){CuDerivModel_CO(dt, v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) );}

 
 
    }
	//This one looks suspicious but leaving it and will check it later.
   for(int recInd =0;recInd<sim.NRecSites;recInd++){
		VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + Nt-WARPSIZE+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
	}
 
}
__global__ void NeuroGPUKernel(Stim stim, MYFTYPE* ParamsM, Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal,MYDTYPE CompDepth,MYDTYPE CompFDepth)
{

	MYFTYPE *amps,*SMemVHot;
	MYDTYPE offset=0;
	
	
	
	runSimulation(InMat,ParamsM,V,stim,sim,VHotGlobal,SMemVHot,amps,offset);

}
void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}
MYFTYPE* ReadAllParams(const char* FN,MYDTYPE NParams,MYDTYPE Nx,MYDTYPE  &ntemp) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	//sprintf(FileName,"%sForC.mat",FN);
	MYFTYPE* ans;
	MYDTYPE tmp;
	FILE *fl = fopen(FN, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return nullptr;
	}
	fread(&tmp, sizeof(MYDTYPE),1, fl);
		ntemp = tmp;

	ans=(MYFTYPE *)malloc(Nx * NPARAMS * tmp * sizeof(MYFTYPE ));
	fread(ans, sizeof(MYFTYPE), tmp*Nx*NParams, fl);
	fclose(fl);
	return ans;
}
void ReadParamsMat(const char* FN,MYFTYPE** ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	for(int i=0;i<NParams;i++){
		ParamsM[i]=(MYFTYPE*)malloc(Nx * sizeof(MYFTYPE));
		fread(ParamsM[i], sizeof(MYFTYPE), Nx, fl);
	}
	fclose(fl);
}


hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) { 

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(1);
	cudaStatus = hipDeviceReset();
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=stim.Nt;
	hipHostMalloc((void**)&VHotsHost,stim.NStimuli*Nt*sim.NRecSites*NSets*sizeof(MYFTYPE));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.NComps=InMat.NComps;
	Mat_d.Depth=InMat.Depth;
	Mat_d.NModels=InMat.NModels;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	Mat_d.nLRel=InMat.nLRel;
	Mat_d.nFLRel=InMat.nFLRel;
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	stim_d.Nt = stim.Nt;
	Sim sim_d; 	 			
	sim_d.Celsius = sim.Celsius; 	 	
	sim_d.dt = sim.dt; 	 	
	sim_d.NRecSites = sim.NRecSites; 	 	
	sim_d.TFinal = sim.TFinal; 
#ifndef STIMFROMFILE
    cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE

    stim_d.numofdts = stim.numofdts;
    cudaStatus = hipMalloc((void**)&stim_d.dtInds, stim_d.numofdts * sizeof(MYDTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.numofdts * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE));
#endif // STIMFROMFILE
	cudaStatus = hipMalloc((void**)&sim_d.RecSites, sim_d.NRecSites * sizeof(MYDTYPE));
	


	//cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * InMat.NModels* sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*Nt *stim.NStimuli* sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*sim_d.NRecSites*Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N +1)* sizeof(MYDTYPE));
#endif
	/*cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (CompDepth)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (CompFDepth)*WARPSIZE* sizeof(MYDTYPE));

	cudaStatus = hipMalloc((void**)&Mat_d.LRelStarts, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.LRelEnds, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelStarts, Mat_d.nFLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelEnds, Mat_d.nFLRel* sizeof(MYDTYPE));
*/
	

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	#ifndef STIMFROMFILE
 
    cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
	
    cudaStatus = hipMemcpy(stim_d.dtInds, stim.dtInds, stim.numofdts * sizeof(MYDTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.numofdts * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // STIMFROMFILE
	cudaStatus = hipMemcpy(sim_d.RecSites, sim.RecSites, sim_d.NRecSites * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cE), InMat.e, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cF), InMat.f, InMat.N * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cKs), InMat.Ks, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(Mat_d.SegToComp, InMat.SegToComp, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegToComp), InMat.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cBoolModel), InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE));

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCm), InMat.Cms, InMat.N * sizeof(MYFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSonNoVec), InMat.SonNoVec, InMat.N * sizeof(MYDTYPE));
	
	
	//cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelStarts), InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelEnds), InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelVec), InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegStartI), InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegEndI), InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFathers), InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFIdxs), InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMemcpy(Mat_d.KsB, InMat.KsB,(InMat.N+1)*sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif

	//cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByLevel32), InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByFLevel32), InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE));


	//cudaStatus = hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelStarts), InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelEnds), InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelStarts), InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelEnds), InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE));


	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.NComps *NSets * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}
	// copying
	
	cudaStatus = hipMemcpy(d_modelParams, ParamsM, NPARAMS * InMat.NComps * NSets *sizeof(MYFTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(NSets);

	MYDTYPE offset=0;
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
#ifdef BKSUB1
	//offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
#endif
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*InMat.NModels*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, SegToComp,boolModel
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	//offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	//offset+=WARPSIZE*(CompDepth)*sizeof(MYDTYPE);
	//offset+=WARPSIZE*(CompFDepth)*sizeof(MYDTYPE);
	//// 4 CompByLevel32 CompByFLevel32
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
#ifdef BKSUB2
	offset+=(Mat_d.N+1)*sizeof(MYDTYPE);
#endif
	// 5 LRelStarts LRelEnds FLRelStarts  FLRelEnds KsB
	//offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	MYDTYPE FrameworkMemSize=offset;
//PerStimulus
	printf("offset is%d\n",offset);

	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);  // uHP (Diag)
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE); // bHP (rhs)
	offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE); // Vs

	offset+=((WARPSIZE)*sim.NRecSites)*sizeof(MYFTYPE);//SMemVHot
#ifdef STIMFROMFILE
    offset+=WARPSIZE*sizeof(MYFTYPE);//amps
#endif // STIMFROMFILE
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+(PerStimulus)*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
		hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
		MYDTYPE currKernelRun = NKERNEL;
	MYDTYPE prevRuns = 0;
	int memSizeForVHotGlobal = Nt*stim.NStimuli*sim.NRecSites;
	MYDTYPE memSizeForModelParams = NPARAMS * InMat.NComps;
	MYDTYPE streamID=0;
	hipStream_t stream0,stream1,stream2,stream3;
	hipStreamCreate (&stream0);
	hipStreamCreate (&stream1);
	hipStreamCreate (&stream2);
	hipStreamCreate (&stream3);
	hipEventRecord(start, 0);
	while (prevRuns<NSets){
	if(currKernelRun>(NSets-prevRuns)){
		currKernelRun = (NSets-prevRuns);
	}
	if(streamID==0){
		NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream0>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR	

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream0); 
		
	}
	if(streamID==1){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream1>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream1); 

	}
	if(streamID==2){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream2>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt *sim.NRecSites *  stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream2); 

	}
	if(streamID==3){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream3>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream3); 

	}
	prevRuns +=currKernelRun;
	if(streamID==3){
		streamID == 0;
	}	else {
		streamID +=1;
	}
	}
	//hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time required : %f", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}
	//cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal,NSets * Nt * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost); 
	printf("stEfork2TimeLoopGPU took %f milliseconds\n",elapsedTime);
	FILE *file = fopen(TIMES_FN, "wb");
	if ( file ) {
		MYDTYPE mul32 = MUL32;
		fwrite(&mul32,sizeof(MYDTYPE),1,file);
		fwrite(&elapsedTime,sizeof(float),1,file);
	} else {
		printf("ERR SaveArrayToFile %s\n",TIMES_FN);
	}
	fclose(file);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,NSets*Nt*stim.NStimuli*sim.NRecSites,VHotsHost);
	
	return cudaStatus;
}


hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(1);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
	  stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V,CompDepth,CompFDepth,NSets); //RRR sim
	  
	
	


	return cudaStatus;
}
