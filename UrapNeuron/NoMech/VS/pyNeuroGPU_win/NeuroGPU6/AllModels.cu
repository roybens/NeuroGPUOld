#include "hip/hip_runtime.h"
// Automatically generated CU for /home/roybens/tneuorGPU/NeuroGPU/UrapNeuron/MainenPy2./runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.h"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)


// GGlobals:
#define celsius (37.0)
#define stoprun (0.0)
#define clamp_resist (0.001)
#define secondorder (0.0)

// NGlobals:
#define q10_ca (2.3)
#define temp_ca (23.0)
#define tadj_ca (3.20936395327)
#define vmin_ca (-120.0)
#define vmax_ca (100.0)
#define vshift_ca (0.0)
#define depth_cad (0.1)
#define cainf_cad (0.0001)
#define taur_cad (200.0)
#define q10_kca (2.3)
#define temp_kca (23.0)
#define tadj_kca (3.20936395327)
#define vmin_kca (-120.0)
#define vmax_kca (100.0)
#define q10_km (2.3)
#define temp_km (23.0)
#define tadj_km (3.20936395327)
#define vmin_km (-120.0)
#define vmax_km (100.0)
#define q10_kv (2.3)
#define temp_kv (23.0)
#define tadj_kv (3.20936395327)
#define vmin_kv (-120.0)
#define vmax_kv (100.0)
#define q10_na (2.3)
#define temp_na (23.0)
#define tadj_na (3.20936395327)
#define vmin_na (-120.0)
#define vmax_na (100.0)
#define vshift_na (-5.0)
// Reversals:
#define ena (60.0f)
#define ek (-90.0f)
#define eca (140.0f)

// Declarations:
__device__ void Cutrates_ca(float v ,float gbar_ca,float cao_ca,float &hinf,float &htau,float &minf,float &mtau);
__device__ void Curates_ca(float vm ,float gbar_ca,float cao_ca,float &hinf,float &htau,float &minf,float &mtau);
__device__ void Curates_kca(float cai,float gbar_kca,float caix_kca,float Ra_kca,float Rb_kca,float &a,float &b,float &ninf,float &ntau);
__device__ void Cutrates_km(float v ,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km,float &a,float &b,float &ninf,float &ntau);
__device__ void Curates_km(float v ,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km,float &a,float &b,float &ninf,float &ntau);
__device__ void Cutrates_kv(float v ,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv,float &a,float &b,float &ninf,float &ntau);
__device__ void Curates_kv(float v ,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv,float &a,float &b,float &ninf,float &ntau);
__device__ void Cutrates_na(float v,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na,float &hinf,float &htau,float &minf,float &mtau);
__device__ void Curates_na(float vm,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na,float &hinf,float &htau,float &minf,float &mtau);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:
__device__ float Cuefun_ca(float z){
	if (fabs(z) < 1e-4) {;
		return  1 - z/2;
	}else{;
		return  z/(exp(z) - 1);
	};
};
__device__ float Cuefun_km(float z){
	if (fabs(z) < 1e-4) {;
		return  1 - z/2;
	}else{;
		return  z/(exp(z) - 1);
	};
};
__device__ float Cuefun_kv(float z){
	if (fabs(z) < 1e-4) {;
		return  1 - z/2;
	}else{;
		return  z/(exp(z) - 1);
	};
};
__device__ float Cutrap0_na(float v,float th,float a,float q){
	if (fabs((v-th)/q) > 1e-6) {;
	        return  a * (v - th) / (1 - exp(-(v - th)/q));
	} else {;
	        return  a * q;
	};
}	;

// Procedures:
__device__ void Cutrates_ca(float v ,float gbar_ca,float cao_ca,float &hinf,float &htau,float &minf,float &mtau) {

	Curates_ca(v,gbar_ca,cao_ca,hinf,htau,minf,mtau);
};
__device__ void Curates_ca(float vm ,float gbar_ca,float cao_ca,float &hinf,float &htau,float &minf,float &mtau) {
float  a, b;
/* removed tadj_ca recalculation */
a = 0.209*Cuefun_ca(-(27+vm)/3.8);
	b = 0.94*exp((-75-vm)/17);
	;
	mtau = 1/tadj_ca/(a+b);
	minf = a/(a+b);
		;
	a = 0.000457*exp((-13-vm)/50);
	b = 0.0065/(exp((-vm-15)/28) + 1);
	htau = 1/tadj_ca/(a+b);
	hinf = a/(a+b);
};
__device__ void Curates_kca(float cai,float gbar_kca,float caix_kca,float Ra_kca,float Rb_kca,float &a,float &b,float &ninf,float &ntau) {

;
a =pow((MYFTYPE) Ra_kca * cai,(MYFTYPE)caix_kca);
b = Rb_kca;
/* removed tadj_kca recalculation */
ntau = 1/tadj_kca/(a+b);
	ninf = a/(a+b);
;
};
__device__ void Cutrates_km(float v ,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km,float &a,float &b,float &ninf,float &ntau) {

;
	Curates_km(v,gbar_km,tha_km,qa_km,Ra_km,Rb_km,a,b,ninf,ntau);
};
__device__ void Curates_km(float v ,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km,float &a,float &b,float &ninf,float &ntau) {

;
;
a = Ra_km * qa_km * Cuefun_km(-(v - tha_km)/qa_km);
;
b = Rb_km * qa_km * Cuefun_km((v - tha_km)/qa_km);
/* removed tadj_km recalculation */
ntau = 1/tadj_km/(a+b);
	ninf = a/(a+b);
};
__device__ void Cutrates_kv(float v ,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv,float &a,float &b,float &ninf,float &ntau) {

;
	Curates_kv(v,gbar_kv,tha_kv,qa_kv,Ra_kv,Rb_kv,a,b,ninf,ntau);
};
__device__ void Curates_kv(float v ,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv,float &a,float &b,float &ninf,float &ntau) {

;
;
a = Ra_kv * qa_kv * Cuefun_kv(-(v - tha_kv)/qa_kv);
;
b = Rb_kv * qa_kv * Cuefun_kv((v - tha_kv)/qa_kv);
/* removed tadj_kv recalculation */
ntau = 1/tadj_kv/(a+b);
	ninf = a/(a+b);
};
__device__ void Cutrates_na(float v,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na,float &hinf,float &htau,float &minf,float &mtau) {

;
;
	;
	Curates_na(v,gbar_na,tha_na,qa_na,Ra_na,Rb_na,thi1_na,thi2_na,qi_na,thinf_na,qinf_na,Rg_na,Rd_na,hinf,htau,minf,mtau);
};
__device__ void Curates_na(float vm,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na,float &hinf,float &htau,float &minf,float &mtau) {
float  a, b;
	a = Cutrap0_na(vm,tha_na,Ra_na,qa_na);
	b = Cutrap0_na(-vm,-tha_na,Rb_na,qa_na);
/* removed tadj_na recalculation */
	mtau = 1/tadj_na/(a+b);
	minf = a/(a+b);
		;
	a = Cutrap0_na(vm,thi1_na,Rd_na,qi_na);
	b = Cutrap0_na(-vm,-thi2_na,Rg_na,qi_na);
	htau = 1/tadj_na/(a+b);
	hinf = 1/(1+exp((vm-thinf_na)/qinf_na));
};

// Inits:


__device__ void CuInitModel_ca(float v,float &m,float &h,float gbar_ca,float cao_ca, float cai, float &ica){
float hinf,htau,minf,mtau;
   /* removed tadj_ca recalculation */
   Cutrates_ca(v+vshift_ca,gbar_ca,cao_ca,hinf,htau,minf,mtau);
   m = minf;
   h = hinf;
};


__device__ void CuInitModel_cad(float v,float &	ca		, float ica, float &cai){
   ca = cainf_cad;
   cai = ca;
};


__device__ void CuInitModel_kca(float v,float &n,float gbar_kca,float caix_kca,float Ra_kca,float Rb_kca, float cai){
float a,b,ninf,ntau;
   Curates_kca(cai,gbar_kca,caix_kca,Ra_kca,Rb_kca,a,b,ninf,ntau);
   n = ninf;
};


__device__ void CuInitModel_km(float v,float &n,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km){
float a,b,ninf,ntau;
   /* removed tadj_km recalculation */
   Cutrates_km(v,gbar_km,tha_km,qa_km,Ra_km,Rb_km,a,b,ninf,ntau);
   n = ninf;
};


__device__ void CuInitModel_kv(float v,float &n,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv){
float a,b,ninf,ntau;
   /* removed tadj_kv recalculation */
   Cutrates_kv(v,gbar_kv,tha_kv,qa_kv,Ra_kv,Rb_kv,a,b,ninf,ntau);
   n = ninf;
};


__device__ void CuInitModel_na(float v,float &m,float &h,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na){
float hinf,htau,minf,mtau;
   /* removed tadj_na recalculation */
   Cutrates_na(v+vshift_na,gbar_na,tha_na,qa_na,Ra_na,Rb_na,thi1_na,thi2_na,qi_na,thinf_na,qinf_na,Rg_na,Rd_na,hinf,htau,minf,mtau);
   m = minf;
   h = hinf;
};

// Derivs:
__device__ void CuDerivModel_ca(float dt, float v,float &m,float &h,float gbar_ca,float cao_ca, float cai, float &ica){

float hinf,htau,minf,mtau;
   Cutrates_ca (   v + vshift_ca ,gbar_ca,cao_ca,hinf,htau,minf,mtau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
}

__device__ void CuDerivModel_cad(float dt, float v,float &	ca		, float ica, float &cai){
float drive_channel;
   drive_channel = - ( 10000.0 ) * ica / ( 2.0 * FARADAY * depth_cad ) ;
   if ( drive_channel <= 0. ) {
     drive_channel = 0. ;
     }
    ca = ca + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / taur_cad)))*(- ( drive_channel + ( ( cainf_cad ) ) / taur_cad ) / ( ( ( ( - 1.0) ) ) / taur_cad ) - ca) ;
   cai = ca ;
}

__device__ void CuDerivModel_kca(float dt, float v,float &n,float gbar_kca,float caix_kca,float Ra_kca,float Rb_kca, float cai){
float ek;
float a,b,ninf,ntau;
   Curates_kca (   cai ,gbar_kca,caix_kca,Ra_kca,Rb_kca,a,b,ninf,ntau);
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
}

__device__ void CuDerivModel_km(float dt, float v,float &n,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km){

float a,b,ninf,ntau;
   Cutrates_km (   v ,gbar_km,tha_km,qa_km,Ra_km,Rb_km,a,b,ninf,ntau);
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
}

__device__ void CuDerivModel_kv(float dt, float v,float &n,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv){

float a,b,ninf,ntau;
   Cutrates_kv (   v ,gbar_kv,tha_kv,qa_kv,Ra_kv,Rb_kv,a,b,ninf,ntau);
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
}

__device__ void CuDerivModel_na(float dt, float v,float &m,float &h,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na){

float hinf,htau,minf,mtau;
   Cutrates_na (   v + vshift_na ,gbar_na,tha_na,qa_na,Ra_na,Rb_na,thi1_na,thi2_na,qi_na,thinf_na,qinf_na,Rg_na,Rd_na,hinf,htau,minf,mtau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
}


// Breaks:


__device__ void CuBreakpointModel_ca(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &m,float &h,float gbar_ca,float cao_ca, float cai, float &ica) {
float gca, mtau, hinf, minf, htau;
float ;
gca=tadj_ca*gbar_ca*m*m*h;
ica=(1e-4)*gca*(v-eca);
sumCurrents+= ica;
sumConductivity+= gca;
};
;


__device__ void CuBreakpointModel_cad(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &	ca		, float ica, float &cai) {
float gca;
float ;
};
;


__device__ void CuBreakpointModel_kca(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &n,float gbar_kca,float caix_kca,float Ra_kca,float Rb_kca, float cai) {
float ntau, gca, gk, ninf;
float ik;
gk=tadj_kca*gbar_kca*n;
ik=(1e-4)*gk*(v-ek);
sumCurrents+= ik;
sumConductivity+= gk;
};
;


__device__ void CuBreakpointModel_km(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &n,float gbar_km,float tha_km,float qa_km,float Ra_km,float Rb_km) {
float ntau, gk, ninf;
float ik;
gk=tadj_km*gbar_km*n;
ik=(1e-4)*gk*(v-ek);
sumCurrents+= ik;
sumConductivity+= gk;
};
;


__device__ void CuBreakpointModel_kv(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &n,float gbar_kv,float tha_kv,float qa_kv,float Ra_kv,float Rb_kv) {
float ntau, gk, ninf;
float ik;
gk=tadj_kv*gbar_kv*n;
ik=(1e-4)*gk*(v-ek);
sumCurrents+= ik;
sumConductivity+= gk;
};
;


__device__ void CuBreakpointModel_na(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, float v,float &m,float &h,float gbar_na,float tha_na,float qa_na,float Ra_na,float Rb_na,float thi1_na,float thi2_na,float qi_na,float thinf_na,float qinf_na,float Rg_na,float Rd_na) {
float mtau, gna, hinf, minf, htau;
float ina;
gna=tadj_na*gbar_na*m*m*m*h;
ina=(1e-4)*gna*(v-ena);
sumCurrents+= ina;
sumConductivity+= gna;
};
;
