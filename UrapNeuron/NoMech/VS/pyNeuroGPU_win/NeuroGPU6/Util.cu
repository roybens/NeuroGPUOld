#include "hip/hip_runtime.h"
#include "Util.h"
//#include "CudaStuff.cuh"
MYFTYPE  maxf(MYFTYPE  a, MYFTYPE  b) {
	if (a>b)
		return a;
	return b;
}

MYFTYPE  MaxAbsDiffVec(MYFTYPE * A, MYFTYPE  *B, int n) {
	MYFTYPE  cmax = -1;
	MYFTYPE  curdiff;
	for (int i = 0; i<n; i++) {
		curdiff = abs(A[i] - B[i]);
		cmax = maxf(cmax, curdiff);
	}
	return cmax;
}

void CopyVec(MYFTYPE * A, MYFTYPE  *B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[i];
	}
}
void CopyVecMYSECONDFTYPE(MYSECONDFTYPE* A, MYSECONDFTYPE  *B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[i];
	}
}
void CopyVecTwoTypes(MYSECONDFTYPE *A, MYFTYPE * B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[i];
	}
}
void FlipVec(MYFTYPE * A, MYFTYPE  *B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[n - i];
	}
}
void ReadShortFromCSV(char* line, unsigned short *ans, int n) {
	int count = 0;
	char* tok;
	unsigned short tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		printf("%s\n", tok);
		tmp = atoi(tok);
		*ans = (unsigned short)tmp;
	}
	else {

		for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ","))
		{
			tmp = atoi(tok);
			ans[count++] = (unsigned short)tmp;
			
		}
	}
}


void ReadDoubleFromCSV(char* line, double *ans, int n) {
	int count=0;
	char* tok;
	double tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		tmp = atof(tok);
		*ans = (double)tmp;
	}	else{

	for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ","))
		{
		tmp = atof(tok);
		ans[count++] = tmp;
		
		}
	}
}
void ReadFloatFromCSV(char* line, float *ans, int n) {
	int count = 0;
	char* tok;
	float tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		tmp = atof(tok);
		*ans = (float)tmp;
	}
	else {

		for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ","))
		{
			tmp = atof(tok);
			ans[count++] = tmp;

		}
	}
}
MYFTYPE* ReadAllParams(const char* FN, MYDTYPE NParams, MYDTYPE Nx, MYDTYPE  &nSets) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	//sprintf(FileName,"%sForC.mat",FN);
	MYFTYPE* ans;
	MYDTYPE tmp;
	FILE *fl = fopen(FN, "r"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read allparmas.csv\n");
		return nullptr;
	}
	char line[1009600];
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, &nSets, 1);
	ans = (MYFTYPE *)malloc(Nx * NParams * nSets * sizeof(MYFTYPE));

	for(int i=0;i<nSets;i++){
		fgets(line, sizeof(line), fl);
		ReadFloatFromCSV(line, &ans[i*Nx*NParams], Nx*NParams);
	}
	fclose(fl);
	return ans;
}
/*function [OurDiag rhs] =solveByNeuron(Nx,Aux,OurDiag,rhs,Neuron)
%TRIANG
% for (i = i3 - 1; i >= i2; --i) {
% 		nd = _nt->_v_node[i];
% 		pnd = _nt->_v_parent[i];
% 		p = NODEA(nd) / NODED(nd);,
% 		NODED(pnd) -= p * NODEB(nd);
% 		NODERHS(pnd) -= p * NODERHS(nd);
% 	}
for seg=Nx:-1:2
parentIndex =  getParentInd(Neuron,seg);
%     parentIndex=Nx+1-Aux.Ks(Nx+2-(seg));
p = Aux.e(Nx+2-seg)/OurDiag(seg);
OurDiag(parentIndex)=OurDiag(parentIndex)-p*Aux.f(Nx+1-seg);
rhs(parentIndex)=rhs(parentIndex)-p*rhs(seg);
end
% %BKSUB
% for (i = i1; i < i2; ++i) {
% 		NODERHS(_nt->_v_node[i]) /= NODED(_nt->_v_node[i]);
% 	}
% 	for (i = i2; i < i3; ++i) {
% 		cnd = _nt->_v_node[i];
% 		nd = _nt->_v_parent[i];
% 		NODERHS(cnd) -= NODEB(cnd) * NODERHS(nd);
% 		NODERHS(cnd) /= NODED(cnd);
% 	}
%     }
rhs(1) = rhs(1)/OurDiag(1);
for seg=2:Nx
parentIndex =  getParentInd(Neuron,seg);
%     parentIndex=Nx+1-Aux.Ks(Nx+2-(seg));
rhs(seg) = rhs(seg)- Aux.f(Nx+1-seg)*rhs(parentIndex);
rhs(seg)=rhs(seg)/OurDiag(seg);
end*/
void solveByNeuron(const HMat &InMat, MYSECONDFTYPE* B, MYSECONDFTYPE* D) {
	FILE *fdebug2;
	double  p;
	int ParentInd;
	for (int seg = InMat.N - 1; seg >= 1; seg--){
		//NEURON: 	p = NODEA(nd) / NODED(nd);
		p = InMat.e[InMat.N + 2 - seg - 2] / D[seg];
		ParentInd = InMat.N - InMat.Ks[InMat.N - seg];
		//NEURON: NODED(pnd) -= p * NODEB(nd);
		D[ParentInd] -= p*InMat.f[InMat.N + 1 - seg - 2];
		B[ParentInd] -= p*B[seg];
	}
	//BKSUB
	B[0] = B[0] / D[0];
	for (int seg = 1; seg<InMat.N; seg++){
		ParentInd = InMat.N - InMat.Ks[InMat.N - seg];
		B[seg] -= InMat.f[InMat.N + 1 - seg - 2] * B[ParentInd];
		B[seg] = B[seg] / D[seg];

	}
}




//void SolveTriDiagonalHinesSerialCPU(const HMat &InMat, MYFTYPE* B, MYFTYPE* HX) {
//	MYDTYPE k;
//	int i;
//	MYFTYPE *uH,*bH;
//	uH = (MYFTYPE*) malloc(InMat.N*sizeof(MYFTYPE));
//	bH = (MYFTYPE*) malloc(InMat.N*sizeof(MYFTYPE));
//	
//	for(i=0;i<InMat.N;i++) {
//		uH[i]=InMat.d[i];
//		bH[i]=B[i];
//	}
//	for(i=1;i<InMat.N;i++) {
//		k=InMat.Ks[i]-1;
//		uH[k]=uH[k]-InMat.e[i]*(InMat.f[i-1]/uH[i-1]);
//		bH[k]=bH[k]-bH[i-1]*InMat.e[i]/uH[i-1];
//	}
//	
//	HX[InMat.N-1]=bH[InMat.N-1]/uH[InMat.N-1];
//	for(i=InMat.N-2;i>=0;i--) {
//		k=InMat.Ks[i+1]-1;
//		HX[i]=(bH[i]-HX[k]*InMat.f[i])/uH[i];
//	}
//}

void FreeSerialNeuronData(HMat &InMat) {
	printf("Freeing\n");
	free(InMat.e);
	free(InMat.f);
	free(InMat.Ks);
	free(InMat.boolModel);
	free(InMat.Cms);
	printf("End\n");
}

void ReadStimData(const char* FN, Stim &stim, MYDTYPE Nx){
	char FileName[300];
	//sprintf(FileName,"%s%d.dat",FN,MUL32*32);
	sprintf(FileName, "%s.dat", FN);
	printf("Start reading file - ReadStimData()\n");

	FILE *fl;
	fl = fopen(FileName, "rb");
	if (!fl)
	{
		printf("Failed to read StimData\n");
		return;
	}
	//fread(&stim.NStimuli, sizeof(MYDTYPE),1, fl);
	stim.NStimuli = 1;
	stim.dels = (MYFTYPE*)malloc(stim.NStimuli*sizeof(MYFTYPE));
	fread(stim.dels, sizeof(MYFTYPE), stim.NStimuli, fl);
	stim.durs = (MYFTYPE*)malloc(stim.NStimuli*sizeof(MYFTYPE));
	fread(stim.durs, sizeof(MYFTYPE), stim.NStimuli, fl);
	stim.amps = (MYFTYPE*)malloc(stim.NStimuli*sizeof(MYFTYPE));
	fread(stim.amps, sizeof(MYFTYPE), stim.NStimuli, fl);
	//fread(&stim.comp, sizeof(MYDTYPE),1, fl);
	fread(&stim.area, sizeof(MYFTYPE), 1, fl);
	fread(&stim.loc, sizeof(MYDTYPE), 1, fl);
	stim.loc = stim.loc - 1;
	fclose(fl);
	return;
}




void CreateStimData(Stim &stim){
	stim.NStimuli = NSTIM;
	stim.dels = (MYFTYPE*)malloc(stim.NStimuli*sizeof(MYFTYPE));
	stim.durs = (MYFTYPE*)malloc(stim.NStimuli*sizeof(MYFTYPE));
	stim.amps = (MYFTYPE*)malloc(stim.NStimuli*sizeof(MYFTYPE));
	stim.comp = 1;
	stim.area = 28.3771;
	stim.loc = 2;
	MYFTYPE  initAmp = -1;
	for (int i = 0; i<stim.NStimuli; i++){
		stim.durs[i] = 20.0;
		stim.dels[i] = 10.0;
		stim.amps[i] = initAmp + 1 * i;
	}
	return;
}

void ReadStimFromFile(const char* FN, Stim &stim){
	FILE *fl;
	fl = fopen(FN, "rb");
	printf("reading stimfromfile - %s", FN);
	if (!fl)
	{
		printf("Failed to read StimData\n");
		return;
	}
	MYDTYPE Nt;
	fread(&stim.NStimuli, sizeof(MYDTYPE), 1, fl);
	fread(&Nt, sizeof(MYDTYPE), 1, fl);
	stim.Nt = MYDTYPE(ceil(MYFTYPE(Nt) / WARPSIZE)*WARPSIZE);
	fread(&stim.comp, sizeof(MYDTYPE), 1, fl);
	fread(&stim.loc, sizeof(MYDTYPE), 1, fl);
	stim.loc -= 1;
	fread(&stim.area, sizeof(MYFTYPE), 1, fl);
	fread(&stim.numofdts, sizeof(MYDTYPE), 1, fl);
	stim.dtInds = (MYDTYPE*)malloc(stim.numofdts*sizeof(MYDTYPE));//dont need it now
	stim.durs = (MYFTYPE*)malloc(stim.numofdts*sizeof(MYFTYPE));//acts as dts.
	stim.amps = (MYFTYPE*)malloc(Nt*stim.NStimuli*sizeof(MYFTYPE));
	fread(stim.dtInds, sizeof(MYDTYPE), stim.numofdts, fl);
	fread(stim.durs, sizeof(MYFTYPE), stim.numofdts, fl);
	fread(stim.amps, sizeof(MYFTYPE), Nt*stim.NStimuli, fl);
	fclose(fl);

}
void FreeStimData(Stim &stim){
	printf("Still need to implement FreeStimData()\n");
}
void ReadSimData(const char* FN, MYDTYPE N, Sim &sim){
	char FileName[300];
	//sprintf(FileName,"%s%d.dat",FN,MUL32*32);
	sprintf(FileName, "%s.dat", FN);
	printf("Start reading file - ReadSimData()\n");
	FILE *fl;
	fl = fopen(FileName, "rb");
	if (!fl)
	{
		printf("Failed to read SimData\n");
		return;
	}
	sim.Vs = (MYFTYPE*)malloc(N*sizeof(MYFTYPE));
	fread(sim.Vs, sizeof(MYFTYPE), N, fl);
	fread(&sim.dt, sizeof(MYFTYPE), 1, fl);
	fread(&sim.TFinal, sizeof(MYFTYPE), 1, fl);
	MYDTYPE Nt = ceil(sim.TFinal / sim.dt);
	MYDTYPE to32 = 32 - Nt % 32;
	sim.TFinal = sim.TFinal + (to32 + 1)*sim.dt;
	fread(&sim.NRecSites, sizeof(MYDTYPE), 1, fl);
	//MYDTYPE Stam;
	sim.RecSites = (MYDTYPE*)malloc(sim.NRecSites*sizeof(MYDTYPE));
	fread(sim.RecSites, sizeof(MYDTYPE), sim.NRecSites, fl);
	//fread(&sim.Celsius, sizeof(MYFTYPE),1, fl);
	fclose(fl);
	return;
}
void FreeSimData(Sim &sim){
	printf("Still need to implement FreeStimData()\n");
}
double diffclock(clock_t clock1, clock_t clock2)
{
	double diffticks = clock1 - clock2;
	double diffms = (diffticks) / CLOCKS_PER_SEC;
	return diffms;
}

void SaveArrayToFile(const char* FN, const int N, const MYFTYPE* Arr) {
	printf("printing %s size is %d\n", FN, N);
	const int prec = 3;
	FILE *file = fopen(FN, "wb");
	if (file) {
		fwrite(&N, sizeof(int), 1, file);
		fwrite(&prec, sizeof(int), 1, file);
		fwrite(Arr, sizeof(MYFTYPE), N, file);
	}
	else {
		printf("ERR SaveArrayToFile %s %d\n", FN, N);
	}
	fclose(file);
}

void CompareArrays(const MYFTYPE * App, const MYFTYPE  *Real, int n, char* Prefix) {
	MYFTYPE MaxDiff = 0, CurDiff, MaxRDiff = 0;
	for (int i = 0; i<n; i++) {
		CurDiff = abs(App[i] - Real[i]);
		MaxDiff = maxf(MaxDiff, CurDiff);
		MaxRDiff = maxf(MaxRDiff, CurDiff / abs(Real[i]));
	}
	printf("Comparing %s,%d: MaxD=%f, MaxRD=%f\n", Prefix, n, MaxDiff, MaxRDiff);
}
void SaveVHots(const char* FN, MYFTYPE** VHotsHost, MYDTYPE Nt, MYDTYPE NStimuli){
	FILE *file = fopen(FN, "wb");
	if (file) {
		for (int i = 0; i<NStimuli; i++){
			fwrite(VHotsHost[i], sizeof(MYFTYPE), Nt, file);
		}
	}
	else {
		printf("ERR SaveArrayToFile %s %d\n", FN);
	}
	fclose(file);
}
/*
void ReadDebugData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt){
	FILE *fl;
	MYDTYPE NP;
	fl = fopen(FN, "rb");

	for (int i = 0; i<NSTATES; i++){
		DebugData[i] = (MYFTYPE*)malloc((Nsegs*Nt)*sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs*Nt, fl);
	}
}

void SetStatesFromDebug(MYFTYPE** StatesM, MYFTYPE** DebugData, MYDTYPE iter, MYDTYPE Nseg){
	MYDTYPE debugIter;
	if (iter>0){
		iter -= 1;
		debugIter = iter*Nseg;
		for (int currSeg = 0; currSeg<Nseg; currSeg++){
			for (int currState = 0; currState<NSTATES; currState++){
				StatesM[currState][currSeg] = DebugData[currState][debugIter + currSeg];
			}
		}
	}
}
*/
void ReadRHSData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt){
	FILE *fl;
	fl = fopen(FN, "rb");
	for (int i = 0; i<Nt; i++){
		DebugData[i] = (MYFTYPE*)malloc((Nsegs)*sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs, fl);
	}
}

void SetRHSFromNeuron(MYFTYPE* rhs, MYFTYPE** DebugData, MYDTYPE i, MYDTYPE N){

	for (int currSeg = 0; currSeg<N; currSeg++){
		rhs[currSeg] = DebugData[i][currSeg];
	}


}
void ReadDData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt){
	FILE *fl;
	fl = fopen(FN, "rb");
	for (int i = 0; i<Nt; i++){
		DebugData[i] = (MYFTYPE*)malloc((Nsegs)*sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs, fl);
	}
}

void SetDFromNeuron(MYFTYPE* D, MYFTYPE** DebugData, MYDTYPE i, MYDTYPE N){

	for (int currSeg = 0; currSeg<N; currSeg++){
		D[currSeg] = DebugData[i][currSeg];

	}

}
void ReadVData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt){
	FILE *fl;
	fl = fopen(FN, "rb");
	for (int i = 0; i<Nt; i++){
		DebugData[i] = (MYFTYPE*)malloc((Nsegs)*sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs, fl);
	}
}
void SetVFromNeuron(MYFTYPE* vs, MYFTYPE** DebugData, MYDTYPE i, MYDTYPE N){
	if (i>0){
		for (int currSeg = 0; currSeg<N; currSeg++){
			vs[currSeg] = DebugData[i - 1][currSeg];
		}
	}

}
void debugPrintMYFTYPE(MYFTYPE* A, MYDTYPE N, FILE* fdebug){
	fwrite(&N, sizeof(MYDTYPE), 1, fdebug);
	fwrite(A, sizeof(MYFTYPE), N, fdebug);
}
void debugPrintMYSECONDFTYPE(MYSECONDFTYPE* A, MYDTYPE N, FILE* fdebug){
	fwrite(&N, sizeof(MYDTYPE), 1, fdebug);
	fwrite(A, sizeof(MYSECONDFTYPE), N, fdebug);
}
