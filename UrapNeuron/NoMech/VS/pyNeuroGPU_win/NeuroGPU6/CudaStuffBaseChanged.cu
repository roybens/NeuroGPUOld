#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "AllModels.cu"
#include "AllModels.cuh"

__constant__ MYFTYPE cCm[NSEG];
__constant__ MYSECONDFTYPE cE[NSEG];
__constant__ MYSECONDFTYPE cF[NSEG];
__constant__ MYDTYPE cFIdxs[NSEG*LOG_N_DEPTH];
__constant__ MYDTYPE cKs[NSEG];
__constant__ MYDTYPE cSegToComp[NSEG];
__constant__ MYDTYPE cBoolModel[NSEG * N_MODELS];//One day change this to bool
__constant__ MYDTYPE cRelStarts[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelEnds[N_FATHERS];//nFathers
__constant__ MYDTYPE cFathers[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelVec[N_CALL_FOR_FATHER];//nCallForFather
__constant__ MYDTYPE cSegStartI[N_CALL_FOR_FATHER+1];//nCallForFather
__constant__ MYDTYPE cSegEndI[N_CALL_FOR_FATHER+1];//nCallForFather

__constant__ MYDTYPE cCompByLevel32[COMP_DEPTH*WARPSIZE];//CompDepth
__constant__ MYDTYPE cCompByFLevel32[COMP_DEPTH*WARPSIZE];//CompFDepth
__constant__ MYDTYPE cLRelStarts[N_L_REL];//nLRel
__constant__ MYDTYPE cLRelEnds[N_L_REL];//nLRel
__constant__ MYDTYPE cFLRelStarts[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cFLRelEnds[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cSonNoVec[NSEG];//InMat.N 


#ifdef ILP1
	#define SUPERILPMACRO(x) ; ## x ## (1);
#endif
#ifdef ILP2
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2);
#endif
#ifdef ILP3
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3);
#endif
#ifdef ILP4
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); 
#endif
#ifdef ILP5
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); 
#endif
#ifdef ILP6
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6);
#endif
#ifdef ILP7
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7);
#endif
#ifdef ILP8
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8);
#endif
#ifdef ILP9
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9);
#endif
#ifdef ILP10
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10);
#endif
#ifdef ILP11
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11);
#endif
#ifdef ILP12
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12);
#endif
#ifdef ILP13
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13);
#endif
#ifdef ILP14
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14);
#endif
#ifdef ILP15
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15);
#endif
#ifdef ILP16
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16);
#endif
#ifdef ILP17
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17);
#endif
#ifdef ILP18
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18);
#endif
#ifdef ILP19
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19);
#endif
#ifdef ILP20
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20);
#endif
#ifdef ILP21
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21);
#endif
#ifdef ILP22
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22);
#endif
#ifdef ILP23
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23);
#endif
#ifdef ILP24
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24);
#endif
#ifdef ILP25
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25);
#endif
#ifdef ILP26
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26);
#endif
#ifdef ILP27
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27);
#endif
#ifdef ILP28
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28);
#endif
#ifdef ILP29
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29);
#endif
#ifdef ILP30
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30);
#endif
#ifdef ILP31
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31);
#endif
#ifdef ILP32
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); 
#endif
#ifdef ILP33
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33);
#endif
#ifdef ILP34
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34);
#endif
#ifdef ILP35
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35);
#endif
#ifdef ILP36
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36);
#endif
#ifdef ILP37
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37);
#endif
#ifdef ILP38
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38);
#endif
#ifdef ILP39
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39);
#endif
#ifdef ILP40
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40);
#endif
#ifdef ILP41
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41);
#endif
#ifdef ILP42
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42);
#endif
#ifdef ILP43
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43);
#endif
#ifdef ILP44
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44);
#endif
#ifdef ILP45
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45);
#endif
#ifdef ILP46
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46);
#endif
#ifdef ILP47
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47);
#endif
#ifdef ILP48
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48);
#endif
#ifdef ILP49
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49);
#endif
#ifdef ILP50
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49); ## x ## (50);
#endif

__device__ void BeforeLU(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel,LRelIndex;
	MYDTYPE JumctionI;
	CurJ = cCompByLevel32[2]-1;
	LRelIndex=cLRelStarts[CurLevel];
	LRelIndex=LRelIndex+cLRelEnds[CurLevel];
	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		
		for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
		//for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx]-1;
			for(i=cSegStartI[JumctionI]-1;i<cSegEndI[JumctionI];i++) {
				MYSECONDFTYPE uHPm1=uHP[i-1];
			
				uHP[i]=uHP[i]-cE[i-1]*(cF[i-1]/uHPm1); // So far same as paper parallel
				uHPm1=uHP[i-1];
				MYSECONDFTYPE bHPm1=bHP[i-1];
				bHP[i]=bHP[i]-bHPm1*cE[i-1]/uHPm1; // bH is y
			}
		}
		if(CurLevel<Depth) {
			for(LRelIndex=cFLRelStarts[CurLevel];LRelIndex<=cFLRelEnds[CurLevel];LRelIndex++){ 
				CurB=cCompByFLevel32[(LRelIndex)*WARPSIZE+PIdx]-1;//RB i inserted another  -1 into the index RB 2 i removed the-1 from the curlevel 
				CurJ=cFathers[CurB]-1;
				MYDTYPE St=cRelStarts[CurB];
				MYDTYPE En=cRelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=cRelVec[j-1]-1;
					MYSECONDFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]-=cE[t-1]*(cF[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYSECONDFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]-=bHPm1*cE[t-1]/uHPm1; 
				}
			}
		}	
	}
}

#ifdef BKSUB1
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* PX, MYSECONDFTYPE* PF,MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE LognDepth)
{
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
#define THISCOMMANDHERE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*(## VARILP -1)); MYDTYPE NextID_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHERE1)
	MYDTYPE i;
	
	PX=bHP;
	PF=uHP;
	// PX[PIdx_1]=PX[PIdx_1]/PF[PIdx_1]; // PF[PIdx_1]=-cF[PIdx_1]/PF[PIdx_1];
#define THISCOMMANDHERE2(VARILP) PX[PIdx_ ## VARILP ]=PX[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ]; PF[PIdx_ ## VARILP ]=-cF[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHERE2)

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		// NextID_1=cFIdxs[i*InMat.N+PIdx_1]-1; // MYFTYPE OldPXj_1=PX[PIdx_1]; // MYFTYPE OldPXNextID_1=PX[NextID_1]; // PX[PIdx_1]=OldPXj_1+OldPXNextID_1*PF[PIdx_1];
#define THISCOMMANDHERE3(VARILP) NextID_ ## VARILP =InMat.FIdxs[i*InMat.N+PIdx_ ## VARILP ]-1; MYFTYPE OldPXj_ ## VARILP =PX[PIdx_ ## VARILP ]; MYFTYPE OldPXNextID_ ## VARILP =PX[NextID_ ## VARILP ]; PX[PIdx_ ## VARILP ]=OldPXj_ ## VARILP +OldPXNextID_ ## VARILP *PF[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHERE3)

		// PX[j]=PX[j]+PX[NextID]*PF[j];
		// PF[j]=PF[j]*PF[NextID];
		// MYFTYPE OldPFj_1=PF[PIdx_1]; // MYFTYPE OldPFNextID_1=PF[NextID_1]; // PF[PIdx_1]=OldPFj_1*OldPFNextID_1;
#define THISCOMMANDHERE4(VARILP) MYFTYPE OldPFj_ ## VARILP =PF[PIdx_ ## VARILP ]; MYFTYPE OldPFNextID_ ## VARILP =PF[NextID_ ## VARILP ]; PF[PIdx_ ## VARILP ]=OldPFj_ ## VARILP *OldPFNextID_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHERE4)
	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP,MYSECONDFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
	SUPERILPMACRO(THISCOMMANDHEREB1)

	MYDTYPE j,CurJ,CurB,t;
	MYDTYPE JumctionI;
	short CurLevel,i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex,k;
	MYFTYPE temp;
 for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
  //     Run all independent set for this level, in parallel
  // for JumctionI=find(Level==CurLevel) % in parallel
  for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
   JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx_1]-1;
   // for i=(cSegEndI(JumctionI)):-1:(cSegStartI(JumctionI)-1)
   for(i=cSegEndI[JumctionI]-1;i>=(cSegStartI[JumctionI]-2);i--) {
    // k=cKsB(i+1);
     k=cKsB[i+1];
	 
    // bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
   Out[i]=(bHP[i]-Out[k]*cF[i])/uHP[i];
   }
  }
 }
 } 

#endif

__device__ void runSimulation(HMat SMemMat,HMat InMat,MYFTYPE* V,Stim stim,Sim sim,MYFTYPE* VHotGlobal,MYFTYPE* SMemVHot,MYFTYPE* amps,MYDTYPE offset)
{
	MYFTYPE *Vs;
	MYSECONDFTYPE *uHP,*bHP;
	MYDTYPE StimID=threadIdx.y;
		
#ifdef BKSUB2
    MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(InMat.N+2*WARPSIZE)*sizeof(MYFTYPE);
#endif
#ifdef BKSUB1

  MYDTYPE PerStimulus=((InMat.N+3)*2*sizeof(MYSECONDFTYPE))+(WARPSIZE + sim.NRecSites*(WARPSIZE+1))*sizeof(MYFTYPE);
  PerStimulus =ceilf(MYSECONDFTYPE(PerStimulus)/sizeof(MYSECONDFTYPE)+1)*sizeof(MYSECONDFTYPE); 
#endif
	//MYDTYPE PerStimulus;
	//PerStimulus = InMat.N+2;
	offset+=PerStimulus*threadIdx.y;
	uHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+3)*sizeof(MYSECONDFTYPE);
	//uHP[InMat.N+2] =0;
	bHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+3)*sizeof(MYSECONDFTYPE);
	//bHP[InMat.N+2] =0;
	offset = ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE)+1)*sizeof(MYSECONDFTYPE);
	//Vs = (MYFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYFTYPE);
	// Vs[PIdx_3]=V[PIdx_3];
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=(WARPSIZE+1)*sim.NRecSites*sizeof(MYFTYPE);
	SMemVHot[WARPSIZE]=0;
    amps = (MYFTYPE*) &smem[offset];
    offset+=WARPSIZE*sizeof(MYFTYPE);
	MYDTYPE NeuronID = blockIdx.x;
	int Nt=stim.Nt;
	MYFTYPE t=0;
	MYSECONDFTYPE *PX,*PF;
	PX=bHP;
	PF=uHP;
	#define THISCOMMANDHERE18(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1));
	SUPERILPMACRO(THISCOMMANDHERE18)

	 #define THISCOMMANDHERE19(VARILP) MYFTYPE isModel_ ## VARILP , dVec_ ## VARILP , Vmid_ ## VARILP ; MYFTYPE ModelStates_ ## VARILP [NSTATES]; MYFTYPE v_ ## VARILP =V[PIdx_ ## VARILP ]; MYFTYPE dv_ ## VARILP;
	SUPERILPMACRO(THISCOMMANDHERE19)

	 #define THISCOMMANDHERE20(VARILP) MYSECONDFTYPE sumCurrents_ ## VARILP=0 ,sumCurrentsDv_ ## VARILP=0; MYFTYPE  sumConductivity_ ## VARILP=0 ,sumConductivityDv_ ## VARILP=0 ;
	SUPERILPMACRO(THISCOMMANDHERE20)
	// end for model in init
	SUPERILPMACRO(SET_STATES)
	#define THISCOMMANDHERE21(VARILP) MYDTYPE SonNo_ ## VARILP =cSonNoVec[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHERE21)
	
	#define THISCOMMANDHERE22(VARILP) bHP[PIdx_ ## VARILP]=0;dv_ ## VARILP =0;
	SUPERILPMACRO(THISCOMMANDHERE22)

	// MYDTYPE parentIndex_3; // MYDTYPE Eidx_3; // Eidx_3 = SMemMat->N-PIdx_3;
	#define THISCOMMANDHERE23(VARILP) MYDTYPE parentIndex_ ## VARILP ; MYDTYPE Eidx_ ## VARILP ; Eidx_ ## VARILP = SMemMat.N-PIdx_ ## VARILP -1;
	SUPERILPMACRO(THISCOMMANDHERE23)
	// RRRXXX This probably should be in ILP too!
	if(Eidx_1>SMemMat.N-1){
		Eidx_1=SMemMat.N-1;
	}
	MYDTYPE perThreadParamMSize = SMemMat.NComps*NPARAMS;
#define param_macro(paramInd,segmentInd) ParamsM[NeuronID*perThreadParamMSize + paramInd*SMemMat.NComps+cSegToComp[segmentInd] ]
	
	SUPERILPMACRO(CALL_TO_INIT_STATES_CU)
	
		// parentIndex_3 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_3];
#define THISCOMMANDHERE24(VARILP) parentIndex_ ## VARILP =SMemMat.N-cKs[SMemMat.N-PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHERE24)
	MYDTYPE stimLoc = stim.loc;
    MYFTYPE stimArea = stim.area;
    MYDTYPE dtCounter=0;
    MYFTYPE  dt = sim.dt;
#ifndef STIMFROMFILE
    MYFTYPE stimDel = stim.dels[StimID];//Should be constant
    MYFTYPE stimDur = stim.durs[StimID];//Should be constant
    MYFTYPE stimAmp = stim.amps[StimID];
#endif // !STIMFROMFILE
	// MYFTYPE rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#define THISCOMMANDHERE25(VARILP) MYSECONDFTYPE rhs_ ## VARILP ,D_ ## VARILP; MYFTYPE gModel_ ## VARILP ,StimCurrent_ ## VARILP;
	SUPERILPMACRO(THISCOMMANDHERE25)

for(int i=0;i<Nt;i++) {
        if(i==stim.dtInds[dtCounter]){
                dt = stim.durs[dtCounter];
                if (dtCounter != stim.numofdts-1){
                    dtCounter++;
                }
        }
        t+=0.5*dt;
 
       if((i%(WARPSIZE)==0)){
            if(i>0){
               for(int recInd =0;recInd<sim.NRecSites;recInd++){
					VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + (i-WARPSIZE)+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
				}
            }
			
            amps[PIdx_1]=stim.amps[threadIdx.y*Nt+i+PIdx_1];
        }
        for(int recInd =0;recInd<sim.NRecSites;recInd++){
			if(sim.RecSites[recInd]%WARPSIZE == threadIdx.x)
				//This is done by all threads why???
				SMemVHot[recInd*WARPSIZE+i%(WARPSIZE)]=v_1;//This is going to be challenging to make it general but possible.
			
		}
		 // rhs_3=0; // D_3=0;// StimCurrent_3=0;
#define THISCOMMANDHERE26(VARILP) rhs_ ## VARILP =0; D_ ## VARILP =0; sumCurrents_ ## VARILP=0; sumConductivity_ ## VARILP=0; sumCurrentsDv_ ## VARILP=0; sumConductivityDv_ ## VARILP=0;StimCurrent_ ## VARILP =0;
	SUPERILPMACRO(THISCOMMANDHERE26)
		// Before matrix
		// if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#ifndef STIMFROMFILE
    #define THISCOMMANDHERE27(VARILP) if(t>=stimDel && t<stimDel+stimDur && PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP  = 100*stimAmp/stimArea;}
	SUPERILPMACRO(THISCOMMANDHERE27)
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
    
    #define THISCOMMANDHERE28(VARILP) if(PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP = 100*amps[i%WARPSIZE]/stimArea;}
	SUPERILPMACRO(THISCOMMANDHERE28)
#endif // STIMFROMFILE
	SUPERILPMACRO(CALL_TO_BREAK_DV_CU)
	SUPERILPMACRO(CALL_TO_BREAK_CU)

	// gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; 
#define THISCOMMANDHERE29(VARILP) gModel_ ## VARILP  = (sumCurrentsDv_ ## VARILP -sumCurrents_ ## VARILP )/EPS_V;  
	SUPERILPMACRO(THISCOMMANDHERE29)
	
#define THISCOMMANDHERE30(VARILP) rhs_ ## VARILP =StimCurrent_ ## VARILP -sumCurrents_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHERE30)
#define THISCOMMANDHERE31(VARILP) D_ ## VARILP =gModel_ ## VARILP +cCm[PIdx_ ## VARILP ]/(dt*1000); D_ ## VARILP -=cF[SMemMat.N-PIdx_ ## VARILP -1];
	SUPERILPMACRO(THISCOMMANDHERE31)
   
    //this should be out and correct cKs
	if(PIdx_1==0) {
		parentIndex_1=0;
	};
#define THISCOMMANDHERE32(VARILP)  dv_ ## VARILP +=bHP[SMemMat.N-parentIndex_ ## VARILP -1]-bHP[SMemMat.N-PIdx_ ## VARILP -1]; rhs_ ## VARILP -=cF[SMemMat.N-PIdx_ ## VARILP -1]*dv_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHERE32)
		// D_3=sumConductivity_3+SMemMat.Cms[PIdx_3]/(sim.dt*1000); // D_3-=SMemMat.f[SMemMat.N-PIdx_3-1];
		// bHP[SMemMat.N-PIdx_3-1]=rhs_3; // uHP[SMemMat.N-PIdx_3-1]=D_3;
#define THISCOMMANDHERE33(VARILP) bHP[SMemMat.N-PIdx_ ## VARILP -1]=rhs_ ## VARILP ; uHP[SMemMat.N-PIdx_ ## VARILP -1]=D_ ## VARILP ;
		SUPERILPMACRO(THISCOMMANDHERE33)
		syncthreads();
		
		// if(SonNo_3==1) {bHP[SMemMat.N-parentIndex_3-1]+=cE[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=cE[Eidx_3];}
		
#define THISCOMMANDHERE34(VARILP) if(cSonNoVec[PIdx_ ## VARILP] ==1) {bHP[SMemMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[SMemMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
		SUPERILPMACRO(THISCOMMANDHERE34)
		// if(SonNo_3==2) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#define THISCOMMANDHERE35(VARILP) if(cSonNoVec[PIdx_ ## VARILP] ==2) {bHP[SMemMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[SMemMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
		SUPERILPMACRO(THISCOMMANDHERE35)
		syncthreads();
		BeforeLU(SMemMat,uHP,bHP,SMemMat.Depth);
#ifdef BKSUB1
		BkSub(SMemMat, PX,PF,uHP,bHP,SMemMat.LognDepth);
		#define THISCOMMANDHERE36(VARILP) Vmid_ ## VARILP =PX[SMemMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ; Vs[PIdx_ ## VARILP ]+=Vmid_ ## VARILP ;
		SUPERILPMACRO(THISCOMMANDHERE36)
#endif

		
#ifdef BKSUB2
		#define THISCOMMANDHERE37(VARILP) MYFTYPE vTemp_ ## VARILP=Vs[PIdx_ ## VARILP];
		SUPERILPMACRO(THISCOMMANDHERE37)
		syncthreads();
		BkSub(SMemMat,uHP,bHP,Vs,SMemMat.Depth);
		syncthreads();
		PX=Vs;

#define THISCOMMANDHERE38(VARILP) Vmid_ ## VARILP =PX[SMemMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ;
		SUPERILPMACRO(THISCOMMANDHERE38)
		syncthreads(); 
#define THISCOMMANDHERE39(VARILP) Vs[PIdx_ ## VARILP ]= vTemp_ ## VARILP +Vmid_ ## VARILP ;
		SUPERILPMACRO(THISCOMMANDHERE39)
#endif		
            t+=0.5*dt;

        

//      if(SMemMat.boolModel[PIdx_1 +0*SMemMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],p0_1 ,p1_1 ,ModelStates_1[8],ModelStates_1[9]);} if(SMemMat.boolModel[PIdx_1 +1*SMemMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(SMemMat.boolModel[PIdx_1 +2*SMemMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],p2_1 ,p3_1 ,p4_1 ,p5_1 ,ModelStates_1[8]);} if(SMemMat.boolModel[PIdx_1 +3*SMemMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],p6_1 ,p7_1 ,p8_1 ,p9_1 ,p10_1 );} if(SMemMat.boolModel[PIdx_1 +4*SMemMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],p11_1 ,p12_1 ,p13_1 ,p14_1 ,p15_1 );} if(SMemMat.boolModel[PIdx_1 +5*SMemMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],p16_1 ,p17_1 ,p18_1 ,p19_1 ,p20_1 ,p21_1 ,p22_1 ,p23_1 ,p24_1 ,p25_1 ,p26_1 ,p27_1 );}  if(SMemMat.boolModel[PIdx_2 +0*SMemMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],p0_2 ,p1_2 ,ModelStates_2[8],ModelStates_2[9]);} if(SMemMat.boolModel[PIdx_2 +1*SMemMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(SMemMat.boolModel[PIdx_2 +2*SMemMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],p2_2 ,p3_2 ,p4_2 ,p5_2 ,ModelStates_2[8]);} if(SMemMat.boolModel[PIdx_2 +3*SMemMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],p6_2 ,p7_2 ,p8_2 ,p9_2 ,p10_2 );} if(SMemMat.boolModel[PIdx_2 +4*SMemMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],p11_2 ,p12_2 ,p13_2 ,p14_2 ,p15_2 );} if(SMemMat.boolModel[PIdx_2 +5*SMemMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],p16_2 ,p17_2 ,p18_2 ,p19_2 ,p20_2 ,p21_2 ,p22_2 ,p23_2 ,p24_2 ,p25_2 ,p26_2 ,p27_2 );}  if(SMemMat.boolModel[PIdx_3 +0*SMemMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],p0_3 ,p1_3 ,ModelStates_3[8],ModelStates_3[9]);} if(SMemMat.boolModel[PIdx_3 +1*SMemMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(SMemMat.boolModel[PIdx_3 +2*SMemMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],p2_3 ,p3_3 ,p4_3 ,p5_3 ,ModelStates_3[8]);} if(SMemMat.boolModel[PIdx_3 +3*SMemMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],p6_3 ,p7_3 ,p8_3 ,p9_3 ,p10_3 );} if(SMemMat.boolModel[PIdx_3 +4*SMemMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],p11_3 ,p12_3 ,p13_3 ,p14_3 ,p15_3 );} if(SMemMat.boolModel[PIdx_3 +5*SMemMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],p16_3 ,p17_3 ,p18_3 ,p19_3 ,p20_3 ,p21_3 ,p22_3 ,p23_3 ,p24_3 ,p25_3 ,p26_3 ,p27_3 );}
		SUPERILPMACRO(CALL_TO_DERIV_CU)

 
 
    }
	//This one looks suspicious but leaving it and will check it later.
   for(int recInd =0;recInd<sim.NRecSites;recInd++){
		VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + Nt-WARPSIZE+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
	}
 
}
__global__ void NeuroGPUKernel(Stim stim, MYFTYPE* ParamsM, Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal,MYDTYPE CompDepth,MYDTYPE CompFDepth)
{
	HMat SMemMat;
	MYFTYPE *Vs,*amps,*SMemVHot;
	MYDTYPE offset;
	
	
	offset = InitializeDeviceMemory(stim,sim,InMat,VHotGlobal,CompDepth,CompFDepth,&SMemMat,Vs,amps,SMemVHot);
	runSimulation(SMemMat,InMat,ParamsM,V,stim,sim,VHotGlobal,SMemVHot,amps,offset);

}
void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}
MYFTYPE* ReadAllParams(const char* FN,MYDTYPE NParams,MYDTYPE Nx,MYDTYPE  &ntemp) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	//sprintf(FileName,"%sForC.mat",FN);
	MYFTYPE* ans;
	MYDTYPE tmp;
	FILE *fl = fopen(FN, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return nullptr;
	}
	fread(&tmp, sizeof(MYDTYPE),1, fl);
		ntemp = tmp;

	ans=(MYFTYPE *)malloc(Nx * NPARAMS * tmp * sizeof(MYFTYPE ));
	fread(ans, sizeof(MYFTYPE), tmp*Nx*NParams, fl);
	fclose(fl);
	return ans;
}
void ReadParamsMat(const char* FN,MYFTYPE** ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	for(int i=0;i<NParams;i++){
		ParamsM[i]=(MYFTYPE*)malloc(Nx * sizeof(MYFTYPE));
		fread(ParamsM[i], sizeof(MYFTYPE), Nx, fl);
	}
	fclose(fl);
}


hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) { 

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(1);
	cudaStatus = hipDeviceReset();
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=stim.Nt;
	hipHostMalloc((void**)&VHotsHost,stim.NStimuli*Nt*sim.NRecSites*NSets*sizeof(MYFTYPE));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.NComps=InMat.NComps;
	Mat_d.Depth=InMat.Depth;
	Mat_d.NModels=InMat.NModels;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	Mat_d.nLRel=InMat.nLRel;
	Mat_d.nFLRel=InMat.nFLRel;
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	stim_d.Nt = stim.Nt;
	Sim sim_d; 	 			
	sim_d.Celsius = sim.Celsius; 	 	
	sim_d.dt = sim.dt; 	 	
	sim_d.NRecSites = sim.NRecSites; 	 	
	sim_d.TFinal = sim.TFinal; 
#ifndef STIMFROMFILE
    cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE

    stim_d.numofdts = stim.numofdts;
    cudaStatus = hipMalloc((void**)&stim_d.dtInds, stim_d.numofdts * sizeof(MYDTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.numofdts * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE));
#endif // STIMFROMFILE
	cudaStatus = hipMalloc((void**)&sim_d.RecSites, sim_d.NRecSites * sizeof(MYDTYPE));
	


	//cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * InMat.NModels* sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*Nt *stim.NStimuli* sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*sim_d.NRecSites*Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N +1)* sizeof(MYDTYPE));
#endif
	/*cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (CompDepth)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (CompFDepth)*WARPSIZE* sizeof(MYDTYPE));

	cudaStatus = hipMalloc((void**)&Mat_d.LRelStarts, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.LRelEnds, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelStarts, Mat_d.nFLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelEnds, Mat_d.nFLRel* sizeof(MYDTYPE));
*/
	

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	#ifndef STIMFROMFILE
 
    cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
	
    cudaStatus = hipMemcpy(stim_d.dtInds, stim.dtInds, stim.numofdts * sizeof(MYDTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.numofdts * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // STIMFROMFILE
	cudaStatus = hipMemcpy(sim_d.RecSites, sim.RecSites, sim_d.NRecSites * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cE), InMat.e, 416 * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cF), InMat.f, 416 * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cKs), InMat.Ks, 416 * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(Mat_d.SegToComp, InMat.SegToComp, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegToComp), InMat.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cBoolModel), InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE));

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCm), InMat.Cms, 416 * sizeof(MYFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSonNoVec), InMat.SonNoVec, InMat.N * sizeof(MYDTYPE));
	
	
	//cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelStarts), InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelEnds), InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelVec), InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegStartI), InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegEndI), InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFathers), InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFIdxs), InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMemcpy(Mat_d.KsB, InMat.KsB,(InMat.N+1)*sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif

	//cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByLevel32), InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByFLevel32), InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE));


	//cudaStatus = hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelStarts), InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelEnds), InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelStarts), InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelEnds), InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE));


	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.NComps *NSets * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}
	// copying
	
	cudaStatus = hipMemcpy(d_modelParams, ParamsM, NPARAMS * InMat.NComps * NSets *sizeof(MYFTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(NSets);

	MYDTYPE offset=0;
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
#ifdef BKSUB1
	//offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
#endif
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*InMat.NModels*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, SegToComp,boolModel
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	//offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	//offset+=WARPSIZE*(CompDepth)*sizeof(MYDTYPE);
	//offset+=WARPSIZE*(CompFDepth)*sizeof(MYDTYPE);
	//// 4 CompByLevel32 CompByFLevel32
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
#ifdef BKSUB2
	offset+=(Mat_d.N+1)*sizeof(MYDTYPE);
#endif
	// 5 LRelStarts LRelEnds FLRelStarts  FLRelEnds KsB
	//offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	MYDTYPE FrameworkMemSize=offset;
//PerStimulus
	printf("offset is%d\n",offset);

	offset+=(InMat.N+3)*sizeof(MYSECONDFTYPE);  // uHP (Diag)
	offset+=(InMat.N+3)*sizeof(MYSECONDFTYPE); // bHP (rhs)
	offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE)+1)*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE); // Vs

	offset+=((WARPSIZE+1)*sim.NRecSites)*sizeof(MYFTYPE);//SMemVHot
#ifdef STIMFROMFILE
    offset+=WARPSIZE*sizeof(MYFTYPE);//amps
#endif // STIMFROMFILE
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+(PerStimulus)*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
	clock_t begin,end;
	begin=clock();
		MYDTYPE currKernelRun = NKERNEL;
	MYDTYPE prevRuns = 0;
	int memSizeForVHotGlobal = Nt*stim.NStimuli*sim.NRecSites;
	MYDTYPE memSizeForModelParams = NPARAMS * InMat.NComps;
	MYDTYPE streamID=0;
	hipStream_t stream0,stream1,stream2,stream3;
	hipStreamCreate (&stream0);
	hipStreamCreate (&stream1);
	hipStreamCreate (&stream2);
	hipStreamCreate (&stream3);
	begin=clock();
	while (prevRuns<NSets){
	if(currKernelRun>(NSets-prevRuns)){
		currKernelRun = (NSets-prevRuns);
	}
	if(streamID==0){
		NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream0>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR	

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream0); 
		
	}
	if(streamID==1){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream1>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream1); 

	}
	if(streamID==2){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream2>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt *sim.NRecSites *  stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream2); 

	}
	if(streamID==3){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream3>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream3); 

	}
	prevRuns +=currKernelRun;
	if(streamID==3){
		streamID == 0;
	}	else {
		streamID +=1;
	}
	}
	hipDeviceSynchronize();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}
	//cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal,NSets * Nt * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost); 

	end=clock();
	double totalT = diffclock(end,begin);
	printf("stEfork2TimeLoopGPU took %g milliseconds\n",totalT);
	FILE *file = fopen(TIMES_FN, "wb");
	if ( file ) {
		MYDTYPE mul32 = MUL32;
		fwrite(&mul32,sizeof(MYDTYPE),1,file);
		fwrite(&totalT,sizeof(double),1,file);
	} else {
		printf("ERR SaveArrayToFile %s\n",TIMES_FN);
	}
	fclose(file);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,NSets*Nt*stim.NStimuli*sim.NRecSites,VHotsHost);
	
	return cudaStatus;
}


hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
	  stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V,CompDepth,CompFDepth,NSets); //RRR sim
	  
	
	


	return cudaStatus;
}
