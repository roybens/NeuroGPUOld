#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "Util.h"
#include "hh.h"


__device__ void BeforeLU(HMat InMat, MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel;
	MYDTYPE JumctionI;

	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		JumctionI=InMat.CompByLevel32[CurLevel*WARPSIZE+PIdx]-1;
		
		for(i=InMat.SegStartI[JumctionI]-1;i<InMat.SegEndI[JumctionI];i++) {
			MYFTYPE uHPm1=uHP[i-1];
			
			uHP[i]=uHP[i]-InMat.e[i]*(InMat.f[i-1]/uHPm1); // So far same as paper parallel
			uHPm1=uHP[i-1];
			MYFTYPE bHPm1=bHP[i-1];
			bHP[i]=bHP[i]-bHPm1*InMat.e[i]/uHPm1; // bH is y
		}
		if(CurLevel<Depth) {
			CurB=InMat.CompByFLevel32[(CurLevel)*WARPSIZE+PIdx]-1;//RBS i inserted another  -1 into the index RBS 2 i removed the-1 from the curlevel 
			CurJ=InMat.Fathers[CurB]-1;
			MYDTYPE St=InMat.RelStarts[CurB];
			MYDTYPE En=InMat.RelEnds[CurB];
			for(j=St;j<=En;j++) {
				t=InMat.RelVec[j-1]-1;
				MYFTYPE uHPm1=uHP[t-1];
				uHP[CurJ]-=InMat.e[t]*(InMat.f[t-1]/uHPm1); 
				uHPm1=uHP[t-1];
				MYFTYPE bHPm1=bHP[t-1];
				bHP[CurJ]-=bHPm1*InMat.e[t]/uHPm1; 
			}
		}	
	}
}
/*__device__ void BkSubB(HMat InMat, MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE PIdx_2=threadIdx.x+WARPSIZE;
#ifdef ILP3
	MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#endif
#ifdef ILP4
	MYDTYPE PIdx_4=threadIdx.x+(WARPSIZE*3);
#endif
#ifdef ILP5
	MYDTYPE PIdx_5=threadIdx.x+(WARPSIZE*4);
#endif
#ifdef ILP6
	MYDTYPE PIdx_6=threadIdx.x+(WARPSIZE*5);
#endif
#ifdef ILP7
	MYDTYPE PIdx_7=threadIdx.x+(WARPSIZE*6);
#endif
#ifdef ILP8
	MYDTYPE PIdx_8=threadIdx.x+(WARPSIZE*7);
#endif
#ifdef ILP9
	MYDTYPE PIdx_9=threadIdx.x+(WARPSIZE*8);
#endif
#ifdef ILP10
	MYDTYPE PIdx_10=threadIdx.x+(WARPSIZE*9);
#endif
#ifdef ILP11
	MYDTYPE PIdx_11=threadIdx.x+(WARPSIZE*10);
#endif
#ifdef ILP12
	MYDTYPE PIdx_12=threadIdx.x+(WARPSIZE*11);
#endif
#ifdef ILP13
	MYDTYPE PIdx_13=threadIdx.x+(WARPSIZE*12);
#endif
#ifdef ILP14
	MYDTYPE PIdx_14=threadIdx.x+(WARPSIZE*13);
#endif
#ifdef ILP15
	MYDTYPE PIdx_15=threadIdx.x+(WARPSIZE*14);
#endif
#ifdef ILP16
	MYDTYPE PIdx_16=threadIdx.x+(WARPSIZE*15);
#endif
#ifdef ILP17
	MYDTYPE PIdx_17=threadIdx.x+(WARPSIZE*16);
#endif
#ifdef ILP18
	MYDTYPE PIdx_18=threadIdx.x+(WARPSIZE*17);
#endif
#ifdef ILP19
	MYDTYPE PIdx_19=threadIdx.x+(WARPSIZE*18);
#endif
#ifdef ILP20
	MYDTYPE PIdx_20=threadIdx.x+(WARPSIZE*19);
#endif
#ifdef ILP21
	MYDTYPE PIdx_21=threadIdx.x+(WARPSIZE*20);
#endif
#ifdef ILP22
	MYDTYPE PIdx_22=threadIdx.x+(WARPSIZE*21);
#endif
#ifdef ILP23
	MYDTYPE PIdx_23=threadIdx.x+(WARPSIZE*22);
#endif
#ifdef ILP24
	MYDTYPE PIdx_24=threadIdx.x+(WARPSIZE*23);
#endif
#ifdef ILP25
	MYDTYPE PIdx_25=threadIdx.x+(WARPSIZE*24);
#endif
#ifdef ILP26
	MYDTYPE PIdx_26=threadIdx.x+(WARPSIZE*25);
#endif
#ifdef ILP27
	MYDTYPE PIdx_27=threadIdx.x+(WARPSIZE*26);
#endif
#ifdef ILP28
	MYDTYPE PIdx_28=threadIdx.x+(WARPSIZE*27);
#endif
#ifdef ILP29
	MYDTYPE PIdx_29=threadIdx.x+(WARPSIZE*28);
#endif
#ifdef ILP30
	MYDTYPE PIdx_30=threadIdx.x+(WARPSIZE*29);
#endif
#ifdef ILP31
	MYDTYPE PIdx_31=threadIdx.x+(WARPSIZE*30);
#endif
#ifdef ILP32
	MYDTYPE PIdx_32=threadIdx.x+(WARPSIZE*31);
#endif
	MYDTYPE i,j,CurJ,CurB,t, CurLevel;
	MYDTYPE JumctionI;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless?
	bHP[PIdx]=0; // might be useless?
	bHP[PIdx_2]=0; // might be useless?
#ifdef ILP3
	bHP[PIdx_3]=0; // might be useless?
#endif
#ifdef ILP4
	bHP[PIdx_4]=0; // might be useless?
#endif
#ifdef ILP5
	bHP[PIdx_5]=0; // might be useless?
#endif
#ifdef ILP6
	bHP[PIdx_6]=0; // might be useless?
#endif
#ifdef ILP7
	bHP[PIdx_7]=0; // might be useless?
#endif
#ifdef ILP8
	bHP[PIdx_8]=0; // might be useless?
#endif
#ifdef ILP9
	bHP[PIdx_9]=0; // might be useless?
#endif
#ifdef ILP10
	bHP[PIdx_10]=0; // might be useless?
#endif
#ifdef ILP11
	bHP[PIdx_11]=0; // might be useless?
#endif
#ifdef ILP12
	bHP[PIdx_12]=0; // might be useless?
#endif
#ifdef ILP13
	bHP[PIdx_13]=0; // might be useless?
#endif
#ifdef ILP14
	bHP[PIdx_14]=0; // might be useless?
#endif
#ifdef ILP15
	bHP[PIdx_15]=0; // might be useless?
#endif
#ifdef ILP16
	bHP[PIdx_16]=0; // might be useless?
#endif
#ifdef ILP17
	bHP[PIdx_17]=0; // might be useless?
#endif
#ifdef ILP18
	bHP[PIdx_18]=0; // might be useless?
#endif
#ifdef ILP19
	bHP[PIdx_19]=0; // might be useless?
#endif
#ifdef ILP20
	bHP[PIdx_20]=0; // might be useless?
#endif
#ifdef ILP21
	bHP[PIdx_21]=0; // might be useless?
#endif
#ifdef ILP22
	bHP[PIdx_22]=0; // might be useless?
#endif
#ifdef ILP23
	bHP[PIdx_23]=0; // might be useless?
#endif
#ifdef ILP24
	bHP[PIdx_24]=0; // might be useless?
#endif
#ifdef ILP25
	bHP[PIdx_25]=0; // might be useless?
#endif
#ifdef ILP26
	bHP[PIdx_26]=0; // might be useless?
#endif
#ifdef ILP27
	bHP[PIdx_27]=0; // might be useless?
#endif
#ifdef ILP28
	bHP[PIdx_28]=0; // might be useless?
#endif
#ifdef ILP29
	bHP[PIdx_29]=0; // might be useless?
#endif
#ifdef ILP30
	bHP[PIdx_30]=0; // might be useless?
#endif
#ifdef ILP31
	bHP[PIdx_31]=0; // might be useless?
#endif
#ifdef ILP32
	bHP[PIdx_32]=0; // might be useless?
#endif

	// for CurLevel=Depth:-1:0
	for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
		//     Run all independent set for this level, in parallel
		// for JumctionI=find(Level==CurLevel) % in parallel
		JumctionI=InMat.CompByLevel32[CurLevel*WARPSIZE+PIdx]-1;
		// for i=(SegEndI(JumctionI)):-1:(SegStartI(JumctionI)-1)
		for(i=InMat.SegEndI[JumctionI]-2;i>=InMat.SegStartI[JumctionI]-2;i--) {
			// k=KsB(i+1);
			MYFTYPE k=KsB[i+1];
			// bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
			bHP[i]=(uHP[i]-bHP[k]*InMat.[i])/uHP[i];
		}
	}
}
*/
__device__ void BkSub(HMat InMat, MYFTYPE* PX, MYFTYPE* PF,MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE LognDepth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE PIdx_2=threadIdx.x+WARPSIZE;
#ifdef ILP3
	MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#endif
#ifdef ILP4
	MYDTYPE PIdx_4=threadIdx.x+(WARPSIZE*3);
#endif
#ifdef ILP5
	MYDTYPE PIdx_5=threadIdx.x+(WARPSIZE*4);
#endif
#ifdef ILP6
	MYDTYPE PIdx_6=threadIdx.x+(WARPSIZE*5);
#endif
#ifdef ILP7
	MYDTYPE PIdx_7=threadIdx.x+(WARPSIZE*6);
#endif
#ifdef ILP8
	MYDTYPE PIdx_8=threadIdx.x+(WARPSIZE*7);
#endif
#ifdef ILP9
	MYDTYPE PIdx_9=threadIdx.x+(WARPSIZE*8);
#endif
#ifdef ILP10
	MYDTYPE PIdx_10=threadIdx.x+(WARPSIZE*9);
#endif
#ifdef ILP11
	MYDTYPE PIdx_11=threadIdx.x+(WARPSIZE*10);
#endif
#ifdef ILP12
	MYDTYPE PIdx_12=threadIdx.x+(WARPSIZE*11);
#endif
#ifdef ILP13
	MYDTYPE PIdx_13=threadIdx.x+(WARPSIZE*12);
#endif
#ifdef ILP14
	MYDTYPE PIdx_14=threadIdx.x+(WARPSIZE*13);
#endif
#ifdef ILP15
	MYDTYPE PIdx_15=threadIdx.x+(WARPSIZE*14);
#endif
#ifdef ILP16
	MYDTYPE PIdx_16=threadIdx.x+(WARPSIZE*15);
#endif
#ifdef ILP17
	MYDTYPE PIdx_17=threadIdx.x+(WARPSIZE*16);
#endif
#ifdef ILP18
	MYDTYPE PIdx_18=threadIdx.x+(WARPSIZE*17);
#endif
#ifdef ILP19
	MYDTYPE PIdx_19=threadIdx.x+(WARPSIZE*18);
#endif
#ifdef ILP20
	MYDTYPE PIdx_20=threadIdx.x+(WARPSIZE*19);
#endif
#ifdef ILP21
	MYDTYPE PIdx_21=threadIdx.x+(WARPSIZE*20);
#endif
#ifdef ILP22
	MYDTYPE PIdx_22=threadIdx.x+(WARPSIZE*21);
#endif
#ifdef ILP23
	MYDTYPE PIdx_23=threadIdx.x+(WARPSIZE*22);
#endif
#ifdef ILP24
	MYDTYPE PIdx_24=threadIdx.x+(WARPSIZE*23);
#endif
#ifdef ILP25
	MYDTYPE PIdx_25=threadIdx.x+(WARPSIZE*24);
#endif
#ifdef ILP26
	MYDTYPE PIdx_26=threadIdx.x+(WARPSIZE*25);
#endif
#ifdef ILP27
	MYDTYPE PIdx_27=threadIdx.x+(WARPSIZE*26);
#endif
#ifdef ILP28
	MYDTYPE PIdx_28=threadIdx.x+(WARPSIZE*27);
#endif
#ifdef ILP29
	MYDTYPE PIdx_29=threadIdx.x+(WARPSIZE*28);
#endif
#ifdef ILP30
	MYDTYPE PIdx_30=threadIdx.x+(WARPSIZE*29);
#endif
#ifdef ILP31
	MYDTYPE PIdx_31=threadIdx.x+(WARPSIZE*30);
#endif
#ifdef ILP32
	MYDTYPE PIdx_32=threadIdx.x+(WARPSIZE*31);
#endif
	MYDTYPE i;
	MYDTYPE NextID;
	MYDTYPE NextID_2;
#ifdef ILP3
	MYDTYPE NextID_3;
#endif
#ifdef ILP4
	MYDTYPE NextID_4;
#endif
#ifdef ILP5
	MYDTYPE NextID_5;
#endif
#ifdef ILP6
	MYDTYPE NextID_6;
#endif
#ifdef ILP7
	MYDTYPE NextID_7;
#endif
#ifdef ILP8
	MYDTYPE NextID_8;
#endif
#ifdef ILP9
	MYDTYPE NextID_9;
#endif
#ifdef ILP10
	MYDTYPE NextID_10;
#endif
#ifdef ILP11
	MYDTYPE NextID_11;
#endif
#ifdef ILP12
	MYDTYPE NextID_12;
#endif
#ifdef ILP13
	MYDTYPE NextID_13;
#endif
#ifdef ILP14
	MYDTYPE NextID_14;
#endif
#ifdef ILP15
	MYDTYPE NextID_15;
#endif
#ifdef ILP16
	MYDTYPE NextID_16;
#endif
#ifdef ILP17
	MYDTYPE NextID_17;
#endif
#ifdef ILP18
	MYDTYPE NextID_18;
#endif
#ifdef ILP19
	MYDTYPE NextID_19;
#endif
#ifdef ILP20
	MYDTYPE NextID_20;
#endif
#ifdef ILP21
	MYDTYPE NextID_21;
#endif
#ifdef ILP22
	MYDTYPE NextID_22;
#endif
#ifdef ILP23
	MYDTYPE NextID_23;
#endif
#ifdef ILP24
	MYDTYPE NextID_24;
#endif
#ifdef ILP25
	MYDTYPE NextID_25;
#endif
#ifdef ILP26
	MYDTYPE NextID_26;
#endif
#ifdef ILP27
	MYDTYPE NextID_27;
#endif
#ifdef ILP28
	MYDTYPE NextID_28;
#endif
#ifdef ILP29
	MYDTYPE NextID_29;
#endif
#ifdef ILP30
	MYDTYPE NextID_30;
#endif
#ifdef ILP31
	MYDTYPE NextID_31;
#endif
#ifdef ILP32
	MYDTYPE NextID_32;
#endif
	
	PX=bHP;
	PF=uHP;
	PX[PIdx]=PX[PIdx]/PF[PIdx];
	PX[PIdx_2]=PX[PIdx_2]/PF[PIdx_2];
#ifdef ILP3
	PX[PIdx_3]=PX[PIdx_3]/PF[PIdx_3];
#endif
#ifdef ILP4
	PX[PIdx_4]=PX[PIdx_4]/PF[PIdx_4];
#endif
#ifdef ILP5
	PX[PIdx_5]=PX[PIdx_5]/PF[PIdx_5];
#endif
#ifdef ILP6
	PX[PIdx_6]=PX[PIdx_6]/PF[PIdx_6];
#endif
#ifdef ILP7
	PX[PIdx_7]=PX[PIdx_7]/PF[PIdx_7];
#endif
#ifdef ILP8
	PX[PIdx_8]=PX[PIdx_8]/PF[PIdx_8];
#endif
#ifdef ILP9
	PX[PIdx_9]=PX[PIdx_9]/PF[PIdx_9];
#endif
#ifdef ILP10
	PX[PIdx_10]=PX[PIdx_10]/PF[PIdx_10];
#endif
#ifdef ILP11
	PX[PIdx_11]=PX[PIdx_11]/PF[PIdx_11];
#endif
#ifdef ILP12
	PX[PIdx_12]=PX[PIdx_12]/PF[PIdx_12];
#endif
#ifdef ILP13
	PX[PIdx_13]=PX[PIdx_13]/PF[PIdx_13];
#endif
#ifdef ILP14
	PX[PIdx_14]=PX[PIdx_14]/PF[PIdx_14];
#endif
#ifdef ILP15
	PX[PIdx_15]=PX[PIdx_15]/PF[PIdx_15];
#endif
#ifdef ILP16
	PX[PIdx_16]=PX[PIdx_16]/PF[PIdx_16];
#endif
#ifdef ILP17
	PX[PIdx_17]=PX[PIdx_17]/PF[PIdx_17];
#endif
#ifdef ILP18
	PX[PIdx_18]=PX[PIdx_18]/PF[PIdx_18];
#endif
#ifdef ILP19
	PX[PIdx_19]=PX[PIdx_19]/PF[PIdx_19];
#endif
#ifdef ILP20
	PX[PIdx_20]=PX[PIdx_20]/PF[PIdx_20];
#endif
#ifdef ILP21
	PX[PIdx_21]=PX[PIdx_21]/PF[PIdx_21];
#endif
#ifdef ILP22
	PX[PIdx_22]=PX[PIdx_22]/PF[PIdx_22];
#endif
#ifdef ILP23
	PX[PIdx_23]=PX[PIdx_23]/PF[PIdx_23];
#endif
#ifdef ILP24
	PX[PIdx_24]=PX[PIdx_24]/PF[PIdx_24];
#endif
#ifdef ILP25
	PX[PIdx_25]=PX[PIdx_25]/PF[PIdx_25];
#endif
#ifdef ILP26
	PX[PIdx_26]=PX[PIdx_26]/PF[PIdx_26];
#endif
#ifdef ILP27
	PX[PIdx_27]=PX[PIdx_27]/PF[PIdx_27];
#endif
#ifdef ILP28
	PX[PIdx_28]=PX[PIdx_28]/PF[PIdx_28];
#endif
#ifdef ILP29
	PX[PIdx_29]=PX[PIdx_29]/PF[PIdx_29];
#endif
#ifdef ILP30
	PX[PIdx_30]=PX[PIdx_30]/PF[PIdx_30];
#endif
#ifdef ILP31
	PX[PIdx_31]=PX[PIdx_31]/PF[PIdx_31];
#endif
#ifdef ILP32
	PX[PIdx_32]=PX[PIdx_32]/PF[PIdx_32];
#endif
	PF[PIdx]=-InMat.f[PIdx]/PF[PIdx];
	PF[PIdx_2]=-InMat.f[PIdx_2]/PF[PIdx_2];
#ifdef ILP3
	PF[PIdx_3]=-InMat.f[PIdx_3]/PF[PIdx_3];
#endif
#ifdef ILP4
	PF[PIdx_4]=-InMat.f[PIdx_4]/PF[PIdx_4];
#endif
#ifdef ILP5
	PF[PIdx_5]=-InMat.f[PIdx_5]/PF[PIdx_5];
#endif
#ifdef ILP6
	PF[PIdx_6]=-InMat.f[PIdx_6]/PF[PIdx_6];
#endif
#ifdef ILP7
	PF[PIdx_7]=-InMat.f[PIdx_7]/PF[PIdx_7];
#endif
#ifdef ILP8
	PF[PIdx_8]=-InMat.f[PIdx_8]/PF[PIdx_8];
#endif
#ifdef ILP9
	PF[PIdx_9]=-InMat.f[PIdx_9]/PF[PIdx_9];
#endif
#ifdef ILP10
	PF[PIdx_10]=-InMat.f[PIdx_10]/PF[PIdx_10];
#endif
#ifdef ILP11
	PF[PIdx_11]=-InMat.f[PIdx_11]/PF[PIdx_11];
#endif
#ifdef ILP12
	PF[PIdx_12]=-InMat.f[PIdx_12]/PF[PIdx_12];
#endif
#ifdef ILP13
	PF[PIdx_13]=-InMat.f[PIdx_13]/PF[PIdx_13];
#endif
#ifdef ILP14
	PF[PIdx_14]=-InMat.f[PIdx_14]/PF[PIdx_14];
#endif
#ifdef ILP15
	PF[PIdx_15]=-InMat.f[PIdx_15]/PF[PIdx_15];
#endif
#ifdef ILP16
	PF[PIdx_16]=-InMat.f[PIdx_16]/PF[PIdx_16];
#endif
#ifdef ILP17
	PF[PIdx_17]=-InMat.f[PIdx_17]/PF[PIdx_17];
#endif
#ifdef ILP18
	PF[PIdx_18]=-InMat.f[PIdx_18]/PF[PIdx_18];
#endif
#ifdef ILP19
	PF[PIdx_19]=-InMat.f[PIdx_19]/PF[PIdx_19];
#endif
#ifdef ILP20
	PF[PIdx_20]=-InMat.f[PIdx_20]/PF[PIdx_20];
#endif
#ifdef ILP21
	PF[PIdx_21]=-InMat.f[PIdx_21]/PF[PIdx_21];
#endif
#ifdef ILP22
	PF[PIdx_22]=-InMat.f[PIdx_22]/PF[PIdx_22];
#endif
#ifdef ILP23
	PF[PIdx_23]=-InMat.f[PIdx_23]/PF[PIdx_23];
#endif
#ifdef ILP24
	PF[PIdx_24]=-InMat.f[PIdx_24]/PF[PIdx_24];
#endif
#ifdef ILP25
	PF[PIdx_25]=-InMat.f[PIdx_25]/PF[PIdx_25];
#endif
#ifdef ILP26
	PF[PIdx_26]=-InMat.f[PIdx_26]/PF[PIdx_26];
#endif
#ifdef ILP27
	PF[PIdx_27]=-InMat.f[PIdx_27]/PF[PIdx_27];
#endif
#ifdef ILP28
	PF[PIdx_28]=-InMat.f[PIdx_28]/PF[PIdx_28];
#endif
#ifdef ILP29
	PF[PIdx_29]=-InMat.f[PIdx_29]/PF[PIdx_29];
#endif
#ifdef ILP30
	PF[PIdx_30]=-InMat.f[PIdx_30]/PF[PIdx_30];
#endif
#ifdef ILP31
	PF[PIdx_31]=-InMat.f[PIdx_31]/PF[PIdx_31];
#endif
#ifdef ILP32
	PF[PIdx_32]=-InMat.f[PIdx_32]/PF[PIdx_32];
#endif

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		NextID=InMat.FIdxs[i*InMat.N+PIdx]-1;
		NextID_2=InMat.FIdxs[i*InMat.N+PIdx_2]-1;
#ifdef ILP3
		NextID_3=InMat.FIdxs[i*InMat.N+PIdx_3]-1;
#endif
#ifdef ILP4
		NextID_4=InMat.FIdxs[i*InMat.N+PIdx_4]-1;
#endif
#ifdef ILP5
		NextID_5=InMat.FIdxs[i*InMat.N+PIdx_5]-1;
#endif
#ifdef ILP6
		NextID_6=InMat.FIdxs[i*InMat.N+PIdx_6]-1;
#endif
#ifdef ILP7
		NextID_7=InMat.FIdxs[i*InMat.N+PIdx_7]-1;
#endif
#ifdef ILP8
		NextID_8=InMat.FIdxs[i*InMat.N+PIdx_8]-1;
#endif
#ifdef ILP9
		NextID_9=InMat.FIdxs[i*InMat.N+PIdx_9]-1;
#endif
#ifdef ILP10
		NextID_10=InMat.FIdxs[i*InMat.N+PIdx_10]-1;
#endif
#ifdef ILP11
		NextID_11=InMat.FIdxs[i*InMat.N+PIdx_11]-1;
#endif
#ifdef ILP12
		NextID_12=InMat.FIdxs[i*InMat.N+PIdx_12]-1;
#endif
#ifdef ILP13
		NextID_13=InMat.FIdxs[i*InMat.N+PIdx_13]-1;
#endif
#ifdef ILP14
		NextID_14=InMat.FIdxs[i*InMat.N+PIdx_14]-1;
#endif
#ifdef ILP15
		NextID_15=InMat.FIdxs[i*InMat.N+PIdx_15]-1;
#endif
#ifdef ILP16
		NextID_16=InMat.FIdxs[i*InMat.N+PIdx_16]-1;
#endif
#ifdef ILP17
		NextID_17=InMat.FIdxs[i*InMat.N+PIdx_17]-1;
#endif
#ifdef ILP18
		NextID_18=InMat.FIdxs[i*InMat.N+PIdx_18]-1;
#endif
#ifdef ILP19
		NextID_19=InMat.FIdxs[i*InMat.N+PIdx_19]-1;
#endif
#ifdef ILP20
		NextID_20=InMat.FIdxs[i*InMat.N+PIdx_20]-1;
#endif
#ifdef ILP21
		NextID_21=InMat.FIdxs[i*InMat.N+PIdx_21]-1;
#endif
#ifdef ILP22
		NextID_22=InMat.FIdxs[i*InMat.N+PIdx_22]-1;
#endif
#ifdef ILP23
		NextID_23=InMat.FIdxs[i*InMat.N+PIdx_23]-1;
#endif
#ifdef ILP24
		NextID_24=InMat.FIdxs[i*InMat.N+PIdx_24]-1;
#endif
#ifdef ILP25
		NextID_25=InMat.FIdxs[i*InMat.N+PIdx_25]-1;
#endif
#ifdef ILP26
		NextID_26=InMat.FIdxs[i*InMat.N+PIdx_26]-1;
#endif
#ifdef ILP27
		NextID_27=InMat.FIdxs[i*InMat.N+PIdx_27]-1;
#endif
#ifdef ILP28
		NextID_28=InMat.FIdxs[i*InMat.N+PIdx_28]-1;
#endif
#ifdef ILP29
		NextID_29=InMat.FIdxs[i*InMat.N+PIdx_29]-1;
#endif
#ifdef ILP30
		NextID_30=InMat.FIdxs[i*InMat.N+PIdx_30]-1;
#endif
#ifdef ILP31
		NextID_31=InMat.FIdxs[i*InMat.N+PIdx_31]-1;
#endif
#ifdef ILP32
		NextID_32=InMat.FIdxs[i*InMat.N+PIdx_32]-1;
#endif
		MYFTYPE OldPXj=PX[PIdx];
		MYFTYPE OldPXj_2=PX[PIdx_2];
#ifdef ILP3
		MYFTYPE OldPXj_3=PX[PIdx_3];
#endif
#ifdef ILP4
		MYFTYPE OldPXj_4=PX[PIdx_4];
#endif
#ifdef ILP5
		MYFTYPE OldPXj_5=PX[PIdx_5];
#endif
#ifdef ILP6
		MYFTYPE OldPXj_6=PX[PIdx_6];
#endif
#ifdef ILP7
		MYFTYPE OldPXj_7=PX[PIdx_7];
#endif
#ifdef ILP8
		MYFTYPE OldPXj_8=PX[PIdx_8];
#endif
#ifdef ILP9
		MYFTYPE OldPXj_9=PX[PIdx_9];
#endif
#ifdef ILP10
		MYFTYPE OldPXj_10=PX[PIdx_10];
#endif
#ifdef ILP11
		MYFTYPE OldPXj_11=PX[PIdx_11];
#endif
#ifdef ILP12
		MYFTYPE OldPXj_12=PX[PIdx_12];
#endif
#ifdef ILP13
		MYFTYPE OldPXj_13=PX[PIdx_13];
#endif
#ifdef ILP14
		MYFTYPE OldPXj_14=PX[PIdx_14];
#endif
#ifdef ILP15
		MYFTYPE OldPXj_15=PX[PIdx_15];
#endif
#ifdef ILP16
		MYFTYPE OldPXj_16=PX[PIdx_16];
#endif
#ifdef ILP17
		MYFTYPE OldPXj_17=PX[PIdx_17];
#endif
#ifdef ILP18
		MYFTYPE OldPXj_18=PX[PIdx_18];
#endif
#ifdef ILP19
		MYFTYPE OldPXj_19=PX[PIdx_19];
#endif
#ifdef ILP20
		MYFTYPE OldPXj_20=PX[PIdx_20];
#endif
#ifdef ILP21
		MYFTYPE OldPXj_21=PX[PIdx_21];
#endif
#ifdef ILP22
		MYFTYPE OldPXj_22=PX[PIdx_22];
#endif
#ifdef ILP23
		MYFTYPE OldPXj_23=PX[PIdx_23];
#endif
#ifdef ILP24
		MYFTYPE OldPXj_24=PX[PIdx_24];
#endif
#ifdef ILP25
		MYFTYPE OldPXj_25=PX[PIdx_25];
#endif
#ifdef ILP26
		MYFTYPE OldPXj_26=PX[PIdx_26];
#endif
#ifdef ILP27
		MYFTYPE OldPXj_27=PX[PIdx_27];
#endif
#ifdef ILP28
		MYFTYPE OldPXj_28=PX[PIdx_28];
#endif
#ifdef ILP29
		MYFTYPE OldPXj_29=PX[PIdx_29];
#endif
#ifdef ILP30
		MYFTYPE OldPXj_30=PX[PIdx_30];
#endif
#ifdef ILP31
		MYFTYPE OldPXj_31=PX[PIdx_31];
#endif
#ifdef ILP32
		MYFTYPE OldPXj_32=PX[PIdx_32];
#endif
		MYFTYPE OldPXNextID=PX[NextID];
		MYFTYPE OldPXNextID_2=PX[NextID_2];
#ifdef ILP3
		MYFTYPE OldPXNextID_3=PX[NextID_3];
#endif
#ifdef ILP4
		MYFTYPE OldPXNextID_4=PX[NextID_4];
#endif
#ifdef ILP5
		MYFTYPE OldPXNextID_5=PX[NextID_5];
#endif
#ifdef ILP6
		MYFTYPE OldPXNextID_6=PX[NextID_6];
#endif
#ifdef ILP7
		MYFTYPE OldPXNextID_7=PX[NextID_7];
#endif
#ifdef ILP8
		MYFTYPE OldPXNextID_8=PX[NextID_8];
#endif
#ifdef ILP9
		MYFTYPE OldPXNextID_9=PX[NextID_9];
#endif
#ifdef ILP10
		MYFTYPE OldPXNextID_10=PX[NextID_10];
#endif
#ifdef ILP11
		MYFTYPE OldPXNextID_11=PX[NextID_11];
#endif
#ifdef ILP12
		MYFTYPE OldPXNextID_12=PX[NextID_12];
#endif
#ifdef ILP13
		MYFTYPE OldPXNextID_13=PX[NextID_13];
#endif
#ifdef ILP14
		MYFTYPE OldPXNextID_14=PX[NextID_14];
#endif
#ifdef ILP15
		MYFTYPE OldPXNextID_15=PX[NextID_15];
#endif
#ifdef ILP16
		MYFTYPE OldPXNextID_16=PX[NextID_16];
#endif
#ifdef ILP17
		MYFTYPE OldPXNextID_17=PX[NextID_17];
#endif
#ifdef ILP18
		MYFTYPE OldPXNextID_18=PX[NextID_18];
#endif
#ifdef ILP19
		MYFTYPE OldPXNextID_19=PX[NextID_19];
#endif
#ifdef ILP20
		MYFTYPE OldPXNextID_20=PX[NextID_20];
#endif
#ifdef ILP21
		MYFTYPE OldPXNextID_21=PX[NextID_21];
#endif
#ifdef ILP22
		MYFTYPE OldPXNextID_22=PX[NextID_22];
#endif
#ifdef ILP23
		MYFTYPE OldPXNextID_23=PX[NextID_23];
#endif
#ifdef ILP24
		MYFTYPE OldPXNextID_24=PX[NextID_24];
#endif
#ifdef ILP25
		MYFTYPE OldPXNextID_25=PX[NextID_25];
#endif
#ifdef ILP26
		MYFTYPE OldPXNextID_26=PX[NextID_26];
#endif
#ifdef ILP27
		MYFTYPE OldPXNextID_27=PX[NextID_27];
#endif
#ifdef ILP28
		MYFTYPE OldPXNextID_28=PX[NextID_28];
#endif
#ifdef ILP29
		MYFTYPE OldPXNextID_29=PX[NextID_29];
#endif
#ifdef ILP30
		MYFTYPE OldPXNextID_30=PX[NextID_30];
#endif
#ifdef ILP31
		MYFTYPE OldPXNextID_31=PX[NextID_31];
#endif
#ifdef ILP32
		MYFTYPE OldPXNextID_32=PX[NextID_32];
#endif
		PX[PIdx]=OldPXj+OldPXNextID*PF[PIdx];
		PX[PIdx_2]=OldPXj_2+OldPXNextID_2*PF[PIdx_2];
#ifdef ILP3
		PX[PIdx_3]=OldPXj_3+OldPXNextID_3*PF[PIdx_3];
#endif
#ifdef ILP4
		PX[PIdx_4]=OldPXj_4+OldPXNextID_4*PF[PIdx_4];
#endif
#ifdef ILP5
		PX[PIdx_5]=OldPXj_5+OldPXNextID_5*PF[PIdx_5];
#endif
#ifdef ILP6
		PX[PIdx_6]=OldPXj_6+OldPXNextID_6*PF[PIdx_6];
#endif
#ifdef ILP7
		PX[PIdx_7]=OldPXj_7+OldPXNextID_7*PF[PIdx_7];
#endif
#ifdef ILP8
		PX[PIdx_8]=OldPXj_8+OldPXNextID_8*PF[PIdx_8];
#endif
#ifdef ILP9
		PX[PIdx_9]=OldPXj_9+OldPXNextID_9*PF[PIdx_9];
#endif
#ifdef ILP10
		PX[PIdx_10]=OldPXj_10+OldPXNextID_10*PF[PIdx_10];
#endif
#ifdef ILP11
		PX[PIdx_11]=OldPXj_11+OldPXNextID_11*PF[PIdx_11];
#endif
#ifdef ILP12
		PX[PIdx_12]=OldPXj_12+OldPXNextID_12*PF[PIdx_12];
#endif
#ifdef ILP13
		PX[PIdx_13]=OldPXj_13+OldPXNextID_13*PF[PIdx_13];
#endif
#ifdef ILP14
		PX[PIdx_14]=OldPXj_14+OldPXNextID_14*PF[PIdx_14];
#endif
#ifdef ILP15
		PX[PIdx_15]=OldPXj_15+OldPXNextID_15*PF[PIdx_15];
#endif
#ifdef ILP16
		PX[PIdx_16]=OldPXj_16+OldPXNextID_16*PF[PIdx_16];
#endif
#ifdef ILP17
		PX[PIdx_17]=OldPXj_17+OldPXNextID_17*PF[PIdx_17];
#endif
#ifdef ILP18
		PX[PIdx_18]=OldPXj_18+OldPXNextID_18*PF[PIdx_18];
#endif
#ifdef ILP19
		PX[PIdx_19]=OldPXj_19+OldPXNextID_19*PF[PIdx_19];
#endif
#ifdef ILP20
		PX[PIdx_20]=OldPXj_20+OldPXNextID_20*PF[PIdx_20];
#endif
#ifdef ILP21
		PX[PIdx_21]=OldPXj_21+OldPXNextID_21*PF[PIdx_21];
#endif
#ifdef ILP22
		PX[PIdx_22]=OldPXj_22+OldPXNextID_22*PF[PIdx_22];
#endif
#ifdef ILP23
		PX[PIdx_23]=OldPXj_23+OldPXNextID_23*PF[PIdx_23];
#endif
#ifdef ILP24
		PX[PIdx_24]=OldPXj_24+OldPXNextID_24*PF[PIdx_24];
#endif
#ifdef ILP25
		PX[PIdx_25]=OldPXj_25+OldPXNextID_25*PF[PIdx_25];
#endif
#ifdef ILP26
		PX[PIdx_26]=OldPXj_26+OldPXNextID_26*PF[PIdx_26];
#endif
#ifdef ILP27
		PX[PIdx_27]=OldPXj_27+OldPXNextID_27*PF[PIdx_27];
#endif
#ifdef ILP28
		PX[PIdx_28]=OldPXj_28+OldPXNextID_28*PF[PIdx_28];
#endif
#ifdef ILP29
		PX[PIdx_29]=OldPXj_29+OldPXNextID_29*PF[PIdx_29];
#endif
#ifdef ILP30
		PX[PIdx_30]=OldPXj_30+OldPXNextID_30*PF[PIdx_30];
#endif
#ifdef ILP31
		PX[PIdx_31]=OldPXj_31+OldPXNextID_31*PF[PIdx_31];
#endif
#ifdef ILP32
		PX[PIdx_32]=OldPXj_32+OldPXNextID_32*PF[PIdx_32];
#endif
		// PX[j]=PX[j]+PX[NextID]*PF[j];
		MYFTYPE OldPFj=PF[PIdx];
		MYFTYPE OldPFj_2=PF[PIdx_2];
#ifdef ILP3
		MYFTYPE OldPFj_3=PF[PIdx_3];
#endif
#ifdef ILP4
		MYFTYPE OldPFj_4=PF[PIdx_4];
#endif
#ifdef ILP5
		MYFTYPE OldPFj_5=PF[PIdx_5];
#endif
#ifdef ILP6
		MYFTYPE OldPFj_6=PF[PIdx_6];
#endif
#ifdef ILP7
		MYFTYPE OldPFj_7=PF[PIdx_7];
#endif
#ifdef ILP8
		MYFTYPE OldPFj_8=PF[PIdx_8];
#endif
#ifdef ILP9
		MYFTYPE OldPFj_9=PF[PIdx_9];
#endif
#ifdef ILP10
		MYFTYPE OldPFj_10=PF[PIdx_10];
#endif
#ifdef ILP11
		MYFTYPE OldPFj_11=PF[PIdx_11];
#endif
#ifdef ILP12
		MYFTYPE OldPFj_12=PF[PIdx_12];
#endif
#ifdef ILP13
		MYFTYPE OldPFj_13=PF[PIdx_13];
#endif
#ifdef ILP14
		MYFTYPE OldPFj_14=PF[PIdx_14];
#endif
#ifdef ILP15
		MYFTYPE OldPFj_15=PF[PIdx_15];
#endif
#ifdef ILP16
		MYFTYPE OldPFj_16=PF[PIdx_16];
#endif
#ifdef ILP17
		MYFTYPE OldPFj_17=PF[PIdx_17];
#endif
#ifdef ILP18
		MYFTYPE OldPFj_18=PF[PIdx_18];
#endif
#ifdef ILP19
		MYFTYPE OldPFj_19=PF[PIdx_19];
#endif
#ifdef ILP20
		MYFTYPE OldPFj_20=PF[PIdx_20];
#endif
#ifdef ILP21
		MYFTYPE OldPFj_21=PF[PIdx_21];
#endif
#ifdef ILP22
		MYFTYPE OldPFj_22=PF[PIdx_22];
#endif
#ifdef ILP23
		MYFTYPE OldPFj_23=PF[PIdx_23];
#endif
#ifdef ILP24
		MYFTYPE OldPFj_24=PF[PIdx_24];
#endif
#ifdef ILP25
		MYFTYPE OldPFj_25=PF[PIdx_25];
#endif
#ifdef ILP26
		MYFTYPE OldPFj_26=PF[PIdx_26];
#endif
#ifdef ILP27
		MYFTYPE OldPFj_27=PF[PIdx_27];
#endif
#ifdef ILP28
		MYFTYPE OldPFj_28=PF[PIdx_28];
#endif
#ifdef ILP29
		MYFTYPE OldPFj_29=PF[PIdx_29];
#endif
#ifdef ILP30
		MYFTYPE OldPFj_30=PF[PIdx_30];
#endif
#ifdef ILP31
		MYFTYPE OldPFj_31=PF[PIdx_31];
#endif
#ifdef ILP32
		MYFTYPE OldPFj_32=PF[PIdx_32];
#endif
		MYFTYPE OldPFNextID=PF[NextID];
		MYFTYPE OldPFNextID_2=PF[NextID_2];
#ifdef ILP3
		MYFTYPE OldPFNextID_3=PF[NextID_3];
#endif
#ifdef ILP4
		MYFTYPE OldPFNextID_4=PF[NextID_4];
#endif
#ifdef ILP5
		MYFTYPE OldPFNextID_5=PF[NextID_5];
#endif
#ifdef ILP6
		MYFTYPE OldPFNextID_6=PF[NextID_6];
#endif
#ifdef ILP7
		MYFTYPE OldPFNextID_7=PF[NextID_7];
#endif
#ifdef ILP8
		MYFTYPE OldPFNextID_8=PF[NextID_8];
#endif
#ifdef ILP9
		MYFTYPE OldPFNextID_9=PF[NextID_9];
#endif
#ifdef ILP10
		MYFTYPE OldPFNextID_10=PF[NextID_10];
#endif
#ifdef ILP11
		MYFTYPE OldPFNextID_11=PF[NextID_11];
#endif
#ifdef ILP12
		MYFTYPE OldPFNextID_12=PF[NextID_12];
#endif
#ifdef ILP13
		MYFTYPE OldPFNextID_13=PF[NextID_13];
#endif
#ifdef ILP14
		MYFTYPE OldPFNextID_14=PF[NextID_14];
#endif
#ifdef ILP15
		MYFTYPE OldPFNextID_15=PF[NextID_15];
#endif
#ifdef ILP16
		MYFTYPE OldPFNextID_16=PF[NextID_16];
#endif
#ifdef ILP17
		MYFTYPE OldPFNextID_17=PF[NextID_17];
#endif
#ifdef ILP18
		MYFTYPE OldPFNextID_18=PF[NextID_18];
#endif
#ifdef ILP19
		MYFTYPE OldPFNextID_19=PF[NextID_19];
#endif
#ifdef ILP20
		MYFTYPE OldPFNextID_20=PF[NextID_20];
#endif
#ifdef ILP21
		MYFTYPE OldPFNextID_21=PF[NextID_21];
#endif
#ifdef ILP22
		MYFTYPE OldPFNextID_22=PF[NextID_22];
#endif
#ifdef ILP23
		MYFTYPE OldPFNextID_23=PF[NextID_23];
#endif
#ifdef ILP24
		MYFTYPE OldPFNextID_24=PF[NextID_24];
#endif
#ifdef ILP25
		MYFTYPE OldPFNextID_25=PF[NextID_25];
#endif
#ifdef ILP26
		MYFTYPE OldPFNextID_26=PF[NextID_26];
#endif
#ifdef ILP27
		MYFTYPE OldPFNextID_27=PF[NextID_27];
#endif
#ifdef ILP28
		MYFTYPE OldPFNextID_28=PF[NextID_28];
#endif
#ifdef ILP29
		MYFTYPE OldPFNextID_29=PF[NextID_29];
#endif
#ifdef ILP30
		MYFTYPE OldPFNextID_30=PF[NextID_30];
#endif
#ifdef ILP31
		MYFTYPE OldPFNextID_31=PF[NextID_31];
#endif
#ifdef ILP32
		MYFTYPE OldPFNextID_32=PF[NextID_32];
#endif
		// PF[j]=PF[j]*PF[NextID];
		PF[PIdx]=OldPFj*OldPFNextID;
		PF[PIdx_2]=OldPFj_2*OldPFNextID_2;
#ifdef ILP3
		PF[PIdx_3]=OldPFj_3*OldPFNextID_3;
#endif
#ifdef ILP4
		PF[PIdx_4]=OldPFj_4*OldPFNextID_4;
#endif
#ifdef ILP5
		PF[PIdx_5]=OldPFj_5*OldPFNextID_5;
#endif
#ifdef ILP6
		PF[PIdx_6]=OldPFj_6*OldPFNextID_6;
#endif
#ifdef ILP7
		PF[PIdx_7]=OldPFj_7*OldPFNextID_7;
#endif
#ifdef ILP8
		PF[PIdx_8]=OldPFj_8*OldPFNextID_8;
#endif
#ifdef ILP9
		PF[PIdx_9]=OldPFj_9*OldPFNextID_9;
#endif
#ifdef ILP10
		PF[PIdx_10]=OldPFj_10*OldPFNextID_10;
#endif
#ifdef ILP11
		PF[PIdx_11]=OldPFj_11*OldPFNextID_11;
#endif
#ifdef ILP12
		PF[PIdx_12]=OldPFj_12*OldPFNextID_12;
#endif
#ifdef ILP13
		PF[PIdx_13]=OldPFj_13*OldPFNextID_13;
#endif
#ifdef ILP14
		PF[PIdx_14]=OldPFj_14*OldPFNextID_14;
#endif
#ifdef ILP15
		PF[PIdx_15]=OldPFj_15*OldPFNextID_15;
#endif
#ifdef ILP16
		PF[PIdx_16]=OldPFj_16*OldPFNextID_16;
#endif
#ifdef ILP17
		PF[PIdx_17]=OldPFj_17*OldPFNextID_17;
#endif
#ifdef ILP18
		PF[PIdx_18]=OldPFj_18*OldPFNextID_18;
#endif
#ifdef ILP19
		PF[PIdx_19]=OldPFj_19*OldPFNextID_19;
#endif
#ifdef ILP20
		PF[PIdx_20]=OldPFj_20*OldPFNextID_20;
#endif
#ifdef ILP21
		PF[PIdx_21]=OldPFj_21*OldPFNextID_21;
#endif
#ifdef ILP22
		PF[PIdx_22]=OldPFj_22*OldPFNextID_22;
#endif
#ifdef ILP23
		PF[PIdx_23]=OldPFj_23*OldPFNextID_23;
#endif
#ifdef ILP24
		PF[PIdx_24]=OldPFj_24*OldPFNextID_24;
#endif
#ifdef ILP25
		PF[PIdx_25]=OldPFj_25*OldPFNextID_25;
#endif
#ifdef ILP26
		PF[PIdx_26]=OldPFj_26*OldPFNextID_26;
#endif
#ifdef ILP27
		PF[PIdx_27]=OldPFj_27*OldPFNextID_27;
#endif
#ifdef ILP28
		PF[PIdx_28]=OldPFj_28*OldPFNextID_28;
#endif
#ifdef ILP29
		PF[PIdx_29]=OldPFj_29*OldPFNextID_29;
#endif
#ifdef ILP30
		PF[PIdx_30]=OldPFj_30*OldPFNextID_30;
#endif
#ifdef ILP31
		PF[PIdx_31]=OldPFj_31*OldPFNextID_31;
#endif
#ifdef ILP32
		PF[PIdx_32]=OldPFj_32*OldPFNextID_32;
#endif
	}
}

int testCode(MYFTYPE* orig, MYFTYPE* sim,int N){
	int i=0,iErr,iRel;
	MYFTYPE absErr,relErr,currErr,currRel;
	absErr = abs(orig[0]-sim[0]);
	relErr = absErr/abs(orig[0]);
	for(i=0;i<N;i++) {
		currErr = abs(orig[i]-sim[i]);
		currRel = currErr/abs(orig[i]);
		if (currErr>absErr){
			absErr=currErr;
			iErr=i;
		}
		if (currRel>relErr){
			relErr=currRel;
			iRel=i;

		}
	}
	printf("Absolute Error is %g at the %d index and Relative Error is %g at the %d index\n",absErr,iErr,relErr,iRel);
	return 1;
}
__device__ MYFTYPE Kvtrap(MYFTYPE x, MYFTYPE y) {
	if (fabs(x/y) < 1e-6) {;
	return y*(1 - x/y/2);
	}
	else
	{;
	return x/(exp(x/y) - 1);
	};
};

__device__ void Krates(MYFTYPE v, MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau,MYFTYPE &ninf,MYFTYPE &ntau) {
	MYFTYPE  alpha, beta, sum, q10;
	;
	// q10=3^((celsius - 6.3)/10); YYY CHANGE THIS to pow
	q10=pow(3,((celsius - 6.3)/10));
	;
	alpha=.1 * Kvtrap(-(v+40),10);
	beta=4 * exp(-(v+65)/18);
	sum=alpha + beta;
	mtau=1/(q10*sum);
	minf=alpha/sum;
	;
	alpha=.07 * exp(-(v+65)/20);
	beta=1 / (exp(-(v+35)/10) + 1);
	sum=alpha + beta;
	htau=1/(q10*sum);
	hinf=alpha/sum;
	;
	alpha=.01*Kvtrap(-(v+55),10) ;
	beta=.125*exp(-(v+65)/80);
	sum=alpha + beta;
	ntau=1/(q10*sum);
	ninf=alpha/sum;
}

__device__ void KInitModel(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE &n) {
	MYFTYPE hinf,htau,minf,mtau,ninf,ntau;
	Krates(v, hinf,htau,minf,mtau,ninf,ntau);
	m=minf;
	h=hinf;
	n=ninf;
}

__device__ void KDerivModel(MYFTYPE dt, MYFTYPE v, MYFTYPE &m,MYFTYPE &h,MYFTYPE &n) {
	MYFTYPE hinf,htau,minf,mtau,ninf,ntau;
	Krates(v, hinf,htau,minf,mtau,ninf,ntau);
	m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
	h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
	n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
};

// YYY add sum conductivity float BreakpointModel(float &m,float &h,float &n,float gnabar,float gkbar,float gl,float el) {
// YYY add float v
// YYY change to void function
__device__ void KBreakpointModel(MYFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v, MYFTYPE m,MYFTYPE h,MYFTYPE n,MYFTYPE gnabar,MYFTYPE gkbar,MYFTYPE gl,MYFTYPE ena,MYFTYPE ek,MYFTYPE el) {
	float gk,gna;
	float ina,ik,il;
	gna=gnabar*m*m*m*h;
	ina=gna*(v-ena);
	gk=gkbar*n*n*n*n;
	ik=gk*(v-(ek));
	il=gl*(v-el);
	sumCurrents= ina+ik+il;
	sumConductivity= gl+gk+gna;
}



// RRR
__global__ void stEfork2TimeLoopGPUKernel(Stim stim, MYFTYPE* ParamsM, Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal)
{
	
	MYDTYPE StimID=threadIdx.y;
	MYFTYPE stimDel = stim.dels[StimID];
	MYFTYPE stimDur = stim.durs[StimID];
	MYFTYPE stimAmp = stim.amps[StimID];
	MYDTYPE stimLoc = stim.loc;
	MYFTYPE stimArea = stim.area;
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE PIdx_2=threadIdx.x+WARPSIZE;
#ifdef ILP3
	MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#endif
#ifdef ILP4
	MYDTYPE PIdx_4=threadIdx.x+(WARPSIZE*3);
#endif
#ifdef ILP5
	MYDTYPE PIdx_5=threadIdx.x+(WARPSIZE*4);
#endif
#ifdef ILP6
	MYDTYPE PIdx_6=threadIdx.x+(WARPSIZE*5);
#endif
#ifdef ILP7
	MYDTYPE PIdx_7=threadIdx.x+(WARPSIZE*6);
#endif
#ifdef ILP8
	MYDTYPE PIdx_8=threadIdx.x+(WARPSIZE*7);
#endif
#ifdef ILP9
	MYDTYPE PIdx_9=threadIdx.x+(WARPSIZE*8);
#endif
#ifdef ILP10
	MYDTYPE PIdx_10=threadIdx.x+(WARPSIZE*9);
#endif
#ifdef ILP11
	MYDTYPE PIdx_11=threadIdx.x+(WARPSIZE*10);
#endif
#ifdef ILP12
	MYDTYPE PIdx_12=threadIdx.x+(WARPSIZE*11);
#endif
#ifdef ILP13
	MYDTYPE PIdx_13=threadIdx.x+(WARPSIZE*12);
#endif
#ifdef ILP14
	MYDTYPE PIdx_14=threadIdx.x+(WARPSIZE*13);
#endif
#ifdef ILP15
	MYDTYPE PIdx_15=threadIdx.x+(WARPSIZE*14);
#endif
#ifdef ILP16
	MYDTYPE PIdx_16=threadIdx.x+(WARPSIZE*15);
#endif
#ifdef ILP17
	MYDTYPE PIdx_17=threadIdx.x+(WARPSIZE*16);
#endif
#ifdef ILP18
	MYDTYPE PIdx_18=threadIdx.x+(WARPSIZE*17);
#endif
#ifdef ILP19
	MYDTYPE PIdx_19=threadIdx.x+(WARPSIZE*18);
#endif
#ifdef ILP20
	MYDTYPE PIdx_20=threadIdx.x+(WARPSIZE*19);
#endif
#ifdef ILP21
	MYDTYPE PIdx_21=threadIdx.x+(WARPSIZE*20);
#endif
#ifdef ILP22
	MYDTYPE PIdx_22=threadIdx.x+(WARPSIZE*21);
#endif
#ifdef ILP23
	MYDTYPE PIdx_23=threadIdx.x+(WARPSIZE*22);
#endif
#ifdef ILP24
	MYDTYPE PIdx_24=threadIdx.x+(WARPSIZE*23);
#endif
#ifdef ILP25
	MYDTYPE PIdx_25=threadIdx.x+(WARPSIZE*24);
#endif
#ifdef ILP26
	MYDTYPE PIdx_26=threadIdx.x+(WARPSIZE*25);
#endif
#ifdef ILP27
	MYDTYPE PIdx_27=threadIdx.x+(WARPSIZE*26);
#endif
#ifdef ILP28
	MYDTYPE PIdx_28=threadIdx.x+(WARPSIZE*27);
#endif
#ifdef ILP29
	MYDTYPE PIdx_29=threadIdx.x+(WARPSIZE*28);
#endif
#ifdef ILP30
	MYDTYPE PIdx_30=threadIdx.x+(WARPSIZE*29);
#endif
#ifdef ILP31
	MYDTYPE PIdx_31=threadIdx.x+(WARPSIZE*30);
#endif
#ifdef ILP32
	MYDTYPE PIdx_32=threadIdx.x+(WARPSIZE*31);
#endif
	hipError_t cudaStatus;
	// for model RRR
	MYFTYPE t;
	MYFTYPE isModel,dVec, Vmid;
	MYFTYPE isModel_2, dVec_2, Vmid_2;
#ifdef ILP3
	MYFTYPE isModel_3, dVec_3, Vmid_3;
#endif
#ifdef ILP4
	MYFTYPE isModel_4, dVec_4, Vmid_4;
#endif
#ifdef ILP5
	MYFTYPE isModel_5, dVec_5, Vmid_5;
#endif
#ifdef ILP6
	MYFTYPE isModel_6, dVec_6, Vmid_6;
#endif
#ifdef ILP7
	MYFTYPE isModel_7, dVec_7, Vmid_7;
#endif
#ifdef ILP8
	MYFTYPE isModel_8, dVec_8, Vmid_8;
#endif
#ifdef ILP9
	MYFTYPE isModel_9, dVec_9, Vmid_9;
#endif
#ifdef ILP10
	MYFTYPE isModel_10, dVec_10, Vmid_10;
#endif
#ifdef ILP11
	MYFTYPE isModel_11, dVec_11, Vmid_11;
#endif
#ifdef ILP12
	MYFTYPE isModel_12, dVec_12, Vmid_12;
#endif
#ifdef ILP13
	MYFTYPE isModel_13, dVec_13, Vmid_13;
#endif
#ifdef ILP14
	MYFTYPE isModel_14, dVec_14, Vmid_14;
#endif
#ifdef ILP15
	MYFTYPE isModel_15, dVec_15, Vmid_15;
#endif
#ifdef ILP16
	MYFTYPE isModel_16, dVec_16, Vmid_16;
#endif
#ifdef ILP17
	MYFTYPE isModel_17, dVec_17, Vmid_17;
#endif
#ifdef ILP18
	MYFTYPE isModel_18, dVec_18, Vmid_18;
#endif
#ifdef ILP19
	MYFTYPE isModel_19, dVec_19, Vmid_19;
#endif
#ifdef ILP20
	MYFTYPE isModel_20, dVec_20, Vmid_20;
#endif
#ifdef ILP21
	MYFTYPE isModel_21, dVec_21, Vmid_21;
#endif
#ifdef ILP22
	MYFTYPE isModel_22, dVec_22, Vmid_22;
#endif
#ifdef ILP23
	MYFTYPE isModel_23, dVec_23, Vmid_23;
#endif
#ifdef ILP24
	MYFTYPE isModel_24, dVec_24, Vmid_24;
#endif
#ifdef ILP25
	MYFTYPE isModel_25, dVec_25, Vmid_25;
#endif
#ifdef ILP26
	MYFTYPE isModel_26, dVec_26, Vmid_26;
#endif
#ifdef ILP27
	MYFTYPE isModel_27, dVec_27, Vmid_27;
#endif
#ifdef ILP28
	MYFTYPE isModel_28, dVec_28, Vmid_28;
#endif
#ifdef ILP29
	MYFTYPE isModel_29, dVec_29, Vmid_29;
#endif
#ifdef ILP30
	MYFTYPE isModel_30, dVec_30, Vmid_30;
#endif
#ifdef ILP31
	MYFTYPE isModel_31, dVec_31, Vmid_31;
#endif
#ifdef ILP32
	MYFTYPE isModel_32, dVec_32, Vmid_32;
#endif
	MYFTYPE ModelStates[NSTATES];
	MYFTYPE ModelStates_2[NSTATES];
#ifdef ILP3
	MYFTYPE ModelStates_3[NSTATES];
#endif
#ifdef ILP4
	MYFTYPE ModelStates_4[NSTATES];
#endif
#ifdef ILP5
	MYFTYPE ModelStates_5[NSTATES];
#endif
#ifdef ILP6
	MYFTYPE ModelStates_6[NSTATES];
#endif
#ifdef ILP7
	MYFTYPE ModelStates_7[NSTATES];
#endif
#ifdef ILP8
	MYFTYPE ModelStates_8[NSTATES];
#endif
#ifdef ILP9
	MYFTYPE ModelStates_9[NSTATES];
#endif
#ifdef ILP10
	MYFTYPE ModelStates_10[NSTATES];
#endif
#ifdef ILP11
	MYFTYPE ModelStates_11[NSTATES];
#endif
#ifdef ILP12
	MYFTYPE ModelStates_12[NSTATES];
#endif
#ifdef ILP13
	MYFTYPE ModelStates_13[NSTATES];
#endif
#ifdef ILP14
	MYFTYPE ModelStates_14[NSTATES];
#endif
#ifdef ILP15
	MYFTYPE ModelStates_15[NSTATES];
#endif
#ifdef ILP16
	MYFTYPE ModelStates_16[NSTATES];
#endif
#ifdef ILP17
	MYFTYPE ModelStates_17[NSTATES];
#endif
#ifdef ILP18
	MYFTYPE ModelStates_18[NSTATES];
#endif
#ifdef ILP19
	MYFTYPE ModelStates_19[NSTATES];
#endif
#ifdef ILP20
	MYFTYPE ModelStates_20[NSTATES];
#endif
#ifdef ILP21
	MYFTYPE ModelStates_21[NSTATES];
#endif
#ifdef ILP22
	MYFTYPE ModelStates_22[NSTATES];
#endif
#ifdef ILP23
	MYFTYPE ModelStates_23[NSTATES];
#endif
#ifdef ILP24
	MYFTYPE ModelStates_24[NSTATES];
#endif
#ifdef ILP25
	MYFTYPE ModelStates_25[NSTATES];
#endif
#ifdef ILP26
	MYFTYPE ModelStates_26[NSTATES];
#endif
#ifdef ILP27
	MYFTYPE ModelStates_27[NSTATES];
#endif
#ifdef ILP28
	MYFTYPE ModelStates_28[NSTATES];
#endif
#ifdef ILP29
	MYFTYPE ModelStates_29[NSTATES];
#endif
#ifdef ILP30
	MYFTYPE ModelStates_30[NSTATES];
#endif
#ifdef ILP31
	MYFTYPE ModelStates_31[NSTATES];
#endif
#ifdef ILP32
	MYFTYPE ModelStates_32[NSTATES];
#endif
	MYFTYPE v=V[PIdx]; // RRR -65?
	MYFTYPE v_2=V[PIdx_2]; // RRR -65?
#ifdef ILP3
	MYFTYPE v_3=V[PIdx_3]; // RRR -65?
#endif
#ifdef ILP4
	MYFTYPE v_4=V[PIdx_4]; // RRR -65?
#endif
#ifdef ILP5
	MYFTYPE v_5=V[PIdx_5]; // RRR -65?
#endif
#ifdef ILP6
	MYFTYPE v_6=V[PIdx_6]; // RRR -65?
#endif
#ifdef ILP7
	MYFTYPE v_7=V[PIdx_7]; // RRR -65?
#endif
#ifdef ILP8
	MYFTYPE v_8=V[PIdx_8]; // RRR -65?
#endif
#ifdef ILP9
	MYFTYPE v_9=V[PIdx_9]; // RRR -65?
#endif
#ifdef ILP10
	MYFTYPE v_10=V[PIdx_10]; // RRR -65?
#endif
#ifdef ILP11
	MYFTYPE v_11=V[PIdx_11]; // RRR -65?
#endif
#ifdef ILP12
	MYFTYPE v_12=V[PIdx_12]; // RRR -65?
#endif
#ifdef ILP13
	MYFTYPE v_13=V[PIdx_13]; // RRR -65?
#endif
#ifdef ILP14
	MYFTYPE v_14=V[PIdx_14]; // RRR -65?
#endif
#ifdef ILP15
	MYFTYPE v_15=V[PIdx_15]; // RRR -65?
#endif
#ifdef ILP16
	MYFTYPE v_16=V[PIdx_16]; // RRR -65?
#endif
#ifdef ILP17
	MYFTYPE v_17=V[PIdx_17]; // RRR -65?
#endif
#ifdef ILP18
	MYFTYPE v_18=V[PIdx_18]; // RRR -65?
#endif
#ifdef ILP19
	MYFTYPE v_19=V[PIdx_19]; // RRR -65?
#endif
#ifdef ILP20
	MYFTYPE v_20=V[PIdx_20]; // RRR -65?
#endif
#ifdef ILP21
	MYFTYPE v_21=V[PIdx_21]; // RRR -65?
#endif
#ifdef ILP22
	MYFTYPE v_22=V[PIdx_22]; // RRR -65?
#endif
#ifdef ILP23
	MYFTYPE v_23=V[PIdx_23]; // RRR -65?
#endif
#ifdef ILP24
	MYFTYPE v_24=V[PIdx_24]; // RRR -65?
#endif
#ifdef ILP25
	MYFTYPE v_25=V[PIdx_25]; // RRR -65?
#endif
#ifdef ILP26
	MYFTYPE v_26=V[PIdx_26]; // RRR -65?
#endif
#ifdef ILP27
	MYFTYPE v_27=V[PIdx_27]; // RRR -65?
#endif
#ifdef ILP28
	MYFTYPE v_28=V[PIdx_28]; // RRR -65?
#endif
#ifdef ILP29
	MYFTYPE v_29=V[PIdx_29]; // RRR -65?
#endif
#ifdef ILP30
	MYFTYPE v_30=V[PIdx_30]; // RRR -65?
#endif
#ifdef ILP31
	MYFTYPE v_31=V[PIdx_31]; // RRR -65?
#endif
#ifdef ILP32
	MYFTYPE v_32=V[PIdx_32]; // RRR -65?
#endif
	
		
	HMat SMemMat;
	SMemMat.N=InMat.N;
	SMemMat.Depth=InMat.Depth;
	SMemMat.LognDepth=InMat.LognDepth;
	SMemMat.nFathers=InMat.nFathers;
	SMemMat.nCallForFather=InMat.nCallForFather;
	MYFTYPE p1,p2,p3,p4,p5,p6;
	MYFTYPE p1_2,p2_2,p3_2,p4_2,p5_2,p6_2;
#ifdef ILP3
	MYFTYPE p1_3,p2_3,p3_3,p4_3,p5_3,p6_3;
#endif
#ifdef ILP4
	MYFTYPE p1_4,p2_4,p3_4,p4_4,p5_4,p6_4;
#endif
#ifdef ILP5
	MYFTYPE p1_5,p2_5,p3_5,p4_5,p5_5,p6_5;
#endif
#ifdef ILP6
	MYFTYPE p1_6,p2_6,p3_6,p4_6,p5_6,p6_6;
#endif
#ifdef ILP7
	MYFTYPE p1_7,p2_7,p3_7,p4_7,p5_7,p6_7;
#endif
#ifdef ILP8
	MYFTYPE p1_8,p2_8,p3_8,p4_8,p5_8,p6_8;
#endif
#ifdef ILP9
	MYFTYPE p1_9,p2_9,p3_9,p4_9,p5_9,p6_9;
#endif
#ifdef ILP10
	MYFTYPE p1_10,p2_10,p3_10,p4_10,p5_10,p6_10;
#endif
#ifdef ILP11
	MYFTYPE p1_11,p2_11,p3_11,p4_11,p5_11,p6_11;
#endif
#ifdef ILP12
	MYFTYPE p1_12,p2_12,p3_12,p4_12,p5_12,p6_12;
#endif
#ifdef ILP13
	MYFTYPE p1_13,p2_13,p3_13,p4_13,p5_13,p6_13;
#endif
#ifdef ILP14
	MYFTYPE p1_14,p2_14,p3_14,p4_14,p5_14,p6_14;
#endif
#ifdef ILP15
	MYFTYPE p1_15,p2_15,p3_15,p4_15,p5_15,p6_15;
#endif
#ifdef ILP16
	MYFTYPE p1_16,p2_16,p3_16,p4_16,p5_16,p6_16;
#endif
#ifdef ILP17
	MYFTYPE p1_17,p2_17,p3_17,p4_17,p5_17,p6_17;
#endif
#ifdef ILP18
	MYFTYPE p1_18,p2_18,p3_18,p4_18,p5_18,p6_18;
#endif
#ifdef ILP19
	MYFTYPE p1_19,p2_19,p3_19,p4_19,p5_19,p6_19;
#endif
#ifdef ILP20
	MYFTYPE p1_20,p2_20,p3_20,p4_20,p5_20,p6_20;
#endif
#ifdef ILP21
	MYFTYPE p1_21,p2_21,p3_21,p4_21,p5_21,p6_21;
#endif
#ifdef ILP22
	MYFTYPE p1_22,p2_22,p3_22,p4_22,p5_22,p6_22;
#endif
#ifdef ILP23
	MYFTYPE p1_23,p2_23,p3_23,p4_23,p5_23,p6_23;
#endif
#ifdef ILP24
	MYFTYPE p1_24,p2_24,p3_24,p4_24,p5_24,p6_24;
#endif
#ifdef ILP25
	MYFTYPE p1_25,p2_25,p3_25,p4_25,p5_25,p6_25;
#endif
#ifdef ILP26
	MYFTYPE p1_26,p2_26,p3_26,p4_26,p5_26,p6_26;
#endif
#ifdef ILP27
	MYFTYPE p1_27,p2_27,p3_27,p4_27,p5_27,p6_27;
#endif
#ifdef ILP28
	MYFTYPE p1_28,p2_28,p3_28,p4_28,p5_28,p6_28;
#endif
#ifdef ILP29
	MYFTYPE p1_29,p2_29,p3_29,p4_29,p5_29,p6_29;
#endif
#ifdef ILP30
	MYFTYPE p1_30,p2_30,p3_30,p4_30,p5_30,p6_30;
#endif
#ifdef ILP31
	MYFTYPE p1_31,p2_31,p3_31,p4_31,p5_31,p6_31;
#endif
#ifdef ILP32
	MYFTYPE p1_32,p2_32,p3_32,p4_32,p5_32,p6_32;
#endif
	p1=ParamsM[0*SMemMat.N+PIdx];
	p1_2=ParamsM[0*SMemMat.N+PIdx_2];
#ifdef ILP3
	p1_3=ParamsM[0*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p1_4=ParamsM[0*SMemMat.N+PIdx_4];
#endif
#ifdef ILP5
	p1_5=ParamsM[0*SMemMat.N+PIdx_5];
#endif
#ifdef ILP6
	p1_6=ParamsM[0*SMemMat.N+PIdx_6];
#endif
#ifdef ILP7
	p1_7=ParamsM[0*SMemMat.N+PIdx_7];
#endif
#ifdef ILP8
	p1_8=ParamsM[0*SMemMat.N+PIdx_8];
#endif
#ifdef ILP9
	p1_9=ParamsM[0*SMemMat.N+PIdx_9];
#endif
#ifdef ILP10
	p1_10=ParamsM[0*SMemMat.N+PIdx_10];
#endif
#ifdef ILP11
	p1_11=ParamsM[0*SMemMat.N+PIdx_11];
#endif
#ifdef ILP12
	p1_12=ParamsM[0*SMemMat.N+PIdx_12];
#endif
#ifdef ILP13
	p1_13=ParamsM[0*SMemMat.N+PIdx_13];
#endif
#ifdef ILP14
	p1_14=ParamsM[0*SMemMat.N+PIdx_14];
#endif
#ifdef ILP15
	p1_15=ParamsM[0*SMemMat.N+PIdx_15];
#endif
#ifdef ILP16
	p1_16=ParamsM[0*SMemMat.N+PIdx_16];
#endif
#ifdef ILP17
	p1_17=ParamsM[0*SMemMat.N+PIdx_17];
#endif
#ifdef ILP18
	p1_18=ParamsM[0*SMemMat.N+PIdx_18];
#endif
#ifdef ILP19
	p1_19=ParamsM[0*SMemMat.N+PIdx_19];
#endif
#ifdef ILP20
	p1_20=ParamsM[0*SMemMat.N+PIdx_20];
#endif
#ifdef ILP21
	p1_21=ParamsM[0*SMemMat.N+PIdx_21];
#endif
#ifdef ILP22
	p1_22=ParamsM[0*SMemMat.N+PIdx_22];
#endif
#ifdef ILP23
	p1_23=ParamsM[0*SMemMat.N+PIdx_23];
#endif
#ifdef ILP24
	p1_24=ParamsM[0*SMemMat.N+PIdx_24];
#endif
#ifdef ILP25
	p1_25=ParamsM[0*SMemMat.N+PIdx_25];
#endif
#ifdef ILP26
	p1_26=ParamsM[0*SMemMat.N+PIdx_26];
#endif
#ifdef ILP27
	p1_27=ParamsM[0*SMemMat.N+PIdx_27];
#endif
#ifdef ILP28
	p1_28=ParamsM[0*SMemMat.N+PIdx_28];
#endif
#ifdef ILP29
	p1_29=ParamsM[0*SMemMat.N+PIdx_29];
#endif
#ifdef ILP30
	p1_30=ParamsM[0*SMemMat.N+PIdx_30];
#endif
#ifdef ILP31
	p1_31=ParamsM[0*SMemMat.N+PIdx_31];
#endif
#ifdef ILP32
	p1_32=ParamsM[0*SMemMat.N+PIdx_32];
#endif
	p2=ParamsM[1*SMemMat.N+PIdx];
	p2_2=ParamsM[1*SMemMat.N+PIdx_2];
#ifdef ILP3
	p2_3=ParamsM[1*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p2_4=ParamsM[1*SMemMat.N+PIdx_4];
#endif
#ifdef ILP5
	p2_5=ParamsM[1*SMemMat.N+PIdx_5];
#endif
#ifdef ILP6
	p2_6=ParamsM[1*SMemMat.N+PIdx_6];
#endif
#ifdef ILP7
	p2_7=ParamsM[1*SMemMat.N+PIdx_7];
#endif
#ifdef ILP8
	p2_8=ParamsM[1*SMemMat.N+PIdx_8];
#endif
#ifdef ILP9
	p2_9=ParamsM[1*SMemMat.N+PIdx_9];
#endif
#ifdef ILP10
	p2_10=ParamsM[1*SMemMat.N+PIdx_10];
#endif
#ifdef ILP11
	p2_11=ParamsM[1*SMemMat.N+PIdx_11];
#endif
#ifdef ILP12
	p2_12=ParamsM[1*SMemMat.N+PIdx_12];
#endif
#ifdef ILP13
	p2_13=ParamsM[1*SMemMat.N+PIdx_13];
#endif
#ifdef ILP14
	p2_14=ParamsM[1*SMemMat.N+PIdx_14];
#endif
#ifdef ILP15
	p2_15=ParamsM[1*SMemMat.N+PIdx_15];
#endif
#ifdef ILP16
	p2_16=ParamsM[1*SMemMat.N+PIdx_16];
#endif
#ifdef ILP17
	p2_17=ParamsM[1*SMemMat.N+PIdx_17];
#endif
#ifdef ILP18
	p2_18=ParamsM[1*SMemMat.N+PIdx_18];
#endif
#ifdef ILP19
	p2_19=ParamsM[1*SMemMat.N+PIdx_19];
#endif
#ifdef ILP20
	p2_20=ParamsM[1*SMemMat.N+PIdx_20];
#endif
#ifdef ILP21
	p2_21=ParamsM[1*SMemMat.N+PIdx_21];
#endif
#ifdef ILP22
	p2_22=ParamsM[1*SMemMat.N+PIdx_22];
#endif
#ifdef ILP23
	p2_23=ParamsM[1*SMemMat.N+PIdx_23];
#endif
#ifdef ILP24
	p2_24=ParamsM[1*SMemMat.N+PIdx_24];
#endif
#ifdef ILP25
	p2_25=ParamsM[1*SMemMat.N+PIdx_25];
#endif
#ifdef ILP26
	p2_26=ParamsM[1*SMemMat.N+PIdx_26];
#endif
#ifdef ILP27
	p2_27=ParamsM[1*SMemMat.N+PIdx_27];
#endif
#ifdef ILP28
	p2_28=ParamsM[1*SMemMat.N+PIdx_28];
#endif
#ifdef ILP29
	p2_29=ParamsM[1*SMemMat.N+PIdx_29];
#endif
#ifdef ILP30
	p2_30=ParamsM[1*SMemMat.N+PIdx_30];
#endif
#ifdef ILP31
	p2_31=ParamsM[1*SMemMat.N+PIdx_31];
#endif
#ifdef ILP32
	p2_32=ParamsM[1*SMemMat.N+PIdx_32];
#endif
	p3=ParamsM[2*SMemMat.N+PIdx];
	p3_2=ParamsM[2*SMemMat.N+PIdx_2];
#ifdef ILP3
	p3_3=ParamsM[2*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p3_4=ParamsM[2*SMemMat.N+PIdx_4];
#endif
#ifdef ILP5
	p3_5=ParamsM[2*SMemMat.N+PIdx_5];
#endif
#ifdef ILP6
	p3_6=ParamsM[2*SMemMat.N+PIdx_6];
#endif
#ifdef ILP7
	p3_7=ParamsM[2*SMemMat.N+PIdx_7];
#endif
#ifdef ILP8
	p3_8=ParamsM[2*SMemMat.N+PIdx_8];
#endif
#ifdef ILP9
	p3_9=ParamsM[2*SMemMat.N+PIdx_9];
#endif
#ifdef ILP10
	p3_10=ParamsM[2*SMemMat.N+PIdx_10];
#endif
#ifdef ILP11
	p3_11=ParamsM[2*SMemMat.N+PIdx_11];
#endif
#ifdef ILP12
	p3_12=ParamsM[2*SMemMat.N+PIdx_12];
#endif
#ifdef ILP13
	p3_13=ParamsM[2*SMemMat.N+PIdx_13];
#endif
#ifdef ILP14
	p3_14=ParamsM[2*SMemMat.N+PIdx_14];
#endif
#ifdef ILP15
	p3_15=ParamsM[2*SMemMat.N+PIdx_15];
#endif
#ifdef ILP16
	p3_16=ParamsM[2*SMemMat.N+PIdx_16];
#endif
#ifdef ILP17
	p3_17=ParamsM[2*SMemMat.N+PIdx_17];
#endif
#ifdef ILP18
	p3_18=ParamsM[2*SMemMat.N+PIdx_18];
#endif
#ifdef ILP19
	p3_19=ParamsM[2*SMemMat.N+PIdx_19];
#endif
#ifdef ILP20
	p3_20=ParamsM[2*SMemMat.N+PIdx_20];
#endif
#ifdef ILP21
	p3_21=ParamsM[2*SMemMat.N+PIdx_21];
#endif
#ifdef ILP22
	p3_22=ParamsM[2*SMemMat.N+PIdx_22];
#endif
#ifdef ILP23
	p3_23=ParamsM[2*SMemMat.N+PIdx_23];
#endif
#ifdef ILP24
	p3_24=ParamsM[2*SMemMat.N+PIdx_24];
#endif
#ifdef ILP25
	p3_25=ParamsM[2*SMemMat.N+PIdx_25];
#endif
#ifdef ILP26
	p3_26=ParamsM[2*SMemMat.N+PIdx_26];
#endif
#ifdef ILP27
	p3_27=ParamsM[2*SMemMat.N+PIdx_27];
#endif
#ifdef ILP28
	p3_28=ParamsM[2*SMemMat.N+PIdx_28];
#endif
#ifdef ILP29
	p3_29=ParamsM[2*SMemMat.N+PIdx_29];
#endif
#ifdef ILP30
	p3_30=ParamsM[2*SMemMat.N+PIdx_30];
#endif
#ifdef ILP31
	p3_31=ParamsM[2*SMemMat.N+PIdx_31];
#endif
#ifdef ILP32
	p3_32=ParamsM[2*SMemMat.N+PIdx_32];
#endif
	p4=ParamsM[3*SMemMat.N+PIdx];
	p4_2=ParamsM[3*SMemMat.N+PIdx_2];
#ifdef ILP3
	p4_3=ParamsM[3*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p4_4=ParamsM[3*SMemMat.N+PIdx_4];
#endif
#ifdef ILP5
	p4_5=ParamsM[3*SMemMat.N+PIdx_5];
#endif
#ifdef ILP6
	p4_6=ParamsM[3*SMemMat.N+PIdx_6];
#endif
#ifdef ILP7
	p4_7=ParamsM[3*SMemMat.N+PIdx_7];
#endif
#ifdef ILP8
	p4_8=ParamsM[3*SMemMat.N+PIdx_8];
#endif
#ifdef ILP9
	p4_9=ParamsM[3*SMemMat.N+PIdx_9];
#endif
#ifdef ILP10
	p4_10=ParamsM[3*SMemMat.N+PIdx_10];
#endif
#ifdef ILP11
	p4_11=ParamsM[3*SMemMat.N+PIdx_11];
#endif
#ifdef ILP12
	p4_12=ParamsM[3*SMemMat.N+PIdx_12];
#endif
#ifdef ILP13
	p4_13=ParamsM[3*SMemMat.N+PIdx_13];
#endif
#ifdef ILP14
	p4_14=ParamsM[3*SMemMat.N+PIdx_14];
#endif
#ifdef ILP15
	p4_15=ParamsM[3*SMemMat.N+PIdx_15];
#endif
#ifdef ILP16
	p4_16=ParamsM[3*SMemMat.N+PIdx_16];
#endif
#ifdef ILP17
	p4_17=ParamsM[3*SMemMat.N+PIdx_17];
#endif
#ifdef ILP18
	p4_18=ParamsM[3*SMemMat.N+PIdx_18];
#endif
#ifdef ILP19
	p4_19=ParamsM[3*SMemMat.N+PIdx_19];
#endif
#ifdef ILP20
	p4_20=ParamsM[3*SMemMat.N+PIdx_20];
#endif
#ifdef ILP21
	p4_21=ParamsM[3*SMemMat.N+PIdx_21];
#endif
#ifdef ILP22
	p4_22=ParamsM[3*SMemMat.N+PIdx_22];
#endif
#ifdef ILP23
	p4_23=ParamsM[3*SMemMat.N+PIdx_23];
#endif
#ifdef ILP24
	p4_24=ParamsM[3*SMemMat.N+PIdx_24];
#endif
#ifdef ILP25
	p4_25=ParamsM[3*SMemMat.N+PIdx_25];
#endif
#ifdef ILP26
	p4_26=ParamsM[3*SMemMat.N+PIdx_26];
#endif
#ifdef ILP27
	p4_27=ParamsM[3*SMemMat.N+PIdx_27];
#endif
#ifdef ILP28
	p4_28=ParamsM[3*SMemMat.N+PIdx_28];
#endif
#ifdef ILP29
	p4_29=ParamsM[3*SMemMat.N+PIdx_29];
#endif
#ifdef ILP30
	p4_30=ParamsM[3*SMemMat.N+PIdx_30];
#endif
#ifdef ILP31
	p4_31=ParamsM[3*SMemMat.N+PIdx_31];
#endif
#ifdef ILP32
	p4_32=ParamsM[3*SMemMat.N+PIdx_32];
#endif
	p5=ParamsM[4*SMemMat.N+PIdx];
	p5_2=ParamsM[4*SMemMat.N+PIdx_2];
#ifdef ILP3
	p5_3=ParamsM[4*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p5_4=ParamsM[4*SMemMat.N+PIdx_4];
#endif
#ifdef ILP5
	p5_5=ParamsM[4*SMemMat.N+PIdx_5];
#endif
#ifdef ILP6
	p5_6=ParamsM[4*SMemMat.N+PIdx_6];
#endif
#ifdef ILP7
	p5_7=ParamsM[4*SMemMat.N+PIdx_7];
#endif
#ifdef ILP8
	p5_8=ParamsM[4*SMemMat.N+PIdx_8];
#endif
#ifdef ILP9
	p5_9=ParamsM[4*SMemMat.N+PIdx_9];
#endif
#ifdef ILP10
	p5_10=ParamsM[4*SMemMat.N+PIdx_10];
#endif
#ifdef ILP11
	p5_11=ParamsM[4*SMemMat.N+PIdx_11];
#endif
#ifdef ILP12
	p5_12=ParamsM[4*SMemMat.N+PIdx_12];
#endif
#ifdef ILP13
	p5_13=ParamsM[4*SMemMat.N+PIdx_13];
#endif
#ifdef ILP14
	p5_14=ParamsM[4*SMemMat.N+PIdx_14];
#endif
#ifdef ILP15
	p5_15=ParamsM[4*SMemMat.N+PIdx_15];
#endif
#ifdef ILP16
	p5_16=ParamsM[4*SMemMat.N+PIdx_16];
#endif
#ifdef ILP17
	p5_17=ParamsM[4*SMemMat.N+PIdx_17];
#endif
#ifdef ILP18
	p5_18=ParamsM[4*SMemMat.N+PIdx_18];
#endif
#ifdef ILP19
	p5_19=ParamsM[4*SMemMat.N+PIdx_19];
#endif
#ifdef ILP20
	p5_20=ParamsM[4*SMemMat.N+PIdx_20];
#endif
#ifdef ILP21
	p5_21=ParamsM[4*SMemMat.N+PIdx_21];
#endif
#ifdef ILP22
	p5_22=ParamsM[4*SMemMat.N+PIdx_22];
#endif
#ifdef ILP23
	p5_23=ParamsM[4*SMemMat.N+PIdx_23];
#endif
#ifdef ILP24
	p5_24=ParamsM[4*SMemMat.N+PIdx_24];
#endif
#ifdef ILP25
	p5_25=ParamsM[4*SMemMat.N+PIdx_25];
#endif
#ifdef ILP26
	p5_26=ParamsM[4*SMemMat.N+PIdx_26];
#endif
#ifdef ILP27
	p5_27=ParamsM[4*SMemMat.N+PIdx_27];
#endif
#ifdef ILP28
	p5_28=ParamsM[4*SMemMat.N+PIdx_28];
#endif
#ifdef ILP29
	p5_29=ParamsM[4*SMemMat.N+PIdx_29];
#endif
#ifdef ILP30
	p5_30=ParamsM[4*SMemMat.N+PIdx_30];
#endif
#ifdef ILP31
	p5_31=ParamsM[4*SMemMat.N+PIdx_31];
#endif
#ifdef ILP32
	p5_32=ParamsM[4*SMemMat.N+PIdx_32];
#endif
	p6=ParamsM[5*SMemMat.N+PIdx];
	p6_2=ParamsM[5*SMemMat.N+PIdx_2];
#ifdef ILP3
	p6_3=ParamsM[5*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p6_4=ParamsM[5*SMemMat.N+PIdx_4];
#endif
#ifdef ILP5
	p6_5=ParamsM[5*SMemMat.N+PIdx_5];
#endif
#ifdef ILP6
	p6_6=ParamsM[5*SMemMat.N+PIdx_6];
#endif
#ifdef ILP7
	p6_7=ParamsM[5*SMemMat.N+PIdx_7];
#endif
#ifdef ILP8
	p6_8=ParamsM[5*SMemMat.N+PIdx_8];
#endif
#ifdef ILP9
	p6_9=ParamsM[5*SMemMat.N+PIdx_9];
#endif
#ifdef ILP10
	p6_10=ParamsM[5*SMemMat.N+PIdx_10];
#endif
#ifdef ILP11
	p6_11=ParamsM[5*SMemMat.N+PIdx_11];
#endif
#ifdef ILP12
	p6_12=ParamsM[5*SMemMat.N+PIdx_12];
#endif
#ifdef ILP13
	p6_13=ParamsM[5*SMemMat.N+PIdx_13];
#endif
#ifdef ILP14
	p6_14=ParamsM[5*SMemMat.N+PIdx_14];
#endif
#ifdef ILP15
	p6_15=ParamsM[5*SMemMat.N+PIdx_15];
#endif
#ifdef ILP16
	p6_16=ParamsM[5*SMemMat.N+PIdx_16];
#endif
#ifdef ILP17
	p6_17=ParamsM[5*SMemMat.N+PIdx_17];
#endif
#ifdef ILP18
	p6_18=ParamsM[5*SMemMat.N+PIdx_18];
#endif
#ifdef ILP19
	p6_19=ParamsM[5*SMemMat.N+PIdx_19];
#endif
#ifdef ILP20
	p6_20=ParamsM[5*SMemMat.N+PIdx_20];
#endif
#ifdef ILP21
	p6_21=ParamsM[5*SMemMat.N+PIdx_21];
#endif
#ifdef ILP22
	p6_22=ParamsM[5*SMemMat.N+PIdx_22];
#endif
#ifdef ILP23
	p6_23=ParamsM[5*SMemMat.N+PIdx_23];
#endif
#ifdef ILP24
	p6_24=ParamsM[5*SMemMat.N+PIdx_24];
#endif
#ifdef ILP25
	p6_25=ParamsM[5*SMemMat.N+PIdx_25];
#endif
#ifdef ILP26
	p6_26=ParamsM[5*SMemMat.N+PIdx_26];
#endif
#ifdef ILP27
	p6_27=ParamsM[5*SMemMat.N+PIdx_27];
#endif
#ifdef ILP28
	p6_28=ParamsM[5*SMemMat.N+PIdx_28];
#endif
#ifdef ILP29
	p6_29=ParamsM[5*SMemMat.N+PIdx_29];
#endif
#ifdef ILP30
	p6_30=ParamsM[5*SMemMat.N+PIdx_30];
#endif
#ifdef ILP31
	p6_31=ParamsM[5*SMemMat.N+PIdx_31];
#endif
#ifdef ILP32
	p6_32=ParamsM[5*SMemMat.N+PIdx_32];
#endif

	
	KInitModel(v,ModelStates[0],ModelStates[1],ModelStates[2]);
	KInitModel(v_2,ModelStates_2[0],ModelStates_2[1],ModelStates_2[2]);
#ifdef ILP3
	KInitModel(v_3,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2]);
#endif
#ifdef ILP4
	KInitModel(v_4,ModelStates_4[0],ModelStates_4[1],ModelStates_4[2]);
#endif
#ifdef ILP5
	KInitModel(v_5,ModelStates_5[0],ModelStates_5[1],ModelStates_5[2]);
#endif
#ifdef ILP6
	KInitModel(v_6,ModelStates_6[0],ModelStates_6[1],ModelStates_6[2]);
#endif
#ifdef ILP7
	KInitModel(v_7,ModelStates_7[0],ModelStates_7[1],ModelStates_7[2]);
#endif
#ifdef ILP8
	KInitModel(v_8,ModelStates_8[0],ModelStates_8[1],ModelStates_8[2]);
#endif
#ifdef ILP9
	KInitModel(v_9,ModelStates_9[0],ModelStates_9[1],ModelStates_9[2]);
#endif
#ifdef ILP10
	KInitModel(v_10,ModelStates_10[0],ModelStates_10[1],ModelStates_10[2]);
#endif
#ifdef ILP11
	KInitModel(v_11,ModelStates_11[0],ModelStates_11[1],ModelStates_11[2]);
#endif
#ifdef ILP12
	KInitModel(v_12,ModelStates_12[0],ModelStates_12[1],ModelStates_12[2]);
#endif
#ifdef ILP13
	KInitModel(v_13,ModelStates_13[0],ModelStates_13[1],ModelStates_13[2]);
#endif
#ifdef ILP14
	KInitModel(v_14,ModelStates_14[0],ModelStates_14[1],ModelStates_14[2]);
#endif
#ifdef ILP15
	KInitModel(v_15,ModelStates_15[0],ModelStates_15[1],ModelStates_15[2]);
#endif
#ifdef ILP16
	KInitModel(v_16,ModelStates_16[0],ModelStates_16[1],ModelStates_16[2]);
#endif
#ifdef ILP17
	KInitModel(v_17,ModelStates_17[0],ModelStates_17[1],ModelStates_17[2]);
#endif
#ifdef ILP18
	KInitModel(v_18,ModelStates_18[0],ModelStates_18[1],ModelStates_18[2]);
#endif
#ifdef ILP19
	KInitModel(v_19,ModelStates_19[0],ModelStates_19[1],ModelStates_19[2]);
#endif
#ifdef ILP20
	KInitModel(v_20,ModelStates_20[0],ModelStates_20[1],ModelStates_20[2]);
#endif
#ifdef ILP21
	KInitModel(v_21,ModelStates_21[0],ModelStates_21[1],ModelStates_21[2]);
#endif
#ifdef ILP22
	KInitModel(v_22,ModelStates_22[0],ModelStates_22[1],ModelStates_22[2]);
#endif
#ifdef ILP23
	KInitModel(v_23,ModelStates_23[0],ModelStates_23[1],ModelStates_23[2]);
#endif
#ifdef ILP24
	KInitModel(v_24,ModelStates_24[0],ModelStates_24[1],ModelStates_24[2]);
#endif
#ifdef ILP25
	KInitModel(v_25,ModelStates_25[0],ModelStates_25[1],ModelStates_25[2]);
#endif
#ifdef ILP26
	KInitModel(v_26,ModelStates_26[0],ModelStates_26[1],ModelStates_26[2]);
#endif
#ifdef ILP27
	KInitModel(v_27,ModelStates_27[0],ModelStates_27[1],ModelStates_27[2]);
#endif
#ifdef ILP28
	KInitModel(v_28,ModelStates_28[0],ModelStates_28[1],ModelStates_28[2]);
#endif
#ifdef ILP29
	KInitModel(v_29,ModelStates_29[0],ModelStates_29[1],ModelStates_29[2]);
#endif
#ifdef ILP30
	KInitModel(v_30,ModelStates_30[0],ModelStates_30[1],ModelStates_30[2]);
#endif
#ifdef ILP31
	KInitModel(v_31,ModelStates_31[0],ModelStates_31[1],ModelStates_31[2]);
#endif
#ifdef ILP32
	KInitModel(v_32,ModelStates_32[0],ModelStates_32[1],ModelStates_32[2]);
#endif
	
	MYFTYPE sumCurrents, sumConductivity;
	MYFTYPE sumCurrents_2, sumConductivity_2;
#ifdef ILP3
	MYFTYPE sumCurrents_3, sumConductivity_3;
#endif
#ifdef ILP4
	MYFTYPE sumCurrents_4, sumConductivity_4;
#endif
#ifdef ILP5
	MYFTYPE sumCurrents_5, sumConductivity_5;
#endif
#ifdef ILP6
	MYFTYPE sumCurrents_6, sumConductivity_6;
#endif
#ifdef ILP7
	MYFTYPE sumCurrents_7, sumConductivity_7;
#endif
#ifdef ILP8
	MYFTYPE sumCurrents_8, sumConductivity_8;
#endif
#ifdef ILP9
	MYFTYPE sumCurrents_9, sumConductivity_9;
#endif
#ifdef ILP10
	MYFTYPE sumCurrents_10, sumConductivity_10;
#endif
#ifdef ILP11
	MYFTYPE sumCurrents_11, sumConductivity_11;
#endif
#ifdef ILP12
	MYFTYPE sumCurrents_12, sumConductivity_12;
#endif
#ifdef ILP13
	MYFTYPE sumCurrents_13, sumConductivity_13;
#endif
#ifdef ILP14
	MYFTYPE sumCurrents_14, sumConductivity_14;
#endif
#ifdef ILP15
	MYFTYPE sumCurrents_15, sumConductivity_15;
#endif
#ifdef ILP16
	MYFTYPE sumCurrents_16, sumConductivity_16;
#endif
#ifdef ILP17
	MYFTYPE sumCurrents_17, sumConductivity_17;
#endif
#ifdef ILP18
	MYFTYPE sumCurrents_18, sumConductivity_18;
#endif
#ifdef ILP19
	MYFTYPE sumCurrents_19, sumConductivity_19;
#endif
#ifdef ILP20
	MYFTYPE sumCurrents_20, sumConductivity_20;
#endif
#ifdef ILP21
	MYFTYPE sumCurrents_21, sumConductivity_21;
#endif
#ifdef ILP22
	MYFTYPE sumCurrents_22, sumConductivity_22;
#endif
#ifdef ILP23
	MYFTYPE sumCurrents_23, sumConductivity_23;
#endif
#ifdef ILP24
	MYFTYPE sumCurrents_24, sumConductivity_24;
#endif
#ifdef ILP25
	MYFTYPE sumCurrents_25, sumConductivity_25;
#endif
#ifdef ILP26
	MYFTYPE sumCurrents_26, sumConductivity_26;
#endif
#ifdef ILP27
	MYFTYPE sumCurrents_27, sumConductivity_27;
#endif
#ifdef ILP28
	MYFTYPE sumCurrents_28, sumConductivity_28;
#endif
#ifdef ILP29
	MYFTYPE sumCurrents_29, sumConductivity_29;
#endif
#ifdef ILP30
	MYFTYPE sumCurrents_30, sumConductivity_30;
#endif
#ifdef ILP31
	MYFTYPE sumCurrents_31, sumConductivity_31;
#endif
#ifdef ILP32
	MYFTYPE sumCurrents_32, sumConductivity_32;
#endif
	MYFTYPE sumCurrentsDv, sumConductivityDv;
	MYFTYPE sumCurrentsDv_2, sumConductivityDv_2;
#ifdef ILP3
	MYFTYPE sumCurrentsDv_3, sumConductivityDv_3;
#endif
#ifdef ILP4
	MYFTYPE sumCurrentsDv_4, sumConductivityDv_4;
#endif
#ifdef ILP5
	MYFTYPE sumCurrentsDv_5, sumConductivityDv_5;
#endif
#ifdef ILP6
	MYFTYPE sumCurrentsDv_6, sumConductivityDv_6;
#endif
#ifdef ILP7
	MYFTYPE sumCurrentsDv_7, sumConductivityDv_7;
#endif
#ifdef ILP8
	MYFTYPE sumCurrentsDv_8, sumConductivityDv_8;
#endif
#ifdef ILP9
	MYFTYPE sumCurrentsDv_9, sumConductivityDv_9;
#endif
#ifdef ILP10
	MYFTYPE sumCurrentsDv_10, sumConductivityDv_10;
#endif
#ifdef ILP11
	MYFTYPE sumCurrentsDv_11, sumConductivityDv_11;
#endif
#ifdef ILP12
	MYFTYPE sumCurrentsDv_12, sumConductivityDv_12;
#endif
#ifdef ILP13
	MYFTYPE sumCurrentsDv_13, sumConductivityDv_13;
#endif
#ifdef ILP14
	MYFTYPE sumCurrentsDv_14, sumConductivityDv_14;
#endif
#ifdef ILP15
	MYFTYPE sumCurrentsDv_15, sumConductivityDv_15;
#endif
#ifdef ILP16
	MYFTYPE sumCurrentsDv_16, sumConductivityDv_16;
#endif
#ifdef ILP17
	MYFTYPE sumCurrentsDv_17, sumConductivityDv_17;
#endif
#ifdef ILP18
	MYFTYPE sumCurrentsDv_18, sumConductivityDv_18;
#endif
#ifdef ILP19
	MYFTYPE sumCurrentsDv_19, sumConductivityDv_19;
#endif
#ifdef ILP20
	MYFTYPE sumCurrentsDv_20, sumConductivityDv_20;
#endif
#ifdef ILP21
	MYFTYPE sumCurrentsDv_21, sumConductivityDv_21;
#endif
#ifdef ILP22
	MYFTYPE sumCurrentsDv_22, sumConductivityDv_22;
#endif
#ifdef ILP23
	MYFTYPE sumCurrentsDv_23, sumConductivityDv_23;
#endif
#ifdef ILP24
	MYFTYPE sumCurrentsDv_24, sumConductivityDv_24;
#endif
#ifdef ILP25
	MYFTYPE sumCurrentsDv_25, sumConductivityDv_25;
#endif
#ifdef ILP26
	MYFTYPE sumCurrentsDv_26, sumConductivityDv_26;
#endif
#ifdef ILP27
	MYFTYPE sumCurrentsDv_27, sumConductivityDv_27;
#endif
#ifdef ILP28
	MYFTYPE sumCurrentsDv_28, sumConductivityDv_28;
#endif
#ifdef ILP29
	MYFTYPE sumCurrentsDv_29, sumConductivityDv_29;
#endif
#ifdef ILP30
	MYFTYPE sumCurrentsDv_30, sumConductivityDv_30;
#endif
#ifdef ILP31
	MYFTYPE sumCurrentsDv_31, sumConductivityDv_31;
#endif
#ifdef ILP32
	MYFTYPE sumCurrentsDv_32, sumConductivityDv_32;
#endif
	// end for model in init

	MYDTYPE offset=0;
	// CONSTANTS
	// floats
	MYFTYPE *Olde,*Oldf;
	Olde=InMat.e;
	Oldf=InMat.f;
	SMemMat.e = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.f = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.e[PIdx]=InMat.e[PIdx];
	SMemMat.e[PIdx_2]=InMat.e[PIdx_2];
#ifdef ILP3
	SMemMat.e[PIdx_3]=InMat.e[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.e[PIdx_4]=InMat.e[PIdx_4];
#endif
#ifdef ILP5
	SMemMat.e[PIdx_5]=InMat.e[PIdx_5];
#endif
#ifdef ILP6
	SMemMat.e[PIdx_6]=InMat.e[PIdx_6];
#endif
#ifdef ILP7
	SMemMat.e[PIdx_7]=InMat.e[PIdx_7];
#endif
#ifdef ILP8
	SMemMat.e[PIdx_8]=InMat.e[PIdx_8];
#endif
#ifdef ILP9
	SMemMat.e[PIdx_9]=InMat.e[PIdx_9];
#endif
#ifdef ILP10
	SMemMat.e[PIdx_10]=InMat.e[PIdx_10];
#endif
#ifdef ILP11
	SMemMat.e[PIdx_11]=InMat.e[PIdx_11];
#endif
#ifdef ILP12
	SMemMat.e[PIdx_12]=InMat.e[PIdx_12];
#endif
#ifdef ILP13
	SMemMat.e[PIdx_13]=InMat.e[PIdx_13];
#endif
#ifdef ILP14
	SMemMat.e[PIdx_14]=InMat.e[PIdx_14];
#endif
#ifdef ILP15
	SMemMat.e[PIdx_15]=InMat.e[PIdx_15];
#endif
#ifdef ILP16
	SMemMat.e[PIdx_16]=InMat.e[PIdx_16];
#endif
#ifdef ILP17
	SMemMat.e[PIdx_17]=InMat.e[PIdx_17];
#endif
#ifdef ILP18
	SMemMat.e[PIdx_18]=InMat.e[PIdx_18];
#endif
#ifdef ILP19
	SMemMat.e[PIdx_19]=InMat.e[PIdx_19];
#endif
#ifdef ILP20
	SMemMat.e[PIdx_20]=InMat.e[PIdx_20];
#endif
#ifdef ILP21
	SMemMat.e[PIdx_21]=InMat.e[PIdx_21];
#endif
#ifdef ILP22
	SMemMat.e[PIdx_22]=InMat.e[PIdx_22];
#endif
#ifdef ILP23
	SMemMat.e[PIdx_23]=InMat.e[PIdx_23];
#endif
#ifdef ILP24
	SMemMat.e[PIdx_24]=InMat.e[PIdx_24];
#endif
#ifdef ILP25
	SMemMat.e[PIdx_25]=InMat.e[PIdx_25];
#endif
#ifdef ILP26
	SMemMat.e[PIdx_26]=InMat.e[PIdx_26];
#endif
#ifdef ILP27
	SMemMat.e[PIdx_27]=InMat.e[PIdx_27];
#endif
#ifdef ILP28
	SMemMat.e[PIdx_28]=InMat.e[PIdx_28];
#endif
#ifdef ILP29
	SMemMat.e[PIdx_29]=InMat.e[PIdx_29];
#endif
#ifdef ILP30
	SMemMat.e[PIdx_30]=InMat.e[PIdx_30];
#endif
#ifdef ILP31
	SMemMat.e[PIdx_31]=InMat.e[PIdx_31];
#endif
#ifdef ILP32
	SMemMat.e[PIdx_32]=InMat.e[PIdx_32];
#endif
	SMemMat.f[PIdx]=InMat.f[PIdx];
	SMemMat.f[PIdx_2]=InMat.f[PIdx_2];
#ifdef ILP3
	SMemMat.f[PIdx_3]=InMat.f[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.f[PIdx_4]=InMat.f[PIdx_4];
#endif
#ifdef ILP5
	SMemMat.f[PIdx_5]=InMat.f[PIdx_5];
#endif
#ifdef ILP6
	SMemMat.f[PIdx_6]=InMat.f[PIdx_6];
#endif
#ifdef ILP7
	SMemMat.f[PIdx_7]=InMat.f[PIdx_7];
#endif
#ifdef ILP8
	SMemMat.f[PIdx_8]=InMat.f[PIdx_8];
#endif
#ifdef ILP9
	SMemMat.f[PIdx_9]=InMat.f[PIdx_9];
#endif
#ifdef ILP10
	SMemMat.f[PIdx_10]=InMat.f[PIdx_10];
#endif
#ifdef ILP11
	SMemMat.f[PIdx_11]=InMat.f[PIdx_11];
#endif
#ifdef ILP12
	SMemMat.f[PIdx_12]=InMat.f[PIdx_12];
#endif
#ifdef ILP13
	SMemMat.f[PIdx_13]=InMat.f[PIdx_13];
#endif
#ifdef ILP14
	SMemMat.f[PIdx_14]=InMat.f[PIdx_14];
#endif
#ifdef ILP15
	SMemMat.f[PIdx_15]=InMat.f[PIdx_15];
#endif
#ifdef ILP16
	SMemMat.f[PIdx_16]=InMat.f[PIdx_16];
#endif
#ifdef ILP17
	SMemMat.f[PIdx_17]=InMat.f[PIdx_17];
#endif
#ifdef ILP18
	SMemMat.f[PIdx_18]=InMat.f[PIdx_18];
#endif
#ifdef ILP19
	SMemMat.f[PIdx_19]=InMat.f[PIdx_19];
#endif
#ifdef ILP20
	SMemMat.f[PIdx_20]=InMat.f[PIdx_20];
#endif
#ifdef ILP21
	SMemMat.f[PIdx_21]=InMat.f[PIdx_21];
#endif
#ifdef ILP22
	SMemMat.f[PIdx_22]=InMat.f[PIdx_22];
#endif
#ifdef ILP23
	SMemMat.f[PIdx_23]=InMat.f[PIdx_23];
#endif
#ifdef ILP24
	SMemMat.f[PIdx_24]=InMat.f[PIdx_24];
#endif
#ifdef ILP25
	SMemMat.f[PIdx_25]=InMat.f[PIdx_25];
#endif
#ifdef ILP26
	SMemMat.f[PIdx_26]=InMat.f[PIdx_26];
#endif
#ifdef ILP27
	SMemMat.f[PIdx_27]=InMat.f[PIdx_27];
#endif
#ifdef ILP28
	SMemMat.f[PIdx_28]=InMat.f[PIdx_28];
#endif
#ifdef ILP29
	SMemMat.f[PIdx_29]=InMat.f[PIdx_29];
#endif
#ifdef ILP30
	SMemMat.f[PIdx_30]=InMat.f[PIdx_30];
#endif
#ifdef ILP31
	SMemMat.f[PIdx_31]=InMat.f[PIdx_31];
#endif
#ifdef ILP32
	SMemMat.f[PIdx_32]=InMat.f[PIdx_32];
#endif
	InMat.e=SMemMat.e;
	InMat.f=SMemMat.f;
	SMemMat.Cms = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.Cms[PIdx]=InMat.Cms[PIdx];
	SMemMat.Cms[PIdx_2]=InMat.Cms[PIdx_2];
#ifdef ILP3
	SMemMat.Cms[PIdx_3]=InMat.Cms[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.Cms[PIdx_4]=InMat.Cms[PIdx_4];
#endif
#ifdef ILP5
	SMemMat.Cms[PIdx_5]=InMat.Cms[PIdx_5];
#endif
#ifdef ILP6
	SMemMat.Cms[PIdx_6]=InMat.Cms[PIdx_6];
#endif
#ifdef ILP7
	SMemMat.Cms[PIdx_7]=InMat.Cms[PIdx_7];
#endif
#ifdef ILP8
	SMemMat.Cms[PIdx_8]=InMat.Cms[PIdx_8];
#endif
#ifdef ILP9
	SMemMat.Cms[PIdx_9]=InMat.Cms[PIdx_9];
#endif
#ifdef ILP10
	SMemMat.Cms[PIdx_10]=InMat.Cms[PIdx_10];
#endif
#ifdef ILP11
	SMemMat.Cms[PIdx_11]=InMat.Cms[PIdx_11];
#endif
#ifdef ILP12
	SMemMat.Cms[PIdx_12]=InMat.Cms[PIdx_12];
#endif
#ifdef ILP13
	SMemMat.Cms[PIdx_13]=InMat.Cms[PIdx_13];
#endif
#ifdef ILP14
	SMemMat.Cms[PIdx_14]=InMat.Cms[PIdx_14];
#endif
#ifdef ILP15
	SMemMat.Cms[PIdx_15]=InMat.Cms[PIdx_15];
#endif
#ifdef ILP16
	SMemMat.Cms[PIdx_16]=InMat.Cms[PIdx_16];
#endif
#ifdef ILP17
	SMemMat.Cms[PIdx_17]=InMat.Cms[PIdx_17];
#endif
#ifdef ILP18
	SMemMat.Cms[PIdx_18]=InMat.Cms[PIdx_18];
#endif
#ifdef ILP19
	SMemMat.Cms[PIdx_19]=InMat.Cms[PIdx_19];
#endif
#ifdef ILP20
	SMemMat.Cms[PIdx_20]=InMat.Cms[PIdx_20];
#endif
#ifdef ILP21
	SMemMat.Cms[PIdx_21]=InMat.Cms[PIdx_21];
#endif
#ifdef ILP22
	SMemMat.Cms[PIdx_22]=InMat.Cms[PIdx_22];
#endif
#ifdef ILP23
	SMemMat.Cms[PIdx_23]=InMat.Cms[PIdx_23];
#endif
#ifdef ILP24
	SMemMat.Cms[PIdx_24]=InMat.Cms[PIdx_24];
#endif
#ifdef ILP25
	SMemMat.Cms[PIdx_25]=InMat.Cms[PIdx_25];
#endif
#ifdef ILP26
	SMemMat.Cms[PIdx_26]=InMat.Cms[PIdx_26];
#endif
#ifdef ILP27
	SMemMat.Cms[PIdx_27]=InMat.Cms[PIdx_27];
#endif
#ifdef ILP28
	SMemMat.Cms[PIdx_28]=InMat.Cms[PIdx_28];
#endif
#ifdef ILP29
	SMemMat.Cms[PIdx_29]=InMat.Cms[PIdx_29];
#endif
#ifdef ILP30
	SMemMat.Cms[PIdx_30]=InMat.Cms[PIdx_30];
#endif
#ifdef ILP31
	SMemMat.Cms[PIdx_31]=InMat.Cms[PIdx_31];
#endif
#ifdef ILP32
	SMemMat.Cms[PIdx_32]=InMat.Cms[PIdx_32];
#endif
	// 1

	// discrete
	SMemMat.FIdxs = (MYDTYPE*) &smem[offset];
	offset+=InMat.N*SMemMat.LognDepth*sizeof(MYDTYPE);
	for(int i=0;i<SMemMat.LognDepth;i++) {
		SMemMat.FIdxs[i*InMat.N+PIdx]=InMat.FIdxs[i*InMat.N+PIdx];
		SMemMat.FIdxs[i*InMat.N+PIdx_2]=InMat.FIdxs[i*InMat.N+PIdx_2];
#ifdef ILP3
		SMemMat.FIdxs[i*InMat.N+PIdx_3]=InMat.FIdxs[i*InMat.N+PIdx_3];
#endif
#ifdef ILP4
		SMemMat.FIdxs[i*InMat.N+PIdx_4]=InMat.FIdxs[i*InMat.N+PIdx_4];
#endif
#ifdef ILP5
		SMemMat.FIdxs[i*InMat.N+PIdx_5]=InMat.FIdxs[i*InMat.N+PIdx_5];
#endif
#ifdef ILP6
		SMemMat.FIdxs[i*InMat.N+PIdx_6]=InMat.FIdxs[i*InMat.N+PIdx_6];
#endif
#ifdef ILP7
		SMemMat.FIdxs[i*InMat.N+PIdx_7]=InMat.FIdxs[i*InMat.N+PIdx_7];
#endif
#ifdef ILP8
		SMemMat.FIdxs[i*InMat.N+PIdx_8]=InMat.FIdxs[i*InMat.N+PIdx_8];
#endif
#ifdef ILP9
		SMemMat.FIdxs[i*InMat.N+PIdx_9]=InMat.FIdxs[i*InMat.N+PIdx_9];
#endif
#ifdef ILP10
		SMemMat.FIdxs[i*InMat.N+PIdx_10]=InMat.FIdxs[i*InMat.N+PIdx_10];
#endif
#ifdef ILP11
		SMemMat.FIdxs[i*InMat.N+PIdx_11]=InMat.FIdxs[i*InMat.N+PIdx_11];
#endif
#ifdef ILP12
		SMemMat.FIdxs[i*InMat.N+PIdx_12]=InMat.FIdxs[i*InMat.N+PIdx_12];
#endif
#ifdef ILP13
		SMemMat.FIdxs[i*InMat.N+PIdx_13]=InMat.FIdxs[i*InMat.N+PIdx_13];
#endif
#ifdef ILP14
		SMemMat.FIdxs[i*InMat.N+PIdx_14]=InMat.FIdxs[i*InMat.N+PIdx_14];
#endif
#ifdef ILP15
		SMemMat.FIdxs[i*InMat.N+PIdx_15]=InMat.FIdxs[i*InMat.N+PIdx_15];
#endif
#ifdef ILP16
		SMemMat.FIdxs[i*InMat.N+PIdx_16]=InMat.FIdxs[i*InMat.N+PIdx_16];
#endif
#ifdef ILP17
		SMemMat.FIdxs[i*InMat.N+PIdx_17]=InMat.FIdxs[i*InMat.N+PIdx_17];
#endif
#ifdef ILP18
		SMemMat.FIdxs[i*InMat.N+PIdx_18]=InMat.FIdxs[i*InMat.N+PIdx_18];
#endif
#ifdef ILP19
		SMemMat.FIdxs[i*InMat.N+PIdx_19]=InMat.FIdxs[i*InMat.N+PIdx_19];
#endif
#ifdef ILP20
		SMemMat.FIdxs[i*InMat.N+PIdx_20]=InMat.FIdxs[i*InMat.N+PIdx_20];
#endif
#ifdef ILP21
		SMemMat.FIdxs[i*InMat.N+PIdx_21]=InMat.FIdxs[i*InMat.N+PIdx_21];
#endif
#ifdef ILP22
		SMemMat.FIdxs[i*InMat.N+PIdx_22]=InMat.FIdxs[i*InMat.N+PIdx_22];
#endif
#ifdef ILP23
		SMemMat.FIdxs[i*InMat.N+PIdx_23]=InMat.FIdxs[i*InMat.N+PIdx_23];
#endif
#ifdef ILP24
		SMemMat.FIdxs[i*InMat.N+PIdx_24]=InMat.FIdxs[i*InMat.N+PIdx_24];
#endif
#ifdef ILP25
		SMemMat.FIdxs[i*InMat.N+PIdx_25]=InMat.FIdxs[i*InMat.N+PIdx_25];
#endif
#ifdef ILP26
		SMemMat.FIdxs[i*InMat.N+PIdx_26]=InMat.FIdxs[i*InMat.N+PIdx_26];
#endif
#ifdef ILP27
		SMemMat.FIdxs[i*InMat.N+PIdx_27]=InMat.FIdxs[i*InMat.N+PIdx_27];
#endif
#ifdef ILP28
		SMemMat.FIdxs[i*InMat.N+PIdx_28]=InMat.FIdxs[i*InMat.N+PIdx_28];
#endif
#ifdef ILP29
		SMemMat.FIdxs[i*InMat.N+PIdx_29]=InMat.FIdxs[i*InMat.N+PIdx_29];
#endif
#ifdef ILP30
		SMemMat.FIdxs[i*InMat.N+PIdx_30]=InMat.FIdxs[i*InMat.N+PIdx_30];
#endif
#ifdef ILP31
		SMemMat.FIdxs[i*InMat.N+PIdx_31]=InMat.FIdxs[i*InMat.N+PIdx_31];
#endif
#ifdef ILP32
		SMemMat.FIdxs[i*InMat.N+PIdx_32]=InMat.FIdxs[i*InMat.N+PIdx_32];
#endif
	}
	
	SMemMat.Ks = (MYDTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYDTYPE);
	SMemMat.Ks[PIdx]=InMat.Ks[PIdx];
	SMemMat.Ks[PIdx_2]=InMat.Ks[PIdx_2];
#ifdef ILP3
	SMemMat.Ks[PIdx_3]=InMat.Ks[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.Ks[PIdx_4]=InMat.Ks[PIdx_4];
#endif
#ifdef ILP5
	SMemMat.Ks[PIdx_5]=InMat.Ks[PIdx_5];
#endif
#ifdef ILP6
	SMemMat.Ks[PIdx_6]=InMat.Ks[PIdx_6];
#endif
#ifdef ILP7
	SMemMat.Ks[PIdx_7]=InMat.Ks[PIdx_7];
#endif
#ifdef ILP8
	SMemMat.Ks[PIdx_8]=InMat.Ks[PIdx_8];
#endif
#ifdef ILP9
	SMemMat.Ks[PIdx_9]=InMat.Ks[PIdx_9];
#endif
#ifdef ILP10
	SMemMat.Ks[PIdx_10]=InMat.Ks[PIdx_10];
#endif
#ifdef ILP11
	SMemMat.Ks[PIdx_11]=InMat.Ks[PIdx_11];
#endif
#ifdef ILP12
	SMemMat.Ks[PIdx_12]=InMat.Ks[PIdx_12];
#endif
#ifdef ILP13
	SMemMat.Ks[PIdx_13]=InMat.Ks[PIdx_13];
#endif
#ifdef ILP14
	SMemMat.Ks[PIdx_14]=InMat.Ks[PIdx_14];
#endif
#ifdef ILP15
	SMemMat.Ks[PIdx_15]=InMat.Ks[PIdx_15];
#endif
#ifdef ILP16
	SMemMat.Ks[PIdx_16]=InMat.Ks[PIdx_16];
#endif
#ifdef ILP17
	SMemMat.Ks[PIdx_17]=InMat.Ks[PIdx_17];
#endif
#ifdef ILP18
	SMemMat.Ks[PIdx_18]=InMat.Ks[PIdx_18];
#endif
#ifdef ILP19
	SMemMat.Ks[PIdx_19]=InMat.Ks[PIdx_19];
#endif
#ifdef ILP20
	SMemMat.Ks[PIdx_20]=InMat.Ks[PIdx_20];
#endif
#ifdef ILP21
	SMemMat.Ks[PIdx_21]=InMat.Ks[PIdx_21];
#endif
#ifdef ILP22
	SMemMat.Ks[PIdx_22]=InMat.Ks[PIdx_22];
#endif
#ifdef ILP23
	SMemMat.Ks[PIdx_23]=InMat.Ks[PIdx_23];
#endif
#ifdef ILP24
	SMemMat.Ks[PIdx_24]=InMat.Ks[PIdx_24];
#endif
#ifdef ILP25
	SMemMat.Ks[PIdx_25]=InMat.Ks[PIdx_25];
#endif
#ifdef ILP26
	SMemMat.Ks[PIdx_26]=InMat.Ks[PIdx_26];
#endif
#ifdef ILP27
	SMemMat.Ks[PIdx_27]=InMat.Ks[PIdx_27];
#endif
#ifdef ILP28
	SMemMat.Ks[PIdx_28]=InMat.Ks[PIdx_28];
#endif
#ifdef ILP29
	SMemMat.Ks[PIdx_29]=InMat.Ks[PIdx_29];
#endif
#ifdef ILP30
	SMemMat.Ks[PIdx_30]=InMat.Ks[PIdx_30];
#endif
#ifdef ILP31
	SMemMat.Ks[PIdx_31]=InMat.Ks[PIdx_31];
#endif
#ifdef ILP32
	SMemMat.Ks[PIdx_32]=InMat.Ks[PIdx_32];
#endif
	
	SMemMat.boolModel= (MYDTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYDTYPE);
	SMemMat.boolModel[PIdx]=InMat.boolModel[PIdx];
	SMemMat.boolModel[PIdx_2]=InMat.boolModel[PIdx_2];
#ifdef ILP3
	SMemMat.boolModel[PIdx_3]=InMat.boolModel[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.boolModel[PIdx_4]=InMat.boolModel[PIdx_4];
#endif
#ifdef ILP5
	SMemMat.boolModel[PIdx_5]=InMat.boolModel[PIdx_5];
#endif
#ifdef ILP6
	SMemMat.boolModel[PIdx_6]=InMat.boolModel[PIdx_6];
#endif
#ifdef ILP7
	SMemMat.boolModel[PIdx_7]=InMat.boolModel[PIdx_7];
#endif
#ifdef ILP8
	SMemMat.boolModel[PIdx_8]=InMat.boolModel[PIdx_8];
#endif
#ifdef ILP9
	SMemMat.boolModel[PIdx_9]=InMat.boolModel[PIdx_9];
#endif
#ifdef ILP10
	SMemMat.boolModel[PIdx_10]=InMat.boolModel[PIdx_10];
#endif
#ifdef ILP11
	SMemMat.boolModel[PIdx_11]=InMat.boolModel[PIdx_11];
#endif
#ifdef ILP12
	SMemMat.boolModel[PIdx_12]=InMat.boolModel[PIdx_12];
#endif
#ifdef ILP13
	SMemMat.boolModel[PIdx_13]=InMat.boolModel[PIdx_13];
#endif
#ifdef ILP14
	SMemMat.boolModel[PIdx_14]=InMat.boolModel[PIdx_14];
#endif
#ifdef ILP15
	SMemMat.boolModel[PIdx_15]=InMat.boolModel[PIdx_15];
#endif
#ifdef ILP16
	SMemMat.boolModel[PIdx_16]=InMat.boolModel[PIdx_16];
#endif
#ifdef ILP17
	SMemMat.boolModel[PIdx_17]=InMat.boolModel[PIdx_17];
#endif
#ifdef ILP18
	SMemMat.boolModel[PIdx_18]=InMat.boolModel[PIdx_18];
#endif
#ifdef ILP19
	SMemMat.boolModel[PIdx_19]=InMat.boolModel[PIdx_19];
#endif
#ifdef ILP20
	SMemMat.boolModel[PIdx_20]=InMat.boolModel[PIdx_20];
#endif
#ifdef ILP21
	SMemMat.boolModel[PIdx_21]=InMat.boolModel[PIdx_21];
#endif
#ifdef ILP22
	SMemMat.boolModel[PIdx_22]=InMat.boolModel[PIdx_22];
#endif
#ifdef ILP23
	SMemMat.boolModel[PIdx_23]=InMat.boolModel[PIdx_23];
#endif
#ifdef ILP24
	SMemMat.boolModel[PIdx_24]=InMat.boolModel[PIdx_24];
#endif
#ifdef ILP25
	SMemMat.boolModel[PIdx_25]=InMat.boolModel[PIdx_25];
#endif
#ifdef ILP26
	SMemMat.boolModel[PIdx_26]=InMat.boolModel[PIdx_26];
#endif
#ifdef ILP27
	SMemMat.boolModel[PIdx_27]=InMat.boolModel[PIdx_27];
#endif
#ifdef ILP28
	SMemMat.boolModel[PIdx_28]=InMat.boolModel[PIdx_28];
#endif
#ifdef ILP29
	SMemMat.boolModel[PIdx_29]=InMat.boolModel[PIdx_29];
#endif
#ifdef ILP30
	SMemMat.boolModel[PIdx_30]=InMat.boolModel[PIdx_30];
#endif
#ifdef ILP31
	SMemMat.boolModel[PIdx_31]=InMat.boolModel[PIdx_31];
#endif
#ifdef ILP32
	SMemMat.boolModel[PIdx_32]=InMat.boolModel[PIdx_32];
#endif
	// 2
	MYDTYPE SonNo=InMat.SonNoVec[PIdx];
	MYDTYPE SonNo_2=InMat.SonNoVec[PIdx_2];
#ifdef ILP3
	MYDTYPE SonNo_3=InMat.SonNoVec[PIdx_3];
#endif
#ifdef ILP4
	MYDTYPE SonNo_4=InMat.SonNoVec[PIdx_4];
#endif
#ifdef ILP5
	MYDTYPE SonNo_5=InMat.SonNoVec[PIdx_5];
#endif
#ifdef ILP6
	MYDTYPE SonNo_6=InMat.SonNoVec[PIdx_6];
#endif
#ifdef ILP7
	MYDTYPE SonNo_7=InMat.SonNoVec[PIdx_7];
#endif
#ifdef ILP8
	MYDTYPE SonNo_8=InMat.SonNoVec[PIdx_8];
#endif
#ifdef ILP9
	MYDTYPE SonNo_9=InMat.SonNoVec[PIdx_9];
#endif
#ifdef ILP10
	MYDTYPE SonNo_10=InMat.SonNoVec[PIdx_10];
#endif
#ifdef ILP11
	MYDTYPE SonNo_11=InMat.SonNoVec[PIdx_11];
#endif
#ifdef ILP12
	MYDTYPE SonNo_12=InMat.SonNoVec[PIdx_12];
#endif
#ifdef ILP13
	MYDTYPE SonNo_13=InMat.SonNoVec[PIdx_13];
#endif
#ifdef ILP14
	MYDTYPE SonNo_14=InMat.SonNoVec[PIdx_14];
#endif
#ifdef ILP15
	MYDTYPE SonNo_15=InMat.SonNoVec[PIdx_15];
#endif
#ifdef ILP16
	MYDTYPE SonNo_16=InMat.SonNoVec[PIdx_16];
#endif
#ifdef ILP17
	MYDTYPE SonNo_17=InMat.SonNoVec[PIdx_17];
#endif
#ifdef ILP18
	MYDTYPE SonNo_18=InMat.SonNoVec[PIdx_18];
#endif
#ifdef ILP19
	MYDTYPE SonNo_19=InMat.SonNoVec[PIdx_19];
#endif
#ifdef ILP20
	MYDTYPE SonNo_20=InMat.SonNoVec[PIdx_20];
#endif
#ifdef ILP21
	MYDTYPE SonNo_21=InMat.SonNoVec[PIdx_21];
#endif
#ifdef ILP22
	MYDTYPE SonNo_22=InMat.SonNoVec[PIdx_22];
#endif
#ifdef ILP23
	MYDTYPE SonNo_23=InMat.SonNoVec[PIdx_23];
#endif
#ifdef ILP24
	MYDTYPE SonNo_24=InMat.SonNoVec[PIdx_24];
#endif
#ifdef ILP25
	MYDTYPE SonNo_25=InMat.SonNoVec[PIdx_25];
#endif
#ifdef ILP26
	MYDTYPE SonNo_26=InMat.SonNoVec[PIdx_26];
#endif
#ifdef ILP27
	MYDTYPE SonNo_27=InMat.SonNoVec[PIdx_27];
#endif
#ifdef ILP28
	MYDTYPE SonNo_28=InMat.SonNoVec[PIdx_28];
#endif
#ifdef ILP29
	MYDTYPE SonNo_29=InMat.SonNoVec[PIdx_29];
#endif
#ifdef ILP30
	MYDTYPE SonNo_30=InMat.SonNoVec[PIdx_30];
#endif
#ifdef ILP31
	MYDTYPE SonNo_31=InMat.SonNoVec[PIdx_31];
#endif
#ifdef ILP32
	MYDTYPE SonNo_32=InMat.SonNoVec[PIdx_32];
#endif
	
	SMemMat.RelStarts = (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	SMemMat.RelEnds = (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	SMemMat.RelVec = (MYDTYPE*) &smem[offset];
	offset+=InMat.nCallForFather*sizeof(MYDTYPE);
	SMemMat.SegStartI = (MYDTYPE*) &smem[offset];
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);
	SMemMat.SegEndI = (MYDTYPE*) &smem[offset];
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);

	SMemMat.Fathers= (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	// 3
	SMemMat.CompByLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	offset+=WARPSIZE*(SMemMat.Depth+1)*sizeof(MYDTYPE);
	SMemMat.CompByFLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	offset+=WARPSIZE*(SMemMat.Depth)*sizeof(MYDTYPE);
	
	// Moving back to floats - make sure that the address is good for float (is modulu 4).
	offset=ceilf(float(offset)/sizeof(MYFTYPE))*sizeof(MYFTYPE);
	MYDTYPE ConstantMemSize=offset;
	
	if(PIdx<InMat.nFathers) { // up to 32 !!!
		SMemMat.RelStarts[PIdx]=InMat.RelStarts[PIdx];
		SMemMat.RelEnds[PIdx]=InMat.RelEnds[PIdx];
		SMemMat.Fathers[PIdx]=InMat.Fathers[PIdx];
	}
	if(PIdx<InMat.nCallForFather) { // up to 32 !!!
		SMemMat.RelVec[PIdx]=InMat.RelVec[PIdx];
	}
	if(PIdx<InMat.nCallForFather+1) { // up to 32 !!!
		SMemMat.SegStartI[PIdx]=InMat.SegStartI[PIdx];
		SMemMat.SegEndI[PIdx]=InMat.SegEndI[PIdx];
	}
	syncthreads();
	
	for(int i=0;i<=SMemMat.Depth;i++) {
		SMemMat.CompByLevel32[WARPSIZE*i+PIdx]=InMat.CompByLevel32[WARPSIZE*i+PIdx];
	}
	for(int i=0;i<SMemMat.Depth;i++) {
		SMemMat.CompByFLevel32[WARPSIZE*i+PIdx]=InMat.CompByFLevel32[WARPSIZE*i+PIdx];
	}

	// Per STIMULUS
	// floats
	int Nt=ceil(sim.TFinal/sim.dt);
	// For solving the matrix
	MYDTYPE PerStimulus=((InMat.N+2)*2+InMat.N+WARPSIZE)*sizeof(MYFTYPE);
	offset+=PerStimulus*threadIdx.y;
	MYFTYPE *uHP,*bHP,*PX,*PF, *Vs;
	uHP = (MYFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYFTYPE);
	bHP = (MYFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYFTYPE);
	Vs = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	Vs[PIdx]=V[PIdx];
	Vs[PIdx_2]=V[PIdx_2];
#ifdef ILP3
	Vs[PIdx_3]=V[PIdx_3];
#endif
#ifdef ILP4
	Vs[PIdx_4]=V[PIdx_4];
#endif
#ifdef ILP5
	Vs[PIdx_5]=V[PIdx_5];
#endif
#ifdef ILP6
	Vs[PIdx_6]=V[PIdx_6];
#endif
#ifdef ILP7
	Vs[PIdx_7]=V[PIdx_7];
#endif
#ifdef ILP8
	Vs[PIdx_8]=V[PIdx_8];
#endif
#ifdef ILP9
	Vs[PIdx_9]=V[PIdx_9];
#endif
#ifdef ILP10
	Vs[PIdx_10]=V[PIdx_10];
#endif
#ifdef ILP11
	Vs[PIdx_11]=V[PIdx_11];
#endif
#ifdef ILP12
	Vs[PIdx_12]=V[PIdx_12];
#endif
#ifdef ILP13
	Vs[PIdx_13]=V[PIdx_13];
#endif
#ifdef ILP14
	Vs[PIdx_14]=V[PIdx_14];
#endif
#ifdef ILP15
	Vs[PIdx_15]=V[PIdx_15];
#endif
#ifdef ILP16
	Vs[PIdx_16]=V[PIdx_16];
#endif
#ifdef ILP17
	Vs[PIdx_17]=V[PIdx_17];
#endif
#ifdef ILP18
	Vs[PIdx_18]=V[PIdx_18];
#endif
#ifdef ILP19
	Vs[PIdx_19]=V[PIdx_19];
#endif
#ifdef ILP20
	Vs[PIdx_20]=V[PIdx_20];
#endif
#ifdef ILP21
	Vs[PIdx_21]=V[PIdx_21];
#endif
#ifdef ILP22
	Vs[PIdx_22]=V[PIdx_22];
#endif
#ifdef ILP23
	Vs[PIdx_23]=V[PIdx_23];
#endif
#ifdef ILP24
	Vs[PIdx_24]=V[PIdx_24];
#endif
#ifdef ILP25
	Vs[PIdx_25]=V[PIdx_25];
#endif
#ifdef ILP26
	Vs[PIdx_26]=V[PIdx_26];
#endif
#ifdef ILP27
	Vs[PIdx_27]=V[PIdx_27];
#endif
#ifdef ILP28
	Vs[PIdx_28]=V[PIdx_28];
#endif
#ifdef ILP29
	Vs[PIdx_29]=V[PIdx_29];
#endif
#ifdef ILP30
	Vs[PIdx_30]=V[PIdx_30];
#endif
#ifdef ILP31
	Vs[PIdx_31]=V[PIdx_31];
#endif
#ifdef ILP32
	Vs[PIdx_32]=V[PIdx_32];
#endif

	PX=bHP;
	PF=uHP;
	MYFTYPE *SMemVHot;
	
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=WARPSIZE*sizeof(MYFTYPE);

	MYDTYPE parentIndex;
	MYDTYPE parentIndex_2;
#ifdef ILP3
	MYDTYPE parentIndex_3;
#endif
#ifdef ILP4
	MYDTYPE parentIndex_4;
#endif
#ifdef ILP5
	MYDTYPE parentIndex_5;
#endif
#ifdef ILP6
	MYDTYPE parentIndex_6;
#endif
#ifdef ILP7
	MYDTYPE parentIndex_7;
#endif
#ifdef ILP8
	MYDTYPE parentIndex_8;
#endif
#ifdef ILP9
	MYDTYPE parentIndex_9;
#endif
#ifdef ILP10
	MYDTYPE parentIndex_10;
#endif
#ifdef ILP11
	MYDTYPE parentIndex_11;
#endif
#ifdef ILP12
	MYDTYPE parentIndex_12;
#endif
#ifdef ILP13
	MYDTYPE parentIndex_13;
#endif
#ifdef ILP14
	MYDTYPE parentIndex_14;
#endif
#ifdef ILP15
	MYDTYPE parentIndex_15;
#endif
#ifdef ILP16
	MYDTYPE parentIndex_16;
#endif
#ifdef ILP17
	MYDTYPE parentIndex_17;
#endif
#ifdef ILP18
	MYDTYPE parentIndex_18;
#endif
#ifdef ILP19
	MYDTYPE parentIndex_19;
#endif
#ifdef ILP20
	MYDTYPE parentIndex_20;
#endif
#ifdef ILP21
	MYDTYPE parentIndex_21;
#endif
#ifdef ILP22
	MYDTYPE parentIndex_22;
#endif
#ifdef ILP23
	MYDTYPE parentIndex_23;
#endif
#ifdef ILP24
	MYDTYPE parentIndex_24;
#endif
#ifdef ILP25
	MYDTYPE parentIndex_25;
#endif
#ifdef ILP26
	MYDTYPE parentIndex_26;
#endif
#ifdef ILP27
	MYDTYPE parentIndex_27;
#endif
#ifdef ILP28
	MYDTYPE parentIndex_28;
#endif
#ifdef ILP29
	MYDTYPE parentIndex_29;
#endif
#ifdef ILP30
	MYDTYPE parentIndex_30;
#endif
#ifdef ILP31
	MYDTYPE parentIndex_31;
#endif
#ifdef ILP32
	MYDTYPE parentIndex_32;
#endif
	MYDTYPE Eidx;
	MYDTYPE Eidx_2;
#ifdef ILP3
	MYDTYPE Eidx_3;
#endif
#ifdef ILP4
	MYDTYPE Eidx_4;
#endif
#ifdef ILP5
	MYDTYPE Eidx_5;
#endif
#ifdef ILP6
	MYDTYPE Eidx_6;
#endif
#ifdef ILP7
	MYDTYPE Eidx_7;
#endif
#ifdef ILP8
	MYDTYPE Eidx_8;
#endif
#ifdef ILP9
	MYDTYPE Eidx_9;
#endif
#ifdef ILP10
	MYDTYPE Eidx_10;
#endif
#ifdef ILP11
	MYDTYPE Eidx_11;
#endif
#ifdef ILP12
	MYDTYPE Eidx_12;
#endif
#ifdef ILP13
	MYDTYPE Eidx_13;
#endif
#ifdef ILP14
	MYDTYPE Eidx_14;
#endif
#ifdef ILP15
	MYDTYPE Eidx_15;
#endif
#ifdef ILP16
	MYDTYPE Eidx_16;
#endif
#ifdef ILP17
	MYDTYPE Eidx_17;
#endif
#ifdef ILP18
	MYDTYPE Eidx_18;
#endif
#ifdef ILP19
	MYDTYPE Eidx_19;
#endif
#ifdef ILP20
	MYDTYPE Eidx_20;
#endif
#ifdef ILP21
	MYDTYPE Eidx_21;
#endif
#ifdef ILP22
	MYDTYPE Eidx_22;
#endif
#ifdef ILP23
	MYDTYPE Eidx_23;
#endif
#ifdef ILP24
	MYDTYPE Eidx_24;
#endif
#ifdef ILP25
	MYDTYPE Eidx_25;
#endif
#ifdef ILP26
	MYDTYPE Eidx_26;
#endif
#ifdef ILP27
	MYDTYPE Eidx_27;
#endif
#ifdef ILP28
	MYDTYPE Eidx_28;
#endif
#ifdef ILP29
	MYDTYPE Eidx_29;
#endif
#ifdef ILP30
	MYDTYPE Eidx_30;
#endif
#ifdef ILP31
	MYDTYPE Eidx_31;
#endif
#ifdef ILP32
	MYDTYPE Eidx_32;
#endif
	Eidx = SMemMat.N-PIdx;
	Eidx_2 = SMemMat.N-PIdx_2;
#ifdef ILP3
	Eidx_3 = SMemMat.N-PIdx_3;
#endif
#ifdef ILP4
	Eidx_4 = SMemMat.N-PIdx_4;
#endif
#ifdef ILP5
	Eidx_5 = SMemMat.N-PIdx_5;
#endif
#ifdef ILP6
	Eidx_6 = SMemMat.N-PIdx_6;
#endif
#ifdef ILP7
	Eidx_7 = SMemMat.N-PIdx_7;
#endif
#ifdef ILP8
	Eidx_8 = SMemMat.N-PIdx_8;
#endif
#ifdef ILP9
	Eidx_9 = SMemMat.N-PIdx_9;
#endif
#ifdef ILP10
	Eidx_10 = SMemMat.N-PIdx_10;
#endif
#ifdef ILP11
	Eidx_11 = SMemMat.N-PIdx_11;
#endif
#ifdef ILP12
	Eidx_12 = SMemMat.N-PIdx_12;
#endif
#ifdef ILP13
	Eidx_13 = SMemMat.N-PIdx_13;
#endif
#ifdef ILP14
	Eidx_14 = SMemMat.N-PIdx_14;
#endif
#ifdef ILP15
	Eidx_15 = SMemMat.N-PIdx_15;
#endif
#ifdef ILP16
	Eidx_16 = SMemMat.N-PIdx_16;
#endif
#ifdef ILP17
	Eidx_17 = SMemMat.N-PIdx_17;
#endif
#ifdef ILP18
	Eidx_18 = SMemMat.N-PIdx_18;
#endif
#ifdef ILP19
	Eidx_19 = SMemMat.N-PIdx_19;
#endif
#ifdef ILP20
	Eidx_20 = SMemMat.N-PIdx_20;
#endif
#ifdef ILP21
	Eidx_21 = SMemMat.N-PIdx_21;
#endif
#ifdef ILP22
	Eidx_22 = SMemMat.N-PIdx_22;
#endif
#ifdef ILP23
	Eidx_23 = SMemMat.N-PIdx_23;
#endif
#ifdef ILP24
	Eidx_24 = SMemMat.N-PIdx_24;
#endif
#ifdef ILP25
	Eidx_25 = SMemMat.N-PIdx_25;
#endif
#ifdef ILP26
	Eidx_26 = SMemMat.N-PIdx_26;
#endif
#ifdef ILP27
	Eidx_27 = SMemMat.N-PIdx_27;
#endif
#ifdef ILP28
	Eidx_28 = SMemMat.N-PIdx_28;
#endif
#ifdef ILP29
	Eidx_29 = SMemMat.N-PIdx_29;
#endif
#ifdef ILP30
	Eidx_30 = SMemMat.N-PIdx_30;
#endif
#ifdef ILP31
	Eidx_31 = SMemMat.N-PIdx_31;
#endif
#ifdef ILP32
	Eidx_32 = SMemMat.N-PIdx_32;
#endif
	if(Eidx>SMemMat.N-1){
		Eidx=SMemMat.N-1;
	}
	if(PIdx==0) {
		parentIndex=0; }
	else {
		parentIndex =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx];
	}
	
	parentIndex_2 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_2];
#ifdef ILP3
	parentIndex_3 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_3];
#endif
#ifdef ILP4
	parentIndex_4 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_4];
#endif
#ifdef ILP5
	parentIndex_5 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_5];
#endif
#ifdef ILP6
	parentIndex_6 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_6];
#endif
#ifdef ILP7
	parentIndex_7 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_7];
#endif
#ifdef ILP8
	parentIndex_8 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_8];
#endif
#ifdef ILP9
	parentIndex_9 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_9];
#endif
#ifdef ILP10
	parentIndex_10 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_10];
#endif
#ifdef ILP11
	parentIndex_11 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_11];
#endif
#ifdef ILP12
	parentIndex_12 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_12];
#endif
#ifdef ILP13
	parentIndex_13 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_13];
#endif
#ifdef ILP14
	parentIndex_14 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_14];
#endif
#ifdef ILP15
	parentIndex_15 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_15];
#endif
#ifdef ILP16
	parentIndex_16 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_16];
#endif
#ifdef ILP17
	parentIndex_17 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_17];
#endif
#ifdef ILP18
	parentIndex_18 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_18];
#endif
#ifdef ILP19
	parentIndex_19 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_19];
#endif
#ifdef ILP20
	parentIndex_20 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_20];
#endif
#ifdef ILP21
	parentIndex_21 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_21];
#endif
#ifdef ILP22
	parentIndex_22 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_22];
#endif
#ifdef ILP23
	parentIndex_23 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_23];
#endif
#ifdef ILP24
	parentIndex_24 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_24];
#endif
#ifdef ILP25
	parentIndex_25 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_25];
#endif
#ifdef ILP26
	parentIndex_26 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_26];
#endif
#ifdef ILP27
	parentIndex_27 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_27];
#endif
#ifdef ILP28
	parentIndex_28 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_28];
#endif
#ifdef ILP29
	parentIndex_29 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_29];
#endif
#ifdef ILP30
	parentIndex_30 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_30];
#endif
#ifdef ILP31
	parentIndex_31 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_31];
#endif
#ifdef ILP32
	parentIndex_32 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_32];
#endif

	float rhs,D,gModel,StimCurrent,dv;
	float rhs_2,D_2,gModel_2,StimCurrent_2,dv_2;
#ifdef ILP3
	float rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#endif
#ifdef ILP4
	float rhs_4,D_4,gModel_4,StimCurrent_4,dv_4;
#endif
#ifdef ILP5
	float rhs_5,D_5,gModel_5,StimCurrent_5,dv_5;
#endif
#ifdef ILP6
	float rhs_6,D_6,gModel_6,StimCurrent_6,dv_6;
#endif
#ifdef ILP7
	float rhs_7,D_7,gModel_7,StimCurrent_7,dv_7;
#endif
#ifdef ILP8
	float rhs_8,D_8,gModel_8,StimCurrent_8,dv_8;
#endif
#ifdef ILP9
	float rhs_9,D_9,gModel_9,StimCurrent_9,dv_9;
#endif
#ifdef ILP10
	float rhs_10,D_10,gModel_10,StimCurrent_10,dv_10;
#endif
#ifdef ILP11
	float rhs_11,D_11,gModel_11,StimCurrent_11,dv_11;
#endif
#ifdef ILP12
	float rhs_12,D_12,gModel_12,StimCurrent_12,dv_12;
#endif
#ifdef ILP13
	float rhs_13,D_13,gModel_13,StimCurrent_13,dv_13;
#endif
#ifdef ILP14
	float rhs_14,D_14,gModel_14,StimCurrent_14,dv_14;
#endif
#ifdef ILP15
	float rhs_15,D_15,gModel_15,StimCurrent_15,dv_15;
#endif
#ifdef ILP16
	float rhs_16,D_16,gModel_16,StimCurrent_16,dv_16;
#endif
#ifdef ILP17
	float rhs_17,D_17,gModel_17,StimCurrent_17,dv_17;
#endif
#ifdef ILP18
	float rhs_18,D_18,gModel_18,StimCurrent_18,dv_18;
#endif
#ifdef ILP19
	float rhs_19,D_19,gModel_19,StimCurrent_19,dv_19;
#endif
#ifdef ILP20
	float rhs_20,D_20,gModel_20,StimCurrent_20,dv_20;
#endif
#ifdef ILP21
	float rhs_21,D_21,gModel_21,StimCurrent_21,dv_21;
#endif
#ifdef ILP22
	float rhs_22,D_22,gModel_22,StimCurrent_22,dv_22;
#endif
#ifdef ILP23
	float rhs_23,D_23,gModel_23,StimCurrent_23,dv_23;
#endif
#ifdef ILP24
	float rhs_24,D_24,gModel_24,StimCurrent_24,dv_24;
#endif
#ifdef ILP25
	float rhs_25,D_25,gModel_25,StimCurrent_25,dv_25;
#endif
#ifdef ILP26
	float rhs_26,D_26,gModel_26,StimCurrent_26,dv_26;
#endif
#ifdef ILP27
	float rhs_27,D_27,gModel_27,StimCurrent_27,dv_27;
#endif
#ifdef ILP28
	float rhs_28,D_28,gModel_28,StimCurrent_28,dv_28;
#endif
#ifdef ILP29
	float rhs_29,D_29,gModel_29,StimCurrent_29,dv_29;
#endif
#ifdef ILP30
	float rhs_30,D_30,gModel_30,StimCurrent_30,dv_30;
#endif
#ifdef ILP31
	float rhs_31,D_31,gModel_31,StimCurrent_31,dv_31;
#endif
#ifdef ILP32
	float rhs_32,D_32,gModel_32,StimCurrent_32,dv_32;
#endif
	for(int i=0;i<Nt;i++) {
		t = i*sim.dt;

		// Output
		rhs=0;
		rhs_2=0;
#ifdef ILP3
		rhs_3=0;
#endif
#ifdef ILP4
		rhs_4=0;
#endif
#ifdef ILP5
		rhs_5=0;
#endif
#ifdef ILP6
		rhs_6=0;
#endif
#ifdef ILP7
		rhs_7=0;
#endif
#ifdef ILP8
		rhs_8=0;
#endif
#ifdef ILP9
		rhs_9=0;
#endif
#ifdef ILP10
		rhs_10=0;
#endif
#ifdef ILP11
		rhs_11=0;
#endif
#ifdef ILP12
		rhs_12=0;
#endif
#ifdef ILP13
		rhs_13=0;
#endif
#ifdef ILP14
		rhs_14=0;
#endif
#ifdef ILP15
		rhs_15=0;
#endif
#ifdef ILP16
		rhs_16=0;
#endif
#ifdef ILP17
		rhs_17=0;
#endif
#ifdef ILP18
		rhs_18=0;
#endif
#ifdef ILP19
		rhs_19=0;
#endif
#ifdef ILP20
		rhs_20=0;
#endif
#ifdef ILP21
		rhs_21=0;
#endif
#ifdef ILP22
		rhs_22=0;
#endif
#ifdef ILP23
		rhs_23=0;
#endif
#ifdef ILP24
		rhs_24=0;
#endif
#ifdef ILP25
		rhs_25=0;
#endif
#ifdef ILP26
		rhs_26=0;
#endif
#ifdef ILP27
		rhs_27=0;
#endif
#ifdef ILP28
		rhs_28=0;
#endif
#ifdef ILP29
		rhs_29=0;
#endif
#ifdef ILP30
		rhs_30=0;
#endif
#ifdef ILP31
		rhs_31=0;
#endif
#ifdef ILP32
		rhs_32=0;
#endif
		D=0;
		D_2=0;
#ifdef ILP3
		D_3=0;
#endif
#ifdef ILP4
		D_4=0;
#endif
#ifdef ILP5
		D_5=0;
#endif
#ifdef ILP6
		D_6=0;
#endif
#ifdef ILP7
		D_7=0;
#endif
#ifdef ILP8
		D_8=0;
#endif
#ifdef ILP9
		D_9=0;
#endif
#ifdef ILP10
		D_10=0;
#endif
#ifdef ILP11
		D_11=0;
#endif
#ifdef ILP12
		D_12=0;
#endif
#ifdef ILP13
		D_13=0;
#endif
#ifdef ILP14
		D_14=0;
#endif
#ifdef ILP15
		D_15=0;
#endif
#ifdef ILP16
		D_16=0;
#endif
#ifdef ILP17
		D_17=0;
#endif
#ifdef ILP18
		D_18=0;
#endif
#ifdef ILP19
		D_19=0;
#endif
#ifdef ILP20
		D_20=0;
#endif
#ifdef ILP21
		D_21=0;
#endif
#ifdef ILP22
		D_22=0;
#endif
#ifdef ILP23
		D_23=0;
#endif
#ifdef ILP24
		D_24=0;
#endif
#ifdef ILP25
		D_25=0;
#endif
#ifdef ILP26
		D_26=0;
#endif
#ifdef ILP27
		D_27=0;
#endif
#ifdef ILP28
		D_28=0;
#endif
#ifdef ILP29
		D_29=0;
#endif
#ifdef ILP30
		D_30=0;
#endif
#ifdef ILP31
		D_31=0;
#endif
#ifdef ILP32
		D_32=0;
#endif

		// Before matrix

		//RRR here some work - this should do it    
		// CALL_TO_KERNEL_BREAK
		// CALL_TO_KERNEL_BREAK_dV

		KBreakpointModel(sumCurrents, sumConductivity,v,ModelStates[0],ModelStates[1],ModelStates[2],p1,p2,p3,p4,p5,p6);
		KBreakpointModel(sumCurrents_2, sumConductivity_2,v_2,ModelStates_2[0],ModelStates_2[1],ModelStates_2[2],p1_2,p2_2,p3_2,p4_2,p5_2,p6_2);
#ifdef ILP3
		KBreakpointModel(sumCurrents_3, sumConductivity_3,v_3,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2],p1_3,p2_3,p3_3,p4_3,p5_3,p6_3);
#endif
#ifdef ILP4
		KBreakpointModel(sumCurrents_4, sumConductivity_4,v_4,ModelStates_4[0],ModelStates_4[1],ModelStates_4[2],p1_4,p2_4,p3_4,p4_4,p5_4,p6_4);
#endif
#ifdef ILP5
		KBreakpointModel(sumCurrents_5, sumConductivity_5,v_5,ModelStates_5[0],ModelStates_5[1],ModelStates_5[2],p1_5,p2_5,p3_5,p4_5,p5_5,p6_5);
#endif
#ifdef ILP6
		KBreakpointModel(sumCurrents_6, sumConductivity_6,v_6,ModelStates_6[0],ModelStates_6[1],ModelStates_6[2],p1_6,p2_6,p3_6,p4_6,p5_6,p6_6);
#endif
#ifdef ILP7
		KBreakpointModel(sumCurrents_7, sumConductivity_7,v_7,ModelStates_7[0],ModelStates_7[1],ModelStates_7[2],p1_7,p2_7,p3_7,p4_7,p5_7,p6_7);
#endif
#ifdef ILP8
		KBreakpointModel(sumCurrents_8, sumConductivity_8,v_8,ModelStates_8[0],ModelStates_8[1],ModelStates_8[2],p1_8,p2_8,p3_8,p4_8,p5_8,p6_8);
#endif
#ifdef ILP9
		KBreakpointModel(sumCurrents_9, sumConductivity_9,v_9,ModelStates_9[0],ModelStates_9[1],ModelStates_9[2],p1_9,p2_9,p3_9,p4_9,p5_9,p6_9);
#endif
#ifdef ILP10
		KBreakpointModel(sumCurrents_10, sumConductivity_10,v_10,ModelStates_10[0],ModelStates_10[1],ModelStates_10[2],p1_10,p2_10,p3_10,p4_10,p5_10,p6_10);
#endif
#ifdef ILP11
		KBreakpointModel(sumCurrents_11, sumConductivity_11,v_11,ModelStates_11[0],ModelStates_11[1],ModelStates_11[2],p1_11,p2_11,p3_11,p4_11,p5_11,p6_11);
#endif
#ifdef ILP12
		KBreakpointModel(sumCurrents_12, sumConductivity_12,v_12,ModelStates_12[0],ModelStates_12[1],ModelStates_12[2],p1_12,p2_12,p3_12,p4_12,p5_12,p6_12);
#endif
#ifdef ILP13
		KBreakpointModel(sumCurrents_13, sumConductivity_13,v_13,ModelStates_13[0],ModelStates_13[1],ModelStates_13[2],p1_13,p2_13,p3_13,p4_13,p5_13,p6_13);
#endif
#ifdef ILP14
		KBreakpointModel(sumCurrents_14, sumConductivity_14,v_14,ModelStates_14[0],ModelStates_14[1],ModelStates_14[2],p1_14,p2_14,p3_14,p4_14,p5_14,p6_14);
#endif
#ifdef ILP15
		KBreakpointModel(sumCurrents_15, sumConductivity_15,v_15,ModelStates_15[0],ModelStates_15[1],ModelStates_15[2],p1_15,p2_15,p3_15,p4_15,p5_15,p6_15);
#endif
#ifdef ILP16
		KBreakpointModel(sumCurrents_16, sumConductivity_16,v_16,ModelStates_16[0],ModelStates_16[1],ModelStates_16[2],p1_16,p2_16,p3_16,p4_16,p5_16,p6_16);
#endif
#ifdef ILP17
		KBreakpointModel(sumCurrents_17, sumConductivity_17,v_17,ModelStates_17[0],ModelStates_17[1],ModelStates_17[2],p1_17,p2_17,p3_17,p4_17,p5_17,p6_17);
#endif
#ifdef ILP18
		KBreakpointModel(sumCurrents_18, sumConductivity_18,v_18,ModelStates_18[0],ModelStates_18[1],ModelStates_18[2],p1_18,p2_18,p3_18,p4_18,p5_18,p6_18);
#endif
#ifdef ILP19
		KBreakpointModel(sumCurrents_19, sumConductivity_19,v_19,ModelStates_19[0],ModelStates_19[1],ModelStates_19[2],p1_19,p2_19,p3_19,p4_19,p5_19,p6_19);
#endif
#ifdef ILP20
		KBreakpointModel(sumCurrents_20, sumConductivity_20,v_20,ModelStates_20[0],ModelStates_20[1],ModelStates_20[2],p1_20,p2_20,p3_20,p4_20,p5_20,p6_20);
#endif
#ifdef ILP21
		KBreakpointModel(sumCurrents_21, sumConductivity_21,v_21,ModelStates_21[0],ModelStates_21[1],ModelStates_21[2],p1_21,p2_21,p3_21,p4_21,p5_21,p6_21);
#endif
#ifdef ILP22
		KBreakpointModel(sumCurrents_22, sumConductivity_22,v_22,ModelStates_22[0],ModelStates_22[1],ModelStates_22[2],p1_22,p2_22,p3_22,p4_22,p5_22,p6_22);
#endif
#ifdef ILP23
		KBreakpointModel(sumCurrents_23, sumConductivity_23,v_23,ModelStates_23[0],ModelStates_23[1],ModelStates_23[2],p1_23,p2_23,p3_23,p4_23,p5_23,p6_23);
#endif
#ifdef ILP24
		KBreakpointModel(sumCurrents_24, sumConductivity_24,v_24,ModelStates_24[0],ModelStates_24[1],ModelStates_24[2],p1_24,p2_24,p3_24,p4_24,p5_24,p6_24);
#endif
#ifdef ILP25
		KBreakpointModel(sumCurrents_25, sumConductivity_25,v_25,ModelStates_25[0],ModelStates_25[1],ModelStates_25[2],p1_25,p2_25,p3_25,p4_25,p5_25,p6_25);
#endif
#ifdef ILP26
		KBreakpointModel(sumCurrents_26, sumConductivity_26,v_26,ModelStates_26[0],ModelStates_26[1],ModelStates_26[2],p1_26,p2_26,p3_26,p4_26,p5_26,p6_26);
#endif
#ifdef ILP27
		KBreakpointModel(sumCurrents_27, sumConductivity_27,v_27,ModelStates_27[0],ModelStates_27[1],ModelStates_27[2],p1_27,p2_27,p3_27,p4_27,p5_27,p6_27);
#endif
#ifdef ILP28
		KBreakpointModel(sumCurrents_28, sumConductivity_28,v_28,ModelStates_28[0],ModelStates_28[1],ModelStates_28[2],p1_28,p2_28,p3_28,p4_28,p5_28,p6_28);
#endif
#ifdef ILP29
		KBreakpointModel(sumCurrents_29, sumConductivity_29,v_29,ModelStates_29[0],ModelStates_29[1],ModelStates_29[2],p1_29,p2_29,p3_29,p4_29,p5_29,p6_29);
#endif
#ifdef ILP30
		KBreakpointModel(sumCurrents_30, sumConductivity_30,v_30,ModelStates_30[0],ModelStates_30[1],ModelStates_30[2],p1_30,p2_30,p3_30,p4_30,p5_30,p6_30);
#endif
#ifdef ILP31
		KBreakpointModel(sumCurrents_31, sumConductivity_31,v_31,ModelStates_31[0],ModelStates_31[1],ModelStates_31[2],p1_31,p2_31,p3_31,p4_31,p5_31,p6_31);
#endif
#ifdef ILP32
		KBreakpointModel(sumCurrents_32, sumConductivity_32,v_32,ModelStates_32[0],ModelStates_32[1],ModelStates_32[2],p1_32,p2_32,p3_32,p4_32,p5_32,p6_32);
#endif
		KBreakpointModel(sumCurrentsDv, sumConductivityDv,v+0.001,ModelStates[0],ModelStates[1],ModelStates[2],p1,p2,p3,p4,p5,p6);
		KBreakpointModel(sumCurrentsDv_2, sumConductivityDv_2,v_2+0.001,ModelStates_2[0],ModelStates_2[1],ModelStates_2[2],p1_2,p2_2,p3_2,p4_2,p5_2,p6_2);
#ifdef ILP3
		KBreakpointModel(sumCurrentsDv_3, sumConductivityDv_3,v_3+0.001,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2],p1_3,p2_3,p3_3,p4_3,p5_3,p6_3);
#endif
#ifdef ILP4
		KBreakpointModel(sumCurrentsDv_4, sumConductivityDv_4,v_4+0.001,ModelStates_4[0],ModelStates_4[1],ModelStates_4[2],p1_4,p2_4,p3_4,p4_4,p5_4,p6_4);
#endif
#ifdef ILP5
		KBreakpointModel(sumCurrentsDv_5, sumConductivityDv_5,v_5+0.001,ModelStates_5[0],ModelStates_5[1],ModelStates_5[2],p1_5,p2_5,p3_5,p4_5,p5_5,p6_5);
#endif
#ifdef ILP6
		KBreakpointModel(sumCurrentsDv_6, sumConductivityDv_6,v_6+0.001,ModelStates_6[0],ModelStates_6[1],ModelStates_6[2],p1_6,p2_6,p3_6,p4_6,p5_6,p6_6);
#endif
#ifdef ILP7
		KBreakpointModel(sumCurrentsDv_7, sumConductivityDv_7,v_7+0.001,ModelStates_7[0],ModelStates_7[1],ModelStates_7[2],p1_7,p2_7,p3_7,p4_7,p5_7,p6_7);
#endif
#ifdef ILP8
		KBreakpointModel(sumCurrentsDv_8, sumConductivityDv_8,v_8+0.001,ModelStates_8[0],ModelStates_8[1],ModelStates_8[2],p1_8,p2_8,p3_8,p4_8,p5_8,p6_8);
#endif
#ifdef ILP9
		KBreakpointModel(sumCurrentsDv_9, sumConductivityDv_9,v_9+0.001,ModelStates_9[0],ModelStates_9[1],ModelStates_9[2],p1_9,p2_9,p3_9,p4_9,p5_9,p6_9);
#endif
#ifdef ILP10
		KBreakpointModel(sumCurrentsDv_10, sumConductivityDv_10,v_10+0.001,ModelStates_10[0],ModelStates_10[1],ModelStates_10[2],p1_10,p2_10,p3_10,p4_10,p5_10,p6_10);
#endif
#ifdef ILP11
		KBreakpointModel(sumCurrentsDv_11, sumConductivityDv_11,v_11+0.001,ModelStates_11[0],ModelStates_11[1],ModelStates_11[2],p1_11,p2_11,p3_11,p4_11,p5_11,p6_11);
#endif
#ifdef ILP12
		KBreakpointModel(sumCurrentsDv_12, sumConductivityDv_12,v_12+0.001,ModelStates_12[0],ModelStates_12[1],ModelStates_12[2],p1_12,p2_12,p3_12,p4_12,p5_12,p6_12);
#endif
#ifdef ILP13
		KBreakpointModel(sumCurrentsDv_13, sumConductivityDv_13,v_13+0.001,ModelStates_13[0],ModelStates_13[1],ModelStates_13[2],p1_13,p2_13,p3_13,p4_13,p5_13,p6_13);
#endif
#ifdef ILP14
		KBreakpointModel(sumCurrentsDv_14, sumConductivityDv_14,v_14+0.001,ModelStates_14[0],ModelStates_14[1],ModelStates_14[2],p1_14,p2_14,p3_14,p4_14,p5_14,p6_14);
#endif
#ifdef ILP15
		KBreakpointModel(sumCurrentsDv_15, sumConductivityDv_15,v_15+0.001,ModelStates_15[0],ModelStates_15[1],ModelStates_15[2],p1_15,p2_15,p3_15,p4_15,p5_15,p6_15);
#endif
#ifdef ILP16
		KBreakpointModel(sumCurrentsDv_16, sumConductivityDv_16,v_16+0.001,ModelStates_16[0],ModelStates_16[1],ModelStates_16[2],p1_16,p2_16,p3_16,p4_16,p5_16,p6_16);
#endif
#ifdef ILP17
		KBreakpointModel(sumCurrentsDv_17, sumConductivityDv_17,v_17+0.001,ModelStates_17[0],ModelStates_17[1],ModelStates_17[2],p1_17,p2_17,p3_17,p4_17,p5_17,p6_17);
#endif
#ifdef ILP18
		KBreakpointModel(sumCurrentsDv_18, sumConductivityDv_18,v_18+0.001,ModelStates_18[0],ModelStates_18[1],ModelStates_18[2],p1_18,p2_18,p3_18,p4_18,p5_18,p6_18);
#endif
#ifdef ILP19
		KBreakpointModel(sumCurrentsDv_19, sumConductivityDv_19,v_19+0.001,ModelStates_19[0],ModelStates_19[1],ModelStates_19[2],p1_19,p2_19,p3_19,p4_19,p5_19,p6_19);
#endif
#ifdef ILP20
		KBreakpointModel(sumCurrentsDv_20, sumConductivityDv_20,v_20+0.001,ModelStates_20[0],ModelStates_20[1],ModelStates_20[2],p1_20,p2_20,p3_20,p4_20,p5_20,p6_20);
#endif
#ifdef ILP21
		KBreakpointModel(sumCurrentsDv_21, sumConductivityDv_21,v_21+0.001,ModelStates_21[0],ModelStates_21[1],ModelStates_21[2],p1_21,p2_21,p3_21,p4_21,p5_21,p6_21);
#endif
#ifdef ILP22
		KBreakpointModel(sumCurrentsDv_22, sumConductivityDv_22,v_22+0.001,ModelStates_22[0],ModelStates_22[1],ModelStates_22[2],p1_22,p2_22,p3_22,p4_22,p5_22,p6_22);
#endif
#ifdef ILP23
		KBreakpointModel(sumCurrentsDv_23, sumConductivityDv_23,v_23+0.001,ModelStates_23[0],ModelStates_23[1],ModelStates_23[2],p1_23,p2_23,p3_23,p4_23,p5_23,p6_23);
#endif
#ifdef ILP24
		KBreakpointModel(sumCurrentsDv_24, sumConductivityDv_24,v_24+0.001,ModelStates_24[0],ModelStates_24[1],ModelStates_24[2],p1_24,p2_24,p3_24,p4_24,p5_24,p6_24);
#endif
#ifdef ILP25
		KBreakpointModel(sumCurrentsDv_25, sumConductivityDv_25,v_25+0.001,ModelStates_25[0],ModelStates_25[1],ModelStates_25[2],p1_25,p2_25,p3_25,p4_25,p5_25,p6_25);
#endif
#ifdef ILP26
		KBreakpointModel(sumCurrentsDv_26, sumConductivityDv_26,v_26+0.001,ModelStates_26[0],ModelStates_26[1],ModelStates_26[2],p1_26,p2_26,p3_26,p4_26,p5_26,p6_26);
#endif
#ifdef ILP27
		KBreakpointModel(sumCurrentsDv_27, sumConductivityDv_27,v_27+0.001,ModelStates_27[0],ModelStates_27[1],ModelStates_27[2],p1_27,p2_27,p3_27,p4_27,p5_27,p6_27);
#endif
#ifdef ILP28
		KBreakpointModel(sumCurrentsDv_28, sumConductivityDv_28,v_28+0.001,ModelStates_28[0],ModelStates_28[1],ModelStates_28[2],p1_28,p2_28,p3_28,p4_28,p5_28,p6_28);
#endif
#ifdef ILP29
		KBreakpointModel(sumCurrentsDv_29, sumConductivityDv_29,v_29+0.001,ModelStates_29[0],ModelStates_29[1],ModelStates_29[2],p1_29,p2_29,p3_29,p4_29,p5_29,p6_29);
#endif
#ifdef ILP30
		KBreakpointModel(sumCurrentsDv_30, sumConductivityDv_30,v_30+0.001,ModelStates_30[0],ModelStates_30[1],ModelStates_30[2],p1_30,p2_30,p3_30,p4_30,p5_30,p6_30);
#endif
#ifdef ILP31
		KBreakpointModel(sumCurrentsDv_31, sumConductivityDv_31,v_31+0.001,ModelStates_31[0],ModelStates_31[1],ModelStates_31[2],p1_31,p2_31,p3_31,p4_31,p5_31,p6_31);
#endif
#ifdef ILP32
		KBreakpointModel(sumCurrentsDv_32, sumConductivityDv_32,v_32+0.001,ModelStates_32[0],ModelStates_32[1],ModelStates_32[2],p1_32,p2_32,p3_32,p4_32,p5_32,p6_32);
#endif

		gModel=0;
		gModel_2=0;
#ifdef ILP3
		gModel_3=0;
#endif
#ifdef ILP4
		gModel_4=0;
#endif
#ifdef ILP5
		gModel_5=0;
#endif
#ifdef ILP6
		gModel_6=0;
#endif
#ifdef ILP7
		gModel_7=0;
#endif
#ifdef ILP8
		gModel_8=0;
#endif
#ifdef ILP9
		gModel_9=0;
#endif
#ifdef ILP10
		gModel_10=0;
#endif
#ifdef ILP11
		gModel_11=0;
#endif
#ifdef ILP12
		gModel_12=0;
#endif
#ifdef ILP13
		gModel_13=0;
#endif
#ifdef ILP14
		gModel_14=0;
#endif
#ifdef ILP15
		gModel_15=0;
#endif
#ifdef ILP16
		gModel_16=0;
#endif
#ifdef ILP17
		gModel_17=0;
#endif
#ifdef ILP18
		gModel_18=0;
#endif
#ifdef ILP19
		gModel_19=0;
#endif
#ifdef ILP20
		gModel_20=0;
#endif
#ifdef ILP21
		gModel_21=0;
#endif
#ifdef ILP22
		gModel_22=0;
#endif
#ifdef ILP23
		gModel_23=0;
#endif
#ifdef ILP24
		gModel_24=0;
#endif
#ifdef ILP25
		gModel_25=0;
#endif
#ifdef ILP26
		gModel_26=0;
#endif
#ifdef ILP27
		gModel_27=0;
#endif
#ifdef ILP28
		gModel_28=0;
#endif
#ifdef ILP29
		gModel_29=0;
#endif
#ifdef ILP30
		gModel_30=0;
#endif
#ifdef ILP31
		gModel_31=0;
#endif
#ifdef ILP32
		gModel_32=0;
#endif
		if(SMemMat.boolModel[PIdx]==1) {
			gModel = (sumCurrentsDv-sumCurrents)/0.001; }
		else {
			sumConductivity=0;
			sumConductivityDv=0;
			sumCurrents=0;
			sumCurrentsDv=0;
		}
		if(SMemMat.boolModel[PIdx_2]==1) {gModel_2 = (sumCurrentsDv_2-sumCurrents_2)/0.001; }else {sumConductivity_2=0;sumConductivityDv_2=0;sumCurrents_2=0;sumCurrentsDv_2=0;}
#ifdef ILP3
		if(SMemMat.boolModel[PIdx_3]==1) {gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; }else {sumConductivity_3=0;sumConductivityDv_3=0;sumCurrents_3=0;sumCurrentsDv_3=0;}
#endif
#ifdef ILP4
		if(SMemMat.boolModel[PIdx_4]==1) {gModel_4 = (sumCurrentsDv_4-sumCurrents_4)/0.001; }else {sumConductivity_4=0;sumConductivityDv_4=0;sumCurrents_4=0;sumCurrentsDv_4=0;}
#endif
#ifdef ILP5
		if(SMemMat.boolModel[PIdx_5]==1) {gModel_5 = (sumCurrentsDv_5-sumCurrents_5)/0.001; }else {sumConductivity_5=0;sumConductivityDv_5=0;sumCurrents_5=0;sumCurrentsDv_5=0;}
#endif
#ifdef ILP6
		if(SMemMat.boolModel[PIdx_6]==1) {gModel_6 = (sumCurrentsDv_6-sumCurrents_6)/0.001; }else {sumConductivity_6=0;sumConductivityDv_6=0;sumCurrents_6=0;sumCurrentsDv_6=0;}
#endif
#ifdef ILP7
		if(SMemMat.boolModel[PIdx_7]==1) {gModel_7 = (sumCurrentsDv_7-sumCurrents_7)/0.001; }else {sumConductivity_7=0;sumConductivityDv_7=0;sumCurrents_7=0;sumCurrentsDv_7=0;}
#endif
#ifdef ILP8
		if(SMemMat.boolModel[PIdx_8]==1) {gModel_8 = (sumCurrentsDv_8-sumCurrents_8)/0.001; }else {sumConductivity_8=0;sumConductivityDv_8=0;sumCurrents_8=0;sumCurrentsDv_8=0;}
#endif
#ifdef ILP9
		if(SMemMat.boolModel[PIdx_9]==1) {gModel_9 = (sumCurrentsDv_9-sumCurrents_9)/0.001; }else {sumConductivity_9=0;sumConductivityDv_9=0;sumCurrents_9=0;sumCurrentsDv_9=0;}
#endif
#ifdef ILP10
		if(SMemMat.boolModel[PIdx_10]==1) {gModel_10 = (sumCurrentsDv_10-sumCurrents_10)/0.001; }else {sumConductivity_10=0;sumConductivityDv_10=0;sumCurrents_10=0;sumCurrentsDv_10=0;}
#endif
#ifdef ILP11
		if(SMemMat.boolModel[PIdx_11]==1) {gModel_11 = (sumCurrentsDv_11-sumCurrents_11)/0.001; }else {sumConductivity_11=0;sumConductivityDv_11=0;sumCurrents_11=0;sumCurrentsDv_11=0;}
#endif
#ifdef ILP12
		if(SMemMat.boolModel[PIdx_12]==1) {gModel_12 = (sumCurrentsDv_12-sumCurrents_12)/0.001; }else {sumConductivity_12=0;sumConductivityDv_12=0;sumCurrents_12=0;sumCurrentsDv_12=0;}
#endif
#ifdef ILP13
		if(SMemMat.boolModel[PIdx_13]==1) {gModel_13 = (sumCurrentsDv_13-sumCurrents_13)/0.001; }else {sumConductivity_13=0;sumConductivityDv_13=0;sumCurrents_13=0;sumCurrentsDv_13=0;}
#endif
#ifdef ILP14
		if(SMemMat.boolModel[PIdx_14]==1) {gModel_14 = (sumCurrentsDv_14-sumCurrents_14)/0.001; }else {sumConductivity_14=0;sumConductivityDv_14=0;sumCurrents_14=0;sumCurrentsDv_14=0;}
#endif
#ifdef ILP15
		if(SMemMat.boolModel[PIdx_15]==1) {gModel_15 = (sumCurrentsDv_15-sumCurrents_15)/0.001; }else {sumConductivity_15=0;sumConductivityDv_15=0;sumCurrents_15=0;sumCurrentsDv_15=0;}
#endif
#ifdef ILP16
		if(SMemMat.boolModel[PIdx_16]==1) {gModel_16 = (sumCurrentsDv_16-sumCurrents_16)/0.001; }else {sumConductivity_16=0;sumConductivityDv_16=0;sumCurrents_16=0;sumCurrentsDv_16=0;}
#endif
#ifdef ILP17
		if(SMemMat.boolModel[PIdx_17]==1) {gModel_17 = (sumCurrentsDv_17-sumCurrents_17)/0.001; }else {sumConductivity_17=0;sumConductivityDv_17=0;sumCurrents_17=0;sumCurrentsDv_17=0;}
#endif
#ifdef ILP18
		if(SMemMat.boolModel[PIdx_18]==1) {gModel_18 = (sumCurrentsDv_18-sumCurrents_18)/0.001; }else {sumConductivity_18=0;sumConductivityDv_18=0;sumCurrents_18=0;sumCurrentsDv_18=0;}
#endif
#ifdef ILP19
		if(SMemMat.boolModel[PIdx_19]==1) {gModel_19 = (sumCurrentsDv_19-sumCurrents_19)/0.001; }else {sumConductivity_19=0;sumConductivityDv_19=0;sumCurrents_19=0;sumCurrentsDv_19=0;}
#endif
#ifdef ILP20
		if(SMemMat.boolModel[PIdx_20]==1) {gModel_20 = (sumCurrentsDv_20-sumCurrents_20)/0.001; }else {sumConductivity_20=0;sumConductivityDv_20=0;sumCurrents_20=0;sumCurrentsDv_20=0;}
#endif
#ifdef ILP21
		if(SMemMat.boolModel[PIdx_21]==1) {gModel_21 = (sumCurrentsDv_21-sumCurrents_21)/0.001; }else {sumConductivity_21=0;sumConductivityDv_21=0;sumCurrents_21=0;sumCurrentsDv_21=0;}
#endif
#ifdef ILP22
		if(SMemMat.boolModel[PIdx_22]==1) {gModel_22 = (sumCurrentsDv_22-sumCurrents_22)/0.001; }else {sumConductivity_22=0;sumConductivityDv_22=0;sumCurrents_22=0;sumCurrentsDv_22=0;}
#endif
#ifdef ILP23
		if(SMemMat.boolModel[PIdx_23]==1) {gModel_23 = (sumCurrentsDv_23-sumCurrents_23)/0.001; }else {sumConductivity_23=0;sumConductivityDv_23=0;sumCurrents_23=0;sumCurrentsDv_23=0;}
#endif
#ifdef ILP24
		if(SMemMat.boolModel[PIdx_24]==1) {gModel_24 = (sumCurrentsDv_24-sumCurrents_24)/0.001; }else {sumConductivity_24=0;sumConductivityDv_24=0;sumCurrents_24=0;sumCurrentsDv_24=0;}
#endif
#ifdef ILP25
		if(SMemMat.boolModel[PIdx_25]==1) {gModel_25 = (sumCurrentsDv_25-sumCurrents_25)/0.001; }else {sumConductivity_25=0;sumConductivityDv_25=0;sumCurrents_25=0;sumCurrentsDv_25=0;}
#endif
#ifdef ILP26
		if(SMemMat.boolModel[PIdx_26]==1) {gModel_26 = (sumCurrentsDv_26-sumCurrents_26)/0.001; }else {sumConductivity_26=0;sumConductivityDv_26=0;sumCurrents_26=0;sumCurrentsDv_26=0;}
#endif
#ifdef ILP27
		if(SMemMat.boolModel[PIdx_27]==1) {gModel_27 = (sumCurrentsDv_27-sumCurrents_27)/0.001; }else {sumConductivity_27=0;sumConductivityDv_27=0;sumCurrents_27=0;sumCurrentsDv_27=0;}
#endif
#ifdef ILP28
		if(SMemMat.boolModel[PIdx_28]==1) {gModel_28 = (sumCurrentsDv_28-sumCurrents_28)/0.001; }else {sumConductivity_28=0;sumConductivityDv_28=0;sumCurrents_28=0;sumCurrentsDv_28=0;}
#endif
#ifdef ILP29
		if(SMemMat.boolModel[PIdx_29]==1) {gModel_29 = (sumCurrentsDv_29-sumCurrents_29)/0.001; }else {sumConductivity_29=0;sumConductivityDv_29=0;sumCurrents_29=0;sumCurrentsDv_29=0;}
#endif
#ifdef ILP30
		if(SMemMat.boolModel[PIdx_30]==1) {gModel_30 = (sumCurrentsDv_30-sumCurrents_30)/0.001; }else {sumConductivity_30=0;sumConductivityDv_30=0;sumCurrents_30=0;sumCurrentsDv_30=0;}
#endif
#ifdef ILP31
		if(SMemMat.boolModel[PIdx_31]==1) {gModel_31 = (sumCurrentsDv_31-sumCurrents_31)/0.001; }else {sumConductivity_31=0;sumConductivityDv_31=0;sumCurrents_31=0;sumCurrentsDv_31=0;}
#endif
#ifdef ILP32
		if(SMemMat.boolModel[PIdx_32]==1) {gModel_32 = (sumCurrentsDv_32-sumCurrents_32)/0.001; }else {sumConductivity_32=0;sumConductivityDv_32=0;sumCurrents_32=0;sumCurrentsDv_32=0;}
#endif
		StimCurrent=0;
		StimCurrent_2=0;
#ifdef ILP3
		StimCurrent_3=0;
#endif
#ifdef ILP4
		StimCurrent_4=0;
#endif
#ifdef ILP5
		StimCurrent_5=0;
#endif
#ifdef ILP6
		StimCurrent_6=0;
#endif
#ifdef ILP7
		StimCurrent_7=0;
#endif
#ifdef ILP8
		StimCurrent_8=0;
#endif
#ifdef ILP9
		StimCurrent_9=0;
#endif
#ifdef ILP10
		StimCurrent_10=0;
#endif
#ifdef ILP11
		StimCurrent_11=0;
#endif
#ifdef ILP12
		StimCurrent_12=0;
#endif
#ifdef ILP13
		StimCurrent_13=0;
#endif
#ifdef ILP14
		StimCurrent_14=0;
#endif
#ifdef ILP15
		StimCurrent_15=0;
#endif
#ifdef ILP16
		StimCurrent_16=0;
#endif
#ifdef ILP17
		StimCurrent_17=0;
#endif
#ifdef ILP18
		StimCurrent_18=0;
#endif
#ifdef ILP19
		StimCurrent_19=0;
#endif
#ifdef ILP20
		StimCurrent_20=0;
#endif
#ifdef ILP21
		StimCurrent_21=0;
#endif
#ifdef ILP22
		StimCurrent_22=0;
#endif
#ifdef ILP23
		StimCurrent_23=0;
#endif
#ifdef ILP24
		StimCurrent_24=0;
#endif
#ifdef ILP25
		StimCurrent_25=0;
#endif
#ifdef ILP26
		StimCurrent_26=0;
#endif
#ifdef ILP27
		StimCurrent_27=0;
#endif
#ifdef ILP28
		StimCurrent_28=0;
#endif
#ifdef ILP29
		StimCurrent_29=0;
#endif
#ifdef ILP30
		StimCurrent_30=0;
#endif
#ifdef ILP31
		StimCurrent_31=0;
#endif
#ifdef ILP32
		StimCurrent_32=0;
#endif

		if(t>=stimDel && t<stimDel+stimDur && PIdx == stimLoc){
			StimCurrent = 100*stimAmp/stimArea;
		}
		if(t>=stimDel && t<stimDel+stimDur && PIdx_2 == stimLoc){StimCurrent_2 = 100*stimAmp/stimArea;}
#ifdef ILP3
		if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP4
		if(t>=stimDel && t<stimDel+stimDur && PIdx_4 == stimLoc){StimCurrent_4 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP5
		if(t>=stimDel && t<stimDel+stimDur && PIdx_5 == stimLoc){StimCurrent_5 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP6
		if(t>=stimDel && t<stimDel+stimDur && PIdx_6 == stimLoc){StimCurrent_6 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP7
		if(t>=stimDel && t<stimDel+stimDur && PIdx_7 == stimLoc){StimCurrent_7 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP8
		if(t>=stimDel && t<stimDel+stimDur && PIdx_8 == stimLoc){StimCurrent_8 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP9
		if(t>=stimDel && t<stimDel+stimDur && PIdx_9 == stimLoc){StimCurrent_9 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP10
		if(t>=stimDel && t<stimDel+stimDur && PIdx_10 == stimLoc){StimCurrent_10 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP11
		if(t>=stimDel && t<stimDel+stimDur && PIdx_11 == stimLoc){StimCurrent_11 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP12
		if(t>=stimDel && t<stimDel+stimDur && PIdx_12 == stimLoc){StimCurrent_12 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP13
		if(t>=stimDel && t<stimDel+stimDur && PIdx_13 == stimLoc){StimCurrent_13 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP14
		if(t>=stimDel && t<stimDel+stimDur && PIdx_14 == stimLoc){StimCurrent_14 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP15
		if(t>=stimDel && t<stimDel+stimDur && PIdx_15 == stimLoc){StimCurrent_15 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP16
		if(t>=stimDel && t<stimDel+stimDur && PIdx_16 == stimLoc){StimCurrent_16 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP17
		if(t>=stimDel && t<stimDel+stimDur && PIdx_17 == stimLoc){StimCurrent_17 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP18
		if(t>=stimDel && t<stimDel+stimDur && PIdx_18 == stimLoc){StimCurrent_18 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP19
		if(t>=stimDel && t<stimDel+stimDur && PIdx_19 == stimLoc){StimCurrent_19 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP20
		if(t>=stimDel && t<stimDel+stimDur && PIdx_20 == stimLoc){StimCurrent_20 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP21
		if(t>=stimDel && t<stimDel+stimDur && PIdx_21 == stimLoc){StimCurrent_21 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP22
		if(t>=stimDel && t<stimDel+stimDur && PIdx_22 == stimLoc){StimCurrent_22 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP23
		if(t>=stimDel && t<stimDel+stimDur && PIdx_23 == stimLoc){StimCurrent_23 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP24
		if(t>=stimDel && t<stimDel+stimDur && PIdx_24 == stimLoc){StimCurrent_24 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP25
		if(t>=stimDel && t<stimDel+stimDur && PIdx_25 == stimLoc){StimCurrent_25 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP26
		if(t>=stimDel && t<stimDel+stimDur && PIdx_26 == stimLoc){StimCurrent_26 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP27
		if(t>=stimDel && t<stimDel+stimDur && PIdx_27 == stimLoc){StimCurrent_27 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP28
		if(t>=stimDel && t<stimDel+stimDur && PIdx_28 == stimLoc){StimCurrent_28 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP29
		if(t>=stimDel && t<stimDel+stimDur && PIdx_29 == stimLoc){StimCurrent_29 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP30
		if(t>=stimDel && t<stimDel+stimDur && PIdx_30 == stimLoc){StimCurrent_30 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP31
		if(t>=stimDel && t<stimDel+stimDur && PIdx_31 == stimLoc){StimCurrent_31 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP32
		if(t>=stimDel && t<stimDel+stimDur && PIdx_32 == stimLoc){StimCurrent_32 = 100*stimAmp/stimArea;}
#endif
		
		
		rhs=StimCurrent-sumCurrents;
		rhs_2=StimCurrent_2-sumCurrents_2;
#ifdef ILP3
		rhs_3=StimCurrent_3-sumCurrents_3;
#endif
#ifdef ILP4
		rhs_4=StimCurrent_4-sumCurrents_4;
#endif
#ifdef ILP5
		rhs_5=StimCurrent_5-sumCurrents_5;
#endif
#ifdef ILP6
		rhs_6=StimCurrent_6-sumCurrents_6;
#endif
#ifdef ILP7
		rhs_7=StimCurrent_7-sumCurrents_7;
#endif
#ifdef ILP8
		rhs_8=StimCurrent_8-sumCurrents_8;
#endif
#ifdef ILP9
		rhs_9=StimCurrent_9-sumCurrents_9;
#endif
#ifdef ILP10
		rhs_10=StimCurrent_10-sumCurrents_10;
#endif
#ifdef ILP11
		rhs_11=StimCurrent_11-sumCurrents_11;
#endif
#ifdef ILP12
		rhs_12=StimCurrent_12-sumCurrents_12;
#endif
#ifdef ILP13
		rhs_13=StimCurrent_13-sumCurrents_13;
#endif
#ifdef ILP14
		rhs_14=StimCurrent_14-sumCurrents_14;
#endif
#ifdef ILP15
		rhs_15=StimCurrent_15-sumCurrents_15;
#endif
#ifdef ILP16
		rhs_16=StimCurrent_16-sumCurrents_16;
#endif
#ifdef ILP17
		rhs_17=StimCurrent_17-sumCurrents_17;
#endif
#ifdef ILP18
		rhs_18=StimCurrent_18-sumCurrents_18;
#endif
#ifdef ILP19
		rhs_19=StimCurrent_19-sumCurrents_19;
#endif
#ifdef ILP20
		rhs_20=StimCurrent_20-sumCurrents_20;
#endif
#ifdef ILP21
		rhs_21=StimCurrent_21-sumCurrents_21;
#endif
#ifdef ILP22
		rhs_22=StimCurrent_22-sumCurrents_22;
#endif
#ifdef ILP23
		rhs_23=StimCurrent_23-sumCurrents_23;
#endif
#ifdef ILP24
		rhs_24=StimCurrent_24-sumCurrents_24;
#endif
#ifdef ILP25
		rhs_25=StimCurrent_25-sumCurrents_25;
#endif
#ifdef ILP26
		rhs_26=StimCurrent_26-sumCurrents_26;
#endif
#ifdef ILP27
		rhs_27=StimCurrent_27-sumCurrents_27;
#endif
#ifdef ILP28
		rhs_28=StimCurrent_28-sumCurrents_28;
#endif
#ifdef ILP29
		rhs_29=StimCurrent_29-sumCurrents_29;
#endif
#ifdef ILP30
		rhs_30=StimCurrent_30-sumCurrents_30;
#endif
#ifdef ILP31
		rhs_31=StimCurrent_31-sumCurrents_31;
#endif
#ifdef ILP32
		rhs_32=StimCurrent_32-sumCurrents_32;
#endif


		dv=Vs[parentIndex]-Vs[PIdx];
		dv_2=Vs[parentIndex_2]-Vs[PIdx_2];
#ifdef ILP3
		dv_3=Vs[parentIndex_3]-Vs[PIdx_3];
#endif
#ifdef ILP4
		dv_4=Vs[parentIndex_4]-Vs[PIdx_4];
#endif
#ifdef ILP5
		dv_5=Vs[parentIndex_5]-Vs[PIdx_5];
#endif
#ifdef ILP6
		dv_6=Vs[parentIndex_6]-Vs[PIdx_6];
#endif
#ifdef ILP7
		dv_7=Vs[parentIndex_7]-Vs[PIdx_7];
#endif
#ifdef ILP8
		dv_8=Vs[parentIndex_8]-Vs[PIdx_8];
#endif
#ifdef ILP9
		dv_9=Vs[parentIndex_9]-Vs[PIdx_9];
#endif
#ifdef ILP10
		dv_10=Vs[parentIndex_10]-Vs[PIdx_10];
#endif
#ifdef ILP11
		dv_11=Vs[parentIndex_11]-Vs[PIdx_11];
#endif
#ifdef ILP12
		dv_12=Vs[parentIndex_12]-Vs[PIdx_12];
#endif
#ifdef ILP13
		dv_13=Vs[parentIndex_13]-Vs[PIdx_13];
#endif
#ifdef ILP14
		dv_14=Vs[parentIndex_14]-Vs[PIdx_14];
#endif
#ifdef ILP15
		dv_15=Vs[parentIndex_15]-Vs[PIdx_15];
#endif
#ifdef ILP16
		dv_16=Vs[parentIndex_16]-Vs[PIdx_16];
#endif
#ifdef ILP17
		dv_17=Vs[parentIndex_17]-Vs[PIdx_17];
#endif
#ifdef ILP18
		dv_18=Vs[parentIndex_18]-Vs[PIdx_18];
#endif
#ifdef ILP19
		dv_19=Vs[parentIndex_19]-Vs[PIdx_19];
#endif
#ifdef ILP20
		dv_20=Vs[parentIndex_20]-Vs[PIdx_20];
#endif
#ifdef ILP21
		dv_21=Vs[parentIndex_21]-Vs[PIdx_21];
#endif
#ifdef ILP22
		dv_22=Vs[parentIndex_22]-Vs[PIdx_22];
#endif
#ifdef ILP23
		dv_23=Vs[parentIndex_23]-Vs[PIdx_23];
#endif
#ifdef ILP24
		dv_24=Vs[parentIndex_24]-Vs[PIdx_24];
#endif
#ifdef ILP25
		dv_25=Vs[parentIndex_25]-Vs[PIdx_25];
#endif
#ifdef ILP26
		dv_26=Vs[parentIndex_26]-Vs[PIdx_26];
#endif
#ifdef ILP27
		dv_27=Vs[parentIndex_27]-Vs[PIdx_27];
#endif
#ifdef ILP28
		dv_28=Vs[parentIndex_28]-Vs[PIdx_28];
#endif
#ifdef ILP29
		dv_29=Vs[parentIndex_29]-Vs[PIdx_29];
#endif
#ifdef ILP30
		dv_30=Vs[parentIndex_30]-Vs[PIdx_30];
#endif
#ifdef ILP31
		dv_31=Vs[parentIndex_31]-Vs[PIdx_31];
#endif
#ifdef ILP32
		dv_32=Vs[parentIndex_32]-Vs[PIdx_32];
#endif

		rhs-=SMemMat.f[SMemMat.N-PIdx-1]*dv;
		rhs_2-=SMemMat.f[SMemMat.N-PIdx_2-1]*dv_2;
#ifdef ILP3
		rhs_3-=SMemMat.f[SMemMat.N-PIdx_3-1]*dv_3;
#endif
#ifdef ILP4
		rhs_4-=SMemMat.f[SMemMat.N-PIdx_4-1]*dv_4;
#endif
#ifdef ILP5
		rhs_5-=SMemMat.f[SMemMat.N-PIdx_5-1]*dv_5;
#endif
#ifdef ILP6
		rhs_6-=SMemMat.f[SMemMat.N-PIdx_6-1]*dv_6;
#endif
#ifdef ILP7
		rhs_7-=SMemMat.f[SMemMat.N-PIdx_7-1]*dv_7;
#endif
#ifdef ILP8
		rhs_8-=SMemMat.f[SMemMat.N-PIdx_8-1]*dv_8;
#endif
#ifdef ILP9
		rhs_9-=SMemMat.f[SMemMat.N-PIdx_9-1]*dv_9;
#endif
#ifdef ILP10
		rhs_10-=SMemMat.f[SMemMat.N-PIdx_10-1]*dv_10;
#endif
#ifdef ILP11
		rhs_11-=SMemMat.f[SMemMat.N-PIdx_11-1]*dv_11;
#endif
#ifdef ILP12
		rhs_12-=SMemMat.f[SMemMat.N-PIdx_12-1]*dv_12;
#endif
#ifdef ILP13
		rhs_13-=SMemMat.f[SMemMat.N-PIdx_13-1]*dv_13;
#endif
#ifdef ILP14
		rhs_14-=SMemMat.f[SMemMat.N-PIdx_14-1]*dv_14;
#endif
#ifdef ILP15
		rhs_15-=SMemMat.f[SMemMat.N-PIdx_15-1]*dv_15;
#endif
#ifdef ILP16
		rhs_16-=SMemMat.f[SMemMat.N-PIdx_16-1]*dv_16;
#endif
#ifdef ILP17
		rhs_17-=SMemMat.f[SMemMat.N-PIdx_17-1]*dv_17;
#endif
#ifdef ILP18
		rhs_18-=SMemMat.f[SMemMat.N-PIdx_18-1]*dv_18;
#endif
#ifdef ILP19
		rhs_19-=SMemMat.f[SMemMat.N-PIdx_19-1]*dv_19;
#endif
#ifdef ILP20
		rhs_20-=SMemMat.f[SMemMat.N-PIdx_20-1]*dv_20;
#endif
#ifdef ILP21
		rhs_21-=SMemMat.f[SMemMat.N-PIdx_21-1]*dv_21;
#endif
#ifdef ILP22
		rhs_22-=SMemMat.f[SMemMat.N-PIdx_22-1]*dv_22;
#endif
#ifdef ILP23
		rhs_23-=SMemMat.f[SMemMat.N-PIdx_23-1]*dv_23;
#endif
#ifdef ILP24
		rhs_24-=SMemMat.f[SMemMat.N-PIdx_24-1]*dv_24;
#endif
#ifdef ILP25
		rhs_25-=SMemMat.f[SMemMat.N-PIdx_25-1]*dv_25;
#endif
#ifdef ILP26
		rhs_26-=SMemMat.f[SMemMat.N-PIdx_26-1]*dv_26;
#endif
#ifdef ILP27
		rhs_27-=SMemMat.f[SMemMat.N-PIdx_27-1]*dv_27;
#endif
#ifdef ILP28
		rhs_28-=SMemMat.f[SMemMat.N-PIdx_28-1]*dv_28;
#endif
#ifdef ILP29
		rhs_29-=SMemMat.f[SMemMat.N-PIdx_29-1]*dv_29;
#endif
#ifdef ILP30
		rhs_30-=SMemMat.f[SMemMat.N-PIdx_30-1]*dv_30;
#endif
#ifdef ILP31
		rhs_31-=SMemMat.f[SMemMat.N-PIdx_31-1]*dv_31;
#endif
#ifdef ILP32
		rhs_32-=SMemMat.f[SMemMat.N-PIdx_32-1]*dv_32;
#endif

		D=sumConductivity+SMemMat.Cms[PIdx]/(sim.dt*1000);
		D_2=sumConductivity_2+SMemMat.Cms[PIdx_2]/(sim.dt*1000);
#ifdef ILP3
		D_3=sumConductivity_3+SMemMat.Cms[PIdx_3]/(sim.dt*1000);
#endif
#ifdef ILP4
		D_4=sumConductivity_4+SMemMat.Cms[PIdx_4]/(sim.dt*1000);
#endif
#ifdef ILP5
		D_5=sumConductivity_5+SMemMat.Cms[PIdx_5]/(sim.dt*1000);
#endif
#ifdef ILP6
		D_6=sumConductivity_6+SMemMat.Cms[PIdx_6]/(sim.dt*1000);
#endif
#ifdef ILP7
		D_7=sumConductivity_7+SMemMat.Cms[PIdx_7]/(sim.dt*1000);
#endif
#ifdef ILP8
		D_8=sumConductivity_8+SMemMat.Cms[PIdx_8]/(sim.dt*1000);
#endif
#ifdef ILP9
		D_9=sumConductivity_9+SMemMat.Cms[PIdx_9]/(sim.dt*1000);
#endif
#ifdef ILP10
		D_10=sumConductivity_10+SMemMat.Cms[PIdx_10]/(sim.dt*1000);
#endif
#ifdef ILP11
		D_11=sumConductivity_11+SMemMat.Cms[PIdx_11]/(sim.dt*1000);
#endif
#ifdef ILP12
		D_12=sumConductivity_12+SMemMat.Cms[PIdx_12]/(sim.dt*1000);
#endif
#ifdef ILP13
		D_13=sumConductivity_13+SMemMat.Cms[PIdx_13]/(sim.dt*1000);
#endif
#ifdef ILP14
		D_14=sumConductivity_14+SMemMat.Cms[PIdx_14]/(sim.dt*1000);
#endif
#ifdef ILP15
		D_15=sumConductivity_15+SMemMat.Cms[PIdx_15]/(sim.dt*1000);
#endif
#ifdef ILP16
		D_16=sumConductivity_16+SMemMat.Cms[PIdx_16]/(sim.dt*1000);
#endif
#ifdef ILP17
		D_17=sumConductivity_17+SMemMat.Cms[PIdx_17]/(sim.dt*1000);
#endif
#ifdef ILP18
		D_18=sumConductivity_18+SMemMat.Cms[PIdx_18]/(sim.dt*1000);
#endif
#ifdef ILP19
		D_19=sumConductivity_19+SMemMat.Cms[PIdx_19]/(sim.dt*1000);
#endif
#ifdef ILP20
		D_20=sumConductivity_20+SMemMat.Cms[PIdx_20]/(sim.dt*1000);
#endif
#ifdef ILP21
		D_21=sumConductivity_21+SMemMat.Cms[PIdx_21]/(sim.dt*1000);
#endif
#ifdef ILP22
		D_22=sumConductivity_22+SMemMat.Cms[PIdx_22]/(sim.dt*1000);
#endif
#ifdef ILP23
		D_23=sumConductivity_23+SMemMat.Cms[PIdx_23]/(sim.dt*1000);
#endif
#ifdef ILP24
		D_24=sumConductivity_24+SMemMat.Cms[PIdx_24]/(sim.dt*1000);
#endif
#ifdef ILP25
		D_25=sumConductivity_25+SMemMat.Cms[PIdx_25]/(sim.dt*1000);
#endif
#ifdef ILP26
		D_26=sumConductivity_26+SMemMat.Cms[PIdx_26]/(sim.dt*1000);
#endif
#ifdef ILP27
		D_27=sumConductivity_27+SMemMat.Cms[PIdx_27]/(sim.dt*1000);
#endif
#ifdef ILP28
		D_28=sumConductivity_28+SMemMat.Cms[PIdx_28]/(sim.dt*1000);
#endif
#ifdef ILP29
		D_29=sumConductivity_29+SMemMat.Cms[PIdx_29]/(sim.dt*1000);
#endif
#ifdef ILP30
		D_30=sumConductivity_30+SMemMat.Cms[PIdx_30]/(sim.dt*1000);
#endif
#ifdef ILP31
		D_31=sumConductivity_31+SMemMat.Cms[PIdx_31]/(sim.dt*1000);
#endif
#ifdef ILP32
		D_32=sumConductivity_32+SMemMat.Cms[PIdx_32]/(sim.dt*1000);
#endif

		D-=SMemMat.f[SMemMat.N-PIdx-1];
		D_2-=SMemMat.f[SMemMat.N-PIdx_2-1];
#ifdef ILP3
		D_3-=SMemMat.f[SMemMat.N-PIdx_3-1];
#endif
#ifdef ILP4
		D_4-=SMemMat.f[SMemMat.N-PIdx_4-1];
#endif
#ifdef ILP5
		D_5-=SMemMat.f[SMemMat.N-PIdx_5-1];
#endif
#ifdef ILP6
		D_6-=SMemMat.f[SMemMat.N-PIdx_6-1];
#endif
#ifdef ILP7
		D_7-=SMemMat.f[SMemMat.N-PIdx_7-1];
#endif
#ifdef ILP8
		D_8-=SMemMat.f[SMemMat.N-PIdx_8-1];
#endif
#ifdef ILP9
		D_9-=SMemMat.f[SMemMat.N-PIdx_9-1];
#endif
#ifdef ILP10
		D_10-=SMemMat.f[SMemMat.N-PIdx_10-1];
#endif
#ifdef ILP11
		D_11-=SMemMat.f[SMemMat.N-PIdx_11-1];
#endif
#ifdef ILP12
		D_12-=SMemMat.f[SMemMat.N-PIdx_12-1];
#endif
#ifdef ILP13
		D_13-=SMemMat.f[SMemMat.N-PIdx_13-1];
#endif
#ifdef ILP14
		D_14-=SMemMat.f[SMemMat.N-PIdx_14-1];
#endif
#ifdef ILP15
		D_15-=SMemMat.f[SMemMat.N-PIdx_15-1];
#endif
#ifdef ILP16
		D_16-=SMemMat.f[SMemMat.N-PIdx_16-1];
#endif
#ifdef ILP17
		D_17-=SMemMat.f[SMemMat.N-PIdx_17-1];
#endif
#ifdef ILP18
		D_18-=SMemMat.f[SMemMat.N-PIdx_18-1];
#endif
#ifdef ILP19
		D_19-=SMemMat.f[SMemMat.N-PIdx_19-1];
#endif
#ifdef ILP20
		D_20-=SMemMat.f[SMemMat.N-PIdx_20-1];
#endif
#ifdef ILP21
		D_21-=SMemMat.f[SMemMat.N-PIdx_21-1];
#endif
#ifdef ILP22
		D_22-=SMemMat.f[SMemMat.N-PIdx_22-1];
#endif
#ifdef ILP23
		D_23-=SMemMat.f[SMemMat.N-PIdx_23-1];
#endif
#ifdef ILP24
		D_24-=SMemMat.f[SMemMat.N-PIdx_24-1];
#endif
#ifdef ILP25
		D_25-=SMemMat.f[SMemMat.N-PIdx_25-1];
#endif
#ifdef ILP26
		D_26-=SMemMat.f[SMemMat.N-PIdx_26-1];
#endif
#ifdef ILP27
		D_27-=SMemMat.f[SMemMat.N-PIdx_27-1];
#endif
#ifdef ILP28
		D_28-=SMemMat.f[SMemMat.N-PIdx_28-1];
#endif
#ifdef ILP29
		D_29-=SMemMat.f[SMemMat.N-PIdx_29-1];
#endif
#ifdef ILP30
		D_30-=SMemMat.f[SMemMat.N-PIdx_30-1];
#endif
#ifdef ILP31
		D_31-=SMemMat.f[SMemMat.N-PIdx_31-1];
#endif
#ifdef ILP32
		D_32-=SMemMat.f[SMemMat.N-PIdx_32-1];
#endif
		if(PIdx==0){
			rhs=0;
			D=0;
		}

		bHP[SMemMat.N-PIdx-1]=rhs;
		bHP[SMemMat.N-PIdx_2-1]=rhs_2;
#ifdef ILP3
		bHP[SMemMat.N-PIdx_3-1]=rhs_3;
#endif
#ifdef ILP4
		bHP[SMemMat.N-PIdx_4-1]=rhs_4;
#endif
#ifdef ILP5
		bHP[SMemMat.N-PIdx_5-1]=rhs_5;
#endif
#ifdef ILP6
		bHP[SMemMat.N-PIdx_6-1]=rhs_6;
#endif
#ifdef ILP7
		bHP[SMemMat.N-PIdx_7-1]=rhs_7;
#endif
#ifdef ILP8
		bHP[SMemMat.N-PIdx_8-1]=rhs_8;
#endif
#ifdef ILP9
		bHP[SMemMat.N-PIdx_9-1]=rhs_9;
#endif
#ifdef ILP10
		bHP[SMemMat.N-PIdx_10-1]=rhs_10;
#endif
#ifdef ILP11
		bHP[SMemMat.N-PIdx_11-1]=rhs_11;
#endif
#ifdef ILP12
		bHP[SMemMat.N-PIdx_12-1]=rhs_12;
#endif
#ifdef ILP13
		bHP[SMemMat.N-PIdx_13-1]=rhs_13;
#endif
#ifdef ILP14
		bHP[SMemMat.N-PIdx_14-1]=rhs_14;
#endif
#ifdef ILP15
		bHP[SMemMat.N-PIdx_15-1]=rhs_15;
#endif
#ifdef ILP16
		bHP[SMemMat.N-PIdx_16-1]=rhs_16;
#endif
#ifdef ILP17
		bHP[SMemMat.N-PIdx_17-1]=rhs_17;
#endif
#ifdef ILP18
		bHP[SMemMat.N-PIdx_18-1]=rhs_18;
#endif
#ifdef ILP19
		bHP[SMemMat.N-PIdx_19-1]=rhs_19;
#endif
#ifdef ILP20
		bHP[SMemMat.N-PIdx_20-1]=rhs_20;
#endif
#ifdef ILP21
		bHP[SMemMat.N-PIdx_21-1]=rhs_21;
#endif
#ifdef ILP22
		bHP[SMemMat.N-PIdx_22-1]=rhs_22;
#endif
#ifdef ILP23
		bHP[SMemMat.N-PIdx_23-1]=rhs_23;
#endif
#ifdef ILP24
		bHP[SMemMat.N-PIdx_24-1]=rhs_24;
#endif
#ifdef ILP25
		bHP[SMemMat.N-PIdx_25-1]=rhs_25;
#endif
#ifdef ILP26
		bHP[SMemMat.N-PIdx_26-1]=rhs_26;
#endif
#ifdef ILP27
		bHP[SMemMat.N-PIdx_27-1]=rhs_27;
#endif
#ifdef ILP28
		bHP[SMemMat.N-PIdx_28-1]=rhs_28;
#endif
#ifdef ILP29
		bHP[SMemMat.N-PIdx_29-1]=rhs_29;
#endif
#ifdef ILP30
		bHP[SMemMat.N-PIdx_30-1]=rhs_30;
#endif
#ifdef ILP31
		bHP[SMemMat.N-PIdx_31-1]=rhs_31;
#endif
#ifdef ILP32
		bHP[SMemMat.N-PIdx_32-1]=rhs_32;
#endif
		uHP[SMemMat.N-PIdx-1]=D;
		uHP[SMemMat.N-PIdx_2-1]=D_2;
#ifdef ILP3
		uHP[SMemMat.N-PIdx_3-1]=D_3;
#endif
#ifdef ILP4
		uHP[SMemMat.N-PIdx_4-1]=D_4;
#endif
#ifdef ILP5
		uHP[SMemMat.N-PIdx_5-1]=D_5;
#endif
#ifdef ILP6
		uHP[SMemMat.N-PIdx_6-1]=D_6;
#endif
#ifdef ILP7
		uHP[SMemMat.N-PIdx_7-1]=D_7;
#endif
#ifdef ILP8
		uHP[SMemMat.N-PIdx_8-1]=D_8;
#endif
#ifdef ILP9
		uHP[SMemMat.N-PIdx_9-1]=D_9;
#endif
#ifdef ILP10
		uHP[SMemMat.N-PIdx_10-1]=D_10;
#endif
#ifdef ILP11
		uHP[SMemMat.N-PIdx_11-1]=D_11;
#endif
#ifdef ILP12
		uHP[SMemMat.N-PIdx_12-1]=D_12;
#endif
#ifdef ILP13
		uHP[SMemMat.N-PIdx_13-1]=D_13;
#endif
#ifdef ILP14
		uHP[SMemMat.N-PIdx_14-1]=D_14;
#endif
#ifdef ILP15
		uHP[SMemMat.N-PIdx_15-1]=D_15;
#endif
#ifdef ILP16
		uHP[SMemMat.N-PIdx_16-1]=D_16;
#endif
#ifdef ILP17
		uHP[SMemMat.N-PIdx_17-1]=D_17;
#endif
#ifdef ILP18
		uHP[SMemMat.N-PIdx_18-1]=D_18;
#endif
#ifdef ILP19
		uHP[SMemMat.N-PIdx_19-1]=D_19;
#endif
#ifdef ILP20
		uHP[SMemMat.N-PIdx_20-1]=D_20;
#endif
#ifdef ILP21
		uHP[SMemMat.N-PIdx_21-1]=D_21;
#endif
#ifdef ILP22
		uHP[SMemMat.N-PIdx_22-1]=D_22;
#endif
#ifdef ILP23
		uHP[SMemMat.N-PIdx_23-1]=D_23;
#endif
#ifdef ILP24
		uHP[SMemMat.N-PIdx_24-1]=D_24;
#endif
#ifdef ILP25
		uHP[SMemMat.N-PIdx_25-1]=D_25;
#endif
#ifdef ILP26
		uHP[SMemMat.N-PIdx_26-1]=D_26;
#endif
#ifdef ILP27
		uHP[SMemMat.N-PIdx_27-1]=D_27;
#endif
#ifdef ILP28
		uHP[SMemMat.N-PIdx_28-1]=D_28;
#endif
#ifdef ILP29
		uHP[SMemMat.N-PIdx_29-1]=D_29;
#endif
#ifdef ILP30
		uHP[SMemMat.N-PIdx_30-1]=D_30;
#endif
#ifdef ILP31
		uHP[SMemMat.N-PIdx_31-1]=D_31;
#endif
#ifdef ILP32
		uHP[SMemMat.N-PIdx_32-1]=D_32;
#endif
		
		syncthreads();
		
		if(SonNo==1) {
		
			bHP[SMemMat.N-parentIndex-1]+=SMemMat.e[Eidx]*dv;
		
			uHP[SMemMat.N-parentIndex-1]-=SMemMat.e[Eidx];
		}
		if(SonNo_2==1) {bHP[SMemMat.N-parentIndex_2-1]+=SMemMat.e[Eidx_2]*dv_2;uHP[SMemMat.N-parentIndex_2-1]-=SMemMat.e[Eidx_2];}
#ifdef ILP3
		if(SonNo_3==1) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#endif
#ifdef ILP4
		if(SonNo_4==1) {bHP[SMemMat.N-parentIndex_4-1]+=SMemMat.e[Eidx_4]*dv_4;uHP[SMemMat.N-parentIndex_4-1]-=SMemMat.e[Eidx_4];}
#endif
#ifdef ILP5
		if(SonNo_5==1) {bHP[SMemMat.N-parentIndex_5-1]+=SMemMat.e[Eidx_5]*dv_5;uHP[SMemMat.N-parentIndex_5-1]-=SMemMat.e[Eidx_5];}
#endif
#ifdef ILP6
		if(SonNo_6==1) {bHP[SMemMat.N-parentIndex_6-1]+=SMemMat.e[Eidx_6]*dv_6;uHP[SMemMat.N-parentIndex_6-1]-=SMemMat.e[Eidx_6];}
#endif
#ifdef ILP7
		if(SonNo_7==1) {bHP[SMemMat.N-parentIndex_7-1]+=SMemMat.e[Eidx_7]*dv_7;uHP[SMemMat.N-parentIndex_7-1]-=SMemMat.e[Eidx_7];}
#endif
#ifdef ILP8
		if(SonNo_8==1) {bHP[SMemMat.N-parentIndex_8-1]+=SMemMat.e[Eidx_8]*dv_8;uHP[SMemMat.N-parentIndex_8-1]-=SMemMat.e[Eidx_8];}
#endif
#ifdef ILP9
		if(SonNo_9==1) {bHP[SMemMat.N-parentIndex_9-1]+=SMemMat.e[Eidx_9]*dv_9;uHP[SMemMat.N-parentIndex_9-1]-=SMemMat.e[Eidx_9];}
#endif
#ifdef ILP10
		if(SonNo_10==1) {bHP[SMemMat.N-parentIndex_10-1]+=SMemMat.e[Eidx_10]*dv_10;uHP[SMemMat.N-parentIndex_10-1]-=SMemMat.e[Eidx_10];}
#endif
#ifdef ILP11
		if(SonNo_11==1) {bHP[SMemMat.N-parentIndex_11-1]+=SMemMat.e[Eidx_11]*dv_11;uHP[SMemMat.N-parentIndex_11-1]-=SMemMat.e[Eidx_11];}
#endif
#ifdef ILP12
		if(SonNo_12==1) {bHP[SMemMat.N-parentIndex_12-1]+=SMemMat.e[Eidx_12]*dv_12;uHP[SMemMat.N-parentIndex_12-1]-=SMemMat.e[Eidx_12];}
#endif
#ifdef ILP13
		if(SonNo_13==1) {bHP[SMemMat.N-parentIndex_13-1]+=SMemMat.e[Eidx_13]*dv_13;uHP[SMemMat.N-parentIndex_13-1]-=SMemMat.e[Eidx_13];}
#endif
#ifdef ILP14
		if(SonNo_14==1) {bHP[SMemMat.N-parentIndex_14-1]+=SMemMat.e[Eidx_14]*dv_14;uHP[SMemMat.N-parentIndex_14-1]-=SMemMat.e[Eidx_14];}
#endif
#ifdef ILP15
		if(SonNo_15==1) {bHP[SMemMat.N-parentIndex_15-1]+=SMemMat.e[Eidx_15]*dv_15;uHP[SMemMat.N-parentIndex_15-1]-=SMemMat.e[Eidx_15];}
#endif
#ifdef ILP16
		if(SonNo_16==1) {bHP[SMemMat.N-parentIndex_16-1]+=SMemMat.e[Eidx_16]*dv_16;uHP[SMemMat.N-parentIndex_16-1]-=SMemMat.e[Eidx_16];}
#endif
#ifdef ILP17
		if(SonNo_17==1) {bHP[SMemMat.N-parentIndex_17-1]+=SMemMat.e[Eidx_17]*dv_17;uHP[SMemMat.N-parentIndex_17-1]-=SMemMat.e[Eidx_17];}
#endif
#ifdef ILP18
		if(SonNo_18==1) {bHP[SMemMat.N-parentIndex_18-1]+=SMemMat.e[Eidx_18]*dv_18;uHP[SMemMat.N-parentIndex_18-1]-=SMemMat.e[Eidx_18];}
#endif
#ifdef ILP19
		if(SonNo_19==1) {bHP[SMemMat.N-parentIndex_19-1]+=SMemMat.e[Eidx_19]*dv_19;uHP[SMemMat.N-parentIndex_19-1]-=SMemMat.e[Eidx_19];}
#endif
#ifdef ILP20
		if(SonNo_20==1) {bHP[SMemMat.N-parentIndex_20-1]+=SMemMat.e[Eidx_20]*dv_20;uHP[SMemMat.N-parentIndex_20-1]-=SMemMat.e[Eidx_20];}
#endif
#ifdef ILP21
		if(SonNo_21==1) {bHP[SMemMat.N-parentIndex_21-1]+=SMemMat.e[Eidx_21]*dv_21;uHP[SMemMat.N-parentIndex_21-1]-=SMemMat.e[Eidx_21];}
#endif
#ifdef ILP22
		if(SonNo_22==1) {bHP[SMemMat.N-parentIndex_22-1]+=SMemMat.e[Eidx_22]*dv_22;uHP[SMemMat.N-parentIndex_22-1]-=SMemMat.e[Eidx_22];}
#endif
#ifdef ILP23
		if(SonNo_23==1) {bHP[SMemMat.N-parentIndex_23-1]+=SMemMat.e[Eidx_23]*dv_23;uHP[SMemMat.N-parentIndex_23-1]-=SMemMat.e[Eidx_23];}
#endif
#ifdef ILP24
		if(SonNo_24==1) {bHP[SMemMat.N-parentIndex_24-1]+=SMemMat.e[Eidx_24]*dv_24;uHP[SMemMat.N-parentIndex_24-1]-=SMemMat.e[Eidx_24];}
#endif
#ifdef ILP25
		if(SonNo_25==1) {bHP[SMemMat.N-parentIndex_25-1]+=SMemMat.e[Eidx_25]*dv_25;uHP[SMemMat.N-parentIndex_25-1]-=SMemMat.e[Eidx_25];}
#endif
#ifdef ILP26
		if(SonNo_26==1) {bHP[SMemMat.N-parentIndex_26-1]+=SMemMat.e[Eidx_26]*dv_26;uHP[SMemMat.N-parentIndex_26-1]-=SMemMat.e[Eidx_26];}
#endif
#ifdef ILP27
		if(SonNo_27==1) {bHP[SMemMat.N-parentIndex_27-1]+=SMemMat.e[Eidx_27]*dv_27;uHP[SMemMat.N-parentIndex_27-1]-=SMemMat.e[Eidx_27];}
#endif
#ifdef ILP28
		if(SonNo_28==1) {bHP[SMemMat.N-parentIndex_28-1]+=SMemMat.e[Eidx_28]*dv_28;uHP[SMemMat.N-parentIndex_28-1]-=SMemMat.e[Eidx_28];}
#endif
#ifdef ILP29
		if(SonNo_29==1) {bHP[SMemMat.N-parentIndex_29-1]+=SMemMat.e[Eidx_29]*dv_29;uHP[SMemMat.N-parentIndex_29-1]-=SMemMat.e[Eidx_29];}
#endif
#ifdef ILP30
		if(SonNo_30==1) {bHP[SMemMat.N-parentIndex_30-1]+=SMemMat.e[Eidx_30]*dv_30;uHP[SMemMat.N-parentIndex_30-1]-=SMemMat.e[Eidx_30];}
#endif
#ifdef ILP31
		if(SonNo_31==1) {bHP[SMemMat.N-parentIndex_31-1]+=SMemMat.e[Eidx_31]*dv_31;uHP[SMemMat.N-parentIndex_31-1]-=SMemMat.e[Eidx_31];}
#endif
#ifdef ILP32
		if(SonNo_32==1) {bHP[SMemMat.N-parentIndex_32-1]+=SMemMat.e[Eidx_32]*dv_32;uHP[SMemMat.N-parentIndex_32-1]-=SMemMat.e[Eidx_32];}
#endif

		if(SonNo==2) {bHP[SMemMat.N-parentIndex-1]+=SMemMat.e[Eidx]*dv;uHP[SMemMat.N-parentIndex-1]-=SMemMat.e[Eidx];}
		if(SonNo_2==2) {bHP[SMemMat.N-parentIndex_2-1]+=SMemMat.e[Eidx_2]*dv_2;uHP[SMemMat.N-parentIndex_2-1]-=SMemMat.e[Eidx_2];}
#ifdef ILP3
		if(SonNo_3==2) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#endif
#ifdef ILP4
		if(SonNo_4==2) {bHP[SMemMat.N-parentIndex_4-1]+=SMemMat.e[Eidx_4]*dv_4;uHP[SMemMat.N-parentIndex_4-1]-=SMemMat.e[Eidx_4];}
#endif
#ifdef ILP5
		if(SonNo_5==2) {bHP[SMemMat.N-parentIndex_5-1]+=SMemMat.e[Eidx_5]*dv_5;uHP[SMemMat.N-parentIndex_5-1]-=SMemMat.e[Eidx_5];}
#endif
#ifdef ILP6
		if(SonNo_6==2) {bHP[SMemMat.N-parentIndex_6-1]+=SMemMat.e[Eidx_6]*dv_6;uHP[SMemMat.N-parentIndex_6-1]-=SMemMat.e[Eidx_6];}
#endif
#ifdef ILP7
		if(SonNo_7==2) {bHP[SMemMat.N-parentIndex_7-1]+=SMemMat.e[Eidx_7]*dv_7;uHP[SMemMat.N-parentIndex_7-1]-=SMemMat.e[Eidx_7];}
#endif
#ifdef ILP8
		if(SonNo_8==2) {bHP[SMemMat.N-parentIndex_8-1]+=SMemMat.e[Eidx_8]*dv_8;uHP[SMemMat.N-parentIndex_8-1]-=SMemMat.e[Eidx_8];}
#endif
#ifdef ILP9
		if(SonNo_9==2) {bHP[SMemMat.N-parentIndex_9-1]+=SMemMat.e[Eidx_9]*dv_9;uHP[SMemMat.N-parentIndex_9-1]-=SMemMat.e[Eidx_9];}
#endif
#ifdef ILP10
		if(SonNo_10==2) {bHP[SMemMat.N-parentIndex_10-1]+=SMemMat.e[Eidx_10]*dv_10;uHP[SMemMat.N-parentIndex_10-1]-=SMemMat.e[Eidx_10];}
#endif
#ifdef ILP11
		if(SonNo_11==2) {bHP[SMemMat.N-parentIndex_11-1]+=SMemMat.e[Eidx_11]*dv_11;uHP[SMemMat.N-parentIndex_11-1]-=SMemMat.e[Eidx_11];}
#endif
#ifdef ILP12
		if(SonNo_12==2) {bHP[SMemMat.N-parentIndex_12-1]+=SMemMat.e[Eidx_12]*dv_12;uHP[SMemMat.N-parentIndex_12-1]-=SMemMat.e[Eidx_12];}
#endif
#ifdef ILP13
		if(SonNo_13==2) {bHP[SMemMat.N-parentIndex_13-1]+=SMemMat.e[Eidx_13]*dv_13;uHP[SMemMat.N-parentIndex_13-1]-=SMemMat.e[Eidx_13];}
#endif
#ifdef ILP14
		if(SonNo_14==2) {bHP[SMemMat.N-parentIndex_14-1]+=SMemMat.e[Eidx_14]*dv_14;uHP[SMemMat.N-parentIndex_14-1]-=SMemMat.e[Eidx_14];}
#endif
#ifdef ILP15
		if(SonNo_15==2) {bHP[SMemMat.N-parentIndex_15-1]+=SMemMat.e[Eidx_15]*dv_15;uHP[SMemMat.N-parentIndex_15-1]-=SMemMat.e[Eidx_15];}
#endif
#ifdef ILP16
		if(SonNo_16==2) {bHP[SMemMat.N-parentIndex_16-1]+=SMemMat.e[Eidx_16]*dv_16;uHP[SMemMat.N-parentIndex_16-1]-=SMemMat.e[Eidx_16];}
#endif
#ifdef ILP17
		if(SonNo_17==2) {bHP[SMemMat.N-parentIndex_17-1]+=SMemMat.e[Eidx_17]*dv_17;uHP[SMemMat.N-parentIndex_17-1]-=SMemMat.e[Eidx_17];}
#endif
#ifdef ILP18
		if(SonNo_18==2) {bHP[SMemMat.N-parentIndex_18-1]+=SMemMat.e[Eidx_18]*dv_18;uHP[SMemMat.N-parentIndex_18-1]-=SMemMat.e[Eidx_18];}
#endif
#ifdef ILP19
		if(SonNo_19==2) {bHP[SMemMat.N-parentIndex_19-1]+=SMemMat.e[Eidx_19]*dv_19;uHP[SMemMat.N-parentIndex_19-1]-=SMemMat.e[Eidx_19];}
#endif
#ifdef ILP20
		if(SonNo_20==2) {bHP[SMemMat.N-parentIndex_20-1]+=SMemMat.e[Eidx_20]*dv_20;uHP[SMemMat.N-parentIndex_20-1]-=SMemMat.e[Eidx_20];}
#endif
#ifdef ILP21
		if(SonNo_21==2) {bHP[SMemMat.N-parentIndex_21-1]+=SMemMat.e[Eidx_21]*dv_21;uHP[SMemMat.N-parentIndex_21-1]-=SMemMat.e[Eidx_21];}
#endif
#ifdef ILP22
		if(SonNo_22==2) {bHP[SMemMat.N-parentIndex_22-1]+=SMemMat.e[Eidx_22]*dv_22;uHP[SMemMat.N-parentIndex_22-1]-=SMemMat.e[Eidx_22];}
#endif
#ifdef ILP23
		if(SonNo_23==2) {bHP[SMemMat.N-parentIndex_23-1]+=SMemMat.e[Eidx_23]*dv_23;uHP[SMemMat.N-parentIndex_23-1]-=SMemMat.e[Eidx_23];}
#endif
#ifdef ILP24
		if(SonNo_24==2) {bHP[SMemMat.N-parentIndex_24-1]+=SMemMat.e[Eidx_24]*dv_24;uHP[SMemMat.N-parentIndex_24-1]-=SMemMat.e[Eidx_24];}
#endif
#ifdef ILP25
		if(SonNo_25==2) {bHP[SMemMat.N-parentIndex_25-1]+=SMemMat.e[Eidx_25]*dv_25;uHP[SMemMat.N-parentIndex_25-1]-=SMemMat.e[Eidx_25];}
#endif
#ifdef ILP26
		if(SonNo_26==2) {bHP[SMemMat.N-parentIndex_26-1]+=SMemMat.e[Eidx_26]*dv_26;uHP[SMemMat.N-parentIndex_26-1]-=SMemMat.e[Eidx_26];}
#endif
#ifdef ILP27
		if(SonNo_27==2) {bHP[SMemMat.N-parentIndex_27-1]+=SMemMat.e[Eidx_27]*dv_27;uHP[SMemMat.N-parentIndex_27-1]-=SMemMat.e[Eidx_27];}
#endif
#ifdef ILP28
		if(SonNo_28==2) {bHP[SMemMat.N-parentIndex_28-1]+=SMemMat.e[Eidx_28]*dv_28;uHP[SMemMat.N-parentIndex_28-1]-=SMemMat.e[Eidx_28];}
#endif
#ifdef ILP29
		if(SonNo_29==2) {bHP[SMemMat.N-parentIndex_29-1]+=SMemMat.e[Eidx_29]*dv_29;uHP[SMemMat.N-parentIndex_29-1]-=SMemMat.e[Eidx_29];}
#endif
#ifdef ILP30
		if(SonNo_30==2) {bHP[SMemMat.N-parentIndex_30-1]+=SMemMat.e[Eidx_30]*dv_30;uHP[SMemMat.N-parentIndex_30-1]-=SMemMat.e[Eidx_30];}
#endif
#ifdef ILP31
		if(SonNo_31==2) {bHP[SMemMat.N-parentIndex_31-1]+=SMemMat.e[Eidx_31]*dv_31;uHP[SMemMat.N-parentIndex_31-1]-=SMemMat.e[Eidx_31];}
#endif
#ifdef ILP32
		if(SonNo_32==2) {bHP[SMemMat.N-parentIndex_32-1]+=SMemMat.e[Eidx_32]*dv_32;uHP[SMemMat.N-parentIndex_32-1]-=SMemMat.e[Eidx_32];}
#endif

		
		BeforeLU(SMemMat,uHP,bHP,SMemMat.Depth);
		BkSub(SMemMat, PX,PF,uHP,bHP,SMemMat.LognDepth);
		Vmid=PX[SMemMat.N-PIdx-1];
		Vmid_2=PX[SMemMat.N-PIdx_2-1];
#ifdef ILP3
		Vmid_3=PX[SMemMat.N-PIdx_3-1];
#endif
#ifdef ILP4
		Vmid_4=PX[SMemMat.N-PIdx_4-1];
#endif
#ifdef ILP5
		Vmid_5=PX[SMemMat.N-PIdx_5-1];
#endif
#ifdef ILP6
		Vmid_6=PX[SMemMat.N-PIdx_6-1];
#endif
#ifdef ILP7
		Vmid_7=PX[SMemMat.N-PIdx_7-1];
#endif
#ifdef ILP8
		Vmid_8=PX[SMemMat.N-PIdx_8-1];
#endif
#ifdef ILP9
		Vmid_9=PX[SMemMat.N-PIdx_9-1];
#endif
#ifdef ILP10
		Vmid_10=PX[SMemMat.N-PIdx_10-1];
#endif
#ifdef ILP11
		Vmid_11=PX[SMemMat.N-PIdx_11-1];
#endif
#ifdef ILP12
		Vmid_12=PX[SMemMat.N-PIdx_12-1];
#endif
#ifdef ILP13
		Vmid_13=PX[SMemMat.N-PIdx_13-1];
#endif
#ifdef ILP14
		Vmid_14=PX[SMemMat.N-PIdx_14-1];
#endif
#ifdef ILP15
		Vmid_15=PX[SMemMat.N-PIdx_15-1];
#endif
#ifdef ILP16
		Vmid_16=PX[SMemMat.N-PIdx_16-1];
#endif
#ifdef ILP17
		Vmid_17=PX[SMemMat.N-PIdx_17-1];
#endif
#ifdef ILP18
		Vmid_18=PX[SMemMat.N-PIdx_18-1];
#endif
#ifdef ILP19
		Vmid_19=PX[SMemMat.N-PIdx_19-1];
#endif
#ifdef ILP20
		Vmid_20=PX[SMemMat.N-PIdx_20-1];
#endif
#ifdef ILP21
		Vmid_21=PX[SMemMat.N-PIdx_21-1];
#endif
#ifdef ILP22
		Vmid_22=PX[SMemMat.N-PIdx_22-1];
#endif
#ifdef ILP23
		Vmid_23=PX[SMemMat.N-PIdx_23-1];
#endif
#ifdef ILP24
		Vmid_24=PX[SMemMat.N-PIdx_24-1];
#endif
#ifdef ILP25
		Vmid_25=PX[SMemMat.N-PIdx_25-1];
#endif
#ifdef ILP26
		Vmid_26=PX[SMemMat.N-PIdx_26-1];
#endif
#ifdef ILP27
		Vmid_27=PX[SMemMat.N-PIdx_27-1];
#endif
#ifdef ILP28
		Vmid_28=PX[SMemMat.N-PIdx_28-1];
#endif
#ifdef ILP29
		Vmid_29=PX[SMemMat.N-PIdx_29-1];
#endif
#ifdef ILP30
		Vmid_30=PX[SMemMat.N-PIdx_30-1];
#endif
#ifdef ILP31
		Vmid_31=PX[SMemMat.N-PIdx_31-1];
#endif
#ifdef ILP32
		Vmid_32=PX[SMemMat.N-PIdx_32-1];
#endif

		v+=Vmid;
		v_2+=Vmid_2;
#ifdef ILP3
		v_3+=Vmid_3;
#endif
#ifdef ILP4
		v_4+=Vmid_4;
#endif
#ifdef ILP5
		v_5+=Vmid_5;
#endif
#ifdef ILP6
		v_6+=Vmid_6;
#endif
#ifdef ILP7
		v_7+=Vmid_7;
#endif
#ifdef ILP8
		v_8+=Vmid_8;
#endif
#ifdef ILP9
		v_9+=Vmid_9;
#endif
#ifdef ILP10
		v_10+=Vmid_10;
#endif
#ifdef ILP11
		v_11+=Vmid_11;
#endif
#ifdef ILP12
		v_12+=Vmid_12;
#endif
#ifdef ILP13
		v_13+=Vmid_13;
#endif
#ifdef ILP14
		v_14+=Vmid_14;
#endif
#ifdef ILP15
		v_15+=Vmid_15;
#endif
#ifdef ILP16
		v_16+=Vmid_16;
#endif
#ifdef ILP17
		v_17+=Vmid_17;
#endif
#ifdef ILP18
		v_18+=Vmid_18;
#endif
#ifdef ILP19
		v_19+=Vmid_19;
#endif
#ifdef ILP20
		v_20+=Vmid_20;
#endif
#ifdef ILP21
		v_21+=Vmid_21;
#endif
#ifdef ILP22
		v_22+=Vmid_22;
#endif
#ifdef ILP23
		v_23+=Vmid_23;
#endif
#ifdef ILP24
		v_24+=Vmid_24;
#endif
#ifdef ILP25
		v_25+=Vmid_25;
#endif
#ifdef ILP26
		v_26+=Vmid_26;
#endif
#ifdef ILP27
		v_27+=Vmid_27;
#endif
#ifdef ILP28
		v_28+=Vmid_28;
#endif
#ifdef ILP29
		v_29+=Vmid_29;
#endif
#ifdef ILP30
		v_30+=Vmid_30;
#endif
#ifdef ILP31
		v_31+=Vmid_31;
#endif
#ifdef ILP32
		v_32+=Vmid_32;
#endif
		Vs[PIdx]+=Vmid;
		Vs[PIdx_2]+=Vmid_2;
#ifdef ILP3
		Vs[PIdx_3]+=Vmid_3;
#endif
#ifdef ILP4
		Vs[PIdx_4]+=Vmid_4;
#endif
#ifdef ILP5
		Vs[PIdx_5]+=Vmid_5;
#endif
#ifdef ILP6
		Vs[PIdx_6]+=Vmid_6;
#endif
#ifdef ILP7
		Vs[PIdx_7]+=Vmid_7;
#endif
#ifdef ILP8
		Vs[PIdx_8]+=Vmid_8;
#endif
#ifdef ILP9
		Vs[PIdx_9]+=Vmid_9;
#endif
#ifdef ILP10
		Vs[PIdx_10]+=Vmid_10;
#endif
#ifdef ILP11
		Vs[PIdx_11]+=Vmid_11;
#endif
#ifdef ILP12
		Vs[PIdx_12]+=Vmid_12;
#endif
#ifdef ILP13
		Vs[PIdx_13]+=Vmid_13;
#endif
#ifdef ILP14
		Vs[PIdx_14]+=Vmid_14;
#endif
#ifdef ILP15
		Vs[PIdx_15]+=Vmid_15;
#endif
#ifdef ILP16
		Vs[PIdx_16]+=Vmid_16;
#endif
#ifdef ILP17
		Vs[PIdx_17]+=Vmid_17;
#endif
#ifdef ILP18
		Vs[PIdx_18]+=Vmid_18;
#endif
#ifdef ILP19
		Vs[PIdx_19]+=Vmid_19;
#endif
#ifdef ILP20
		Vs[PIdx_20]+=Vmid_20;
#endif
#ifdef ILP21
		Vs[PIdx_21]+=Vmid_21;
#endif
#ifdef ILP22
		Vs[PIdx_22]+=Vmid_22;
#endif
#ifdef ILP23
		Vs[PIdx_23]+=Vmid_23;
#endif
#ifdef ILP24
		Vs[PIdx_24]+=Vmid_24;
#endif
#ifdef ILP25
		Vs[PIdx_25]+=Vmid_25;
#endif
#ifdef ILP26
		Vs[PIdx_26]+=Vmid_26;
#endif
#ifdef ILP27
		Vs[PIdx_27]+=Vmid_27;
#endif
#ifdef ILP28
		Vs[PIdx_28]+=Vmid_28;
#endif
#ifdef ILP29
		Vs[PIdx_29]+=Vmid_29;
#endif
#ifdef ILP30
		Vs[PIdx_30]+=Vmid_30;
#endif
#ifdef ILP31
		Vs[PIdx_31]+=Vmid_31;
#endif
#ifdef ILP32
		Vs[PIdx_32]+=Vmid_32;
#endif

		if(SMemMat.boolModel[PIdx]==1) {KDerivModel(sim.dt, Vs[PIdx], ModelStates[0],ModelStates[1],ModelStates[2]);}
		if(SMemMat.boolModel[PIdx_2]==1) {KDerivModel(sim.dt, Vs[PIdx_2], ModelStates_2[0],ModelStates_2[1],ModelStates_2[2]); 		}		
#ifdef ILP3
		if(SMemMat.boolModel[PIdx_3]==1) {KDerivModel(sim.dt, Vs[PIdx_3], ModelStates_3[0],ModelStates_3[1],ModelStates_3[2]); 		}		
#endif
#ifdef ILP4
		if(SMemMat.boolModel[PIdx_4]==1) {KDerivModel(sim.dt, Vs[PIdx_4], ModelStates_4[0],ModelStates_4[1],ModelStates_4[2]); 		}		
#endif
#ifdef ILP5
		if(SMemMat.boolModel[PIdx_5]==1) {KDerivModel(sim.dt, Vs[PIdx_5], ModelStates_5[0],ModelStates_5[1],ModelStates_5[2]); 		}		
#endif
#ifdef ILP6
		if(SMemMat.boolModel[PIdx_6]==1) {KDerivModel(sim.dt, Vs[PIdx_6], ModelStates_6[0],ModelStates_6[1],ModelStates_6[2]); 		}		
#endif
#ifdef ILP7
		if(SMemMat.boolModel[PIdx_7]==1) {KDerivModel(sim.dt, Vs[PIdx_7], ModelStates_7[0],ModelStates_7[1],ModelStates_7[2]); 		}		
#endif
#ifdef ILP8
		if(SMemMat.boolModel[PIdx_8]==1) {KDerivModel(sim.dt, Vs[PIdx_8], ModelStates_8[0],ModelStates_8[1],ModelStates_8[2]); 		}		
#endif
#ifdef ILP9
		if(SMemMat.boolModel[PIdx_9]==1) {KDerivModel(sim.dt, Vs[PIdx_9], ModelStates_9[0],ModelStates_9[1],ModelStates_9[2]); 		}		
#endif
#ifdef ILP10
		if(SMemMat.boolModel[PIdx_10]==1) {KDerivModel(sim.dt, Vs[PIdx_10], ModelStates_10[0],ModelStates_10[1],ModelStates_10[2]); 		}		
#endif
#ifdef ILP11
		if(SMemMat.boolModel[PIdx_11]==1) {KDerivModel(sim.dt, Vs[PIdx_11], ModelStates_11[0],ModelStates_11[1],ModelStates_11[2]); 		}		
#endif
#ifdef ILP12
		if(SMemMat.boolModel[PIdx_12]==1) {KDerivModel(sim.dt, Vs[PIdx_12], ModelStates_12[0],ModelStates_12[1],ModelStates_12[2]); 		}		
#endif
#ifdef ILP13
		if(SMemMat.boolModel[PIdx_13]==1) {KDerivModel(sim.dt, Vs[PIdx_13], ModelStates_13[0],ModelStates_13[1],ModelStates_13[2]); 		}		
#endif
#ifdef ILP14
		if(SMemMat.boolModel[PIdx_14]==1) {KDerivModel(sim.dt, Vs[PIdx_14], ModelStates_14[0],ModelStates_14[1],ModelStates_14[2]); 		}		
#endif
#ifdef ILP15
		if(SMemMat.boolModel[PIdx_15]==1) {KDerivModel(sim.dt, Vs[PIdx_15], ModelStates_15[0],ModelStates_15[1],ModelStates_15[2]); 		}		
#endif
#ifdef ILP16
		if(SMemMat.boolModel[PIdx_16]==1) {KDerivModel(sim.dt, Vs[PIdx_16], ModelStates_16[0],ModelStates_16[1],ModelStates_16[2]); 		}		
#endif
#ifdef ILP17
		if(SMemMat.boolModel[PIdx_17]==1) {KDerivModel(sim.dt, Vs[PIdx_17], ModelStates_17[0],ModelStates_17[1],ModelStates_17[2]); 		}		
#endif
#ifdef ILP18
		if(SMemMat.boolModel[PIdx_18]==1) {KDerivModel(sim.dt, Vs[PIdx_18], ModelStates_18[0],ModelStates_18[1],ModelStates_18[2]); 		}		
#endif
#ifdef ILP19
		if(SMemMat.boolModel[PIdx_19]==1) {KDerivModel(sim.dt, Vs[PIdx_19], ModelStates_19[0],ModelStates_19[1],ModelStates_19[2]); 		}		
#endif
#ifdef ILP20
		if(SMemMat.boolModel[PIdx_20]==1) {KDerivModel(sim.dt, Vs[PIdx_20], ModelStates_20[0],ModelStates_20[1],ModelStates_20[2]); 		}		
#endif
#ifdef ILP21
		if(SMemMat.boolModel[PIdx_21]==1) {KDerivModel(sim.dt, Vs[PIdx_21], ModelStates_21[0],ModelStates_21[1],ModelStates_21[2]); 		}		
#endif
#ifdef ILP22
		if(SMemMat.boolModel[PIdx_22]==1) {KDerivModel(sim.dt, Vs[PIdx_22], ModelStates_22[0],ModelStates_22[1],ModelStates_22[2]); 		}		
#endif
#ifdef ILP23
		if(SMemMat.boolModel[PIdx_23]==1) {KDerivModel(sim.dt, Vs[PIdx_23], ModelStates_23[0],ModelStates_23[1],ModelStates_23[2]); 		}		
#endif
#ifdef ILP24
		if(SMemMat.boolModel[PIdx_24]==1) {KDerivModel(sim.dt, Vs[PIdx_24], ModelStates_24[0],ModelStates_24[1],ModelStates_24[2]); 		}		
#endif
#ifdef ILP25
		if(SMemMat.boolModel[PIdx_25]==1) {KDerivModel(sim.dt, Vs[PIdx_25], ModelStates_25[0],ModelStates_25[1],ModelStates_25[2]); 		}		
#endif
#ifdef ILP26
		if(SMemMat.boolModel[PIdx_26]==1) {KDerivModel(sim.dt, Vs[PIdx_26], ModelStates_26[0],ModelStates_26[1],ModelStates_26[2]); 		}		
#endif
#ifdef ILP27
		if(SMemMat.boolModel[PIdx_27]==1) {KDerivModel(sim.dt, Vs[PIdx_27], ModelStates_27[0],ModelStates_27[1],ModelStates_27[2]); 		}		
#endif
#ifdef ILP28
		if(SMemMat.boolModel[PIdx_28]==1) {KDerivModel(sim.dt, Vs[PIdx_28], ModelStates_28[0],ModelStates_28[1],ModelStates_28[2]); 		}		
#endif
#ifdef ILP29
		if(SMemMat.boolModel[PIdx_29]==1) {KDerivModel(sim.dt, Vs[PIdx_29], ModelStates_29[0],ModelStates_29[1],ModelStates_29[2]); 		}		
#endif
#ifdef ILP30
		if(SMemMat.boolModel[PIdx_30]==1) {KDerivModel(sim.dt, Vs[PIdx_30], ModelStates_30[0],ModelStates_30[1],ModelStates_30[2]); 		}		
#endif
#ifdef ILP31
		if(SMemMat.boolModel[PIdx_31]==1) {KDerivModel(sim.dt, Vs[PIdx_31], ModelStates_31[0],ModelStates_31[1],ModelStates_31[2]); 		}		
#endif
#ifdef ILP32
		if(SMemMat.boolModel[PIdx_32]==1) {KDerivModel(sim.dt, Vs[PIdx_32], ModelStates_32[0],ModelStates_32[1],ModelStates_32[2]); 		}		
#endif
		
		if((i%(WARPSIZE)==0)&&i>0){
			VHotGlobal[threadIdx.y*Nt+(i-WARPSIZE)+PIdx]=SMemVHot[PIdx];
		}
		SMemVHot[i%(WARPSIZE)]=Vs[stimLoc];
	}

}

void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	FILE *fl = fopen(FN, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}

__constant__ MYDTYPE* constKs;

hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, float* V) { 
	hipError_t cudaStatus;
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=ceil(sim.TFinal/sim.dt);
	VHotsHost=(float*)malloc(stim.NStimuli*Nt*sizeof(float));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.Depth=InMat.Depth;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
	
	


	cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
	cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (Mat_d.Depth+1)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (Mat_d.Depth)*WARPSIZE* sizeof(MYDTYPE));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);


	cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.Cms, InMat.Cms, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);

	
	
	// 32 data
	cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (Mat_d.Depth+1)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (Mat_d.Depth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}
	// copying
	MYFTYPE* ParamsForCuda;
	ParamsForCuda=(float*)malloc(InMat.N * NPARAMS *  sizeof(float));
	ReadParamsMatX(ParamsMat_FN,ParamsForCuda,NPARAMS,InMat.N);
	cudaStatus = hipMemcpy(d_modelParams, ParamsForCuda, NPARAMS * InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(1,NNEURONS);

	MYDTYPE offset=0;
	offset+=InMat.N*sizeof(MYFTYPE);
	offset+=InMat.N*sizeof(MYFTYPE);
	offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
	offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
	offset+=InMat.N*sizeof(MYDTYPE);
	offset+=InMat.N*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, boolModel
	offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	offset+=WARPSIZE*(InMat.Depth+1)*sizeof(MYDTYPE);
	offset+=WARPSIZE*(InMat.Depth)*sizeof(MYDTYPE);
	// 4 CompByLevel32 CompByFLevel32
	
	offset=ceilf(float(offset)/sizeof(MYFTYPE))*sizeof(MYFTYPE);
	MYDTYPE FrameworkMemSize=offset;

	offset+=(InMat.N+2)*sizeof(MYFTYPE);  // uHP (Diag)
	offset+=(InMat.N+2)*sizeof(MYFTYPE); // bHP (rhs)
	offset+=InMat.N*sizeof(MYFTYPE); // Vs
	offset+=Nt*sizeof(MYFTYPE);//SMemVHot
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+PerStimulus*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
	stEfork2TimeLoopGPUKernel<<<gridDim, blockDim,TotalSMem>>>(stim_d, d_modelParams, sim, Mat_d, V_d,VHotsGlobal); // RRR

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal, Nt * stim.NStimuli* sizeof(float), hipMemcpyDeviceToHost); 

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,Nt*stim.NStimuli,VHotsHost);
	
	return cudaStatus;
}

hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, float* V) {
	hipError_t cudaStatus;

	clock_t begin,end;
	
	begin=clock();
	stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V); //RRR sim
	end=clock();
	printf("stEfork2TimeLoopGPU took %g seconds\n",double(diffclock(end,begin)));
	


	return cudaStatus;
}
