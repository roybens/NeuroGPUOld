#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "Util.h"
#include "hh.h"


__device__ void BeforeLU(HMat InMat, MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel;
	MYDTYPE JumctionI;

	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		JumctionI=InMat.CompByLevel32[CurLevel*WARPSIZE+PIdx]-1;
		
		for(i=InMat.SegStartI[JumctionI]-1;i<InMat.SegEndI[JumctionI];i++) {
			MYFTYPE uHPm1=uHP[i-1];
			
			uHP[i]=uHP[i]-InMat.e[i]*(InMat.f[i-1]/uHPm1); // So far same as paper parallel
			uHPm1=uHP[i-1];
			MYFTYPE bHPm1=bHP[i-1];
			bHP[i]=bHP[i]-bHPm1*InMat.e[i]/uHPm1; // bH is y
		}
		if(CurLevel<Depth) {
			CurB=InMat.CompByFLevel32[(CurLevel)*WARPSIZE+PIdx]-1;//RBS i inserted another  -1 into the index RBS 2 i removed the-1 from the curlevel 
			CurJ=InMat.Fathers[CurB]-1;
			MYDTYPE St=InMat.RelStarts[CurB];
			MYDTYPE En=InMat.RelEnds[CurB];
			for(j=St;j<=En;j++) {
				t=InMat.RelVec[j-1]-1;
				MYFTYPE uHPm1=uHP[t-1];
				uHP[CurJ]-=InMat.e[t]*(InMat.f[t-1]/uHPm1); 
				uHPm1=uHP[t-1];
				MYFTYPE bHPm1=bHP[t-1];
				bHP[CurJ]-=bHPm1*InMat.e[t]/uHPm1; 
			}
		}	
	}
}
/*__device__ void BkSubB(HMat InMat, MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE PIdx_2=threadIdx.x+WARPSIZE;
#ifdef ILP3
	MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#endif
#ifdef ILP4
	MYDTYPE PIdx_4=threadIdx.x+(WARPSIZE*3);
#endif
	MYDTYPE i,j,CurJ,CurB,t, CurLevel;
	MYDTYPE JumctionI;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless?
	bHP[PIdx]=0; // might be useless?
	bHP[PIdx_2]=0; // might be useless?
#ifdef ILP3
	bHP[PIdx_3]=0; // might be useless?
#endif
#ifdef ILP4
	bHP[PIdx_4]=0; // might be useless?
#endif

	// for CurLevel=Depth:-1:0
	for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
		//     Run all independent set for this level, in parallel
		// for JumctionI=find(Level==CurLevel) % in parallel
		JumctionI=InMat.CompByLevel32[CurLevel*WARPSIZE+PIdx]-1;
		// for i=(SegEndI(JumctionI)):-1:(SegStartI(JumctionI)-1)
		for(i=InMat.SegEndI[JumctionI]-2;i>=InMat.SegStartI[JumctionI]-2;i--) {
			// k=KsB(i+1);
			MYFTYPE k=KsB[i+1];
			// bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
			bHP[i]=(uHP[i]-bHP[k]*InMat.[i])/uHP[i];
		}
	}
}
*/
__device__ void BkSub(HMat InMat, MYFTYPE* PX, MYFTYPE* PF,MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE LognDepth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE PIdx_2=threadIdx.x+WARPSIZE;
#ifdef ILP3
	MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#endif
#ifdef ILP4
	MYDTYPE PIdx_4=threadIdx.x+(WARPSIZE*3);
#endif
	MYDTYPE i;
	MYDTYPE NextID;
	MYDTYPE NextID_2;
#ifdef ILP3
	MYDTYPE NextID_3;
#endif
#ifdef ILP4
	MYDTYPE NextID_4;
#endif
	
	PX=bHP;
	PF=uHP;
	PX[PIdx]=PX[PIdx]/PF[PIdx];
	PX[PIdx_2]=PX[PIdx_2]/PF[PIdx_2];
#ifdef ILP3
	PX[PIdx_3]=PX[PIdx_3]/PF[PIdx_3];
#endif
#ifdef ILP4
	PX[PIdx_4]=PX[PIdx_4]/PF[PIdx_4];
#endif
	PF[PIdx]=-InMat.f[PIdx]/PF[PIdx];
	PF[PIdx_2]=-InMat.f[PIdx_2]/PF[PIdx_2];
#ifdef ILP3
	PF[PIdx_3]=-InMat.f[PIdx_3]/PF[PIdx_3];
#endif
#ifdef ILP4
	PF[PIdx_4]=-InMat.f[PIdx_4]/PF[PIdx_4];
#endif

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		NextID=InMat.FIdxs[i*InMat.N+PIdx]-1;
		NextID_2=InMat.FIdxs[i*InMat.N+PIdx_2]-1;
#ifdef ILP3
		NextID_3=InMat.FIdxs[i*InMat.N+PIdx_3]-1;
#endif
#ifdef ILP4
		NextID_4=InMat.FIdxs[i*InMat.N+PIdx_4]-1;
#endif
		MYFTYPE OldPXj=PX[PIdx];
		MYFTYPE OldPXj_2=PX[PIdx_2];
#ifdef ILP3
		MYFTYPE OldPXj_3=PX[PIdx_3];
#endif
#ifdef ILP4
		MYFTYPE OldPXj_4=PX[PIdx_4];
#endif
		MYFTYPE OldPXNextID=PX[NextID];
		MYFTYPE OldPXNextID_2=PX[NextID_2];
#ifdef ILP3
		MYFTYPE OldPXNextID_3=PX[NextID_3];
#endif
#ifdef ILP4
		MYFTYPE OldPXNextID_4=PX[NextID_4];
#endif
		PX[PIdx]=OldPXj+OldPXNextID*PF[PIdx];
		PX[PIdx_2]=OldPXj_2+OldPXNextID_2*PF[PIdx_2];
#ifdef ILP3
		PX[PIdx_3]=OldPXj_3+OldPXNextID_3*PF[PIdx_3];
#endif
#ifdef ILP4
		PX[PIdx_4]=OldPXj_4+OldPXNextID_4*PF[PIdx_4];
#endif
		// PX[j]=PX[j]+PX[NextID]*PF[j];
		MYFTYPE OldPFj=PF[PIdx];
		MYFTYPE OldPFj_2=PF[PIdx_2];
#ifdef ILP3
		MYFTYPE OldPFj_3=PF[PIdx_3];
#endif
#ifdef ILP4
		MYFTYPE OldPFj_4=PF[PIdx_4];
#endif
		MYFTYPE OldPFNextID=PF[NextID];
		MYFTYPE OldPFNextID_2=PF[NextID_2];
#ifdef ILP3
		MYFTYPE OldPFNextID_3=PF[NextID_3];
#endif
#ifdef ILP4
		MYFTYPE OldPFNextID_4=PF[NextID_4];
#endif
		// PF[j]=PF[j]*PF[NextID];
		PF[PIdx]=OldPFj*OldPFNextID;
		PF[PIdx_2]=OldPFj_2*OldPFNextID_2;
#ifdef ILP3
		PF[PIdx_3]=OldPFj_3*OldPFNextID_3;
#endif
#ifdef ILP4
		PF[PIdx_4]=OldPFj_4*OldPFNextID_4;
#endif
	}
}

int testCode(MYFTYPE* orig, MYFTYPE* sim,int N){
	int i=0,iErr,iRel;
	MYFTYPE absErr,relErr,currErr,currRel;
	absErr = abs(orig[0]-sim[0]);
	relErr = absErr/abs(orig[0]);
	for(i=0;i<N;i++) {
		currErr = abs(orig[i]-sim[i]);
		currRel = currErr/abs(orig[i]);
		if (currErr>absErr){
			absErr=currErr;
			iErr=i;
		}
		if (currRel>relErr){
			relErr=currRel;
			iRel=i;

		}
	}
	printf("Absolute Error is %g at the %d index and Relative Error is %g at the %d index\n",absErr,iErr,relErr,iRel);
	return 1;
}
__device__ MYFTYPE Kvtrap(MYFTYPE x, MYFTYPE y) {
	if (fabs(x/y) < 1e-6) {;
	return y*(1 - x/y/2);
	}
	else
	{;
	return x/(exp(x/y) - 1);
	};
};

__device__ void Krates(MYFTYPE v, MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau,MYFTYPE &ninf,MYFTYPE &ntau) {
	MYFTYPE  alpha, beta, sum, q10;
	;
	// q10=3^((celsius - 6.3)/10); YYY CHANGE THIS to pow
	q10=pow(3,((celsius - 6.3)/10));
	;
	alpha=.1 * Kvtrap(-(v+40),10);
	beta=4 * exp(-(v+65)/18);
	sum=alpha + beta;
	mtau=1/(q10*sum);
	minf=alpha/sum;
	;
	alpha=.07 * exp(-(v+65)/20);
	beta=1 / (exp(-(v+35)/10) + 1);
	sum=alpha + beta;
	htau=1/(q10*sum);
	hinf=alpha/sum;
	;
	alpha=.01*Kvtrap(-(v+55),10) ;
	beta=.125*exp(-(v+65)/80);
	sum=alpha + beta;
	ntau=1/(q10*sum);
	ninf=alpha/sum;
}

__device__ void KInitModel(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE &n) {
	MYFTYPE hinf,htau,minf,mtau,ninf,ntau;
	Krates(v, hinf,htau,minf,mtau,ninf,ntau);
	m=minf;
	h=hinf;
	n=ninf;
}

__device__ void KDerivModel(MYFTYPE dt, MYFTYPE v, MYFTYPE &m,MYFTYPE &h,MYFTYPE &n) {
	MYFTYPE hinf,htau,minf,mtau,ninf,ntau;
	Krates(v, hinf,htau,minf,mtau,ninf,ntau);
	m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
	h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
	n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
};

// YYY add sum conductivity float BreakpointModel(float &m,float &h,float &n,float gnabar,float gkbar,float gl,float el) {
// YYY add float v
// YYY change to void function
__device__ void KBreakpointModel(MYFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v, MYFTYPE m,MYFTYPE h,MYFTYPE n,MYFTYPE gnabar,MYFTYPE gkbar,MYFTYPE gl,MYFTYPE ena,MYFTYPE ek,MYFTYPE el) {
	float gk,gna;
	float ina,ik,il;
	gna=gnabar*m*m*m*h;
	ina=gna*(v-ena);
	gk=gkbar*n*n*n*n;
	ik=gk*(v-(ek));
	il=gl*(v-el);
	sumCurrents= ina+ik+il;
	sumConductivity= gl+gk+gna;
}



// RRR
__global__ void stEfork2TimeLoopGPUKernel(Stim stim, MYFTYPE* ParamsM, Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal)
{
	
	MYDTYPE StimID=threadIdx.y;
	MYFTYPE stimDel = stim.dels[StimID];
	MYFTYPE stimDur = stim.durs[StimID];
	MYFTYPE stimAmp = stim.amps[StimID];
	MYDTYPE stimLoc = stim.loc;
	MYFTYPE stimArea = stim.area;
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE PIdx_2=threadIdx.x+WARPSIZE;
#ifdef ILP3
	MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#endif
#ifdef ILP4
	MYDTYPE PIdx_4=threadIdx.x+(WARPSIZE*3);
#endif
	hipError_t cudaStatus;
	// for model RRR
	MYFTYPE t;
	MYFTYPE isModel,dVec, Vmid;
	MYFTYPE isModel_2, dVec_2, Vmid_2;
#ifdef ILP3
	MYFTYPE isModel_3, dVec_3, Vmid_3;
#endif
#ifdef ILP4
	MYFTYPE isModel_4, dVec_4, Vmid_4;
#endif
	MYFTYPE ModelStates[NSTATES];
	MYFTYPE ModelStates_2[NSTATES];
#ifdef ILP3
	MYFTYPE ModelStates_3[NSTATES];
#endif
#ifdef ILP4
	MYFTYPE ModelStates_4[NSTATES];
#endif
	MYFTYPE v=V[PIdx]; // RRR -65?
	MYFTYPE v_2=V[PIdx_2]; // RRR -65?
#ifdef ILP3
	MYFTYPE v_3=V[PIdx_3]; // RRR -65?
#endif
#ifdef ILP4
	MYFTYPE v_4=V[PIdx_4]; // RRR -65?
#endif
	
		
	HMat SMemMat;
	SMemMat.N=InMat.N;
	SMemMat.Depth=InMat.Depth;
	SMemMat.LognDepth=InMat.LognDepth;
	SMemMat.nFathers=InMat.nFathers;
	SMemMat.nCallForFather=InMat.nCallForFather;
	MYFTYPE p1,p2,p3,p4,p5,p6;
	MYFTYPE p1_2,p2_2,p3_2,p4_2,p5_2,p6_2;
#ifdef ILP3
	MYFTYPE p1_3,p2_3,p3_3,p4_3,p5_3,p6_3;
#endif
#ifdef ILP4
	MYFTYPE p1_4,p2_4,p3_4,p4_4,p5_4,p6_4;
#endif
	p1=ParamsM[0*SMemMat.N+PIdx];
	p1_2=ParamsM[0*SMemMat.N+PIdx_2];
#ifdef ILP3
	p1_3=ParamsM[0*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p1_4=ParamsM[0*SMemMat.N+PIdx_4];
#endif
	p2=ParamsM[1*SMemMat.N+PIdx];
	p2_2=ParamsM[1*SMemMat.N+PIdx_2];
#ifdef ILP3
	p2_3=ParamsM[1*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p2_4=ParamsM[1*SMemMat.N+PIdx_4];
#endif
	p3=ParamsM[2*SMemMat.N+PIdx];
	p3_2=ParamsM[2*SMemMat.N+PIdx_2];
#ifdef ILP3
	p3_3=ParamsM[2*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p3_4=ParamsM[2*SMemMat.N+PIdx_4];
#endif
	p4=ParamsM[3*SMemMat.N+PIdx];
	p4_2=ParamsM[3*SMemMat.N+PIdx_2];
#ifdef ILP3
	p4_3=ParamsM[3*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p4_4=ParamsM[3*SMemMat.N+PIdx_4];
#endif
	p5=ParamsM[4*SMemMat.N+PIdx];
	p5_2=ParamsM[4*SMemMat.N+PIdx_2];
#ifdef ILP3
	p5_3=ParamsM[4*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p5_4=ParamsM[4*SMemMat.N+PIdx_4];
#endif
	p6=ParamsM[5*SMemMat.N+PIdx];
	p6_2=ParamsM[5*SMemMat.N+PIdx_2];
#ifdef ILP3
	p6_3=ParamsM[5*SMemMat.N+PIdx_3];
#endif
#ifdef ILP4
	p6_4=ParamsM[5*SMemMat.N+PIdx_4];
#endif

	
	KInitModel(v,ModelStates[0],ModelStates[1],ModelStates[2]);
	KInitModel(v_2,ModelStates_2[0],ModelStates_2[1],ModelStates_2[2]);
#ifdef ILP3
	KInitModel(v_3,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2]);
#endif
#ifdef ILP4
	KInitModel(v_4,ModelStates_4[0],ModelStates_4[1],ModelStates_4[2]);
#endif
	
	MYFTYPE sumCurrents, sumConductivity;
	MYFTYPE sumCurrents_2, sumConductivity_2;
#ifdef ILP3
	MYFTYPE sumCurrents_3, sumConductivity_3;
#endif
#ifdef ILP4
	MYFTYPE sumCurrents_4, sumConductivity_4;
#endif
	MYFTYPE sumCurrentsDv, sumConductivityDv;
	MYFTYPE sumCurrentsDv_2, sumConductivityDv_2;
#ifdef ILP3
	MYFTYPE sumCurrentsDv_3, sumConductivityDv_3;
#endif
#ifdef ILP4
	MYFTYPE sumCurrentsDv_4, sumConductivityDv_4;
#endif
	// end for model in init

	MYDTYPE offset=0;
	// CONSTANTS
	// floats
	MYFTYPE *Olde,*Oldf;
	Olde=InMat.e;
	Oldf=InMat.f;
	SMemMat.e = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.f = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.e[PIdx]=InMat.e[PIdx];
	SMemMat.e[PIdx_2]=InMat.e[PIdx_2];
#ifdef ILP3
	SMemMat.e[PIdx_3]=InMat.e[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.e[PIdx_4]=InMat.e[PIdx_4];
#endif
	SMemMat.f[PIdx]=InMat.f[PIdx];
	SMemMat.f[PIdx_2]=InMat.f[PIdx_2];
#ifdef ILP3
	SMemMat.f[PIdx_3]=InMat.f[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.f[PIdx_4]=InMat.f[PIdx_4];
#endif
	InMat.e=SMemMat.e;
	InMat.f=SMemMat.f;
	SMemMat.Cms = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.Cms[PIdx]=InMat.Cms[PIdx];
	SMemMat.Cms[PIdx_2]=InMat.Cms[PIdx_2];
#ifdef ILP3
	SMemMat.Cms[PIdx_3]=InMat.Cms[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.Cms[PIdx_4]=InMat.Cms[PIdx_4];
#endif
	// 1

	// discrete
	SMemMat.FIdxs = (MYDTYPE*) &smem[offset];
	offset+=InMat.N*SMemMat.LognDepth*sizeof(MYDTYPE);
	for(int i=0;i<SMemMat.LognDepth;i++) {
		SMemMat.FIdxs[i*InMat.N+PIdx]=InMat.FIdxs[i*InMat.N+PIdx];
		SMemMat.FIdxs[i*InMat.N+PIdx_2]=InMat.FIdxs[i*InMat.N+PIdx_2];
#ifdef ILP3
		SMemMat.FIdxs[i*InMat.N+PIdx_3]=InMat.FIdxs[i*InMat.N+PIdx_3];
#endif
#ifdef ILP4
		SMemMat.FIdxs[i*InMat.N+PIdx_4]=InMat.FIdxs[i*InMat.N+PIdx_4];
#endif
	}
	
	SMemMat.Ks = (MYDTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYDTYPE);
	SMemMat.Ks[PIdx]=InMat.Ks[PIdx];
	SMemMat.Ks[PIdx_2]=InMat.Ks[PIdx_2];
#ifdef ILP3
	SMemMat.Ks[PIdx_3]=InMat.Ks[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.Ks[PIdx_4]=InMat.Ks[PIdx_4];
#endif
	
	SMemMat.boolModel= (MYDTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYDTYPE);
	SMemMat.boolModel[PIdx]=InMat.boolModel[PIdx];
	SMemMat.boolModel[PIdx_2]=InMat.boolModel[PIdx_2];
#ifdef ILP3
	SMemMat.boolModel[PIdx_3]=InMat.boolModel[PIdx_3];
#endif
#ifdef ILP4
	SMemMat.boolModel[PIdx_4]=InMat.boolModel[PIdx_4];
#endif
	// 2
	MYDTYPE SonNo=InMat.SonNoVec[PIdx];
	MYDTYPE SonNo_2=InMat.SonNoVec[PIdx_2];
#ifdef ILP3
	MYDTYPE SonNo_3=InMat.SonNoVec[PIdx_3];
#endif
#ifdef ILP4
	MYDTYPE SonNo_4=InMat.SonNoVec[PIdx_4];
#endif
	
	SMemMat.RelStarts = (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	SMemMat.RelEnds = (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	SMemMat.RelVec = (MYDTYPE*) &smem[offset];
	offset+=InMat.nCallForFather*sizeof(MYDTYPE);
	SMemMat.SegStartI = (MYDTYPE*) &smem[offset];
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);
	SMemMat.SegEndI = (MYDTYPE*) &smem[offset];
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);

	SMemMat.Fathers= (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	// 3
	SMemMat.CompByLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	offset+=WARPSIZE*(SMemMat.Depth+1)*sizeof(MYDTYPE);
	SMemMat.CompByFLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	offset+=WARPSIZE*(SMemMat.Depth)*sizeof(MYDTYPE);
	
	// Moving back to floats - make sure that the address is good for float (is modulu 4).
	offset=ceilf(float(offset)/sizeof(MYFTYPE))*sizeof(MYFTYPE);
	MYDTYPE ConstantMemSize=offset;
	
	if(PIdx<InMat.nFathers) { // up to 32 !!!
		SMemMat.RelStarts[PIdx]=InMat.RelStarts[PIdx];
		SMemMat.RelEnds[PIdx]=InMat.RelEnds[PIdx];
		SMemMat.Fathers[PIdx]=InMat.Fathers[PIdx];
	}
	if(PIdx<InMat.nCallForFather) { // up to 32 !!!
		SMemMat.RelVec[PIdx]=InMat.RelVec[PIdx];
	}
	if(PIdx<InMat.nCallForFather+1) { // up to 32 !!!
		SMemMat.SegStartI[PIdx]=InMat.SegStartI[PIdx];
		SMemMat.SegEndI[PIdx]=InMat.SegEndI[PIdx];
	}
	syncthreads();
	
	for(int i=0;i<=SMemMat.Depth;i++) {
		SMemMat.CompByLevel32[WARPSIZE*i+PIdx]=InMat.CompByLevel32[WARPSIZE*i+PIdx];
	}
	for(int i=0;i<SMemMat.Depth;i++) {
		SMemMat.CompByFLevel32[WARPSIZE*i+PIdx]=InMat.CompByFLevel32[WARPSIZE*i+PIdx];
	}

	// Per STIMULUS
	// floats
	int Nt=ceil(sim.TFinal/sim.dt);
	// For solving the matrix
	MYDTYPE PerStimulus=((InMat.N+2)*2+InMat.N+WARPSIZE)*sizeof(MYFTYPE);
	offset+=PerStimulus*threadIdx.y;
	MYFTYPE *uHP,*bHP,*PX,*PF, *Vs;
	uHP = (MYFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYFTYPE);
	bHP = (MYFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYFTYPE);
	Vs = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	Vs[PIdx]=V[PIdx];
	Vs[PIdx_2]=V[PIdx_2];
#ifdef ILP3
	Vs[PIdx_3]=V[PIdx_3];
#endif
#ifdef ILP4
	Vs[PIdx_4]=V[PIdx_4];
#endif

	PX=bHP;
	PF=uHP;
	MYFTYPE *SMemVHot;
	
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=WARPSIZE*sizeof(MYFTYPE);

	MYDTYPE parentIndex;
	MYDTYPE parentIndex_2;
#ifdef ILP3
	MYDTYPE parentIndex_3;
#endif
#ifdef ILP4
	MYDTYPE parentIndex_4;
#endif
	MYDTYPE Eidx;
	MYDTYPE Eidx_2;
#ifdef ILP3
	MYDTYPE Eidx_3;
#endif
#ifdef ILP4
	MYDTYPE Eidx_4;
#endif
	Eidx = SMemMat.N-PIdx;
	Eidx_2 = SMemMat.N-PIdx_2;
#ifdef ILP3
	Eidx_3 = SMemMat.N-PIdx_3;
#endif
#ifdef ILP4
	Eidx_4 = SMemMat.N-PIdx_4;
#endif
	if(Eidx>SMemMat.N-1){
		Eidx=SMemMat.N-1;
	}
	if(PIdx==0) {
		parentIndex=0; }
	else {
		parentIndex =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx];
	}
	
	parentIndex_2 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_2];
#ifdef ILP3
	parentIndex_3 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_3];
#endif
#ifdef ILP4
	parentIndex_4 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_4];
#endif

	float rhs,D,gModel,StimCurrent,dv;
	float rhs_2,D_2,gModel_2,StimCurrent_2,dv_2;
#ifdef ILP3
	float rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#endif
#ifdef ILP4
	float rhs_4,D_4,gModel_4,StimCurrent_4,dv_4;
#endif
	for(int i=0;i<Nt;i++) {
		t = i*sim.dt;

		// Output
		rhs=0;
		rhs_2=0;
#ifdef ILP3
		rhs_3=0;
#endif
#ifdef ILP4
		rhs_4=0;
#endif
		D=0;
		D_2=0;
#ifdef ILP3
		D_3=0;
#endif
#ifdef ILP4
		D_4=0;
#endif

		// Before matrix

		//RRR here some work - this should do it    
		// CALL_TO_KERNEL_BREAK
		// CALL_TO_KERNEL_BREAK_dV

		KBreakpointModel(sumCurrents, sumConductivity,v,ModelStates[0],ModelStates[1],ModelStates[2],p1,p2,p3,p4,p5,p6);
		KBreakpointModel(sumCurrents_2, sumConductivity_2,v_2,ModelStates_2[0],ModelStates_2[1],ModelStates_2[2],p1_2,p2_2,p3_2,p4_2,p5_2,p6_2);
#ifdef ILP3
		KBreakpointModel(sumCurrents_3, sumConductivity_3,v_3,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2],p1_3,p2_3,p3_3,p4_3,p5_3,p6_3);
#endif
#ifdef ILP4
		KBreakpointModel(sumCurrents_4, sumConductivity_4,v_4,ModelStates_4[0],ModelStates_4[1],ModelStates_4[2],p1_4,p2_4,p3_4,p4_4,p5_4,p6_4);
#endif
		KBreakpointModel(sumCurrentsDv, sumConductivityDv,v+0.001,ModelStates[0],ModelStates[1],ModelStates[2],p1,p2,p3,p4,p5,p6);
		KBreakpointModel(sumCurrentsDv_2, sumConductivityDv_2,v_2+0.001,ModelStates_2[0],ModelStates_2[1],ModelStates_2[2],p1_2,p2_2,p3_2,p4_2,p5_2,p6_2);
#ifdef ILP3
		KBreakpointModel(sumCurrentsDv_3, sumConductivityDv_3,v_3+0.001,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2],p1_3,p2_3,p3_3,p4_3,p5_3,p6_3);
#endif
#ifdef ILP4
		KBreakpointModel(sumCurrentsDv_4, sumConductivityDv_4,v_4+0.001,ModelStates_4[0],ModelStates_4[1],ModelStates_4[2],p1_4,p2_4,p3_4,p4_4,p5_4,p6_4);
#endif

		gModel=0;
		gModel_2=0;
#ifdef ILP3
		gModel_3=0;
#endif
#ifdef ILP4
		gModel_4=0;
#endif
		if(SMemMat.boolModel[PIdx]==1) {
			gModel = (sumCurrentsDv-sumCurrents)/0.001; }
		else {
			sumConductivity=0;
			sumConductivityDv=0;
			sumCurrents=0;
			sumCurrentsDv=0;
		}
		if(SMemMat.boolModel[PIdx_2]==1) {gModel_2 = (sumCurrentsDv_2-sumCurrents_2)/0.001; }else {sumConductivity_2=0;sumConductivityDv_2=0;sumCurrents_2=0;sumCurrentsDv_2=0;}
#ifdef ILP3
		if(SMemMat.boolModel[PIdx_3]==1) {gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; }else {sumConductivity_3=0;sumConductivityDv_3=0;sumCurrents_3=0;sumCurrentsDv_3=0;}
#endif
#ifdef ILP4
		if(SMemMat.boolModel[PIdx_4]==1) {gModel_4 = (sumCurrentsDv_4-sumCurrents_4)/0.001; }else {sumConductivity_4=0;sumConductivityDv_4=0;sumCurrents_4=0;sumCurrentsDv_4=0;}
#endif
		StimCurrent=0;
		StimCurrent_2=0;
#ifdef ILP3
		StimCurrent_3=0;
#endif
#ifdef ILP4
		StimCurrent_4=0;
#endif

		if(t>=stimDel && t<stimDel+stimDur && PIdx == stimLoc){
			StimCurrent = 100*stimAmp/stimArea;
		}
		if(t>=stimDel && t<stimDel+stimDur && PIdx_2 == stimLoc){StimCurrent_2 = 100*stimAmp/stimArea;}
#ifdef ILP3
		if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#endif
#ifdef ILP4
		if(t>=stimDel && t<stimDel+stimDur && PIdx_4 == stimLoc){StimCurrent_4 = 100*stimAmp/stimArea;}
#endif
		
		
		rhs=StimCurrent-sumCurrents;
		rhs_2=StimCurrent_2-sumCurrents_2;
#ifdef ILP3
		rhs_3=StimCurrent_3-sumCurrents_3;
#endif
#ifdef ILP4
		rhs_4=StimCurrent_4-sumCurrents_4;
#endif


		dv=Vs[parentIndex]-Vs[PIdx];
		dv_2=Vs[parentIndex_2]-Vs[PIdx_2];
#ifdef ILP3
		dv_3=Vs[parentIndex_3]-Vs[PIdx_3];
#endif
#ifdef ILP4
		dv_4=Vs[parentIndex_4]-Vs[PIdx_4];
#endif

		rhs-=SMemMat.f[SMemMat.N-PIdx-1]*dv;
		rhs_2-=SMemMat.f[SMemMat.N-PIdx_2-1]*dv_2;
#ifdef ILP3
		rhs_3-=SMemMat.f[SMemMat.N-PIdx_3-1]*dv_3;
#endif
#ifdef ILP4
		rhs_4-=SMemMat.f[SMemMat.N-PIdx_4-1]*dv_4;
#endif

		D=sumConductivity+SMemMat.Cms[PIdx]/(sim.dt*1000);
		D_2=sumConductivity_2+SMemMat.Cms[PIdx_2]/(sim.dt*1000);
#ifdef ILP3
		D_3=sumConductivity_3+SMemMat.Cms[PIdx_3]/(sim.dt*1000);
#endif
#ifdef ILP4
		D_4=sumConductivity_4+SMemMat.Cms[PIdx_4]/(sim.dt*1000);
#endif

		D-=SMemMat.f[SMemMat.N-PIdx-1];
		D_2-=SMemMat.f[SMemMat.N-PIdx_2-1];
#ifdef ILP3
		D_3-=SMemMat.f[SMemMat.N-PIdx_3-1];
#endif
#ifdef ILP4
		D_4-=SMemMat.f[SMemMat.N-PIdx_4-1];
#endif
		if(PIdx==0){
			rhs=0;
			D=0;
		}

		bHP[SMemMat.N-PIdx-1]=rhs;
		bHP[SMemMat.N-PIdx_2-1]=rhs_2;
#ifdef ILP3
		bHP[SMemMat.N-PIdx_3-1]=rhs_3;
#endif
#ifdef ILP4
		bHP[SMemMat.N-PIdx_4-1]=rhs_4;
#endif
		uHP[SMemMat.N-PIdx-1]=D;
		uHP[SMemMat.N-PIdx_2-1]=D_2;
#ifdef ILP3
		uHP[SMemMat.N-PIdx_3-1]=D_3;
#endif
#ifdef ILP4
		uHP[SMemMat.N-PIdx_4-1]=D_4;
#endif
		
		syncthreads();
		
		if(SonNo==1) {
		
			bHP[SMemMat.N-parentIndex-1]+=SMemMat.e[Eidx]*dv;
		
			uHP[SMemMat.N-parentIndex-1]-=SMemMat.e[Eidx];
		}
		if(SonNo_2==1) {bHP[SMemMat.N-parentIndex_2-1]+=SMemMat.e[Eidx_2]*dv_2;uHP[SMemMat.N-parentIndex_2-1]-=SMemMat.e[Eidx_2];}
#ifdef ILP3
		if(SonNo_3==1) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#endif
#ifdef ILP4
		if(SonNo_4==1) {bHP[SMemMat.N-parentIndex_4-1]+=SMemMat.e[Eidx_4]*dv_4;uHP[SMemMat.N-parentIndex_4-1]-=SMemMat.e[Eidx_4];}
#endif

		if(SonNo==2) {bHP[SMemMat.N-parentIndex-1]+=SMemMat.e[Eidx]*dv;uHP[SMemMat.N-parentIndex-1]-=SMemMat.e[Eidx];}
		if(SonNo_2==2) {bHP[SMemMat.N-parentIndex_2-1]+=SMemMat.e[Eidx_2]*dv_2;uHP[SMemMat.N-parentIndex_2-1]-=SMemMat.e[Eidx_2];}
#ifdef ILP3
		if(SonNo_3==2) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#endif
#ifdef ILP4
		if(SonNo_4==2) {bHP[SMemMat.N-parentIndex_4-1]+=SMemMat.e[Eidx_4]*dv_4;uHP[SMemMat.N-parentIndex_4-1]-=SMemMat.e[Eidx_4];}
#endif

		
		BeforeLU(SMemMat,uHP,bHP,SMemMat.Depth);
		BkSub(SMemMat, PX,PF,uHP,bHP,SMemMat.LognDepth);
		Vmid=PX[SMemMat.N-PIdx-1];
		Vmid_2=PX[SMemMat.N-PIdx_2-1];
#ifdef ILP3
		Vmid_3=PX[SMemMat.N-PIdx_3-1];
#endif
#ifdef ILP4
		Vmid_4=PX[SMemMat.N-PIdx_4-1];
#endif

		v+=Vmid;
		v_2+=Vmid_2;
#ifdef ILP3
		v_3+=Vmid_3;
#endif
#ifdef ILP4
		v_4+=Vmid_4;
#endif
		Vs[PIdx]+=Vmid;
		Vs[PIdx_2]+=Vmid_2;
#ifdef ILP3
		Vs[PIdx_3]+=Vmid_3;
#endif
#ifdef ILP4
		Vs[PIdx_4]+=Vmid_4;
#endif

		if(SMemMat.boolModel[PIdx]==1) {KDerivModel(sim.dt, Vs[PIdx], ModelStates[0],ModelStates[1],ModelStates[2]);}
		if(SMemMat.boolModel[PIdx_2]==1) {KDerivModel(sim.dt, Vs[PIdx_2], ModelStates_2[0],ModelStates_2[1],ModelStates_2[2]); 		}		
#ifdef ILP3
		if(SMemMat.boolModel[PIdx_3]==1) {KDerivModel(sim.dt, Vs[PIdx_3], ModelStates_3[0],ModelStates_3[1],ModelStates_3[2]); 		}		
#endif
#ifdef ILP4
		if(SMemMat.boolModel[PIdx_4]==1) {KDerivModel(sim.dt, Vs[PIdx_4], ModelStates_4[0],ModelStates_4[1],ModelStates_4[2]); 		}		
#endif
		
		if((i%(WARPSIZE)==0)&&i>0){
			VHotGlobal[threadIdx.y*Nt+(i-WARPSIZE)+PIdx]=SMemVHot[PIdx];
		}
		SMemVHot[i%(WARPSIZE)]=Vs[stimLoc];
	}

}

void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	FILE *fl = fopen(FN, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}

__constant__ MYDTYPE* constKs;

hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, float* V) { 
	hipError_t cudaStatus;
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=ceil(sim.TFinal/sim.dt);
	VHotsHost=(float*)malloc(stim.NStimuli*Nt*sizeof(float));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.Depth=InMat.Depth;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
	
	


	cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
	cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (Mat_d.Depth+1)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (Mat_d.Depth)*WARPSIZE* sizeof(MYDTYPE));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);


	cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.Cms, InMat.Cms, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);

	
	
	// 32 data
	cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (Mat_d.Depth+1)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (Mat_d.Depth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}
	// copying
	MYFTYPE* ParamsForCuda;
	ParamsForCuda=(float*)malloc(InMat.N * NPARAMS *  sizeof(float));
	ReadParamsMatX(ParamsMat_FN,ParamsForCuda,NPARAMS,InMat.N);
	cudaStatus = hipMemcpy(d_modelParams, ParamsForCuda, NPARAMS * InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(1,NNEURONS);

	MYDTYPE offset=0;
	offset+=InMat.N*sizeof(MYFTYPE);
	offset+=InMat.N*sizeof(MYFTYPE);
	offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
	offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
	offset+=InMat.N*sizeof(MYDTYPE);
	offset+=InMat.N*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, boolModel
	offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	offset+=WARPSIZE*(InMat.Depth+1)*sizeof(MYDTYPE);
	offset+=WARPSIZE*(InMat.Depth)*sizeof(MYDTYPE);
	// 4 CompByLevel32 CompByFLevel32
	
	offset=ceilf(float(offset)/sizeof(MYFTYPE))*sizeof(MYFTYPE);
	MYDTYPE FrameworkMemSize=offset;

	offset+=(InMat.N+2)*sizeof(MYFTYPE);  // uHP (Diag)
	offset+=(InMat.N+2)*sizeof(MYFTYPE); // bHP (rhs)
	offset+=InMat.N*sizeof(MYFTYPE); // Vs
	offset+=Nt*sizeof(MYFTYPE);//SMemVHot
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+PerStimulus*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
	stEfork2TimeLoopGPUKernel<<<gridDim, blockDim,TotalSMem>>>(stim_d, d_modelParams, sim, Mat_d, V_d,VHotsGlobal); // RRR

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal, Nt * stim.NStimuli* sizeof(float), hipMemcpyDeviceToHost); 

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,Nt*stim.NStimuli,VHotsHost);
	
	return cudaStatus;
}

hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, float* V) {
	hipError_t cudaStatus;

	clock_t begin,end;
	
	begin=clock();
	stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V); //RRR sim
	end=clock();
	printf("stEfork2TimeLoopGPU took %g seconds\n",double(diffclock(end,begin)));
	


	return cudaStatus;
}
