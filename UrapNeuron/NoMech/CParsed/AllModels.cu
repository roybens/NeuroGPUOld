#include "hip/hip_runtime.h"
// Automatically generated CU for C:\Users\bensr\Dropbox\NeuroGPU\NeuroGPU4NoMech\UrapNeuron\NoMech./runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)


// GGlobals:
#define celsius (6.3)
#define stoprun (0.0)
#define clamp_resist (0.001)
#define secondorder (0.0)

// NGlobals:

// Declarations:
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:

// Procedures:

// Inits:

// Derivs:

// Breaks:
