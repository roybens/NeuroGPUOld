#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "Util.h"

	#define SUPERILPMACRO(x) ; ## x ## (1);
#ifdef ILP2
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2);
#endif
#ifdef ILP3
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3);
#endif
#ifdef ILP4
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); 
#endif
#ifdef ILP5
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); 
#endif
#ifdef ILP6
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6);
#endif
#ifdef ILP7
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7);
#endif
#ifdef ILP8
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8);
#endif
#ifdef ILP9
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9);
#endif
#ifdef ILP10
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10);
#endif
#ifdef ILP11
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11);
#endif
#ifdef ILP12
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12);
#endif
#ifdef ILP13
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13);
#endif
#ifdef ILP14
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14);
#endif
#ifdef ILP15
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15);
#endif
#ifdef ILP16
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16);
#endif
#ifdef ILP17
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17);
#endif
#ifdef ILP18
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18);
#endif
#ifdef ILP19
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19);
#endif
#ifdef ILP20
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20);
#endif
#ifdef ILP21
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21);
#endif
#ifdef ILP22
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22);
#endif
#ifdef ILP23
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23);
#endif
#ifdef ILP24
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24);
#endif
#ifdef ILP25
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25);
#endif
#ifdef ILP26
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26);
#endif
#ifdef ILP27
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27);
#endif
#ifdef ILP28
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28);
#endif
#ifdef ILP29
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29);
#endif
#ifdef ILP30
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30);
#endif
#ifdef ILP31
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31);
#endif
#ifdef ILP32
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); 
#endif
#ifdef ILP33
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33);
#endif
#ifdef ILP34
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34);
#endif
#ifdef ILP35
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35);
#endif
#ifdef ILP36
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36);
#endif
#ifdef ILP37
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37);
#endif
#ifdef ILP38
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38);
#endif
#ifdef ILP39
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39);
#endif
#ifdef ILP40
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40);
#endif
#ifdef ILP41
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41);
#endif
#ifdef ILP42
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42);
#endif
#ifdef ILP43
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43);
#endif
#ifdef ILP44
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44);
#endif
#ifdef ILP45
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45);
#endif
#ifdef ILP46
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46);
#endif
#ifdef ILP47
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47);
#endif
#ifdef ILP48
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48);
#endif
#ifdef ILP49
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49);
#endif
#ifdef ILP50
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49); ## x ## (50);
#endif


__device__ void BeforeLU(HMat InMat, MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel,LRelIndex;

	MYDTYPE s = InMat.CompByLevel32[0];//justforchecking
	MYDTYPE JumctionI;
	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		//s = InMat.LRelStarts[CurLevel];
		MYDTYPE e = InMat.LRelEnds[CurLevel];
		for(LRelIndex=s;LRelIndex<=e;LRelIndex++){ 
		//for(LRelIndex=InMat.LRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI=InMat.CompByLevel32[LRelIndex*WARPSIZE+PIdx]-1;
			for(i=InMat.SegStartI[JumctionI]-1;i<InMat.SegEndI[JumctionI];i++) {
				MYFTYPE uHPm1=uHP[i-1];
			
				uHP[i]=uHP[i]-InMat.e[i]*(InMat.f[i-1]/uHPm1); // So far same as paper parallel
				uHPm1=uHP[i-1];
				MYFTYPE bHPm1=bHP[i-1];
				bHP[i]=bHP[i]-bHPm1*InMat.e[i]/uHPm1; // bH is y
			}
		}
		if(CurLevel<Depth) {
			for(LRelIndex=InMat.FLRelStarts[CurLevel];LRelIndex<=InMat.FLRelEnds[CurLevel];LRelIndex++){ 
				CurB=InMat.CompByFLevel32[(LRelIndex)*WARPSIZE+PIdx]-1;//RBS i inserted another  -1 into the index RBS 2 i removed the-1 from the curlevel 
				CurJ=InMat.Fathers[CurB]-1;
				MYDTYPE St=InMat.RelStarts[CurB];
				MYDTYPE En=InMat.RelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=InMat.RelVec[j-1]-1;
					MYFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]-=InMat.e[t]*(InMat.f[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]-=bHPm1*InMat.e[t]/uHPm1; 
				}
			}
		}	
	}
}

#ifdef BKSUB1
__device__ void BkSub(HMat InMat, MYFTYPE* PX, MYFTYPE* PF,MYFTYPE* uHP, MYFTYPE* bHP, MYDTYPE LognDepth)
{
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
#define THISCOMMANDHERE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*(## VARILP -1)); MYDTYPE NextID_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHERE1)
	MYDTYPE i;
	
	PX=bHP;
	PF=uHP;
	// PX[PIdx_1]=PX[PIdx_1]/PF[PIdx_1]; // PF[PIdx_1]=-InMat.f[PIdx_1]/PF[PIdx_1];
#define THISCOMMANDHERE2(VARILP) PX[PIdx_ ## VARILP ]=PX[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ]; PF[PIdx_ ## VARILP ]=-InMat.f[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHERE2)

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		// NextID_1=InMat.FIdxs[i*InMat.N+PIdx_1]-1; // MYFTYPE OldPXj_1=PX[PIdx_1]; // MYFTYPE OldPXNextID_1=PX[NextID_1]; // PX[PIdx_1]=OldPXj_1+OldPXNextID_1*PF[PIdx_1];
#define THISCOMMANDHERE3(VARILP) NextID_ ## VARILP =InMat.FIdxs[i*InMat.N+PIdx_ ## VARILP ]-1; MYFTYPE OldPXj_ ## VARILP =PX[PIdx_ ## VARILP ]; MYFTYPE OldPXNextID_ ## VARILP =PX[NextID_ ## VARILP ]; PX[PIdx_ ## VARILP ]=OldPXj_ ## VARILP +OldPXNextID_ ## VARILP *PF[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHERE3)

		// PX[j]=PX[j]+PX[NextID]*PF[j];
		// PF[j]=PF[j]*PF[NextID];
		
		// MYFTYPE OldPFj_1=PF[PIdx_1]; // MYFTYPE OldPFNextID_1=PF[NextID_1]; // PF[PIdx_1]=OldPFj_1*OldPFNextID_1;
#define THISCOMMANDHERE4(VARILP) MYFTYPE OldPFj_ ## VARILP =PF[PIdx_ ## VARILP ]; MYFTYPE OldPFNextID_ ## VARILP =PF[NextID_ ## VARILP ]; PF[PIdx_ ## VARILP ]=OldPFj_ ## VARILP *OldPFNextID_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHERE4)
	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYFTYPE* uHP, MYFTYPE* bHP,MYFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
	SUPERILPMACRO(THISCOMMANDHEREB1)

	MYDTYPE j,CurJ,CurB,t;
	MYDTYPE JumctionI;
	short CurLevel,i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex,k;
	MYFTYPE temp;
 for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
  //     Run all independent set for this level, in parallel
  // for JumctionI=find(Level==CurLevel) % in parallel
  for(LRelIndex=InMat.LRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
   JumctionI=InMat.CompByLevel32[LRelIndex*WARPSIZE+PIdx_1]-1;
   // for i=(SegEndI(JumctionI)):-1:(SegStartI(JumctionI)-1)
   for(i=InMat.SegEndI[JumctionI]-1;i>=(InMat.SegStartI[JumctionI]-2);i--) {
    // k=KsB(i+1);
     k=InMat.KsB[i+1];
	 
    // bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
   Out[i]=(bHP[i]-Out[k]*InMat.f[i])/uHP[i];
   }
  }
 }
 } 

#endif
int testCode(MYFTYPE* orig, MYFTYPE* sim,int N){
	int i=0,iErr,iRel;
	MYFTYPE absErr,relErr,currErr,currRel;
	absErr = abs(orig[0]-sim[0]);
	relErr = absErr/abs(orig[0]);
	for(i=0;i<N;i++) {
		currErr = abs(orig[i]-sim[i]);
		currRel = currErr/abs(orig[i]);
		if (currErr>absErr){
			absErr=currErr;
			iErr=i;
		}
		if (currRel>relErr){
			relErr=currRel;
			iRel=i;

		}
	}
	printf("Absolute Error is %g at the %d index and Relative Error is %g at the %d index\n",absErr,iErr,relErr,iRel);
	return 1;
}
__device__ MYFTYPE Kvtrap(MYFTYPE x, MYFTYPE y) {
	if (fabs(x/y) < 1e-6) {;
	return y*(1 - x/y/2);
	}
	else
	{;
	return x/(exp(x/y) - 1);
	};
};

#ifdef HH
__device__ void Krates(MYFTYPE v, MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau,MYFTYPE &ninf,MYFTYPE &ntau) {
	MYFTYPE  alpha, beta, sum, q10;
	;
	// q10=3^((celsius - 6.3)/10); YYY CHANGE THIS to pow
	q10=pow(3,((celsius - 6.3)/10));
	;
	alpha=.1 * Kvtrap(-(v+40),10);
	beta=4 * exp(-(v+65)/18);
	sum=alpha + beta;
	mtau=1/(q10*sum);
	minf=alpha/sum;
	;
	alpha=.07 * exp(-(v+65)/20);
	beta=1 / (exp(-(v+35)/10) + 1);
	sum=alpha + beta;
	htau=1/(q10*sum);
	hinf=alpha/sum;
	;
	alpha=.01*Kvtrap(-(v+55),10) ;
	beta=.125*exp(-(v+65)/80);
	sum=alpha + beta;
	ntau=1/(q10*sum);
	ninf=alpha/sum;
}
#endif
#ifdef HH
__device__ void KInitModel(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE &n) {
	MYFTYPE hinf,htau,minf,mtau,ninf,ntau;
	Krates(v, hinf,htau,minf,mtau,ninf,ntau);
	m=minf;
	h=hinf;
	n=ninf;
}
#endif
#ifdef PAS
__device__ void KInitModel() {
}
#endif
#ifdef HH
__device__ void KDerivModel(MYFTYPE dt, MYFTYPE v, MYFTYPE &m,MYFTYPE &h,MYFTYPE &n) {
	MYFTYPE hinf,htau,minf,mtau,ninf,ntau;
	Krates(v, hinf,htau,minf,mtau,ninf,ntau);
	m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
	h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
	n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
};
#endif
#ifdef PAS
__device__ void KDerivModel() {
	
};
#endif
// YYY add sum conductivity float BreakpointModel(float &m,float &h,float &n,float gnabar,float gkbar,float gl,float el) {
// YYY add float v
// YYY change to void function
#ifdef HH
__device__ void KBreakpointModel(MYFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v, MYFTYPE m,MYFTYPE h,MYFTYPE n,MYFTYPE gnabar,MYFTYPE gkbar,MYFTYPE gl,MYFTYPE ena,MYFTYPE ek,MYFTYPE el) {
	float gk,gna;
	float ina,ik,il;
	gna=gnabar*m*m*m*h;
	ina=gna*(v-ena);
	gk=gkbar*n*n*n*n;
	ik=gk*(v-(ek));
	il=gl*(v-el);
	sumCurrents= ina+ik+il;
	sumConductivity= gl+gk+gna;
}
#endif
#ifdef PAS
__device__ void KBreakpointModel(MYFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v, MYFTYPE g,MYFTYPE e) {
	
	for(int i=0;i<PASLOOP;i++){
		g=g+0.001;
		
	}
	g=g-0.001*PASLOOP;
	sumCurrents= g*(v-e);
	sumConductivity= g;
}
#endif

// RRR
__global__ void stEfork2TimeLoopGPUKernel(Stim stim, MYFTYPE* ParamsM, Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal)
{
	MYDTYPE StimID=threadIdx.y;
	MYFTYPE stimDel = stim.dels[StimID];
	MYFTYPE stimDur = stim.durs[StimID];
	MYFTYPE stimAmp = stim.amps[StimID];
	MYDTYPE stimLoc = stim.loc;
	MYFTYPE stimArea = stim.area;
	
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2);
#define THISCOMMANDHEREE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1));
	SUPERILPMACRO(THISCOMMANDHEREE1)

	hipError_t cudaStatus;
	// for model RRR
	MYFTYPE t;
	
	// MYFTYPE isModel_3, dVec_3, Vmid_3; // MYFTYPE ModelStates_3[NSTATES]; // MYFTYPE v_3=V[PIdx_3]; // RRR -65?
#ifdef HH
#define THISCOMMANDHEREE2(VARILP) MYFTYPE isModel_ ## VARILP , dVec_ ## VARILP , Vmid_ ## VARILP ; MYFTYPE ModelStates_ ## VARILP [NSTATES]; MYFTYPE v_ ## VARILP =V[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE2)
#endif
#ifdef PAS
#define THISCOMMANDHEREE2(VARILP) MYFTYPE isModel_ ## VARILP , dVec_ ## VARILP , Vmid_ ## VARILP ;  MYFTYPE v_ ## VARILP =V[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE2)
#endif


	HMat SMemMat;
	SMemMat.N=InMat.N;
	SMemMat.Depth=InMat.Depth;
	SMemMat.CompDepth32=InMat.CompDepth32;
	SMemMat.CompFDepth32=InMat.CompFDepth32;
	SMemMat.LognDepth=InMat.LognDepth;
	SMemMat.nFathers=InMat.nFathers;
	SMemMat.nCallForFather=InMat.nCallForFather;
	SMemMat.nLRel=InMat.nLRel;
	SMemMat.nFLRel=InMat.nFLRel;


	SMemMat.boolModel=(MYDTYPE*)&smem[0];
	SMemMat.Cms=(MYFTYPE*)&smem[0];
	SMemMat.CompByFLevel32=(MYDTYPE*)&smem[0];
	SMemMat.e=(MYFTYPE*)&smem[0];
	SMemMat.f=(MYFTYPE*)&smem[0];
	SMemMat.Fathers=(MYDTYPE*)&smem[0];
	SMemMat.FIdxs=(MYDTYPE*)&smem[0];
	SMemMat.FLRelEnds=(MYDTYPE*)&smem[0];
	SMemMat.FLRelStarts=(MYDTYPE*)&smem[0];
	SMemMat.Ks=(MYDTYPE*)&smem[0];
	SMemMat.LRelEnds=(MYDTYPE*)&smem[0];
	SMemMat.LRelStarts=(MYDTYPE*)&smem[0];
	SMemMat.RelEnds=(MYDTYPE*)&smem[0];
	SMemMat.RelStarts=(MYDTYPE*)&smem[0];
	SMemMat.RelVec=(MYDTYPE*)&smem[0];
	SMemMat.SegEndI=(MYDTYPE*)&smem[0];
	SMemMat.SegStartI=(MYDTYPE*)&smem[0];
	SMemMat.SonNoVec=(MYDTYPE*)&smem[0];
	SMemMat.CompByLevel32=(MYDTYPE*)&smem[0];
	SMemMat.CompByLevel32[0]=100;
		BeforeLU(SMemMat,SMemMat.Cms,SMemMat.e,SMemMat.Depth);


	// RRRXXX Put in a FOR loop, up to NPARAM or something like that
	// MYFTYPE p1_3,p2_3,p3_3,p4_3,p5_3,p6_3; // p1_3=ParamsM[0*SMemMat.N+PIdx_3]; // p2_3=ParamsM[1*SMemMat.N+PIdx_3]; // p3_3=ParamsM[2*SMemMat.N+PIdx_3]; // p5_3=ParamsM[4*SMemMat.N+PIdx_3]; // p6_3=ParamsM[5*SMemMat.N+PIdx_3];
#ifdef HH
#define THISCOMMANDHEREE3(VARILP) MYFTYPE p1_ ## VARILP ,p2_ ## VARILP ,p3_ ## VARILP ,p4_ ## VARILP ,p5_ ## VARILP ,p6_ ## VARILP ; p1_ ## VARILP =ParamsM[0*SMemMat.N+PIdx_ ## VARILP ]; p2_ ## VARILP =ParamsM[1*SMemMat.N+PIdx_ ## VARILP ]; p3_ ## VARILP =ParamsM[2*SMemMat.N+PIdx_ ## VARILP ]; p4_ ## VARILP =ParamsM[3*SMemMat.N+PIdx_ ## VARILP ]; p5_ ## VARILP =ParamsM[4*SMemMat.N+PIdx_ ## VARILP ]; p6_ ## VARILP =ParamsM[5*SMemMat.N+PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE3)
#endif

#ifdef PAS
#define THISCOMMANDHEREE3(VARILP) MYFTYPE p1_ ## VARILP ,p2_ ## VARILP; p1_ ## VARILP =ParamsM[0*SMemMat.N+PIdx_ ## VARILP ]; p2_ ## VARILP =ParamsM[1*SMemMat.N+PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE3)
#endif
	// KInitModel(v_3,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2]);
#ifdef HH
#define THISCOMMANDHEREE4(VARILP) KInitModel(v_ ## VARILP ,ModelStates_ ## VARILP [0],ModelStates_ ## VARILP [1],ModelStates_ ## VARILP [2]);
	SUPERILPMACRO(THISCOMMANDHEREE4)
#endif
#ifdef PAS
	#define THISCOMMANDHEREE4(VARILP) KInitModel();
	SUPERILPMACRO(THISCOMMANDHEREE4)
#endif


	// MYFTYPE sumCurrents_3, sumConductivity_3; // MYFTYPE sumCurrentsDv_3, sumConductivityDv_3;
#define THISCOMMANDHEREE5(VARILP) MYFTYPE sumCurrents_ ## VARILP , sumConductivity_ ## VARILP ; MYFTYPE sumCurrentsDv_ ## VARILP , sumConductivityDv_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHEREE5)
	// end for model in init

	MYDTYPE offset=0;
	// CONSTANTS
	// floats
	MYFTYPE *Olde,*Oldf;
	Olde=InMat.e;
	Oldf=InMat.f;
	SMemMat.e = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	SMemMat.f = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);
	
	// SMemMat.e[PIdx_3]=InMat.e[PIdx_3]; // SMemMat.f[PIdx_3]=InMat.f[PIdx_3];
#define THISCOMMANDHEREE6(VARILP) SMemMat.e[PIdx_ ## VARILP ]=InMat.e[PIdx_ ## VARILP ]; SMemMat.f[PIdx_ ## VARILP ]=InMat.f[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE6)

	InMat.e=SMemMat.e;
	InMat.f=SMemMat.f;
	SMemMat.Cms = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);

	// SMemMat.Cms[PIdx_3]=InMat.Cms[PIdx_3];
#define THISCOMMANDHEREE7(VARILP) SMemMat.Cms[PIdx_ ## VARILP ]=InMat.Cms[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE7)
	
	// 1

	// discrete
#ifdef BKSUB1
		SMemMat.FIdxs = (MYDTYPE*) &smem[offset];
		offset+=InMat.N*SMemMat.LognDepth*sizeof(MYDTYPE);
		for(int i=0;i<SMemMat.LognDepth;i++) {

			// SMemMat.FIdxs[i*InMat.N+PIdx_3]=InMat.FIdxs[i*InMat.N+PIdx_3];
#define THISCOMMANDHEREE8(VARILP) SMemMat.FIdxs[i*InMat.N+PIdx_ ## VARILP ]=InMat.FIdxs[i*InMat.N+PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE8)

	}
#endif 
#ifdef BKSUB2
		SMemMat.KsB =  (MYDTYPE*) &smem[offset];
		offset+=(SMemMat.N+1)*sizeof(MYDTYPE);		
		SMemMat.KsB[InMat.N]=InMat.KsB[InMat.N];

		// SMemMat.KsB[PIdx_3]=InMat.KsB[PIdx_3];
#define THISCOMMANDHEREE9(VARILP) SMemMat.KsB[PIdx_ ## VARILP ]=InMat.KsB[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE9)

#endif
		
	SMemMat.Ks = (MYDTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYDTYPE);

	// SMemMat.Ks[PIdx_3]=InMat.Ks[PIdx_3];
#define THISCOMMANDHEREE10(VARILP) SMemMat.Ks[PIdx_ ## VARILP ]=InMat.Ks[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE10)

	
	SMemMat.boolModel= (MYDTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYDTYPE);
	
	// SMemMat.boolModel[PIdx_3]=InMat.boolModel[PIdx_3];
#define THISCOMMANDHEREE11(VARILP) SMemMat.boolModel[PIdx_ ## VARILP ]=InMat.boolModel[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE11)

	// 2
	// MYDTYPE SonNo_3=InMat.SonNoVec[PIdx_3];
#define THISCOMMANDHEREE12(VARILP) MYDTYPE SonNo_ ## VARILP =InMat.SonNoVec[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE12)

	
	SMemMat.RelStarts = (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	SMemMat.RelEnds = (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	SMemMat.RelVec = (MYDTYPE*) &smem[offset];
	offset+=InMat.nCallForFather*sizeof(MYDTYPE);
	SMemMat.SegStartI = (MYDTYPE*) &smem[offset];
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);
	SMemMat.SegEndI = (MYDTYPE*) &smem[offset];
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);

	SMemMat.Fathers= (MYDTYPE*) &smem[offset];
	offset+=InMat.nFathers*sizeof(MYDTYPE);
	// 3
	
	SMemMat.CompByLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	offset+=WARPSIZE*(SMemMat.CompDepth32)*sizeof(MYDTYPE);
	/*
	SMemMat.CompByFLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	offset+=WARPSIZE*(SMemMat.CompFDepth32)*sizeof(MYDTYPE);
	
	SMemMat.LRelStarts = (MYDTYPE*) &smem[offset];
	offset+=SMemMat.nLRel*sizeof(MYDTYPE);
	SMemMat.LRelEnds = (MYDTYPE*) &smem[offset];
	offset+=SMemMat.nLRel*sizeof(MYDTYPE);
	SMemMat.FLRelStarts = (MYDTYPE*) &smem[offset];
	offset+=SMemMat.nFLRel*sizeof(MYDTYPE);
	SMemMat.FLRelEnds = (MYDTYPE*) &smem[offset];
	offset+=SMemMat.nFLRel*sizeof(MYDTYPE);
	
	
	// Moving back to floats - make sure that the address is good for float (is modulu 4).
	offset=ceilf(float(offset)/sizeof(MYFTYPE))*sizeof(MYFTYPE);
	MYDTYPE ConstantMemSize=offset;
	
	//if(PIdx_1<InMat.nFathers) { SMemMat.RelStarts[PIdx_1]=InMat.RelStarts[PIdx_1];SMemMat.RelEnds[PIdx_1]=InMat.RelEnds[PIdx_1];SMemMat.Fathers[PIdx_1]=InMat.Fathers[PIdx_1];}
	#define THISCOMMANDHEREE120(VARILP) if(PIdx_ ## VARILP<InMat.nFathers){ SMemMat.RelStarts[PIdx_ ## VARILP]=InMat.RelStarts[PIdx_ ## VARILP];SMemMat.RelEnds[PIdx_ ## VARILP]=InMat.RelEnds[PIdx_ ## VARILP];SMemMat.Fathers[PIdx_ ## VARILP]=InMat.Fathers[PIdx_ ## VARILP];}
	SUPERILPMACRO(THISCOMMANDHEREE120)
	//if(PIdx_1<InMat.nCallForFather) {SMemMat.RelVec[PIdx_1]=InMat.RelVec[PIdx_1];}
	#define THISCOMMANDHEREE121(VARILP) if(PIdx_ ## VARILP<InMat.nCallForFather){SMemMat.RelVec[PIdx_ ## VARILP]=InMat.RelVec[PIdx_ ## VARILP];}
	SUPERILPMACRO(THISCOMMANDHEREE121)
	
	//if(PIdx_1<InMat.nCallForFather+1) {SMemMat.SegStartI[PIdx_1]=InMat.SegStartI[PIdx_1];SMemMat.SegEndI[PIdx_1]=InMat.SegEndI[PIdx_1];}
	#define THISCOMMANDHEREE122(VARILP) if(PIdx_ ## VARILP<InMat.nCallForFather+1){SMemMat.SegStartI[PIdx_ ## VARILP]=InMat.SegStartI[PIdx_ ## VARILP];SMemMat.SegEndI[PIdx_ ## VARILP]=InMat.SegEndI[PIdx_ ## VARILP];}
	SUPERILPMACRO(THISCOMMANDHEREE122)
	syncthreads();
	
	for(int i=0;i<SMemMat.CompDepth32;i++) {SMemMat.CompByLevel32[WARPSIZE*i+PIdx_1]=InMat.CompByLevel32[WARPSIZE*i+PIdx_1];}
	//#define THISCOMMANDHEREE123(VARILP) for(int i=0;i<=SMemMat.Depth;i++) {SMemMat.CompByLevel32[WARPSIZE*i+PIdx_ ## VARILP]=InMat.CompByLevel32[WARPSIZE*i+PIdx_ ## VARILP];}
	//SUPERILPMACRO(THISCOMMANDHEREE123)
	
	for(int i=0;i<SMemMat.CompFDepth32;i++) {SMemMat.CompByFLevel32[WARPSIZE*i+PIdx_1]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_1];}
	//#define THISCOMMANDHEREE124(VARILP) for(int i=0;i<SMemMat.Depth;i++) {SMemMat.CompByFLevel32[WARPSIZE*i+PIdx_ ## VARILP]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_ ## VARILP];}
	//SUPERILPMACRO(THISCOMMANDHEREE124)

	//if(PIdx_1<InMat.nLRel) {SMemMat.LRelStarts[PIdx_1]=InMat.LRelStarts[PIdx_1];SMemMat.LRelEnds[PIdx_1]=InMat.LRelEnds[PIdx_1];}
	#define THISCOMMANDHEREE125(VARILP) if(PIdx_ ## VARILP<InMat.nLRel){SMemMat.LRelStarts[PIdx_ ## VARILP]=InMat.LRelStarts[PIdx_ ## VARILP];SMemMat.LRelEnds[PIdx_ ## VARILP]=InMat.LRelEnds[PIdx_ ## VARILP];}
	SUPERILPMACRO(THISCOMMANDHEREE125)
	syncthreads();


	//if(PIdx_1<InMat.nFLRel) {SMemMat.FLRelStarts[PIdx_1]=InMat.FLRelStarts[PIdx_1];SMemMat.FLRelEnds[PIdx_1]=InMat.FLRelEnds[PIdx_1];}
	#define THISCOMMANDHEREE126(VARILP) if(PIdx_ ## VARILP<InMat.nFLRel) {SMemMat.FLRelStarts[PIdx_ ## VARILP]=InMat.FLRelStarts[PIdx_ ## VARILP];SMemMat.FLRelEnds[PIdx_ ## VARILP]=InMat.FLRelEnds[PIdx_ ## VARILP];}
	SUPERILPMACRO(THISCOMMANDHEREE126)

	*/
	// Per STIMULUS
	// floats
	int Nt=ceil(sim.TFinal/sim.dt);
	// For solving the matrix
#ifdef BKSUB2
	MYDTYPE PerStimulus=((InMat.N+2)*2+InMat.N+WARPSIZE)*sizeof(MYFTYPE);
#endif
#ifdef BKSUB1
	MYDTYPE PerStimulus=((InMat.N+2)*2+InMat.N+WARPSIZE)*sizeof(MYFTYPE);
#endif
	
	offset+=PerStimulus*threadIdx.y;
	MYFTYPE *uHP,*bHP,*PX,*PF, *Vs;
	uHP = (MYFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYFTYPE);
	bHP = (MYFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYFTYPE);
	Vs = (MYFTYPE*) &smem[offset];
	offset+=InMat.N*sizeof(MYFTYPE);

	// Vs[PIdx_3]=V[PIdx_3];
	/*
#define THISCOMMANDHEREE13(VARILP) Vs[PIdx_ ## VARILP ]=V[PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE13)

	*/
	PX=bHP;
	PF=uHP;
	MYFTYPE *SMemVHot;
	/*
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=WARPSIZE*sizeof(MYFTYPE);
	*/
	
	// MYDTYPE parentIndex_3; // MYDTYPE Eidx_3; // Eidx_3 = SMemMat.N-PIdx_3;
#define THISCOMMANDHEREE14(VARILP) MYDTYPE parentIndex_ ## VARILP ; MYDTYPE Eidx_ ## VARILP ; Eidx_ ## VARILP = SMemMat.N-PIdx_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHEREE14)
	
	// RRRXXX This probably should be in ILP too!
	if(Eidx_1>SMemMat.N-1){
		Eidx_1=SMemMat.N-1;
	}
	
	// parentIndex_3 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_3];
#define THISCOMMANDHEREE15(VARILP) parentIndex_ ## VARILP =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_ ## VARILP ];
	SUPERILPMACRO(THISCOMMANDHEREE15)

	// RRRXXX This probably should be in ILP too!
	if(PIdx_1==0) {
		parentIndex_1=0; };
	// This part is redundant now, with the macro
	// else { 		parentIndex_1 =SMemMat.N-SMemMat.Ks[SMemMat.N-PIdx_1]; 	}
	

	// float rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#define THISCOMMANDHEREE16(VARILP) float rhs_ ## VARILP ,D_ ## VARILP ,gModel_ ## VARILP ,StimCurrent_ ## VARILP ,dv_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHEREE16)
		SMemMat.CompByLevel32[0]=100;
		BeforeLU(SMemMat,uHP,bHP,SMemMat.Depth);
	for(int i=0;i<Nt;i++) {
		t = i*sim.dt;

		// Output
		// rhs_3=0; // D_3=0;
#define THISCOMMANDHEREE17(VARILP) rhs_ ## VARILP =0; D_ ## VARILP =0;
	SUPERILPMACRO(THISCOMMANDHEREE17)


		// Before matrix

		//RRR here some work - this should do it    
		// CALL_TO_KERNEL_BREAK
		// CALL_TO_KERNEL_BREAK_dV

		// RRRXXX This should be generified as well\!
		// KBreakpointModel(sumCurrents_3, sumConductivity_3,v_3,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2],p1_3,p2_3,p3_3,p4_3,p5_3,p6_3);
#ifdef HH
#define THISCOMMANDHEREE18(VARILP) KBreakpointModel(sumCurrents_ ## VARILP , sumConductivity_ ## VARILP ,v_ ## VARILP ,ModelStates_ ## VARILP [0],ModelStates_ ## VARILP [1],ModelStates_ ## VARILP [2],p1_ ## VARILP ,p2_ ## VARILP ,p3_ ## VARILP ,p4_ ## VARILP ,p5_ ## VARILP ,p6_ ## VARILP );
	SUPERILPMACRO(THISCOMMANDHEREE18)
#endif
#ifdef PAS
#define THISCOMMANDHEREE18(VARILP) KBreakpointModel(sumCurrents_ ## VARILP , sumConductivity_ ## VARILP ,v_ ## VARILP ,p1_ ## VARILP ,p2_ ## VARILP);
	SUPERILPMACRO(THISCOMMANDHEREE18)
#endif
		// RRRXXX This should be generified as well\!
		// KBreakpointModel(sumCurrentsDv_3, sumConductivityDv_3,v_3+0.001,ModelStates_3[0],ModelStates_3[1],ModelStates_3[2],p1_3,p2_3,p3_3,p4_3,p5_3,p6_3);
#ifdef HH
#define THISCOMMANDHEREE19(VARILP) KBreakpointModel(sumCurrentsDv_ ## VARILP , sumConductivityDv_ ## VARILP ,v_ ## VARILP +0.001,ModelStates_ ## VARILP [0],ModelStates_ ## VARILP [1],ModelStates_ ## VARILP [2],p1_ ## VARILP ,p2_ ## VARILP ,p3_ ## VARILP ,p4_ ## VARILP ,p5_ ## VARILP ,p6_ ## VARILP );
	SUPERILPMACRO(THISCOMMANDHEREE19)
#endif
#ifdef PAS
#define THISCOMMANDHEREE19(VARILP) KBreakpointModel(sumCurrentsDv_ ## VARILP , sumConductivityDv_ ## VARILP ,v_ ## VARILP +0.001,p1_ ## VARILP ,p2_ ## VARILP);
	SUPERILPMACRO(THISCOMMANDHEREE19)
#endif

		// gModel_3=0;
#define THISCOMMANDHEREE20(VARILP) gModel_ ## VARILP =0;
	SUPERILPMACRO(THISCOMMANDHEREE20)

		// if(SMemMat.boolModel[PIdx_3]==1) {gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; }else {sumConductivity_3=0;sumConductivityDv_3=0;sumCurrents_3=0;sumCurrentsDv_3=0;}
#define THISCOMMANDHEREE21(VARILP) if(SMemMat.boolModel[PIdx_ ## VARILP ]==1) {gModel_ ## VARILP  = (sumCurrentsDv_ ## VARILP -sumCurrents_ ## VARILP )/0.001; }else {sumConductivity_ ## VARILP =0;sumConductivityDv_ ## VARILP =0;sumCurrents_ ## VARILP =0;sumCurrentsDv_ ## VARILP =0;}
	SUPERILPMACRO(THISCOMMANDHEREE21)

		// StimCurrent_3=0;
#define THISCOMMANDHEREE22(VARILP) StimCurrent_ ## VARILP =0;
	SUPERILPMACRO(THISCOMMANDHEREE22)


		// if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#define THISCOMMANDHEREE23(VARILP) if(t>=stimDel && t<stimDel+stimDur && PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP  = 100*stimAmp/stimArea;}
	SUPERILPMACRO(THISCOMMANDHEREE23)
		
		
		// rhs_3=StimCurrent_3-sumCurrents_3; // dv_3=Vs[parentIndex_3]-Vs[PIdx_3]; // rhs_3-=SMemMat.f[SMemMat.N-PIdx_3-1]*dv_3;
#define THISCOMMANDHEREE24(VARILP) rhs_ ## VARILP =StimCurrent_ ## VARILP -sumCurrents_ ## VARILP ; dv_ ## VARILP =Vs[parentIndex_ ## VARILP ]-Vs[PIdx_ ## VARILP ]; rhs_ ## VARILP -=SMemMat.f[SMemMat.N-PIdx_ ## VARILP -1]*dv_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHEREE24)


		// D_3=sumConductivity_3+SMemMat.Cms[PIdx_3]/(sim.dt*1000); // D_3-=SMemMat.f[SMemMat.N-PIdx_3-1];
#define THISCOMMANDHEREE25(VARILP) D_ ## VARILP =sumConductivity_ ## VARILP +SMemMat.Cms[PIdx_ ## VARILP ]/(sim.dt*1000); D_ ## VARILP -=SMemMat.f[SMemMat.N-PIdx_ ## VARILP -1];
	SUPERILPMACRO(THISCOMMANDHEREE25)

		// RRRXXX Probably should be ILPed too.
		if(PIdx_1==0){
			rhs_1=0;
			D_1=0;
		}

		// bHP[SMemMat.N-PIdx_3-1]=rhs_3; // uHP[SMemMat.N-PIdx_3-1]=D_3;
#define THISCOMMANDHEREE26(VARILP) bHP[SMemMat.N-PIdx_ ## VARILP -1]=rhs_ ## VARILP ; uHP[SMemMat.N-PIdx_ ## VARILP -1]=D_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHEREE26)
				
		syncthreads();
		
		// if(SonNo_3==1) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#define THISCOMMANDHEREE27(VARILP) if(SonNo_ ## VARILP ==1) {bHP[SMemMat.N-parentIndex_ ## VARILP -1]+=SMemMat.e[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[SMemMat.N-parentIndex_ ## VARILP -1]-=SMemMat.e[Eidx_ ## VARILP ];}
	SUPERILPMACRO(THISCOMMANDHEREE27)


		// if(SonNo_3==2) {bHP[SMemMat.N-parentIndex_3-1]+=SMemMat.e[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=SMemMat.e[Eidx_3];}
#define THISCOMMANDHEREE28(VARILP) if(SonNo_ ## VARILP ==2) {bHP[SMemMat.N-parentIndex_ ## VARILP -1]+=SMemMat.e[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[SMemMat.N-parentIndex_ ## VARILP -1]-=SMemMat.e[Eidx_ ## VARILP ];}
	SUPERILPMACRO(THISCOMMANDHEREE28)

		syncthreads();
		BeforeLU(SMemMat,uHP,bHP,SMemMat.Depth);
#ifdef BKSUB1
		BkSub(SMemMat, PX,PF,uHP,bHP,SMemMat.LognDepth);
		#define THISCOMMANDHEREE29(VARILP) Vmid_ ## VARILP =PX[SMemMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ; Vs[PIdx_ ## VARILP ]+=Vmid_ ## VARILP ;
	SUPERILPMACRO(THISCOMMANDHEREE29)
#endif

		
#ifdef BKSUB2
		#define THISCOMMANDHEREE290(VARILP) MYFTYPE vTemp_ ## VARILP=Vs[PIdx_ ## VARILP];
		SUPERILPMACRO(THISCOMMANDHEREE290);
		syncthreads();
		BkSub(SMemMat,uHP,bHP,Vs,SMemMat.Depth);
		syncthreads();
		PX=Vs;

#define THISCOMMANDHEREE29(VARILP) Vmid_ ## VARILP =PX[SMemMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ;
		SUPERILPMACRO(THISCOMMANDHEREE29)
		syncthreads(); 
#define THISCOMMANDHEREE291(VARILP) Vs[PIdx_ ## VARILP ]= vTemp_ ## VARILP +Vmid_ ## VARILP ;
		SUPERILPMACRO(THISCOMMANDHEREE291)
#endif		
		// RRRXXX at som point, this should be generalized to NSTATES, by some super weird macro, I guess. Leave that for next level.
		// if(SMemMat.boolModel[PIdx_3]==1) {KDerivModel(sim.dt, Vs[PIdx_3], ModelStates_3[0],ModelStates_3[1],ModelStates_3[2]); 		}		
#ifdef HH
#define THISCOMMANDHEREE30(VARILP) if(SMemMat.boolModel[PIdx_ ## VARILP ]==1) {KDerivModel(sim.dt, Vs[PIdx_ ## VARILP ], ModelStates_ ## VARILP [0],ModelStates_ ## VARILP [1],ModelStates_ ## VARILP [2]); 		}
	SUPERILPMACRO(THISCOMMANDHEREE30)
#endif
#ifdef PAS
#define THISCOMMANDHEREE30(VARILP) if(SMemMat.boolModel[PIdx_ ## VARILP ]==1) {KDerivModel(); }
	SUPERILPMACRO(THISCOMMANDHEREE30)
#endif

		if((i%(WARPSIZE)==0)&&i>0){
			VHotGlobal[threadIdx.y*Nt+(i-WARPSIZE)+PIdx_1]=SMemVHot[PIdx_1];
		}
		SMemVHot[i%(WARPSIZE)]=Vs[stimLoc];
}
}

void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}

__constant__ MYDTYPE* constKs;

hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, float* V) { 

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipDeviceReset();
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=ceil(sim.TFinal/sim.dt);
	VHotsHost=(float*)malloc(stim.NStimuli*Nt*sizeof(float));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.Depth=InMat.Depth;
	Mat_d.CompDepth32=InMat.CompDepth32;
	Mat_d.CompFDepth32=InMat.CompFDepth32;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	Mat_d.nLRel=InMat.nLRel;
	Mat_d.nFLRel=InMat.nFLRel;
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
	
	


	cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
#ifdef BKSUB1
	cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N +1)* sizeof(MYDTYPE));
#endif
	cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (Mat_d.CompDepth32)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (Mat_d.CompFDepth32)*WARPSIZE* sizeof(MYDTYPE));

	cudaStatus = hipMalloc((void**)&Mat_d.LRelStarts, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.LRelEnds, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelStarts, Mat_d.nFLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelEnds, Mat_d.nFLRel* sizeof(MYDTYPE));

	

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);


	cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.Cms, InMat.Cms, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);

	
	
	// 32 data
#ifdef BKSUB1
	cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif
#ifdef BKSUB2
	cudaStatus = hipMemcpy(Mat_d.KsB, InMat.KsB,(InMat.N+1)*sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif

	cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (Mat_d.CompDepth32)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (Mat_d.CompFDepth32)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);

	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}
	// copying
	MYFTYPE* ParamsForCuda;
	ParamsForCuda=(float*)malloc(InMat.N * NPARAMS *  sizeof(float));
	ReadParamsMatX(ParamsMat_FN,ParamsForCuda,NPARAMS,InMat.N);
	cudaStatus = hipMemcpy(d_modelParams, ParamsForCuda, NPARAMS * InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(1,NNEURONS);

	MYDTYPE offset=0;
	offset+=InMat.N*sizeof(MYFTYPE);
	offset+=InMat.N*sizeof(MYFTYPE);
	offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
#ifdef BKSUB1
	offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
#endif
	offset+=InMat.N*sizeof(MYDTYPE);
	offset+=InMat.N*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, boolModel
	offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	offset+=WARPSIZE*(InMat.CompDepth32)*sizeof(MYDTYPE);
	offset+=WARPSIZE*(InMat.CompFDepth32)*sizeof(MYDTYPE);
	// 4 CompByLevel32 CompByFLevel32
	offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
	offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
#ifdef BKSUB2
	offset+=(Mat_d.N+1)*sizeof(MYDTYPE);
#endif
	// 5 LRelStarts LRelEnds FLRelStarts  FLRelEnds KsB
	offset=ceilf(float(offset)/sizeof(MYFTYPE))*sizeof(MYFTYPE);
	MYDTYPE FrameworkMemSize=offset;
//PerStimulus

	offset+=(InMat.N+2)*sizeof(MYFTYPE);  // uHP (Diag)
	offset+=(InMat.N+2)*sizeof(MYFTYPE); // bHP (rhs)
	offset+=InMat.N*sizeof(MYFTYPE); // Vs

	offset+=Nt*sizeof(MYFTYPE);//SMemVHot
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+PerStimulus*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
	clock_t begin,end;
	
	begin=clock();
	stEfork2TimeLoopGPUKernel<<<gridDim, blockDim,TotalSMem>>>(stim_d, d_modelParams, sim, Mat_d, V_d,VHotsGlobal); // RRR
	cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal, Nt * stim.NStimuli* sizeof(float), hipMemcpyDeviceToHost); 
	end=clock();
	double totalT = diffclock(end,begin);
	printf("stEfork2TimeLoopGPU took %g seconds\n",totalT);
	FILE *file = fopen(TIMES_FN, "wb");
	if ( file ) {
		MYDTYPE mul32 = MUL32;
		fwrite(&mul32,sizeof(MYDTYPE),1,file);
		fwrite(&totalT,sizeof(double),1,file);
	} else {
		printf("ERR SaveArrayToFile %s\n",TIMES_FN);
	}
	fclose(file);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,Nt*stim.NStimuli,VHotsHost);
	
	return cudaStatus;
}

hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, float* V) {
	hipError_t cudaStatus;

	
	stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V); //RRR sim
	
	


	return cudaStatus;
}
