#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "AllModels.cu"
#include "AllModels.cuh"

__constant__ MYFTYPE cCm[416];
__constant__ MYSECONDFTYPE cE[416];
__constant__ MYSECONDFTYPE cF[416];
__constant__ MYDTYPE cFIdxs[416*7];
__constant__ MYDTYPE cKs[416];
__constant__ MYDTYPE cSegToComp[416];
__constant__ MYDTYPE cBoolModel[416 * 7];//One day change this to bool
__constant__ MYDTYPE cRelStarts[99];//nFathers
__constant__ MYDTYPE cRelEnds[99];//nFathers
__constant__ MYDTYPE cFathers[99];//nFathers
__constant__ MYDTYPE cRelVec[186];//nCallForFather
__constant__ MYDTYPE cSegStartI[186+1];//nCallForFather
__constant__ MYDTYPE cSegEndI[186+1];//nCallForFather

__constant__ MYDTYPE cCompByLevel32[26*WARPSIZE];//CompDepth
__constant__ MYDTYPE cCompByFLevel32[23*WARPSIZE];//CompFDepth
__constant__ MYDTYPE cLRelStarts[24];//nLRel
__constant__ MYDTYPE cLRelEnds[24];//nLRel
__constant__ MYDTYPE cFLRelStarts[23];//nFLRel
__constant__ MYDTYPE cFLRelEnds[23];//nFLRel
__constant__ MYDTYPE cSonNoVec[416];//InMat.N 


#ifdef ILP1
	#define SUPERILPMACRO(x) ; ## x ## (1);
#endif
#ifdef ILP2
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2);
#endif
#ifdef ILP3
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3);
#endif
#ifdef ILP4
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); 
#endif
#ifdef ILP5
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); 
#endif
#ifdef ILP6
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6);
#endif
#ifdef ILP7
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7);
#endif
#ifdef ILP8
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8);
#endif
#ifdef ILP9
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9);
#endif
#ifdef ILP10
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10);
#endif
#ifdef ILP11
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11);
#endif
#ifdef ILP12
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12);
#endif
#ifdef ILP13
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13);
#endif
#ifdef ILP14
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14);
#endif
#ifdef ILP15
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15);
#endif
#ifdef ILP16
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16);
#endif
#ifdef ILP17
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17);
#endif
#ifdef ILP18
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18);
#endif
#ifdef ILP19
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19);
#endif
#ifdef ILP20
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20);
#endif
#ifdef ILP21
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21);
#endif
#ifdef ILP22
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22);
#endif
#ifdef ILP23
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23);
#endif
#ifdef ILP24
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24);
#endif
#ifdef ILP25
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25);
#endif
#ifdef ILP26
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26);
#endif
#ifdef ILP27
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27);
#endif
#ifdef ILP28
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28);
#endif
#ifdef ILP29
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29);
#endif
#ifdef ILP30
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30);
#endif
#ifdef ILP31
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31);
#endif
#ifdef ILP32
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); 
#endif
#ifdef ILP33
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33);
#endif
#ifdef ILP34
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34);
#endif
#ifdef ILP35
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35);
#endif
#ifdef ILP36
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36);
#endif
#ifdef ILP37
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37);
#endif
#ifdef ILP38
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38);
#endif
#ifdef ILP39
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39);
#endif
#ifdef ILP40
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40);
#endif
#ifdef ILP41
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41);
#endif
#ifdef ILP42
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42);
#endif
#ifdef ILP43
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43);
#endif
#ifdef ILP44
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44);
#endif
#ifdef ILP45
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45);
#endif
#ifdef ILP46
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46);
#endif
#ifdef ILP47
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47);
#endif
#ifdef ILP48
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48);
#endif
#ifdef ILP49
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49);
#endif
#ifdef ILP50
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49); ## x ## (50);
#endif


__device__ void BeforeLU(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel,LRelIndex;
	MYDTYPE JumctionI;
	CurJ = cCompByLevel32[2]-1;
	LRelIndex=cLRelStarts[CurLevel];
	LRelIndex=LRelIndex+cLRelEnds[CurLevel];
	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		
		for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
		//for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx]-1;
			for(i=cSegStartI[JumctionI]-1;i<cSegEndI[JumctionI];i++) {
				MYSECONDFTYPE uHPm1=uHP[i-1];
			
				uHP[i]=uHP[i]-cE[i-1]*(cF[i-1]/uHPm1); // So far same as paper parallel
				uHPm1=uHP[i-1];
				MYSECONDFTYPE bHPm1=bHP[i-1];
				bHP[i]=bHP[i]-bHPm1*cE[i-1]/uHPm1; // bH is y
			}
		}
		if(CurLevel<Depth) {
			for(LRelIndex=cFLRelStarts[CurLevel];LRelIndex<=cFLRelEnds[CurLevel];LRelIndex++){ 
				CurB=cCompByFLevel32[(LRelIndex)*WARPSIZE+PIdx]-1;//RB i inserted another  -1 into the index RB 2 i removed the-1 from the curlevel 
				CurJ=cFathers[CurB]-1;
				MYDTYPE St=cRelStarts[CurB];
				MYDTYPE En=cRelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=cRelVec[j-1]-1;
					MYSECONDFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]-=cE[t-1]*(cF[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYSECONDFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]-=bHPm1*cE[t-1]/uHPm1; 
				}
			}
		}	
	}
}

#ifdef BKSUB1
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* PX, MYSECONDFTYPE* PF,MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE LognDepth)
{
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
#define THISCOMMANDHERE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*(## VARILP -1)); MYDTYPE NextID_ ## VARILP ;
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); MYDTYPE NextID_1 ;MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); MYDTYPE NextID_2 ;MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1)); MYDTYPE NextID_3 ;MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1)); MYDTYPE NextID_4 ;MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1)); MYDTYPE NextID_5 ;MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1)); MYDTYPE NextID_6 ;MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1)); MYDTYPE NextID_7 ;MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1)); MYDTYPE NextID_8 ;MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1)); MYDTYPE NextID_9 ;MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1)); MYDTYPE NextID_10 ;MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1)); MYDTYPE NextID_11 ;MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1)); MYDTYPE NextID_12 ;MYDTYPE PIdx_13 =threadIdx.x+(WARPSIZE*(13 -1)); MYDTYPE NextID_13 ;
	MYDTYPE i;
	
	PX=bHP;
	PF=uHP;
	// PX[PIdx_1]=PX[PIdx_1]/PF[PIdx_1]; // PF[PIdx_1]=-InMat.f[PIdx_1]/PF[PIdx_1];
#define THISCOMMANDHERE2(VARILP) PX[PIdx_ ## VARILP ]=PX[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ]; PF[PIdx_ ## VARILP ]=-cF[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ];
	PX[PIdx_1 ]=PX[PIdx_1 ]/PF[PIdx_1 ]; PF[PIdx_1 ]=-cF[PIdx_1 ]/PF[PIdx_1 ];PX[PIdx_2 ]=PX[PIdx_2 ]/PF[PIdx_2 ]; PF[PIdx_2 ]=-cF[PIdx_2 ]/PF[PIdx_2 ];PX[PIdx_3 ]=PX[PIdx_3 ]/PF[PIdx_3 ]; PF[PIdx_3 ]=-cF[PIdx_3 ]/PF[PIdx_3 ];PX[PIdx_4 ]=PX[PIdx_4 ]/PF[PIdx_4 ]; PF[PIdx_4 ]=-cF[PIdx_4 ]/PF[PIdx_4 ];PX[PIdx_5 ]=PX[PIdx_5 ]/PF[PIdx_5 ]; PF[PIdx_5 ]=-cF[PIdx_5 ]/PF[PIdx_5 ];PX[PIdx_6 ]=PX[PIdx_6 ]/PF[PIdx_6 ]; PF[PIdx_6 ]=-cF[PIdx_6 ]/PF[PIdx_6 ];PX[PIdx_7 ]=PX[PIdx_7 ]/PF[PIdx_7 ]; PF[PIdx_7 ]=-cF[PIdx_7 ]/PF[PIdx_7 ];PX[PIdx_8 ]=PX[PIdx_8 ]/PF[PIdx_8 ]; PF[PIdx_8 ]=-cF[PIdx_8 ]/PF[PIdx_8 ];PX[PIdx_9 ]=PX[PIdx_9 ]/PF[PIdx_9 ]; PF[PIdx_9 ]=-cF[PIdx_9 ]/PF[PIdx_9 ];PX[PIdx_10 ]=PX[PIdx_10 ]/PF[PIdx_10 ]; PF[PIdx_10 ]=-cF[PIdx_10 ]/PF[PIdx_10 ];PX[PIdx_11 ]=PX[PIdx_11 ]/PF[PIdx_11 ]; PF[PIdx_11 ]=-cF[PIdx_11 ]/PF[PIdx_11 ];PX[PIdx_12 ]=PX[PIdx_12 ]/PF[PIdx_12 ]; PF[PIdx_12 ]=-cF[PIdx_12 ]/PF[PIdx_12 ];PX[PIdx_13 ]=PX[PIdx_13 ]/PF[PIdx_13 ]; PF[PIdx_13 ]=-cF[PIdx_13 ]/PF[PIdx_13 ];

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		// NextID_1=cFIdxs[i*InMat.N+PIdx_1]-1; // MYFTYPE OldPXj_1=PX[PIdx_1]; // MYFTYPE OldPXNextID_1=PX[NextID_1]; // PX[PIdx_1]=OldPXj_1+OldPXNextID_1*PF[PIdx_1];
#define THISCOMMANDHERE3(VARILP) NextID_ ## VARILP =cFIdxs[i*InMat.N+PIdx_ ## VARILP ]-1; MYFTYPE OldPXj_ ## VARILP =PX[PIdx_ ## VARILP ]; MYFTYPE OldPXNextID_ ## VARILP =PX[NextID_ ## VARILP ]; PX[PIdx_ ## VARILP ]=OldPXj_ ## VARILP +OldPXNextID_ ## VARILP *PF[PIdx_ ## VARILP ];
	NextID_1 =cFIdxs[i*InMat.N+PIdx_1 ]-1; MYFTYPE OldPXj_1 =PX[PIdx_1 ]; MYFTYPE OldPXNextID_1 =PX[NextID_1 ]; PX[PIdx_1 ]=OldPXj_1 +OldPXNextID_1 *PF[PIdx_1 ];NextID_2 =cFIdxs[i*InMat.N+PIdx_2 ]-1; MYFTYPE OldPXj_2 =PX[PIdx_2 ]; MYFTYPE OldPXNextID_2 =PX[NextID_2 ]; PX[PIdx_2 ]=OldPXj_2 +OldPXNextID_2 *PF[PIdx_2 ];NextID_3 =cFIdxs[i*InMat.N+PIdx_3 ]-1; MYFTYPE OldPXj_3 =PX[PIdx_3 ]; MYFTYPE OldPXNextID_3 =PX[NextID_3 ]; PX[PIdx_3 ]=OldPXj_3 +OldPXNextID_3 *PF[PIdx_3 ];NextID_4 =cFIdxs[i*InMat.N+PIdx_4 ]-1; MYFTYPE OldPXj_4 =PX[PIdx_4 ]; MYFTYPE OldPXNextID_4 =PX[NextID_4 ]; PX[PIdx_4 ]=OldPXj_4 +OldPXNextID_4 *PF[PIdx_4 ];NextID_5 =cFIdxs[i*InMat.N+PIdx_5 ]-1; MYFTYPE OldPXj_5 =PX[PIdx_5 ]; MYFTYPE OldPXNextID_5 =PX[NextID_5 ]; PX[PIdx_5 ]=OldPXj_5 +OldPXNextID_5 *PF[PIdx_5 ];NextID_6 =cFIdxs[i*InMat.N+PIdx_6 ]-1; MYFTYPE OldPXj_6 =PX[PIdx_6 ]; MYFTYPE OldPXNextID_6 =PX[NextID_6 ]; PX[PIdx_6 ]=OldPXj_6 +OldPXNextID_6 *PF[PIdx_6 ];NextID_7 =cFIdxs[i*InMat.N+PIdx_7 ]-1; MYFTYPE OldPXj_7 =PX[PIdx_7 ]; MYFTYPE OldPXNextID_7 =PX[NextID_7 ]; PX[PIdx_7 ]=OldPXj_7 +OldPXNextID_7 *PF[PIdx_7 ];NextID_8 =cFIdxs[i*InMat.N+PIdx_8 ]-1; MYFTYPE OldPXj_8 =PX[PIdx_8 ]; MYFTYPE OldPXNextID_8 =PX[NextID_8 ]; PX[PIdx_8 ]=OldPXj_8 +OldPXNextID_8 *PF[PIdx_8 ];NextID_9 =cFIdxs[i*InMat.N+PIdx_9 ]-1; MYFTYPE OldPXj_9 =PX[PIdx_9 ]; MYFTYPE OldPXNextID_9 =PX[NextID_9 ]; PX[PIdx_9 ]=OldPXj_9 +OldPXNextID_9 *PF[PIdx_9 ];NextID_10 =cFIdxs[i*InMat.N+PIdx_10 ]-1; MYFTYPE OldPXj_10 =PX[PIdx_10 ]; MYFTYPE OldPXNextID_10 =PX[NextID_10 ]; PX[PIdx_10 ]=OldPXj_10 +OldPXNextID_10 *PF[PIdx_10 ];NextID_11 =cFIdxs[i*InMat.N+PIdx_11 ]-1; MYFTYPE OldPXj_11 =PX[PIdx_11 ]; MYFTYPE OldPXNextID_11 =PX[NextID_11 ]; PX[PIdx_11 ]=OldPXj_11 +OldPXNextID_11 *PF[PIdx_11 ];NextID_12 =cFIdxs[i*InMat.N+PIdx_12 ]-1; MYFTYPE OldPXj_12 =PX[PIdx_12 ]; MYFTYPE OldPXNextID_12 =PX[NextID_12 ]; PX[PIdx_12 ]=OldPXj_12 +OldPXNextID_12 *PF[PIdx_12 ];NextID_13 =cFIdxs[i*InMat.N+PIdx_13 ]-1; MYFTYPE OldPXj_13 =PX[PIdx_13 ]; MYFTYPE OldPXNextID_13 =PX[NextID_13 ]; PX[PIdx_13 ]=OldPXj_13 +OldPXNextID_13 *PF[PIdx_13 ];

		// PX[j]=PX[j]+PX[NextID]*PF[j];
		// PF[j]=PF[j]*PF[NextID];
		
		// MYFTYPE OldPFj_1=PF[PIdx_1]; // MYFTYPE OldPFNextID_1=PF[NextID_1]; // PF[PIdx_1]=OldPFj_1*OldPFNextID_1;
#define THISCOMMANDHERE4(VARILP) MYFTYPE OldPFj_ ## VARILP =PF[PIdx_ ## VARILP ]; MYFTYPE OldPFNextID_ ## VARILP =PF[NextID_ ## VARILP ]; PF[PIdx_ ## VARILP ]=OldPFj_ ## VARILP *OldPFNextID_ ## VARILP ;
	MYFTYPE OldPFj_1 =PF[PIdx_1 ]; MYFTYPE OldPFNextID_1 =PF[NextID_1 ]; PF[PIdx_1 ]=OldPFj_1 *OldPFNextID_1 ;MYFTYPE OldPFj_2 =PF[PIdx_2 ]; MYFTYPE OldPFNextID_2 =PF[NextID_2 ]; PF[PIdx_2 ]=OldPFj_2 *OldPFNextID_2 ;MYFTYPE OldPFj_3 =PF[PIdx_3 ]; MYFTYPE OldPFNextID_3 =PF[NextID_3 ]; PF[PIdx_3 ]=OldPFj_3 *OldPFNextID_3 ;MYFTYPE OldPFj_4 =PF[PIdx_4 ]; MYFTYPE OldPFNextID_4 =PF[NextID_4 ]; PF[PIdx_4 ]=OldPFj_4 *OldPFNextID_4 ;MYFTYPE OldPFj_5 =PF[PIdx_5 ]; MYFTYPE OldPFNextID_5 =PF[NextID_5 ]; PF[PIdx_5 ]=OldPFj_5 *OldPFNextID_5 ;MYFTYPE OldPFj_6 =PF[PIdx_6 ]; MYFTYPE OldPFNextID_6 =PF[NextID_6 ]; PF[PIdx_6 ]=OldPFj_6 *OldPFNextID_6 ;MYFTYPE OldPFj_7 =PF[PIdx_7 ]; MYFTYPE OldPFNextID_7 =PF[NextID_7 ]; PF[PIdx_7 ]=OldPFj_7 *OldPFNextID_7 ;MYFTYPE OldPFj_8 =PF[PIdx_8 ]; MYFTYPE OldPFNextID_8 =PF[NextID_8 ]; PF[PIdx_8 ]=OldPFj_8 *OldPFNextID_8 ;MYFTYPE OldPFj_9 =PF[PIdx_9 ]; MYFTYPE OldPFNextID_9 =PF[NextID_9 ]; PF[PIdx_9 ]=OldPFj_9 *OldPFNextID_9 ;MYFTYPE OldPFj_10 =PF[PIdx_10 ]; MYFTYPE OldPFNextID_10 =PF[NextID_10 ]; PF[PIdx_10 ]=OldPFj_10 *OldPFNextID_10 ;MYFTYPE OldPFj_11 =PF[PIdx_11 ]; MYFTYPE OldPFNextID_11 =PF[NextID_11 ]; PF[PIdx_11 ]=OldPFj_11 *OldPFNextID_11 ;MYFTYPE OldPFj_12 =PF[PIdx_12 ]; MYFTYPE OldPFNextID_12 =PF[NextID_12 ]; PF[PIdx_12 ]=OldPFj_12 *OldPFNextID_12 ;MYFTYPE OldPFj_13 =PF[PIdx_13 ]; MYFTYPE OldPFNextID_13 =PF[NextID_13 ]; PF[PIdx_13 ]=OldPFj_13 *OldPFNextID_13 ;
	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP,MYSECONDFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); Out[PIdx_1 ]=0;MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); Out[PIdx_2 ]=0;MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1)); Out[PIdx_3 ]=0;MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1)); Out[PIdx_4 ]=0;MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1)); Out[PIdx_5 ]=0;MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1)); Out[PIdx_6 ]=0;MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1)); Out[PIdx_7 ]=0;MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1)); Out[PIdx_8 ]=0;MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1)); Out[PIdx_9 ]=0;MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1)); Out[PIdx_10 ]=0;MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1)); Out[PIdx_11 ]=0;MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1)); Out[PIdx_12 ]=0;MYDTYPE PIdx_13 =threadIdx.x+(WARPSIZE*(13 -1)); Out[PIdx_13 ]=0;

	MYDTYPE j,CurJ,CurB,t;
	MYDTYPE JumctionI;
	short CurLevel,i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex,k;
	MYFTYPE temp;
 for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
  //     Run all independent set for this level, in parallel
  // for JumctionI=find(Level==CurLevel) % in parallel
  for(LRelIndex=InMat.LRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
   JumctionI=InMat.CompByLevel32[LRelIndex*WARPSIZE+PIdx_1]-1;
   // for i=(SegEndI(JumctionI)):-1:(SegStartI(JumctionI)-1)
   for(i=InMat.SegEndI[JumctionI]-1;i>=(InMat.SegStartI[JumctionI]-2);i--) {
    // k=KsB(i+1);
     k=InMat.KsB[i+1];
	 
    // bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
   Out[i]=(bHP[i]-Out[k]*cF[i])/uHP[i];
   }
  }
 }
 } 

#endif

__device__ MYDTYPE InitializeDeviceMemory(Stim stim,Sim sim, HMat InMat,MYFTYPE* VHotGlobal,MYDTYPE CompDepth,MYDTYPE CompFDepth,HMat *SMemMat, MYFTYPE* Vs,MYFTYPE* amps,MYFTYPE* SMemVHot)
{
	//Configure IDs 

	MYDTYPE StimID=threadIdx.y;
	
#define THISCOMMANDHERE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1));
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); MYDTYPE NextID_1 ;MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); MYDTYPE NextID_2 ;MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1)); MYDTYPE NextID_3 ;MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1)); MYDTYPE NextID_4 ;MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1)); MYDTYPE NextID_5 ;MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1)); MYDTYPE NextID_6 ;MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1)); MYDTYPE NextID_7 ;MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1)); MYDTYPE NextID_8 ;MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1)); MYDTYPE NextID_9 ;MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1)); MYDTYPE NextID_10 ;MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1)); MYDTYPE NextID_11 ;MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1)); MYDTYPE NextID_12 ;MYDTYPE PIdx_13 =threadIdx.x+(WARPSIZE*(13 -1)); MYDTYPE NextID_13 ;
	hipError_t cudaStatus;
	
	//Declaring the shared memory elements
	SMemMat->N=InMat.N;
	SMemMat->NComps=InMat.NComps;
	SMemMat->Depth=InMat.Depth;
	SMemMat->NModels=InMat.NModels;
	SMemMat->LognDepth=InMat.LognDepth;
	SMemMat->nFathers=InMat.nFathers;
	SMemMat->nCallForFather=InMat.nCallForFather;
	SMemMat->nLRel=InMat.nLRel;
	SMemMat->nFLRel=InMat.nFLRel;
	//Offset indicates how far we are into the shared memory
	MYDTYPE offset=0;
	//e indicate the supradiagonal
	//SMemMat->e = (MYSECONDFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//f indicates the subdiagonal.
	//SMemMat->f = (MYSECONDFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);

	//Copy e,f from the device memory to the shared memory. - Move this to constant RBSConstant
	// SMemMat->e[PIdx_3]=InMat.e[PIdx_3]; // SMemMat->f[PIdx_3]=cF[PIdx_3];
//#define THISCOMMANDHERE6(VARILP) SMemMat->e[PIdx_ ## VARILP ]=InMat.e[PIdx_ ## VARILP ]; SMemMat->f[PIdx_ ## VARILP ]=cF[PIdx_ ## VARILP ];
	//SMemMat->e[PIdx_1 ]=InMat.e[PIdx_1 ]; SMemMat->f[PIdx_1 ]=cF[PIdx_1 ];SMemMat->e[PIdx_2 ]=InMat.e[PIdx_2 ]; SMemMat->f[PIdx_2 ]=cF[PIdx_2 ];SMemMat->e[PIdx_3 ]=InMat.e[PIdx_3 ]; SMemMat->f[PIdx_3 ]=cF[PIdx_3 ];SMemMat->e[PIdx_4 ]=InMat.e[PIdx_4 ]; SMemMat->f[PIdx_4 ]=cF[PIdx_4 ];SMemMat->e[PIdx_5 ]=InMat.e[PIdx_5 ]; SMemMat->f[PIdx_5 ]=cF[PIdx_5 ];SMemMat->e[PIdx_6 ]=InMat.e[PIdx_6 ]; SMemMat->f[PIdx_6 ]=cF[PIdx_6 ];SMemMat->e[PIdx_7 ]=InMat.e[PIdx_7 ]; SMemMat->f[PIdx_7 ]=cF[PIdx_7 ];SMemMat->e[PIdx_8 ]=InMat.e[PIdx_8 ]; SMemMat->f[PIdx_8 ]=cF[PIdx_8 ];SMemMat->e[PIdx_9 ]=InMat.e[PIdx_9 ]; SMemMat->f[PIdx_9 ]=cF[PIdx_9 ];SMemMat->e[PIdx_10 ]=InMat.e[PIdx_10 ]; SMemMat->f[PIdx_10 ]=cF[PIdx_10 ];SMemMat->e[PIdx_11 ]=InMat.e[PIdx_11 ]; SMemMat->f[PIdx_11 ]=cF[PIdx_11 ];SMemMat->e[PIdx_12 ]=InMat.e[PIdx_12 ]; SMemMat->f[PIdx_12 ]=cF[PIdx_12 ];SMemMat->e[PIdx_13 ]=InMat.e[PIdx_13 ]; SMemMat->f[PIdx_13 ]=cF[PIdx_13 ];
	//InMat.e=SMemMat->e;
	//cF=SMemMat->f;
	//Copy e,f from the device memory to the shared memory. - Move this to constant RBSConstant
	//SMemMat->Cms = (MYFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYFTYPE);
	
//#define THISCOMMANDHERE7(VARILP) SMemMat->Cms[PIdx_ ## VARILP ]=InMat.Cms[PIdx_ ## VARILP ];
	//SMemMat->Cms[PIdx_1 ]=InMat.Cms[PIdx_1 ];SMemMat->Cms[PIdx_2 ]=InMat.Cms[PIdx_2 ];SMemMat->Cms[PIdx_3 ]=InMat.Cms[PIdx_3 ];SMemMat->Cms[PIdx_4 ]=InMat.Cms[PIdx_4 ];SMemMat->Cms[PIdx_5 ]=InMat.Cms[PIdx_5 ];SMemMat->Cms[PIdx_6 ]=InMat.Cms[PIdx_6 ];SMemMat->Cms[PIdx_7 ]=InMat.Cms[PIdx_7 ];SMemMat->Cms[PIdx_8 ]=InMat.Cms[PIdx_8 ];SMemMat->Cms[PIdx_9 ]=InMat.Cms[PIdx_9 ];SMemMat->Cms[PIdx_10 ]=InMat.Cms[PIdx_10 ];SMemMat->Cms[PIdx_11 ]=InMat.Cms[PIdx_11 ];SMemMat->Cms[PIdx_12 ]=InMat.Cms[PIdx_12 ];SMemMat->Cms[PIdx_13 ]=InMat.Cms[PIdx_13 ];
	// discrete
	//BKSUB is the different parallelism we use
#ifdef BKSUB1
	//Copying FIdxs (Father indexes) from device to shared mem
		//SMemMat->FIdxs = (MYDTYPE*) &smem[offset];
		//offset+=InMat.N*SMemMat->LognDepth*sizeof(MYDTYPE);
		//for(int i=0;i<SMemMat->LognDepth;i++) {
			// SMemMat->FIdxs[i*InMat.N+PIdx_3]=cFIdxs[i*InMat.N+PIdx_3];
//#define THISCOMMANDHERE8(VARILP) SMemMat->FIdxs[i*InMat.N+PIdx_ ## VARILP ]=cFIdxs[i*InMat.N+PIdx_ ## VARILP ];
	//SMemMat->FIdxs[i*InMat.N+PIdx_1 ]=cFIdxs[i*InMat.N+PIdx_1 ];SMemMat->FIdxs[i*InMat.N+PIdx_2 ]=cFIdxs[i*InMat.N+PIdx_2 ];SMemMat->FIdxs[i*InMat.N+PIdx_3 ]=cFIdxs[i*InMat.N+PIdx_3 ];SMemMat->FIdxs[i*InMat.N+PIdx_4 ]=cFIdxs[i*InMat.N+PIdx_4 ];SMemMat->FIdxs[i*InMat.N+PIdx_5 ]=cFIdxs[i*InMat.N+PIdx_5 ];SMemMat->FIdxs[i*InMat.N+PIdx_6 ]=cFIdxs[i*InMat.N+PIdx_6 ];SMemMat->FIdxs[i*InMat.N+PIdx_7 ]=cFIdxs[i*InMat.N+PIdx_7 ];SMemMat->FIdxs[i*InMat.N+PIdx_8 ]=cFIdxs[i*InMat.N+PIdx_8 ];SMemMat->FIdxs[i*InMat.N+PIdx_9 ]=cFIdxs[i*InMat.N+PIdx_9 ];SMemMat->FIdxs[i*InMat.N+PIdx_10 ]=cFIdxs[i*InMat.N+PIdx_10 ];SMemMat->FIdxs[i*InMat.N+PIdx_11 ]=cFIdxs[i*InMat.N+PIdx_11 ];SMemMat->FIdxs[i*InMat.N+PIdx_12 ]=cFIdxs[i*InMat.N+PIdx_12 ];SMemMat->FIdxs[i*InMat.N+PIdx_13 ]=cFIdxs[i*InMat.N+PIdx_13 ];
//}
#endif 
#ifdef BKSUB2
		SMemMat->KsB =  (MYDTYPE*) &smem[offset];
		offset+=(SMemMat->N+1)*sizeof(MYDTYPE);		
		SMemMat->KsB[InMat.N]=InMat.KsB[InMat.N];

		// SMemMat->KsB[PIdx_3]=InMat.KsB[PIdx_3];
#define THISCOMMANDHERE9(VARILP) SMemMat->KsB[PIdx_ ## VARILP ]=InMat.KsB[PIdx_ ## VARILP ];
	SMemMat->KsB[PIdx_1 ]=InMat.KsB[PIdx_1 ];SMemMat->KsB[PIdx_2 ]=InMat.KsB[PIdx_2 ];SMemMat->KsB[PIdx_3 ]=InMat.KsB[PIdx_3 ];SMemMat->KsB[PIdx_4 ]=InMat.KsB[PIdx_4 ];SMemMat->KsB[PIdx_5 ]=InMat.KsB[PIdx_5 ];SMemMat->KsB[PIdx_6 ]=InMat.KsB[PIdx_6 ];SMemMat->KsB[PIdx_7 ]=InMat.KsB[PIdx_7 ];SMemMat->KsB[PIdx_8 ]=InMat.KsB[PIdx_8 ];SMemMat->KsB[PIdx_9 ]=InMat.KsB[PIdx_9 ];SMemMat->KsB[PIdx_10 ]=InMat.KsB[PIdx_10 ];SMemMat->KsB[PIdx_11 ]=InMat.KsB[PIdx_11 ];SMemMat->KsB[PIdx_12 ]=InMat.KsB[PIdx_12 ];SMemMat->KsB[PIdx_13 ]=InMat.KsB[PIdx_13 ];
#endif	
	//Copying Ks (ParentSegment) from device to shared
	//SMemMat->Ks = (MYDTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYDTYPE);
	// SMemMat->Ks[PIdx_3]=InMat.Ks[PIdx_3];
//#define THISCOMMANDHERE10(VARILP) SMemMat->Ks[PIdx_ ## VARILP ]=InMat.Ks[PIdx_ ## VARILP ];
//	SMemMat->Ks[PIdx_1 ]=InMat.Ks[PIdx_1 ];SMemMat->Ks[PIdx_2 ]=InMat.Ks[PIdx_2 ];SMemMat->Ks[PIdx_3 ]=InMat.Ks[PIdx_3 ];SMemMat->Ks[PIdx_4 ]=InMat.Ks[PIdx_4 ];SMemMat->Ks[PIdx_5 ]=InMat.Ks[PIdx_5 ];SMemMat->Ks[PIdx_6 ]=InMat.Ks[PIdx_6 ];SMemMat->Ks[PIdx_7 ]=InMat.Ks[PIdx_7 ];SMemMat->Ks[PIdx_8 ]=InMat.Ks[PIdx_8 ];SMemMat->Ks[PIdx_9 ]=InMat.Ks[PIdx_9 ];SMemMat->Ks[PIdx_10 ]=InMat.Ks[PIdx_10 ];SMemMat->Ks[PIdx_11 ]=InMat.Ks[PIdx_11 ];SMemMat->Ks[PIdx_12 ]=InMat.Ks[PIdx_12 ];SMemMat->Ks[PIdx_13 ]=InMat.Ks[PIdx_13 ];
	
	//Copying SegToComp from device to shared
	//SMemMat->SegToComp = (MYDTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYDTYPE);
//#define THISCOMMANDHERE11(VARILP) SMemMat->SegToComp[PIdx_ ## VARILP ]=InMat.SegToComp[PIdx_ ## VARILP ];
	//SMemMat->SegToComp[PIdx_1 ]=InMat.SegToComp[PIdx_1 ];SMemMat->SegToComp[PIdx_2 ]=InMat.SegToComp[PIdx_2 ];SMemMat->SegToComp[PIdx_3 ]=InMat.SegToComp[PIdx_3 ];SMemMat->SegToComp[PIdx_4 ]=InMat.SegToComp[PIdx_4 ];SMemMat->SegToComp[PIdx_5 ]=InMat.SegToComp[PIdx_5 ];SMemMat->SegToComp[PIdx_6 ]=InMat.SegToComp[PIdx_6 ];SMemMat->SegToComp[PIdx_7 ]=InMat.SegToComp[PIdx_7 ];SMemMat->SegToComp[PIdx_8 ]=InMat.SegToComp[PIdx_8 ];SMemMat->SegToComp[PIdx_9 ]=InMat.SegToComp[PIdx_9 ];SMemMat->SegToComp[PIdx_10 ]=InMat.SegToComp[PIdx_10 ];SMemMat->SegToComp[PIdx_11 ]=InMat.SegToComp[PIdx_11 ];SMemMat->SegToComp[PIdx_12 ]=InMat.SegToComp[PIdx_12 ];SMemMat->SegToComp[PIdx_13 ]=InMat.SegToComp[PIdx_13 ];
	//SMemMat->boolModel= (MYDTYPE*) &smem[offset];
	//offset+=InMat.N*SMemMat->NModels*sizeof(MYDTYPE);
	// SMemMat->boolModel[PIdx_3]=InMat.boolModel[PIdx_3];
//#define THISCOMMANDHERE12(VARILP) for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_ ## VARILP+ii*SMemMat->N]=InMat.boolModel[PIdx_ ## VARILP+ii*SMemMat->N];}
	//for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_1+ii*SMemMat->N]=InMat.boolModel[PIdx_1+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_2+ii*SMemMat->N]=InMat.boolModel[PIdx_2+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_3+ii*SMemMat->N]=InMat.boolModel[PIdx_3+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_4+ii*SMemMat->N]=InMat.boolModel[PIdx_4+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_5+ii*SMemMat->N]=InMat.boolModel[PIdx_5+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_6+ii*SMemMat->N]=InMat.boolModel[PIdx_6+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_7+ii*SMemMat->N]=InMat.boolModel[PIdx_7+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_8+ii*SMemMat->N]=InMat.boolModel[PIdx_8+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_9+ii*SMemMat->N]=InMat.boolModel[PIdx_9+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_10+ii*SMemMat->N]=InMat.boolModel[PIdx_10+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_11+ii*SMemMat->N]=InMat.boolModel[PIdx_11+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_12+ii*SMemMat->N]=InMat.boolModel[PIdx_12+ii*SMemMat->N];}for(int ii=0;ii<SMemMat->NModels;ii++) {SMemMat->boolModel[PIdx_13+ii*SMemMat->N]=InMat.boolModel[PIdx_13+ii*SMemMat->N];}
	
	//SMemMat->RelStarts = (MYDTYPE*) &smem[offset];
	//offset+=InMat.nFathers*sizeof(MYDTYPE);
	//SMemMat->RelEnds = (MYDTYPE*) &smem[offset];
	//offset+=InMat.nFathers*sizeof(MYDTYPE);
	//SMemMat->RelVec = (MYDTYPE*) &smem[offset];
	//offset+=InMat.nCallForFather*sizeof(MYDTYPE);
	//SMemMat->SegStartI = (MYDTYPE*) &smem[offset];
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);
	//SMemMat->SegEndI = (MYDTYPE*) &smem[offset];
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE);

	//SMemMat->Fathers= (MYDTYPE*) &smem[offset];
	//offset+=InMat.nFathers*sizeof(MYDTYPE);
	// 3
	//SMemMat->CompByLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	//offset+=WARPSIZE*(CompDepth)*sizeof(MYDTYPE);
	//SMemMat->CompByFLevel32 = (MYDTYPE*) &smem[offset]; // WARPSIZE*(Depth+1)
	//offset+=WARPSIZE*(CompFDepth)*sizeof(MYDTYPE);
	//SMemMat->LRelStarts = (MYDTYPE*) &smem[offset];
	//offset+=SMemMat->nLRel*sizeof(MYDTYPE);
	//SMemMat->LRelEnds = (MYDTYPE*) &smem[offset];
	//offset+=SMemMat->nLRel*sizeof(MYDTYPE);
	//SMemMat->FLRelStarts = (MYDTYPE*) &smem[offset];
	//offset+=SMemMat->nFLRel*sizeof(MYDTYPE);
	//SMemMat->FLRelEnds = (MYDTYPE*) &smem[offset];
	//offset+=SMemMat->nFLRel*sizeof(MYDTYPE);
	// Moving back to MYFTYPEs - make sure that the address is good for MYFTYPE (is modulu 4).
	offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	MYDTYPE ConstantMemSize=offset;
	//if(PIdx_1<InMat.nFathers) { SMemMat->RelStarts[PIdx_1]=InMat.RelStarts[PIdx_1];SMemMat->RelEnds[PIdx_1]=InMat.RelEnds[PIdx_1];SMemMat->Fathers[PIdx_1]=InMat.Fathers[PIdx_1];}
	//#define THISCOMMANDHERE13(VARILP) if(PIdx_ ## VARILP<InMat.nFathers){ SMemMat->RelStarts[PIdx_ ## VARILP]=InMat.RelStarts[PIdx_ ## VARILP];SMemMat->RelEnds[PIdx_ ## VARILP]=InMat.RelEnds[PIdx_ ## VARILP];SMemMat->Fathers[PIdx_ ## VARILP]=InMat.Fathers[PIdx_ ## VARILP];}
	//if(PIdx_1<InMat.nFathers){ SMemMat->RelStarts[PIdx_1]=InMat.RelStarts[PIdx_1];SMemMat->RelEnds[PIdx_1]=InMat.RelEnds[PIdx_1];SMemMat->Fathers[PIdx_1]=InMat.Fathers[PIdx_1];}if(PIdx_2<InMat.nFathers){ SMemMat->RelStarts[PIdx_2]=InMat.RelStarts[PIdx_2];SMemMat->RelEnds[PIdx_2]=InMat.RelEnds[PIdx_2];SMemMat->Fathers[PIdx_2]=InMat.Fathers[PIdx_2];}if(PIdx_3<InMat.nFathers){ SMemMat->RelStarts[PIdx_3]=InMat.RelStarts[PIdx_3];SMemMat->RelEnds[PIdx_3]=InMat.RelEnds[PIdx_3];SMemMat->Fathers[PIdx_3]=InMat.Fathers[PIdx_3];}if(PIdx_4<InMat.nFathers){ SMemMat->RelStarts[PIdx_4]=InMat.RelStarts[PIdx_4];SMemMat->RelEnds[PIdx_4]=InMat.RelEnds[PIdx_4];SMemMat->Fathers[PIdx_4]=InMat.Fathers[PIdx_4];}if(PIdx_5<InMat.nFathers){ SMemMat->RelStarts[PIdx_5]=InMat.RelStarts[PIdx_5];SMemMat->RelEnds[PIdx_5]=InMat.RelEnds[PIdx_5];SMemMat->Fathers[PIdx_5]=InMat.Fathers[PIdx_5];}if(PIdx_6<InMat.nFathers){ SMemMat->RelStarts[PIdx_6]=InMat.RelStarts[PIdx_6];SMemMat->RelEnds[PIdx_6]=InMat.RelEnds[PIdx_6];SMemMat->Fathers[PIdx_6]=InMat.Fathers[PIdx_6];}if(PIdx_7<InMat.nFathers){ SMemMat->RelStarts[PIdx_7]=InMat.RelStarts[PIdx_7];SMemMat->RelEnds[PIdx_7]=InMat.RelEnds[PIdx_7];SMemMat->Fathers[PIdx_7]=InMat.Fathers[PIdx_7];}if(PIdx_8<InMat.nFathers){ SMemMat->RelStarts[PIdx_8]=InMat.RelStarts[PIdx_8];SMemMat->RelEnds[PIdx_8]=InMat.RelEnds[PIdx_8];SMemMat->Fathers[PIdx_8]=InMat.Fathers[PIdx_8];}if(PIdx_9<InMat.nFathers){ SMemMat->RelStarts[PIdx_9]=InMat.RelStarts[PIdx_9];SMemMat->RelEnds[PIdx_9]=InMat.RelEnds[PIdx_9];SMemMat->Fathers[PIdx_9]=InMat.Fathers[PIdx_9];}if(PIdx_10<InMat.nFathers){ SMemMat->RelStarts[PIdx_10]=InMat.RelStarts[PIdx_10];SMemMat->RelEnds[PIdx_10]=InMat.RelEnds[PIdx_10];SMemMat->Fathers[PIdx_10]=InMat.Fathers[PIdx_10];}if(PIdx_11<InMat.nFathers){ SMemMat->RelStarts[PIdx_11]=InMat.RelStarts[PIdx_11];SMemMat->RelEnds[PIdx_11]=InMat.RelEnds[PIdx_11];SMemMat->Fathers[PIdx_11]=InMat.Fathers[PIdx_11];}if(PIdx_12<InMat.nFathers){ SMemMat->RelStarts[PIdx_12]=InMat.RelStarts[PIdx_12];SMemMat->RelEnds[PIdx_12]=InMat.RelEnds[PIdx_12];SMemMat->Fathers[PIdx_12]=InMat.Fathers[PIdx_12];}if(PIdx_13<InMat.nFathers){ SMemMat->RelStarts[PIdx_13]=InMat.RelStarts[PIdx_13];SMemMat->RelEnds[PIdx_13]=InMat.RelEnds[PIdx_13];SMemMat->Fathers[PIdx_13]=InMat.Fathers[PIdx_13];}
	//if(PIdx_1<InMat.nCallForFather) {SMemMat->RelVec[PIdx_1]=InMat.RelVec[PIdx_1];}
	//#define THISCOMMANDHERE14(VARILP) if(PIdx_ ## VARILP<InMat.nCallForFather){SMemMat->RelVec[PIdx_ ## VARILP]=InMat.RelVec[PIdx_ ## VARILP];}
	//if(PIdx_1<InMat.nCallForFather){SMemMat->RelVec[PIdx_1]=InMat.RelVec[PIdx_1];}if(PIdx_2<InMat.nCallForFather){SMemMat->RelVec[PIdx_2]=InMat.RelVec[PIdx_2];}if(PIdx_3<InMat.nCallForFather){SMemMat->RelVec[PIdx_3]=InMat.RelVec[PIdx_3];}if(PIdx_4<InMat.nCallForFather){SMemMat->RelVec[PIdx_4]=InMat.RelVec[PIdx_4];}if(PIdx_5<InMat.nCallForFather){SMemMat->RelVec[PIdx_5]=InMat.RelVec[PIdx_5];}if(PIdx_6<InMat.nCallForFather){SMemMat->RelVec[PIdx_6]=InMat.RelVec[PIdx_6];}if(PIdx_7<InMat.nCallForFather){SMemMat->RelVec[PIdx_7]=InMat.RelVec[PIdx_7];}if(PIdx_8<InMat.nCallForFather){SMemMat->RelVec[PIdx_8]=InMat.RelVec[PIdx_8];}if(PIdx_9<InMat.nCallForFather){SMemMat->RelVec[PIdx_9]=InMat.RelVec[PIdx_9];}if(PIdx_10<InMat.nCallForFather){SMemMat->RelVec[PIdx_10]=InMat.RelVec[PIdx_10];}if(PIdx_11<InMat.nCallForFather){SMemMat->RelVec[PIdx_11]=InMat.RelVec[PIdx_11];}if(PIdx_12<InMat.nCallForFather){SMemMat->RelVec[PIdx_12]=InMat.RelVec[PIdx_12];}if(PIdx_13<InMat.nCallForFather){SMemMat->RelVec[PIdx_13]=InMat.RelVec[PIdx_13];}
	//if(PIdx_1<InMat.nCallForFather+1) {SMemMat->SegStartI[PIdx_1]=InMat.SegStartI[PIdx_1];SMemMat->SegEndI[PIdx_1]=InMat.SegEndI[PIdx_1];}
	//#define THISCOMMANDHERE15(VARILP) if(PIdx_ ## VARILP<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_ ## VARILP]=InMat.SegStartI[PIdx_ ## VARILP];SMemMat->SegEndI[PIdx_ ## VARILP]=InMat.SegEndI[PIdx_ ## VARILP];}
	//if(PIdx_1<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_1]=InMat.SegStartI[PIdx_1];SMemMat->SegEndI[PIdx_1]=InMat.SegEndI[PIdx_1];}if(PIdx_2<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_2]=InMat.SegStartI[PIdx_2];SMemMat->SegEndI[PIdx_2]=InMat.SegEndI[PIdx_2];}if(PIdx_3<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_3]=InMat.SegStartI[PIdx_3];SMemMat->SegEndI[PIdx_3]=InMat.SegEndI[PIdx_3];}if(PIdx_4<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_4]=InMat.SegStartI[PIdx_4];SMemMat->SegEndI[PIdx_4]=InMat.SegEndI[PIdx_4];}if(PIdx_5<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_5]=InMat.SegStartI[PIdx_5];SMemMat->SegEndI[PIdx_5]=InMat.SegEndI[PIdx_5];}if(PIdx_6<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_6]=InMat.SegStartI[PIdx_6];SMemMat->SegEndI[PIdx_6]=InMat.SegEndI[PIdx_6];}if(PIdx_7<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_7]=InMat.SegStartI[PIdx_7];SMemMat->SegEndI[PIdx_7]=InMat.SegEndI[PIdx_7];}if(PIdx_8<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_8]=InMat.SegStartI[PIdx_8];SMemMat->SegEndI[PIdx_8]=InMat.SegEndI[PIdx_8];}if(PIdx_9<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_9]=InMat.SegStartI[PIdx_9];SMemMat->SegEndI[PIdx_9]=InMat.SegEndI[PIdx_9];}if(PIdx_10<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_10]=InMat.SegStartI[PIdx_10];SMemMat->SegEndI[PIdx_10]=InMat.SegEndI[PIdx_10];}if(PIdx_11<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_11]=InMat.SegStartI[PIdx_11];SMemMat->SegEndI[PIdx_11]=InMat.SegEndI[PIdx_11];}if(PIdx_12<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_12]=InMat.SegStartI[PIdx_12];SMemMat->SegEndI[PIdx_12]=InMat.SegEndI[PIdx_12];}if(PIdx_13<InMat.nCallForFather+1){SMemMat->SegStartI[PIdx_13]=InMat.SegStartI[PIdx_13];SMemMat->SegEndI[PIdx_13]=InMat.SegEndI[PIdx_13];}
	//__syncthreads();
	//for(int i=0;i<CompDepth;i++) {SMemMat->CompByLevel32[WARPSIZE*i+PIdx_1]=InMat.CompByLevel32[WARPSIZE*i+PIdx_1];}
	//#define THISCOMMANDHERE123(VARILP) for(int i=0;i<=SMemMat->Depth;i++) {SMemMat->CompByLevel32[WARPSIZE*i+PIdx_ ## VARILP]=InMat.CompByLevel32[WARPSIZE*i+PIdx_ ## VARILP];}
	//for(int i=0;i<=SMemMat->Depth;i++) {SMemMat->CompByLevel32[WARPSIZE*i+PIdx_1]=InMat.CompByLevel32[WARPSIZE*i+PIdx_1];}for(int i=0;i<=SMemMat->Depth;i++) {SMemMat->CompByLevel32[WARPSIZE*i+PIdx_2]=InMat.CompByLevel32[WARPSIZE*i+PIdx_2];}for(int i=0;i<=SMemMat->Depth;i++) {SMemMat->CompByLevel32[WARPSIZE*i+PIdx_3]=InMat.CompByLevel32[WARPSIZE*i+PIdx_3];}
	//for(int i=0;i<CompFDepth;i++) {SMemMat->CompByFLevel32[WARPSIZE*i+PIdx_1]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_1];}
	//#define THISCOMMANDHERE124(VARILP) for(int i=0;i<SMemMat->Depth;i++) {SMemMat->CompByFLevel32[WARPSIZE*i+PIdx_ ## VARILP]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_ ## VARILP];}
	//for(int i=0;i<SMemMat->Depth;i++) {SMemMat->CompByFLevel32[WARPSIZE*i+PIdx_1]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_1];}for(int i=0;i<SMemMat->Depth;i++) {SMemMat->CompByFLevel32[WARPSIZE*i+PIdx_2]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_2];}for(int i=0;i<SMemMat->Depth;i++) {SMemMat->CompByFLevel32[WARPSIZE*i+PIdx_3]=InMat.CompByFLevel32[WARPSIZE*i+PIdx_3];}
	//if(PIdx_1<InMat.nLRel) {SMemMat->LRelStarts[PIdx_1]=InMat.LRelStarts[PIdx_1];SMemMat->LRelEnds[PIdx_1]=InMat.LRelEnds[PIdx_1];}
	//#define THISCOMMANDHERE16(VARILP) if(PIdx_ ## VARILP<InMat.nLRel){SMemMat->LRelStarts[PIdx_ ## VARILP]=InMat.LRelStarts[PIdx_ ## VARILP];SMemMat->LRelEnds[PIdx_ ## VARILP]=InMat.LRelEnds[PIdx_ ## VARILP];}
	//if(PIdx_1<InMat.nLRel){SMemMat->LRelStarts[PIdx_1]=InMat.LRelStarts[PIdx_1];SMemMat->LRelEnds[PIdx_1]=InMat.LRelEnds[PIdx_1];}if(PIdx_2<InMat.nLRel){SMemMat->LRelStarts[PIdx_2]=InMat.LRelStarts[PIdx_2];SMemMat->LRelEnds[PIdx_2]=InMat.LRelEnds[PIdx_2];}if(PIdx_3<InMat.nLRel){SMemMat->LRelStarts[PIdx_3]=InMat.LRelStarts[PIdx_3];SMemMat->LRelEnds[PIdx_3]=InMat.LRelEnds[PIdx_3];}if(PIdx_4<InMat.nLRel){SMemMat->LRelStarts[PIdx_4]=InMat.LRelStarts[PIdx_4];SMemMat->LRelEnds[PIdx_4]=InMat.LRelEnds[PIdx_4];}if(PIdx_5<InMat.nLRel){SMemMat->LRelStarts[PIdx_5]=InMat.LRelStarts[PIdx_5];SMemMat->LRelEnds[PIdx_5]=InMat.LRelEnds[PIdx_5];}if(PIdx_6<InMat.nLRel){SMemMat->LRelStarts[PIdx_6]=InMat.LRelStarts[PIdx_6];SMemMat->LRelEnds[PIdx_6]=InMat.LRelEnds[PIdx_6];}if(PIdx_7<InMat.nLRel){SMemMat->LRelStarts[PIdx_7]=InMat.LRelStarts[PIdx_7];SMemMat->LRelEnds[PIdx_7]=InMat.LRelEnds[PIdx_7];}if(PIdx_8<InMat.nLRel){SMemMat->LRelStarts[PIdx_8]=InMat.LRelStarts[PIdx_8];SMemMat->LRelEnds[PIdx_8]=InMat.LRelEnds[PIdx_8];}if(PIdx_9<InMat.nLRel){SMemMat->LRelStarts[PIdx_9]=InMat.LRelStarts[PIdx_9];SMemMat->LRelEnds[PIdx_9]=InMat.LRelEnds[PIdx_9];}if(PIdx_10<InMat.nLRel){SMemMat->LRelStarts[PIdx_10]=InMat.LRelStarts[PIdx_10];SMemMat->LRelEnds[PIdx_10]=InMat.LRelEnds[PIdx_10];}if(PIdx_11<InMat.nLRel){SMemMat->LRelStarts[PIdx_11]=InMat.LRelStarts[PIdx_11];SMemMat->LRelEnds[PIdx_11]=InMat.LRelEnds[PIdx_11];}if(PIdx_12<InMat.nLRel){SMemMat->LRelStarts[PIdx_12]=InMat.LRelStarts[PIdx_12];SMemMat->LRelEnds[PIdx_12]=InMat.LRelEnds[PIdx_12];}if(PIdx_13<InMat.nLRel){SMemMat->LRelStarts[PIdx_13]=InMat.LRelStarts[PIdx_13];SMemMat->LRelEnds[PIdx_13]=InMat.LRelEnds[PIdx_13];}
	//__syncthreads();
	//if(PIdx_1<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_1]=InMat.FLRelStarts[PIdx_1];SMemMat->FLRelEnds[PIdx_1]=InMat.FLRelEnds[PIdx_1];}
	//#define THISCOMMANDHERE17(VARILP) if(PIdx_ ## VARILP<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_ ## VARILP]=InMat.FLRelStarts[PIdx_ ## VARILP];SMemMat->FLRelEnds[PIdx_ ## VARILP]=InMat.FLRelEnds[PIdx_ ## VARILP];}
	//if(PIdx_1<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_1]=InMat.FLRelStarts[PIdx_1];SMemMat->FLRelEnds[PIdx_1]=InMat.FLRelEnds[PIdx_1];}if(PIdx_2<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_2]=InMat.FLRelStarts[PIdx_2];SMemMat->FLRelEnds[PIdx_2]=InMat.FLRelEnds[PIdx_2];}if(PIdx_3<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_3]=InMat.FLRelStarts[PIdx_3];SMemMat->FLRelEnds[PIdx_3]=InMat.FLRelEnds[PIdx_3];}if(PIdx_4<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_4]=InMat.FLRelStarts[PIdx_4];SMemMat->FLRelEnds[PIdx_4]=InMat.FLRelEnds[PIdx_4];}if(PIdx_5<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_5]=InMat.FLRelStarts[PIdx_5];SMemMat->FLRelEnds[PIdx_5]=InMat.FLRelEnds[PIdx_5];}if(PIdx_6<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_6]=InMat.FLRelStarts[PIdx_6];SMemMat->FLRelEnds[PIdx_6]=InMat.FLRelEnds[PIdx_6];}if(PIdx_7<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_7]=InMat.FLRelStarts[PIdx_7];SMemMat->FLRelEnds[PIdx_7]=InMat.FLRelEnds[PIdx_7];}if(PIdx_8<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_8]=InMat.FLRelStarts[PIdx_8];SMemMat->FLRelEnds[PIdx_8]=InMat.FLRelEnds[PIdx_8];}if(PIdx_9<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_9]=InMat.FLRelStarts[PIdx_9];SMemMat->FLRelEnds[PIdx_9]=InMat.FLRelEnds[PIdx_9];}if(PIdx_10<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_10]=InMat.FLRelStarts[PIdx_10];SMemMat->FLRelEnds[PIdx_10]=InMat.FLRelEnds[PIdx_10];}if(PIdx_11<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_11]=InMat.FLRelStarts[PIdx_11];SMemMat->FLRelEnds[PIdx_11]=InMat.FLRelEnds[PIdx_11];}if(PIdx_12<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_12]=InMat.FLRelStarts[PIdx_12];SMemMat->FLRelEnds[PIdx_12]=InMat.FLRelEnds[PIdx_12];}if(PIdx_13<InMat.nFLRel) {SMemMat->FLRelStarts[PIdx_13]=InMat.FLRelStarts[PIdx_13];SMemMat->FLRelEnds[PIdx_13]=InMat.FLRelEnds[PIdx_13];}

	//RBS This works now where all models are in the same comps we need to genreralize it.
	//RBS  This code should be generated with calling all models init and defining all states.
	return offset;
}	
__device__ void runSimulation(HMat SMemMat,HMat InMat,MYFTYPE* V,Stim stim,Sim sim,MYFTYPE* VHotGlobal,MYFTYPE* SMemVHot,MYFTYPE* amps,MYDTYPE offset,hipTextureObject_t tex)
{
	MYFTYPE *Vs;
	MYSECONDFTYPE *uHP,*bHP;
		// For solving the matrix
#ifdef BKSUB2
    MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(InMat.N+2*WARPSIZE)*sizeof(MYFTYPE);
#endif
#ifdef BKSUB1
   MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(WARPSIZE + sim.NRecSites*WARPSIZE)*sizeof(MYFTYPE);
#endif
	//MYDTYPE PerStimulus;
	//PerStimulus = InMat.N+2;
	offset+=PerStimulus*threadIdx.y;
	uHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);
	bHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);
	offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	//Vs = (MYFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYFTYPE);
	// Vs[PIdx_3]=V[PIdx_3];
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=WARPSIZE*sim.NRecSites*sizeof(MYFTYPE);
    amps = (MYFTYPE*) &smem[offset];
    offset+=WARPSIZE*sizeof(MYFTYPE);
	MYDTYPE NeuronID = blockIdx.x;
	int Nt=stim.Nt;
	MYFTYPE t=0;
	MYSECONDFTYPE *PX,*PF;
	PX=bHP;
	PF=uHP;
	#define THISCOMMANDHERE18(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1));
	MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1));MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1));MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1));MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1));MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1));MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1));MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1));MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1));MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1));MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1));MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1));MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1));MYDTYPE PIdx_13 =threadIdx.x+(WARPSIZE*(13 -1));

	 #define THISCOMMANDHERE19(VARILP) MYFTYPE isModel_ ## VARILP , dVec_ ## VARILP , Vmid_ ## VARILP ; MYFTYPE ModelStates_ ## VARILP [NSTATES]; MYFTYPE v_ ## VARILP =V[PIdx_ ## VARILP ]; MYFTYPE dv_ ## VARILP;
	MYFTYPE isModel_1 , dVec_1 , Vmid_1 ; MYFTYPE ModelStates_1 [NSTATES]; MYFTYPE v_1 =V[PIdx_1 ]; MYFTYPE dv_1;MYFTYPE isModel_2 , dVec_2 , Vmid_2 ; MYFTYPE ModelStates_2 [NSTATES]; MYFTYPE v_2 =V[PIdx_2 ]; MYFTYPE dv_2;MYFTYPE isModel_3 , dVec_3 , Vmid_3 ; MYFTYPE ModelStates_3 [NSTATES]; MYFTYPE v_3 =V[PIdx_3 ]; MYFTYPE dv_3;MYFTYPE isModel_4 , dVec_4 , Vmid_4 ; MYFTYPE ModelStates_4 [NSTATES]; MYFTYPE v_4 =V[PIdx_4 ]; MYFTYPE dv_4;MYFTYPE isModel_5 , dVec_5 , Vmid_5 ; MYFTYPE ModelStates_5 [NSTATES]; MYFTYPE v_5 =V[PIdx_5 ]; MYFTYPE dv_5;MYFTYPE isModel_6 , dVec_6 , Vmid_6 ; MYFTYPE ModelStates_6 [NSTATES]; MYFTYPE v_6 =V[PIdx_6 ]; MYFTYPE dv_6;MYFTYPE isModel_7 , dVec_7 , Vmid_7 ; MYFTYPE ModelStates_7 [NSTATES]; MYFTYPE v_7 =V[PIdx_7 ]; MYFTYPE dv_7;MYFTYPE isModel_8 , dVec_8 , Vmid_8 ; MYFTYPE ModelStates_8 [NSTATES]; MYFTYPE v_8 =V[PIdx_8 ]; MYFTYPE dv_8;MYFTYPE isModel_9 , dVec_9 , Vmid_9 ; MYFTYPE ModelStates_9 [NSTATES]; MYFTYPE v_9 =V[PIdx_9 ]; MYFTYPE dv_9;MYFTYPE isModel_10 , dVec_10 , Vmid_10 ; MYFTYPE ModelStates_10 [NSTATES]; MYFTYPE v_10 =V[PIdx_10 ]; MYFTYPE dv_10;MYFTYPE isModel_11 , dVec_11 , Vmid_11 ; MYFTYPE ModelStates_11 [NSTATES]; MYFTYPE v_11 =V[PIdx_11 ]; MYFTYPE dv_11;MYFTYPE isModel_12 , dVec_12 , Vmid_12 ; MYFTYPE ModelStates_12 [NSTATES]; MYFTYPE v_12 =V[PIdx_12 ]; MYFTYPE dv_12;MYFTYPE isModel_13 , dVec_13 , Vmid_13 ; MYFTYPE ModelStates_13 [NSTATES]; MYFTYPE v_13 =V[PIdx_13 ]; MYFTYPE dv_13;

	 #define THISCOMMANDHERE20(VARILP) MYSECONDFTYPE sumCurrents_ ## VARILP=0 ,sumCurrentsDv_ ## VARILP=0; MYFTYPE  sumConductivity_ ## VARILP=0 ,sumConductivityDv_ ## VARILP=0 ;
	MYSECONDFTYPE sumCurrents_1=0 ,sumCurrentsDv_1=0; MYFTYPE  sumConductivity_1=0 ,sumConductivityDv_1=0 ;MYSECONDFTYPE sumCurrents_2=0 ,sumCurrentsDv_2=0; MYFTYPE  sumConductivity_2=0 ,sumConductivityDv_2=0 ;MYSECONDFTYPE sumCurrents_3=0 ,sumCurrentsDv_3=0; MYFTYPE  sumConductivity_3=0 ,sumConductivityDv_3=0 ;MYSECONDFTYPE sumCurrents_4=0 ,sumCurrentsDv_4=0; MYFTYPE  sumConductivity_4=0 ,sumConductivityDv_4=0 ;MYSECONDFTYPE sumCurrents_5=0 ,sumCurrentsDv_5=0; MYFTYPE  sumConductivity_5=0 ,sumConductivityDv_5=0 ;MYSECONDFTYPE sumCurrents_6=0 ,sumCurrentsDv_6=0; MYFTYPE  sumConductivity_6=0 ,sumConductivityDv_6=0 ;MYSECONDFTYPE sumCurrents_7=0 ,sumCurrentsDv_7=0; MYFTYPE  sumConductivity_7=0 ,sumConductivityDv_7=0 ;MYSECONDFTYPE sumCurrents_8=0 ,sumCurrentsDv_8=0; MYFTYPE  sumConductivity_8=0 ,sumConductivityDv_8=0 ;MYSECONDFTYPE sumCurrents_9=0 ,sumCurrentsDv_9=0; MYFTYPE  sumConductivity_9=0 ,sumConductivityDv_9=0 ;MYSECONDFTYPE sumCurrents_10=0 ,sumCurrentsDv_10=0; MYFTYPE  sumConductivity_10=0 ,sumConductivityDv_10=0 ;MYSECONDFTYPE sumCurrents_11=0 ,sumCurrentsDv_11=0; MYFTYPE  sumConductivity_11=0 ,sumConductivityDv_11=0 ;MYSECONDFTYPE sumCurrents_12=0 ,sumCurrentsDv_12=0; MYFTYPE  sumConductivity_12=0 ,sumConductivityDv_12=0 ;MYSECONDFTYPE sumCurrents_13=0 ,sumCurrentsDv_13=0; MYFTYPE  sumConductivity_13=0 ,sumConductivityDv_13=0 ;
	// end for model in init
	ModelStates_1 [0]=0;ModelStates_1 [1]=0;ModelStates_1 [2]=0;ModelStates_1 [3]=0;ModelStates_1 [4]=0;ModelStates_1 [5]=0;ModelStates_1 [6]=0;ModelStates_1 [7]=0;ModelStates_1 [8]=0;ModelStates_1 [9]=0;;ModelStates_2 [0]=0;ModelStates_2 [1]=0;ModelStates_2 [2]=0;ModelStates_2 [3]=0;ModelStates_2 [4]=0;ModelStates_2 [5]=0;ModelStates_2 [6]=0;ModelStates_2 [7]=0;ModelStates_2 [8]=0;ModelStates_2 [9]=0;;ModelStates_3 [0]=0;ModelStates_3 [1]=0;ModelStates_3 [2]=0;ModelStates_3 [3]=0;ModelStates_3 [4]=0;ModelStates_3 [5]=0;ModelStates_3 [6]=0;ModelStates_3 [7]=0;ModelStates_3 [8]=0;ModelStates_3 [9]=0;;ModelStates_4 [0]=0;ModelStates_4 [1]=0;ModelStates_4 [2]=0;ModelStates_4 [3]=0;ModelStates_4 [4]=0;ModelStates_4 [5]=0;ModelStates_4 [6]=0;ModelStates_4 [7]=0;ModelStates_4 [8]=0;ModelStates_4 [9]=0;;ModelStates_5 [0]=0;ModelStates_5 [1]=0;ModelStates_5 [2]=0;ModelStates_5 [3]=0;ModelStates_5 [4]=0;ModelStates_5 [5]=0;ModelStates_5 [6]=0;ModelStates_5 [7]=0;ModelStates_5 [8]=0;ModelStates_5 [9]=0;;ModelStates_6 [0]=0;ModelStates_6 [1]=0;ModelStates_6 [2]=0;ModelStates_6 [3]=0;ModelStates_6 [4]=0;ModelStates_6 [5]=0;ModelStates_6 [6]=0;ModelStates_6 [7]=0;ModelStates_6 [8]=0;ModelStates_6 [9]=0;;ModelStates_7 [0]=0;ModelStates_7 [1]=0;ModelStates_7 [2]=0;ModelStates_7 [3]=0;ModelStates_7 [4]=0;ModelStates_7 [5]=0;ModelStates_7 [6]=0;ModelStates_7 [7]=0;ModelStates_7 [8]=0;ModelStates_7 [9]=0;;ModelStates_8 [0]=0;ModelStates_8 [1]=0;ModelStates_8 [2]=0;ModelStates_8 [3]=0;ModelStates_8 [4]=0;ModelStates_8 [5]=0;ModelStates_8 [6]=0;ModelStates_8 [7]=0;ModelStates_8 [8]=0;ModelStates_8 [9]=0;;ModelStates_9 [0]=0;ModelStates_9 [1]=0;ModelStates_9 [2]=0;ModelStates_9 [3]=0;ModelStates_9 [4]=0;ModelStates_9 [5]=0;ModelStates_9 [6]=0;ModelStates_9 [7]=0;ModelStates_9 [8]=0;ModelStates_9 [9]=0;;ModelStates_10 [0]=0;ModelStates_10 [1]=0;ModelStates_10 [2]=0;ModelStates_10 [3]=0;ModelStates_10 [4]=0;ModelStates_10 [5]=0;ModelStates_10 [6]=0;ModelStates_10 [7]=0;ModelStates_10 [8]=0;ModelStates_10 [9]=0;;ModelStates_11 [0]=0;ModelStates_11 [1]=0;ModelStates_11 [2]=0;ModelStates_11 [3]=0;ModelStates_11 [4]=0;ModelStates_11 [5]=0;ModelStates_11 [6]=0;ModelStates_11 [7]=0;ModelStates_11 [8]=0;ModelStates_11 [9]=0;;ModelStates_12 [0]=0;ModelStates_12 [1]=0;ModelStates_12 [2]=0;ModelStates_12 [3]=0;ModelStates_12 [4]=0;ModelStates_12 [5]=0;ModelStates_12 [6]=0;ModelStates_12 [7]=0;ModelStates_12 [8]=0;ModelStates_12 [9]=0;;ModelStates_13 [0]=0;ModelStates_13 [1]=0;ModelStates_13 [2]=0;ModelStates_13 [3]=0;ModelStates_13 [4]=0;ModelStates_13 [5]=0;ModelStates_13 [6]=0;ModelStates_13 [7]=0;ModelStates_13 [8]=0;ModelStates_13 [9]=0;;
	//#define THISCOMMANDHERE21(VARILP) MYDTYPE SonNo_ ## VARILP =InMat.SonNoVec[PIdx_ ## VARILP ];
	//MYDTYPE SonNo_1 =InMat.SonNoVec[PIdx_1 ];MYDTYPE SonNo_2 =InMat.SonNoVec[PIdx_2 ];MYDTYPE SonNo_3 =InMat.SonNoVec[PIdx_3 ];MYDTYPE SonNo_4 =InMat.SonNoVec[PIdx_4 ];MYDTYPE SonNo_5 =InMat.SonNoVec[PIdx_5 ];MYDTYPE SonNo_6 =InMat.SonNoVec[PIdx_6 ];MYDTYPE SonNo_7 =InMat.SonNoVec[PIdx_7 ];MYDTYPE SonNo_8 =InMat.SonNoVec[PIdx_8 ];MYDTYPE SonNo_9 =InMat.SonNoVec[PIdx_9 ];MYDTYPE SonNo_10 =InMat.SonNoVec[PIdx_10 ];MYDTYPE SonNo_11 =InMat.SonNoVec[PIdx_11 ];MYDTYPE SonNo_12 =InMat.SonNoVec[PIdx_12 ];MYDTYPE SonNo_13 =InMat.SonNoVec[PIdx_13 ];
	//RBS i deleteed  Vs[PIdx_ ## VARILP ]=V[PIdx_ ## VARILP ];
	#define THISCOMMANDHERE22(VARILP) Vs[PIdx_ ## VARILP ]=V[PIdx_ ## VARILP ];bHP[PIdx_ ## VARILP]=0;dv_ ## VARILP =0;
	bHP[PIdx_1]=0;dv_1 =0;;bHP[PIdx_2]=0;dv_2 =0;bHP[PIdx_3]=0;dv_3 =0;bHP[PIdx_4]=0;dv_4 =0;bHP[PIdx_5]=0;dv_5 =0;bHP[PIdx_6]=0;dv_6 =0;bHP[PIdx_7]=0;dv_7 =0;bHP[PIdx_8]=0;dv_8 =0;bHP[PIdx_9]=0;dv_9 =0;bHP[PIdx_10]=0;dv_10 =0;bHP[PIdx_11]=0;dv_11 =0;bHP[PIdx_12]=0;dv_12 =0;bHP[PIdx_13]=0;dv_13 =0;

	// MYDTYPE parentIndex_3; // MYDTYPE Eidx_3; // Eidx_3 = SMemMat->N-PIdx_3;
	#define THISCOMMANDHERE23(VARILP) MYDTYPE parentIndex_ ## VARILP ; MYDTYPE Eidx_ ## VARILP ; Eidx_ ## VARILP = SMemMat.N-PIdx_ ## VARILP -1;
	MYDTYPE parentIndex_1 ; MYDTYPE Eidx_1 ; Eidx_1 = SMemMat.N-PIdx_1 -1;MYDTYPE parentIndex_2 ; MYDTYPE Eidx_2 ; Eidx_2 = SMemMat.N-PIdx_2 -1;MYDTYPE parentIndex_3 ; MYDTYPE Eidx_3 ; Eidx_3 = SMemMat.N-PIdx_3 -1;MYDTYPE parentIndex_4 ; MYDTYPE Eidx_4 ; Eidx_4 = SMemMat.N-PIdx_4 -1;MYDTYPE parentIndex_5 ; MYDTYPE Eidx_5 ; Eidx_5 = SMemMat.N-PIdx_5 -1;MYDTYPE parentIndex_6 ; MYDTYPE Eidx_6 ; Eidx_6 = SMemMat.N-PIdx_6 -1;MYDTYPE parentIndex_7 ; MYDTYPE Eidx_7 ; Eidx_7 = SMemMat.N-PIdx_7 -1;MYDTYPE parentIndex_8 ; MYDTYPE Eidx_8 ; Eidx_8 = SMemMat.N-PIdx_8 -1;MYDTYPE parentIndex_9 ; MYDTYPE Eidx_9 ; Eidx_9 = SMemMat.N-PIdx_9 -1;MYDTYPE parentIndex_10 ; MYDTYPE Eidx_10 ; Eidx_10 = SMemMat.N-PIdx_10 -1;MYDTYPE parentIndex_11 ; MYDTYPE Eidx_11 ; Eidx_11 = SMemMat.N-PIdx_11 -1;MYDTYPE parentIndex_12 ; MYDTYPE Eidx_12 ; Eidx_12 = SMemMat.N-PIdx_12 -1;MYDTYPE parentIndex_13 ; MYDTYPE Eidx_13 ; Eidx_13 = SMemMat.N-PIdx_13 -1;
	// RRRXXX This probably should be in ILP too!
	if(Eidx_1>SMemMat.N-1){
		Eidx_1=SMemMat.N-1;
	}
	MYDTYPE perThreadParamMSize = SMemMat.NComps*NPARAMS;
//#define param_macro(paramInd,segmentInd) ParamsM[NeuronID*perThreadParamMSize + paramInd*SMemMat.NComps+cSegToComp[segmentInd] ]
	#define param_macro(paramInd,segmentInd) tex2D<float>(tex, paramInd*SMemMat.NComps+cSegToComp[segmentInd], NeuronID)


if(cBoolModel[PIdx_1 +0*SMemMat.N]){CuInitModel_ca(v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*SMemMat.N]){CuInitModel_cad(v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*SMemMat.N]){CuInitModel_kca(v_1,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*SMemMat.N]){CuInitModel_km(v_1,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*SMemMat.N]){CuInitModel_kv(v_1,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*SMemMat.N]){CuInitModel_na(v_1,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );} if(cBoolModel[PIdx_1 +6*SMemMat.N]){CuInitModel_pas(v_1,param_macro(28,PIdx_1) ,param_macro(29,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*SMemMat.N]){CuInitModel_ca(v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*SMemMat.N]){CuInitModel_cad(v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*SMemMat.N]){CuInitModel_kca(v_2,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*SMemMat.N]){CuInitModel_km(v_2,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*SMemMat.N]){CuInitModel_kv(v_2,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*SMemMat.N]){CuInitModel_na(v_2,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );} if(cBoolModel[PIdx_2 +6*SMemMat.N]){CuInitModel_pas(v_2,param_macro(28,PIdx_2) ,param_macro(29,PIdx_2) );}  if(cBoolModel[PIdx_3 +0*SMemMat.N]){CuInitModel_ca(v_3,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*SMemMat.N]){CuInitModel_cad(v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*SMemMat.N]){CuInitModel_kca(v_3,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*SMemMat.N]){CuInitModel_km(v_3,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*SMemMat.N]){CuInitModel_kv(v_3,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*SMemMat.N]){CuInitModel_na(v_3,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );} if(cBoolModel[PIdx_3 +6*SMemMat.N]){CuInitModel_pas(v_3,param_macro(28,PIdx_3) ,param_macro(29,PIdx_3) );}  if(cBoolModel[PIdx_4 +0*SMemMat.N]){CuInitModel_ca(v_4,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*SMemMat.N]){CuInitModel_cad(v_4,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*SMemMat.N]){CuInitModel_kca(v_4,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*SMemMat.N]){CuInitModel_km(v_4,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*SMemMat.N]){CuInitModel_kv(v_4,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*SMemMat.N]){CuInitModel_na(v_4,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );} if(cBoolModel[PIdx_4 +6*SMemMat.N]){CuInitModel_pas(v_4,param_macro(28,PIdx_4) ,param_macro(29,PIdx_4) );}  if(cBoolModel[PIdx_5 +0*SMemMat.N]){CuInitModel_ca(v_5,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*SMemMat.N]){CuInitModel_cad(v_5,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*SMemMat.N]){CuInitModel_kca(v_5,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*SMemMat.N]){CuInitModel_km(v_5,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*SMemMat.N]){CuInitModel_kv(v_5,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*SMemMat.N]){CuInitModel_na(v_5,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );} if(cBoolModel[PIdx_5 +6*SMemMat.N]){CuInitModel_pas(v_5,param_macro(28,PIdx_5) ,param_macro(29,PIdx_5) );}  if(cBoolModel[PIdx_6 +0*SMemMat.N]){CuInitModel_ca(v_6,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*SMemMat.N]){CuInitModel_cad(v_6,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*SMemMat.N]){CuInitModel_kca(v_6,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*SMemMat.N]){CuInitModel_km(v_6,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*SMemMat.N]){CuInitModel_kv(v_6,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*SMemMat.N]){CuInitModel_na(v_6,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );} if(cBoolModel[PIdx_6 +6*SMemMat.N]){CuInitModel_pas(v_6,param_macro(28,PIdx_6) ,param_macro(29,PIdx_6) );}  if(cBoolModel[PIdx_7 +0*SMemMat.N]){CuInitModel_ca(v_7,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*SMemMat.N]){CuInitModel_cad(v_7,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*SMemMat.N]){CuInitModel_kca(v_7,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*SMemMat.N]){CuInitModel_km(v_7,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*SMemMat.N]){CuInitModel_kv(v_7,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*SMemMat.N]){CuInitModel_na(v_7,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );} if(cBoolModel[PIdx_7 +6*SMemMat.N]){CuInitModel_pas(v_7,param_macro(28,PIdx_7) ,param_macro(29,PIdx_7) );}  if(cBoolModel[PIdx_8 +0*SMemMat.N]){CuInitModel_ca(v_8,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*SMemMat.N]){CuInitModel_cad(v_8,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*SMemMat.N]){CuInitModel_kca(v_8,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*SMemMat.N]){CuInitModel_km(v_8,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*SMemMat.N]){CuInitModel_kv(v_8,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*SMemMat.N]){CuInitModel_na(v_8,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );} if(cBoolModel[PIdx_8 +6*SMemMat.N]){CuInitModel_pas(v_8,param_macro(28,PIdx_8) ,param_macro(29,PIdx_8) );}  if(cBoolModel[PIdx_9 +0*SMemMat.N]){CuInitModel_ca(v_9,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*SMemMat.N]){CuInitModel_cad(v_9,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*SMemMat.N]){CuInitModel_kca(v_9,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*SMemMat.N]){CuInitModel_km(v_9,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*SMemMat.N]){CuInitModel_kv(v_9,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*SMemMat.N]){CuInitModel_na(v_9,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );} if(cBoolModel[PIdx_9 +6*SMemMat.N]){CuInitModel_pas(v_9,param_macro(28,PIdx_9) ,param_macro(29,PIdx_9) );}  if(cBoolModel[PIdx_10 +0*SMemMat.N]){CuInitModel_ca(v_10,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*SMemMat.N]){CuInitModel_cad(v_10,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*SMemMat.N]){CuInitModel_kca(v_10,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*SMemMat.N]){CuInitModel_km(v_10,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*SMemMat.N]){CuInitModel_kv(v_10,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*SMemMat.N]){CuInitModel_na(v_10,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );} if(cBoolModel[PIdx_10 +6*SMemMat.N]){CuInitModel_pas(v_10,param_macro(28,PIdx_10) ,param_macro(29,PIdx_10) );}  if(cBoolModel[PIdx_11 +0*SMemMat.N]){CuInitModel_ca(v_11,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*SMemMat.N]){CuInitModel_cad(v_11,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*SMemMat.N]){CuInitModel_kca(v_11,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*SMemMat.N]){CuInitModel_km(v_11,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*SMemMat.N]){CuInitModel_kv(v_11,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*SMemMat.N]){CuInitModel_na(v_11,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );} if(cBoolModel[PIdx_11 +6*SMemMat.N]){CuInitModel_pas(v_11,param_macro(28,PIdx_11) ,param_macro(29,PIdx_11) );}  if(cBoolModel[PIdx_12 +0*SMemMat.N]){CuInitModel_ca(v_12,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*SMemMat.N]){CuInitModel_cad(v_12,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*SMemMat.N]){CuInitModel_kca(v_12,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*SMemMat.N]){CuInitModel_km(v_12,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*SMemMat.N]){CuInitModel_kv(v_12,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*SMemMat.N]){CuInitModel_na(v_12,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );} if(cBoolModel[PIdx_12 +6*SMemMat.N]){CuInitModel_pas(v_12,param_macro(28,PIdx_12) ,param_macro(29,PIdx_12) );}  if(cBoolModel[PIdx_13 +0*SMemMat.N]){CuInitModel_ca(v_13,ModelStates_13[0],ModelStates_13[1],param_macro(0,PIdx_13) ,param_macro(1,PIdx_13) ,ModelStates_13[8],ModelStates_13[9]);} if(cBoolModel[PIdx_13 +1*SMemMat.N]){CuInitModel_cad(v_13,ModelStates_13[2],ModelStates_13[9],ModelStates_13[8]);} if(cBoolModel[PIdx_13 +2*SMemMat.N]){CuInitModel_kca(v_13,ModelStates_13[3],param_macro(2,PIdx_13) ,param_macro(3,PIdx_13) ,param_macro(4,PIdx_13) ,param_macro(5,PIdx_13) ,ModelStates_13[8]);} if(cBoolModel[PIdx_13 +3*SMemMat.N]){CuInitModel_km(v_13,ModelStates_13[4],param_macro(6,PIdx_13) ,param_macro(7,PIdx_13) ,param_macro(8,PIdx_13) ,param_macro(9,PIdx_13) ,param_macro(10,PIdx_13) );} if(cBoolModel[PIdx_13 +4*SMemMat.N]){CuInitModel_kv(v_13,ModelStates_13[5],param_macro(11,PIdx_13) ,param_macro(12,PIdx_13) ,param_macro(13,PIdx_13) ,param_macro(14,PIdx_13) ,param_macro(15,PIdx_13) );} if(cBoolModel[PIdx_13 +5*SMemMat.N]){CuInitModel_na(v_13,ModelStates_13[6],ModelStates_13[7],param_macro(16,PIdx_13) ,param_macro(17,PIdx_13) ,param_macro(18,PIdx_13) ,param_macro(19,PIdx_13) ,param_macro(20,PIdx_13) ,param_macro(21,PIdx_13) ,param_macro(22,PIdx_13) ,param_macro(23,PIdx_13) ,param_macro(24,PIdx_13) ,param_macro(25,PIdx_13) ,param_macro(26,PIdx_13) ,param_macro(27,PIdx_13) );} if(cBoolModel[PIdx_13 +6*SMemMat.N]){CuInitModel_pas(v_13,param_macro(28,PIdx_13) ,param_macro(29,PIdx_13) );}
#define THISCOMMANDHERE24(VARILP) parentIndex_ ## VARILP =SMemMat.N-cKs[SMemMat.N-PIdx_ ## VARILP ];
	parentIndex_1 =SMemMat.N-cKs[SMemMat.N-PIdx_1 ];parentIndex_2 =SMemMat.N-cKs[SMemMat.N-PIdx_2 ];parentIndex_3 =SMemMat.N-cKs[SMemMat.N-PIdx_3 ];parentIndex_4 =SMemMat.N-cKs[SMemMat.N-PIdx_4 ];parentIndex_5 =SMemMat.N-cKs[SMemMat.N-PIdx_5 ];parentIndex_6 =SMemMat.N-cKs[SMemMat.N-PIdx_6 ];parentIndex_7 =SMemMat.N-cKs[SMemMat.N-PIdx_7 ];parentIndex_8 =SMemMat.N-cKs[SMemMat.N-PIdx_8 ];parentIndex_9 =SMemMat.N-cKs[SMemMat.N-PIdx_9 ];parentIndex_10 =SMemMat.N-cKs[SMemMat.N-PIdx_10 ];parentIndex_11 =SMemMat.N-cKs[SMemMat.N-PIdx_11 ];parentIndex_12 =SMemMat.N-cKs[SMemMat.N-PIdx_12 ];parentIndex_13 =SMemMat.N-cKs[SMemMat.N-PIdx_13 ];
	MYDTYPE stimLoc = stim.loc;
    MYFTYPE stimArea = stim.area;
    MYDTYPE dtCounter=0;
    MYFTYPE  dt = sim.dt;
#ifndef STIMFROMFILE
    MYFTYPE stimDel = stim.dels[StimID];//Should be constant
    MYFTYPE stimDur = stim.durs[StimID];//Should be constant
    MYFTYPE stimAmp = stim.amps[StimID];
#endif // !STIMFROMFILE
	// MYFTYPE rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#define THISCOMMANDHERE25(VARILP) MYSECONDFTYPE rhs_ ## VARILP ,D_ ## VARILP; MYFTYPE gModel_ ## VARILP ,StimCurrent_ ## VARILP;
	MYSECONDFTYPE rhs_1 ,D_1; MYFTYPE gModel_1 ,StimCurrent_1;MYSECONDFTYPE rhs_2 ,D_2; MYFTYPE gModel_2 ,StimCurrent_2;MYSECONDFTYPE rhs_3 ,D_3; MYFTYPE gModel_3 ,StimCurrent_3;MYSECONDFTYPE rhs_4 ,D_4; MYFTYPE gModel_4 ,StimCurrent_4;MYSECONDFTYPE rhs_5 ,D_5; MYFTYPE gModel_5 ,StimCurrent_5;MYSECONDFTYPE rhs_6 ,D_6; MYFTYPE gModel_6 ,StimCurrent_6;MYSECONDFTYPE rhs_7 ,D_7; MYFTYPE gModel_7 ,StimCurrent_7;MYSECONDFTYPE rhs_8 ,D_8; MYFTYPE gModel_8 ,StimCurrent_8;MYSECONDFTYPE rhs_9 ,D_9; MYFTYPE gModel_9 ,StimCurrent_9;MYSECONDFTYPE rhs_10 ,D_10; MYFTYPE gModel_10 ,StimCurrent_10;MYSECONDFTYPE rhs_11 ,D_11; MYFTYPE gModel_11 ,StimCurrent_11;MYSECONDFTYPE rhs_12 ,D_12; MYFTYPE gModel_12 ,StimCurrent_12;MYSECONDFTYPE rhs_13 ,D_13; MYFTYPE gModel_13 ,StimCurrent_13;

for(int i=0;i<Nt;i++) {
        if(i==stim.dtInds[dtCounter]){
                dt = stim.durs[dtCounter];
                if (dtCounter != stim.numofdts-1){
                    dtCounter++;
                }
        }
        t+=0.5*dt;
 
        if((i%(WARPSIZE)==0)){
            if(i>0){
               for(int recInd =0;recInd<sim.NRecSites;recInd++){
					VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + (i-WARPSIZE)+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
				}
            }
			
            amps[PIdx_1]=stim.amps[threadIdx.y*Nt+i+PIdx_1];
        }
        for(int recInd =0;recInd<sim.NRecSites;recInd++){
			if(sim.RecSites[recInd]%WARPSIZE == threadIdx.x)
				//This is done by all threads why???
				SMemVHot[recInd*WARPSIZE+i%(WARPSIZE)]=v_1;//This is going to be challenging to make it general but possible.
			
		}
		 // rhs_3=0; // D_3=0;// StimCurrent_3=0;
#define THISCOMMANDHERE26(VARILP) rhs_ ## VARILP =0; D_ ## VARILP =0; sumCurrents_ ## VARILP=0; sumConductivity_ ## VARILP=0; sumCurrentsDv_ ## VARILP=0; sumConductivityDv_ ## VARILP=0;StimCurrent_ ## VARILP =0;
	rhs_1 =0; D_1 =0; sumCurrents_1=0; sumConductivity_1=0; sumCurrentsDv_1=0; sumConductivityDv_1=0;StimCurrent_1 =0;rhs_2 =0; D_2 =0; sumCurrents_2=0; sumConductivity_2=0; sumCurrentsDv_2=0; sumConductivityDv_2=0;StimCurrent_2 =0;rhs_3 =0; D_3 =0; sumCurrents_3=0; sumConductivity_3=0; sumCurrentsDv_3=0; sumConductivityDv_3=0;StimCurrent_3 =0;rhs_4 =0; D_4 =0; sumCurrents_4=0; sumConductivity_4=0; sumCurrentsDv_4=0; sumConductivityDv_4=0;StimCurrent_4 =0;rhs_5 =0; D_5 =0; sumCurrents_5=0; sumConductivity_5=0; sumCurrentsDv_5=0; sumConductivityDv_5=0;StimCurrent_5 =0;rhs_6 =0; D_6 =0; sumCurrents_6=0; sumConductivity_6=0; sumCurrentsDv_6=0; sumConductivityDv_6=0;StimCurrent_6 =0;rhs_7 =0; D_7 =0; sumCurrents_7=0; sumConductivity_7=0; sumCurrentsDv_7=0; sumConductivityDv_7=0;StimCurrent_7 =0;rhs_8 =0; D_8 =0; sumCurrents_8=0; sumConductivity_8=0; sumCurrentsDv_8=0; sumConductivityDv_8=0;StimCurrent_8 =0;rhs_9 =0; D_9 =0; sumCurrents_9=0; sumConductivity_9=0; sumCurrentsDv_9=0; sumConductivityDv_9=0;StimCurrent_9 =0;rhs_10 =0; D_10 =0; sumCurrents_10=0; sumConductivity_10=0; sumCurrentsDv_10=0; sumConductivityDv_10=0;StimCurrent_10 =0;rhs_11 =0; D_11 =0; sumCurrents_11=0; sumConductivity_11=0; sumCurrentsDv_11=0; sumConductivityDv_11=0;StimCurrent_11 =0;rhs_12 =0; D_12 =0; sumCurrents_12=0; sumConductivity_12=0; sumCurrentsDv_12=0; sumConductivityDv_12=0;StimCurrent_12 =0;rhs_13 =0; D_13 =0; sumCurrents_13=0; sumConductivity_13=0; sumCurrentsDv_13=0; sumConductivityDv_13=0;StimCurrent_13 =0;
		// Before matrix
		// if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#ifndef STIMFROMFILE
    #define THISCOMMANDHERE27(VARILP) if(t>=stimDel && t<stimDel+stimDur && PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP  = 100*stimAmp/stimArea;}
	if(t>=stimDel && t<stimDel+stimDur && PIdx_1  == stimLoc){StimCurrent_1  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_2  == stimLoc){StimCurrent_2  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_3  == stimLoc){StimCurrent_3  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_4  == stimLoc){StimCurrent_4  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_5  == stimLoc){StimCurrent_5  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_6  == stimLoc){StimCurrent_6  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_7  == stimLoc){StimCurrent_7  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_8  == stimLoc){StimCurrent_8  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_9  == stimLoc){StimCurrent_9  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_10  == stimLoc){StimCurrent_10  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_11  == stimLoc){StimCurrent_11  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_12  == stimLoc){StimCurrent_12  = 100*stimAmp/stimArea;}if(t>=stimDel && t<stimDel+stimDur && PIdx_13  == stimLoc){StimCurrent_13  = 100*stimAmp/stimArea;}
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
    
    #define THISCOMMANDHERE28(VARILP) if(PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP = 100*amps[i%WARPSIZE]/stimArea;}
	if(PIdx_1  == stimLoc){StimCurrent_1 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_2  == stimLoc){StimCurrent_2 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_3  == stimLoc){StimCurrent_3 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_4  == stimLoc){StimCurrent_4 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_5  == stimLoc){StimCurrent_5 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_6  == stimLoc){StimCurrent_6 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_7  == stimLoc){StimCurrent_7 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_8  == stimLoc){StimCurrent_8 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_9  == stimLoc){StimCurrent_9 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_10  == stimLoc){StimCurrent_10 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_11  == stimLoc){StimCurrent_11 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_12  == stimLoc){StimCurrent_12 = 100*amps[i%WARPSIZE]/stimArea;}if(PIdx_13  == stimLoc){StimCurrent_13 = 100*amps[i%WARPSIZE]/stimArea;}
#endif // STIMFROMFILE
	 if(cBoolModel[PIdx_1 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );} if(cBoolModel[PIdx_1 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,param_macro(28,PIdx_1) ,param_macro(29,PIdx_1) );} if(cBoolModel[PIdx_2 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );} if(cBoolModel[PIdx_2 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,param_macro(28,PIdx_2) ,param_macro(29,PIdx_2) );} if(cBoolModel[PIdx_3 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );} if(cBoolModel[PIdx_3 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,param_macro(28,PIdx_3) ,param_macro(29,PIdx_3) );} if(cBoolModel[PIdx_4 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );} if(cBoolModel[PIdx_4 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,param_macro(28,PIdx_4) ,param_macro(29,PIdx_4) );} if(cBoolModel[PIdx_5 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );} if(cBoolModel[PIdx_5 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,param_macro(28,PIdx_5) ,param_macro(29,PIdx_5) );} if(cBoolModel[PIdx_6 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );} if(cBoolModel[PIdx_6 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,param_macro(28,PIdx_6) ,param_macro(29,PIdx_6) );} if(cBoolModel[PIdx_7 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );} if(cBoolModel[PIdx_7 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,param_macro(28,PIdx_7) ,param_macro(29,PIdx_7) );} if(cBoolModel[PIdx_8 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );} if(cBoolModel[PIdx_8 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,param_macro(28,PIdx_8) ,param_macro(29,PIdx_8) );} if(cBoolModel[PIdx_9 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );} if(cBoolModel[PIdx_9 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,param_macro(28,PIdx_9) ,param_macro(29,PIdx_9) );} if(cBoolModel[PIdx_10 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );} if(cBoolModel[PIdx_10 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,param_macro(28,PIdx_10) ,param_macro(29,PIdx_10) );} if(cBoolModel[PIdx_11 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );} if(cBoolModel[PIdx_11 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,param_macro(28,PIdx_11) ,param_macro(29,PIdx_11) );} if(cBoolModel[PIdx_12 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );} if(cBoolModel[PIdx_12 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,param_macro(28,PIdx_12) ,param_macro(29,PIdx_12) );} if(cBoolModel[PIdx_13 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,ModelStates_13[0],ModelStates_13[1],param_macro(0,PIdx_13) ,param_macro(1,PIdx_13) ,ModelStates_13[8],ModelStates_13[9]);} if(cBoolModel[PIdx_13 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,ModelStates_13[2],ModelStates_13[9],ModelStates_13[8]);} if(cBoolModel[PIdx_13 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,ModelStates_13[3],param_macro(2,PIdx_13) ,param_macro(3,PIdx_13) ,param_macro(4,PIdx_13) ,param_macro(5,PIdx_13) ,ModelStates_13[8]);} if(cBoolModel[PIdx_13 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,ModelStates_13[4],param_macro(6,PIdx_13) ,param_macro(7,PIdx_13) ,param_macro(8,PIdx_13) ,param_macro(9,PIdx_13) ,param_macro(10,PIdx_13) );} if(cBoolModel[PIdx_13 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,ModelStates_13[5],param_macro(11,PIdx_13) ,param_macro(12,PIdx_13) ,param_macro(13,PIdx_13) ,param_macro(14,PIdx_13) ,param_macro(15,PIdx_13) );} if(cBoolModel[PIdx_13 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,ModelStates_13[6],ModelStates_13[7],param_macro(16,PIdx_13) ,param_macro(17,PIdx_13) ,param_macro(18,PIdx_13) ,param_macro(19,PIdx_13) ,param_macro(20,PIdx_13) ,param_macro(21,PIdx_13) ,param_macro(22,PIdx_13) ,param_macro(23,PIdx_13) ,param_macro(24,PIdx_13) ,param_macro(25,PIdx_13) ,param_macro(26,PIdx_13) ,param_macro(27,PIdx_13) );} if(cBoolModel[PIdx_13 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrentsDv_13 , sumConductivityDv_13 ,v_13 +0.001,param_macro(28,PIdx_13) ,param_macro(29,PIdx_13) );}
	 if(cBoolModel[PIdx_1 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );} if(cBoolModel[PIdx_1 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_1 , sumConductivity_1,v_1 ,param_macro(28,PIdx_1) ,param_macro(29,PIdx_1) );} if(cBoolModel[PIdx_2 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );} if(cBoolModel[PIdx_2 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_2 , sumConductivity_2,v_2 ,param_macro(28,PIdx_2) ,param_macro(29,PIdx_2) );} if(cBoolModel[PIdx_3 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );} if(cBoolModel[PIdx_3 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_3 , sumConductivity_3,v_3 ,param_macro(28,PIdx_3) ,param_macro(29,PIdx_3) );} if(cBoolModel[PIdx_4 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );} if(cBoolModel[PIdx_4 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_4 , sumConductivity_4,v_4 ,param_macro(28,PIdx_4) ,param_macro(29,PIdx_4) );} if(cBoolModel[PIdx_5 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );} if(cBoolModel[PIdx_5 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_5 , sumConductivity_5,v_5 ,param_macro(28,PIdx_5) ,param_macro(29,PIdx_5) );} if(cBoolModel[PIdx_6 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );} if(cBoolModel[PIdx_6 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_6 , sumConductivity_6,v_6 ,param_macro(28,PIdx_6) ,param_macro(29,PIdx_6) );} if(cBoolModel[PIdx_7 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );} if(cBoolModel[PIdx_7 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_7 , sumConductivity_7,v_7 ,param_macro(28,PIdx_7) ,param_macro(29,PIdx_7) );} if(cBoolModel[PIdx_8 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );} if(cBoolModel[PIdx_8 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_8 , sumConductivity_8,v_8 ,param_macro(28,PIdx_8) ,param_macro(29,PIdx_8) );} if(cBoolModel[PIdx_9 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );} if(cBoolModel[PIdx_9 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_9 , sumConductivity_9,v_9 ,param_macro(28,PIdx_9) ,param_macro(29,PIdx_9) );} if(cBoolModel[PIdx_10 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );} if(cBoolModel[PIdx_10 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_10 , sumConductivity_10,v_10 ,param_macro(28,PIdx_10) ,param_macro(29,PIdx_10) );} if(cBoolModel[PIdx_11 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );} if(cBoolModel[PIdx_11 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_11 , sumConductivity_11,v_11 ,param_macro(28,PIdx_11) ,param_macro(29,PIdx_11) );} if(cBoolModel[PIdx_12 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );} if(cBoolModel[PIdx_12 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_12 , sumConductivity_12,v_12 ,param_macro(28,PIdx_12) ,param_macro(29,PIdx_12) );} if(cBoolModel[PIdx_13 +0*SMemMat.N]){CuBreakpointModel_ca(sumCurrents_13 , sumConductivity_13,v_13 ,ModelStates_13[0],ModelStates_13[1],param_macro(0,PIdx_13) ,param_macro(1,PIdx_13) ,ModelStates_13[8],ModelStates_13[9]);} if(cBoolModel[PIdx_13 +1*SMemMat.N]){CuBreakpointModel_cad(sumCurrents_13 , sumConductivity_13,v_13 ,ModelStates_13[2],ModelStates_13[9],ModelStates_13[8]);} if(cBoolModel[PIdx_13 +2*SMemMat.N]){CuBreakpointModel_kca(sumCurrents_13 , sumConductivity_13,v_13 ,ModelStates_13[3],param_macro(2,PIdx_13) ,param_macro(3,PIdx_13) ,param_macro(4,PIdx_13) ,param_macro(5,PIdx_13) ,ModelStates_13[8]);} if(cBoolModel[PIdx_13 +3*SMemMat.N]){CuBreakpointModel_km(sumCurrents_13 , sumConductivity_13,v_13 ,ModelStates_13[4],param_macro(6,PIdx_13) ,param_macro(7,PIdx_13) ,param_macro(8,PIdx_13) ,param_macro(9,PIdx_13) ,param_macro(10,PIdx_13) );} if(cBoolModel[PIdx_13 +4*SMemMat.N]){CuBreakpointModel_kv(sumCurrents_13 , sumConductivity_13,v_13 ,ModelStates_13[5],param_macro(11,PIdx_13) ,param_macro(12,PIdx_13) ,param_macro(13,PIdx_13) ,param_macro(14,PIdx_13) ,param_macro(15,PIdx_13) );} if(cBoolModel[PIdx_13 +5*SMemMat.N]){CuBreakpointModel_na(sumCurrents_13 , sumConductivity_13,v_13 ,ModelStates_13[6],ModelStates_13[7],param_macro(16,PIdx_13) ,param_macro(17,PIdx_13) ,param_macro(18,PIdx_13) ,param_macro(19,PIdx_13) ,param_macro(20,PIdx_13) ,param_macro(21,PIdx_13) ,param_macro(22,PIdx_13) ,param_macro(23,PIdx_13) ,param_macro(24,PIdx_13) ,param_macro(25,PIdx_13) ,param_macro(26,PIdx_13) ,param_macro(27,PIdx_13) );} if(cBoolModel[PIdx_13 +6*SMemMat.N]){CuBreakpointModel_pas(sumCurrents_13 , sumConductivity_13,v_13 ,param_macro(28,PIdx_13) ,param_macro(29,PIdx_13) );}
	// gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; 
#define THISCOMMANDHERE29(VARILP) gModel_ ## VARILP  = (sumCurrentsDv_ ## VARILP -sumCurrents_ ## VARILP )/EPS_V;  
	gModel_1  = (sumCurrentsDv_1 -sumCurrents_1 )/EPS_V;  gModel_2  = (sumCurrentsDv_2 -sumCurrents_2 )/EPS_V;  gModel_3  = (sumCurrentsDv_3 -sumCurrents_3 )/EPS_V;  gModel_4  = (sumCurrentsDv_4 -sumCurrents_4 )/EPS_V;  gModel_5  = (sumCurrentsDv_5 -sumCurrents_5 )/EPS_V;  gModel_6  = (sumCurrentsDv_6 -sumCurrents_6 )/EPS_V;  gModel_7  = (sumCurrentsDv_7 -sumCurrents_7 )/EPS_V;  gModel_8  = (sumCurrentsDv_8 -sumCurrents_8 )/EPS_V;  gModel_9  = (sumCurrentsDv_9 -sumCurrents_9 )/EPS_V;  gModel_10  = (sumCurrentsDv_10 -sumCurrents_10 )/EPS_V;  gModel_11  = (sumCurrentsDv_11 -sumCurrents_11 )/EPS_V;  gModel_12  = (sumCurrentsDv_12 -sumCurrents_12 )/EPS_V;  gModel_13  = (sumCurrentsDv_13 -sumCurrents_13 )/EPS_V;  
	
#define THISCOMMANDHERE30(VARILP) rhs_ ## VARILP =StimCurrent_ ## VARILP -sumCurrents_ ## VARILP ;
	rhs_1 =StimCurrent_1 -sumCurrents_1 ;rhs_2 =StimCurrent_2 -sumCurrents_2 ;rhs_3 =StimCurrent_3 -sumCurrents_3 ;rhs_4 =StimCurrent_4 -sumCurrents_4 ;rhs_5 =StimCurrent_5 -sumCurrents_5 ;rhs_6 =StimCurrent_6 -sumCurrents_6 ;rhs_7 =StimCurrent_7 -sumCurrents_7 ;rhs_8 =StimCurrent_8 -sumCurrents_8 ;rhs_9 =StimCurrent_9 -sumCurrents_9 ;rhs_10 =StimCurrent_10 -sumCurrents_10 ;rhs_11 =StimCurrent_11 -sumCurrents_11 ;rhs_12 =StimCurrent_12 -sumCurrents_12 ;rhs_13 =StimCurrent_13 -sumCurrents_13 ;
#define THISCOMMANDHERE31(VARILP) D_ ## VARILP =gModel_ ## VARILP +cCm[PIdx_ ## VARILP ]/(dt*1000); D_ ## VARILP -=cF[SMemMat.N-PIdx_ ## VARILP -1];
	D_1 =gModel_1 +cCm[PIdx_1 ]/(dt*1000); D_1 -=cF[SMemMat.N-PIdx_1 -1];D_2 =gModel_2 +cCm[PIdx_2 ]/(dt*1000); D_2 -=cF[SMemMat.N-PIdx_2 -1];D_3 =gModel_3 +cCm[PIdx_3 ]/(dt*1000); D_3 -=cF[SMemMat.N-PIdx_3 -1];D_4 =gModel_4 +cCm[PIdx_4 ]/(dt*1000); D_4 -=cF[SMemMat.N-PIdx_4 -1];D_5 =gModel_5 +cCm[PIdx_5 ]/(dt*1000); D_5 -=cF[SMemMat.N-PIdx_5 -1];D_6 =gModel_6 +cCm[PIdx_6 ]/(dt*1000); D_6 -=cF[SMemMat.N-PIdx_6 -1];D_7 =gModel_7 +cCm[PIdx_7 ]/(dt*1000); D_7 -=cF[SMemMat.N-PIdx_7 -1];D_8 =gModel_8 +cCm[PIdx_8 ]/(dt*1000); D_8 -=cF[SMemMat.N-PIdx_8 -1];D_9 =gModel_9 +cCm[PIdx_9 ]/(dt*1000); D_9 -=cF[SMemMat.N-PIdx_9 -1];D_10 =gModel_10 +cCm[PIdx_10 ]/(dt*1000); D_10 -=cF[SMemMat.N-PIdx_10 -1];D_11 =gModel_11 +cCm[PIdx_11 ]/(dt*1000); D_11 -=cF[SMemMat.N-PIdx_11 -1];D_12 =gModel_12 +cCm[PIdx_12 ]/(dt*1000); D_12 -=cF[SMemMat.N-PIdx_12 -1];D_13 =gModel_13 +cCm[PIdx_13 ]/(dt*1000); D_13 -=cF[SMemMat.N-PIdx_13 -1];
   
    //this should be out and correct cKs
	if(PIdx_1==0) {
		parentIndex_1=0;
	};
#define THISCOMMANDHERE32(VARILP)  dv_ ## VARILP +=bHP[SMemMat.N-parentIndex_ ## VARILP -1]-bHP[SMemMat.N-PIdx_ ## VARILP -1]; rhs_ ## VARILP -=cF[SMemMat.N-PIdx_ ## VARILP -1]*dv_ ## VARILP ;
	 dv_1 +=bHP[SMemMat.N-parentIndex_1 -1]-bHP[SMemMat.N-PIdx_1 -1]; rhs_1 -=cF[SMemMat.N-PIdx_1 -1]*dv_1 ; dv_2 +=bHP[SMemMat.N-parentIndex_2 -1]-bHP[SMemMat.N-PIdx_2 -1]; rhs_2 -=cF[SMemMat.N-PIdx_2 -1]*dv_2 ; dv_3 +=bHP[SMemMat.N-parentIndex_3 -1]-bHP[SMemMat.N-PIdx_3 -1]; rhs_3 -=cF[SMemMat.N-PIdx_3 -1]*dv_3 ; dv_4 +=bHP[SMemMat.N-parentIndex_4 -1]-bHP[SMemMat.N-PIdx_4 -1]; rhs_4 -=cF[SMemMat.N-PIdx_4 -1]*dv_4 ; dv_5 +=bHP[SMemMat.N-parentIndex_5 -1]-bHP[SMemMat.N-PIdx_5 -1]; rhs_5 -=cF[SMemMat.N-PIdx_5 -1]*dv_5 ; dv_6 +=bHP[SMemMat.N-parentIndex_6 -1]-bHP[SMemMat.N-PIdx_6 -1]; rhs_6 -=cF[SMemMat.N-PIdx_6 -1]*dv_6 ; dv_7 +=bHP[SMemMat.N-parentIndex_7 -1]-bHP[SMemMat.N-PIdx_7 -1]; rhs_7 -=cF[SMemMat.N-PIdx_7 -1]*dv_7 ; dv_8 +=bHP[SMemMat.N-parentIndex_8 -1]-bHP[SMemMat.N-PIdx_8 -1]; rhs_8 -=cF[SMemMat.N-PIdx_8 -1]*dv_8 ; dv_9 +=bHP[SMemMat.N-parentIndex_9 -1]-bHP[SMemMat.N-PIdx_9 -1]; rhs_9 -=cF[SMemMat.N-PIdx_9 -1]*dv_9 ; dv_10 +=bHP[SMemMat.N-parentIndex_10 -1]-bHP[SMemMat.N-PIdx_10 -1]; rhs_10 -=cF[SMemMat.N-PIdx_10 -1]*dv_10 ; dv_11 +=bHP[SMemMat.N-parentIndex_11 -1]-bHP[SMemMat.N-PIdx_11 -1]; rhs_11 -=cF[SMemMat.N-PIdx_11 -1]*dv_11 ; dv_12 +=bHP[SMemMat.N-parentIndex_12 -1]-bHP[SMemMat.N-PIdx_12 -1]; rhs_12 -=cF[SMemMat.N-PIdx_12 -1]*dv_12 ; dv_13 +=bHP[SMemMat.N-parentIndex_13 -1]-bHP[SMemMat.N-PIdx_13 -1]; rhs_13 -=cF[SMemMat.N-PIdx_13 -1]*dv_13 ;
		// D_3=sumConductivity_3+SMemMat.Cms[PIdx_3]/(sim.dt*1000); // D_3-=cF[SMemMat.N-PIdx_3-1];
		// bHP[SMemMat.N-PIdx_3-1]=rhs_3; // uHP[SMemMat.N-PIdx_3-1]=D_3;
#define THISCOMMANDHERE33(VARILP) bHP[SMemMat.N-PIdx_ ## VARILP -1]=rhs_ ## VARILP ; uHP[SMemMat.N-PIdx_ ## VARILP -1]=D_ ## VARILP ;
		bHP[SMemMat.N-PIdx_1 -1]=rhs_1 ; uHP[SMemMat.N-PIdx_1 -1]=D_1 ;bHP[SMemMat.N-PIdx_2 -1]=rhs_2 ; uHP[SMemMat.N-PIdx_2 -1]=D_2 ;bHP[SMemMat.N-PIdx_3 -1]=rhs_3 ; uHP[SMemMat.N-PIdx_3 -1]=D_3 ;bHP[SMemMat.N-PIdx_4 -1]=rhs_4 ; uHP[SMemMat.N-PIdx_4 -1]=D_4 ;bHP[SMemMat.N-PIdx_5 -1]=rhs_5 ; uHP[SMemMat.N-PIdx_5 -1]=D_5 ;bHP[SMemMat.N-PIdx_6 -1]=rhs_6 ; uHP[SMemMat.N-PIdx_6 -1]=D_6 ;bHP[SMemMat.N-PIdx_7 -1]=rhs_7 ; uHP[SMemMat.N-PIdx_7 -1]=D_7 ;bHP[SMemMat.N-PIdx_8 -1]=rhs_8 ; uHP[SMemMat.N-PIdx_8 -1]=D_8 ;bHP[SMemMat.N-PIdx_9 -1]=rhs_9 ; uHP[SMemMat.N-PIdx_9 -1]=D_9 ;bHP[SMemMat.N-PIdx_10 -1]=rhs_10 ; uHP[SMemMat.N-PIdx_10 -1]=D_10 ;bHP[SMemMat.N-PIdx_11 -1]=rhs_11 ; uHP[SMemMat.N-PIdx_11 -1]=D_11 ;bHP[SMemMat.N-PIdx_12 -1]=rhs_12 ; uHP[SMemMat.N-PIdx_12 -1]=D_12 ;bHP[SMemMat.N-PIdx_13 -1]=rhs_13 ; uHP[SMemMat.N-PIdx_13 -1]=D_13 ;
		syncthreads();

		// if(SonNo_3==1) {bHP[SMemMat.N-parentIndex_3-1]+=cE[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=cE[Eidx_3];}
		//I
//#define THISCOMMANDHERE34(VARILP) if(SonNo_ ## VARILP ==1) {bHP[SMemMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[SMemMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
	//above line was changed to the no more variable sonno
		if(cSonNoVec[PIdx_1 ] ==1) {bHP[SMemMat.N-parentIndex_1 -1]+=cE[Eidx_1 ]*dv_1 ;uHP[SMemMat.N-parentIndex_1 -1]-=cE[Eidx_1 ];}if(cSonNoVec[PIdx_2 ] ==1) {bHP[SMemMat.N-parentIndex_2 -1]+=cE[Eidx_2 ]*dv_2 ;uHP[SMemMat.N-parentIndex_2 -1]-=cE[Eidx_2 ];}if(cSonNoVec[PIdx_3 ] ==1) {bHP[SMemMat.N-parentIndex_3 -1]+=cE[Eidx_3 ]*dv_3 ;uHP[SMemMat.N-parentIndex_3 -1]-=cE[Eidx_3 ];}if(cSonNoVec[PIdx_4 ] ==1) {bHP[SMemMat.N-parentIndex_4 -1]+=cE[Eidx_4 ]*dv_4 ;uHP[SMemMat.N-parentIndex_4 -1]-=cE[Eidx_4 ];}if(cSonNoVec[PIdx_5 ] ==1) {bHP[SMemMat.N-parentIndex_5 -1]+=cE[Eidx_5 ]*dv_5 ;uHP[SMemMat.N-parentIndex_5 -1]-=cE[Eidx_5 ];}if(cSonNoVec[PIdx_6 ] ==1) {bHP[SMemMat.N-parentIndex_6 -1]+=cE[Eidx_6 ]*dv_6 ;uHP[SMemMat.N-parentIndex_6 -1]-=cE[Eidx_6 ];}if(cSonNoVec[PIdx_7 ] ==1) {bHP[SMemMat.N-parentIndex_7 -1]+=cE[Eidx_7 ]*dv_7 ;uHP[SMemMat.N-parentIndex_7 -1]-=cE[Eidx_7 ];}if(cSonNoVec[PIdx_8 ] ==1) {bHP[SMemMat.N-parentIndex_8 -1]+=cE[Eidx_8 ]*dv_8 ;uHP[SMemMat.N-parentIndex_8 -1]-=cE[Eidx_8 ];}if(cSonNoVec[PIdx_9 ] ==1) {bHP[SMemMat.N-parentIndex_9 -1]+=cE[Eidx_9 ]*dv_9 ;uHP[SMemMat.N-parentIndex_9 -1]-=cE[Eidx_9 ];}if(cSonNoVec[PIdx_10 ] ==1) {bHP[SMemMat.N-parentIndex_10 -1]+=cE[Eidx_10 ]*dv_10 ;uHP[SMemMat.N-parentIndex_10 -1]-=cE[Eidx_10 ];}if(cSonNoVec[PIdx_11 ] ==1) {bHP[SMemMat.N-parentIndex_11 -1]+=cE[Eidx_11 ]*dv_11 ;uHP[SMemMat.N-parentIndex_11 -1]-=cE[Eidx_11 ];}if(cSonNoVec[PIdx_12 ] ==1) {bHP[SMemMat.N-parentIndex_12 -1]+=cE[Eidx_12 ]*dv_12 ;uHP[SMemMat.N-parentIndex_12 -1]-=cE[Eidx_12 ];}if(cSonNoVec[PIdx_13 ] ==1) {bHP[SMemMat.N-parentIndex_13 -1]+=cE[Eidx_13 ]*dv_13 ;uHP[SMemMat.N-parentIndex_13 -1]-=cE[Eidx_13 ];}
		// if(SonNo_3==2) {bHP[SMemMat.N-parentIndex_3-1]+=cE[Eidx_3]*dv_3;uHP[SMemMat.N-parentIndex_3-1]-=cE[Eidx_3];}
//#define THISCOMMANDHERE35(VARILP) if(SonNo_ ## VARILP ==2) {bHP[SMemMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[SMemMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
		//above line was changed to the no more variable sonno
		if(cSonNoVec[PIdx_1 ] ==2) {bHP[SMemMat.N-parentIndex_1 -1]+=cE[Eidx_1 ]*dv_1 ;uHP[SMemMat.N-parentIndex_1 -1]-=cE[Eidx_1 ];}if(cSonNoVec[PIdx_2 ] ==2) {bHP[SMemMat.N-parentIndex_2 -1]+=cE[Eidx_2 ]*dv_2 ;uHP[SMemMat.N-parentIndex_2 -1]-=cE[Eidx_2 ];}if(cSonNoVec[PIdx_3 ] ==2) {bHP[SMemMat.N-parentIndex_3 -1]+=cE[Eidx_3 ]*dv_3 ;uHP[SMemMat.N-parentIndex_3 -1]-=cE[Eidx_3 ];}if(cSonNoVec[PIdx_4 ] ==2) {bHP[SMemMat.N-parentIndex_4 -1]+=cE[Eidx_4 ]*dv_4 ;uHP[SMemMat.N-parentIndex_4 -1]-=cE[Eidx_4 ];}if(cSonNoVec[PIdx_5 ] ==2) {bHP[SMemMat.N-parentIndex_5 -1]+=cE[Eidx_5 ]*dv_5 ;uHP[SMemMat.N-parentIndex_5 -1]-=cE[Eidx_5 ];}if(cSonNoVec[PIdx_6 ] ==2) {bHP[SMemMat.N-parentIndex_6 -1]+=cE[Eidx_6 ]*dv_6 ;uHP[SMemMat.N-parentIndex_6 -1]-=cE[Eidx_6 ];}if(cSonNoVec[PIdx_7 ] ==2) {bHP[SMemMat.N-parentIndex_7 -1]+=cE[Eidx_7 ]*dv_7 ;uHP[SMemMat.N-parentIndex_7 -1]-=cE[Eidx_7 ];}if(cSonNoVec[PIdx_8 ] ==2) {bHP[SMemMat.N-parentIndex_8 -1]+=cE[Eidx_8 ]*dv_8 ;uHP[SMemMat.N-parentIndex_8 -1]-=cE[Eidx_8 ];}if(cSonNoVec[PIdx_9 ] ==2) {bHP[SMemMat.N-parentIndex_9 -1]+=cE[Eidx_9 ]*dv_9 ;uHP[SMemMat.N-parentIndex_9 -1]-=cE[Eidx_9 ];}if(cSonNoVec[PIdx_10 ] ==2) {bHP[SMemMat.N-parentIndex_10 -1]+=cE[Eidx_10 ]*dv_10 ;uHP[SMemMat.N-parentIndex_10 -1]-=cE[Eidx_10 ];}if(cSonNoVec[PIdx_11 ] ==2) {bHP[SMemMat.N-parentIndex_11 -1]+=cE[Eidx_11 ]*dv_11 ;uHP[SMemMat.N-parentIndex_11 -1]-=cE[Eidx_11 ];}if(cSonNoVec[PIdx_12 ] ==2) {bHP[SMemMat.N-parentIndex_12 -1]+=cE[Eidx_12 ]*dv_12 ;uHP[SMemMat.N-parentIndex_12 -1]-=cE[Eidx_12 ];}if(cSonNoVec[PIdx_13 ] ==2) {bHP[SMemMat.N-parentIndex_13 -1]+=cE[Eidx_13 ]*dv_13 ;uHP[SMemMat.N-parentIndex_13 -1]-=cE[Eidx_13 ];}
		syncthreads();
		BeforeLU(SMemMat,uHP,bHP,SMemMat.Depth);
#ifdef BKSUB1
		BkSub(SMemMat, PX,PF,uHP,bHP,SMemMat.LognDepth);
		//RBS Here i removed Vs[PIdx_ ## VARILP ]+=Vmid_ ## VARILP 
		#define THISCOMMANDHERE36(VARILP) Vmid_ ## VARILP =PX[SMemMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ; Vs[PIdx_ ## VARILP ]+=Vmid_ ## VARILP ;
		Vmid_1 =PX[SMemMat.N-PIdx_1 -1]; v_1 +=Vmid_1 ; Vmid_2 =PX[SMemMat.N-PIdx_2 -1]; v_2 +=Vmid_2 ; Vmid_3 =PX[SMemMat.N-PIdx_3 -1]; v_3 +=Vmid_3 ; Vmid_4 =PX[SMemMat.N-PIdx_4 -1]; v_4 +=Vmid_4 ; Vmid_5 =PX[SMemMat.N-PIdx_5 -1]; v_5 +=Vmid_5 ; Vmid_6 =PX[SMemMat.N-PIdx_6 -1]; v_6 +=Vmid_6 ;Vmid_7 =PX[SMemMat.N-PIdx_7 -1]; v_7 +=Vmid_7 ; Vmid_8 =PX[SMemMat.N-PIdx_8 -1]; v_8 +=Vmid_8 ; Vmid_9 =PX[SMemMat.N-PIdx_9 -1]; v_9 +=Vmid_9 ; Vmid_10 =PX[SMemMat.N-PIdx_10 -1]; v_10 +=Vmid_10 ; Vmid_11 =PX[SMemMat.N-PIdx_11 -1]; v_11 +=Vmid_11 ; Vmid_12 =PX[SMemMat.N-PIdx_12 -1]; v_12 +=Vmid_12 ; Vmid_13 =PX[SMemMat.N-PIdx_13 -1]; v_13 +=Vmid_13 ;
#endif

		
#ifdef BKSUB2
		#define THISCOMMANDHERE37(VARILP) MYFTYPE vTemp_ ## VARILP=Vs[PIdx_ ## VARILP];
		MYFTYPE vTemp_1=Vs[PIdx_1];MYFTYPE vTemp_2=Vs[PIdx_2];MYFTYPE vTemp_3=Vs[PIdx_3];MYFTYPE vTemp_4=Vs[PIdx_4];MYFTYPE vTemp_5=Vs[PIdx_5];MYFTYPE vTemp_6=Vs[PIdx_6];MYFTYPE vTemp_7=Vs[PIdx_7];MYFTYPE vTemp_8=Vs[PIdx_8];MYFTYPE vTemp_9=Vs[PIdx_9];MYFTYPE vTemp_10=Vs[PIdx_10];MYFTYPE vTemp_11=Vs[PIdx_11];MYFTYPE vTemp_12=Vs[PIdx_12];MYFTYPE vTemp_13=Vs[PIdx_13];
		syncthreads();
		BkSub(SMemMat,uHP,bHP,Vs,SMemMat.Depth);
		syncthreads();
		PX=Vs;

#define THISCOMMANDHERE38(VARILP) Vmid_ ## VARILP =PX[SMemMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ;
		Vmid_1 =PX[SMemMat.N-PIdx_1 -1]; v_1 +=Vmid_1 ;Vmid_2 =PX[SMemMat.N-PIdx_2 -1]; v_2 +=Vmid_2 ;Vmid_3 =PX[SMemMat.N-PIdx_3 -1]; v_3 +=Vmid_3 ;Vmid_4 =PX[SMemMat.N-PIdx_4 -1]; v_4 +=Vmid_4 ;Vmid_5 =PX[SMemMat.N-PIdx_5 -1]; v_5 +=Vmid_5 ;Vmid_6 =PX[SMemMat.N-PIdx_6 -1]; v_6 +=Vmid_6 ;Vmid_7 =PX[SMemMat.N-PIdx_7 -1]; v_7 +=Vmid_7 ;Vmid_8 =PX[SMemMat.N-PIdx_8 -1]; v_8 +=Vmid_8 ;Vmid_9 =PX[SMemMat.N-PIdx_9 -1]; v_9 +=Vmid_9 ;Vmid_10 =PX[SMemMat.N-PIdx_10 -1]; v_10 +=Vmid_10 ;Vmid_11 =PX[SMemMat.N-PIdx_11 -1]; v_11 +=Vmid_11 ;Vmid_12 =PX[SMemMat.N-PIdx_12 -1]; v_12 +=Vmid_12 ;Vmid_13 =PX[SMemMat.N-PIdx_13 -1]; v_13 +=Vmid_13 ;
		syncthreads(); 
#define THISCOMMANDHERE39(VARILP) Vs[PIdx_ ## VARILP ]= vTemp_ ## VARILP +Vmid_ ## VARILP ;
		Vs[PIdx_1 ]= vTemp_1 +Vmid_1 ;Vs[PIdx_2 ]= vTemp_2 +Vmid_2 ;Vs[PIdx_3 ]= vTemp_3 +Vmid_3 ;Vs[PIdx_4 ]= vTemp_4 +Vmid_4 ;Vs[PIdx_5 ]= vTemp_5 +Vmid_5 ;Vs[PIdx_6 ]= vTemp_6 +Vmid_6 ;Vs[PIdx_7 ]= vTemp_7 +Vmid_7 ;Vs[PIdx_8 ]= vTemp_8 +Vmid_8 ;Vs[PIdx_9 ]= vTemp_9 +Vmid_9 ;Vs[PIdx_10 ]= vTemp_10 +Vmid_10 ;Vs[PIdx_11 ]= vTemp_11 +Vmid_11 ;Vs[PIdx_12 ]= vTemp_12 +Vmid_12 ;Vs[PIdx_13 ]= vTemp_13 +Vmid_13 ;
#endif		
            t+=0.5*dt;

        

//      if(cBoolModel[PIdx_1 +0*SMemMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*SMemMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*SMemMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*SMemMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*SMemMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*SMemMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*SMemMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*SMemMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*SMemMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*SMemMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*SMemMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*SMemMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );}  if(cBoolModel[PIdx_3 +0*SMemMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*SMemMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*SMemMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*SMemMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*SMemMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*SMemMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );}
		  if(cBoolModel[PIdx_1 +0*SMemMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*SMemMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*SMemMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*SMemMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*SMemMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*SMemMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );} if(cBoolModel[PIdx_1 +6*SMemMat.N]){CuDerivModel_pas(dt, v_1,param_macro(28,PIdx_1) ,param_macro(29,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*SMemMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*SMemMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*SMemMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*SMemMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*SMemMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*SMemMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );} if(cBoolModel[PIdx_2 +6*SMemMat.N]){CuDerivModel_pas(dt, v_2,param_macro(28,PIdx_2) ,param_macro(29,PIdx_2) );}  if(cBoolModel[PIdx_3 +0*SMemMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*SMemMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*SMemMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*SMemMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*SMemMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*SMemMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );} if(cBoolModel[PIdx_3 +6*SMemMat.N]){CuDerivModel_pas(dt, v_3,param_macro(28,PIdx_3) ,param_macro(29,PIdx_3) );}  if(cBoolModel[PIdx_4 +0*SMemMat.N]){CuDerivModel_ca(dt, v_4,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*SMemMat.N]){CuDerivModel_cad(dt, v_4,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*SMemMat.N]){CuDerivModel_kca(dt, v_4,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*SMemMat.N]){CuDerivModel_km(dt, v_4,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*SMemMat.N]){CuDerivModel_kv(dt, v_4,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*SMemMat.N]){CuDerivModel_na(dt, v_4,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );} if(cBoolModel[PIdx_4 +6*SMemMat.N]){CuDerivModel_pas(dt, v_4,param_macro(28,PIdx_4) ,param_macro(29,PIdx_4) );}  if(cBoolModel[PIdx_5 +0*SMemMat.N]){CuDerivModel_ca(dt, v_5,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*SMemMat.N]){CuDerivModel_cad(dt, v_5,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*SMemMat.N]){CuDerivModel_kca(dt, v_5,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*SMemMat.N]){CuDerivModel_km(dt, v_5,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*SMemMat.N]){CuDerivModel_kv(dt, v_5,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*SMemMat.N]){CuDerivModel_na(dt, v_5,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );} if(cBoolModel[PIdx_5 +6*SMemMat.N]){CuDerivModel_pas(dt, v_5,param_macro(28,PIdx_5) ,param_macro(29,PIdx_5) );}  if(cBoolModel[PIdx_6 +0*SMemMat.N]){CuDerivModel_ca(dt, v_6,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*SMemMat.N]){CuDerivModel_cad(dt, v_6,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*SMemMat.N]){CuDerivModel_kca(dt, v_6,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*SMemMat.N]){CuDerivModel_km(dt, v_6,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*SMemMat.N]){CuDerivModel_kv(dt, v_6,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*SMemMat.N]){CuDerivModel_na(dt, v_6,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );} if(cBoolModel[PIdx_6 +6*SMemMat.N]){CuDerivModel_pas(dt, v_6,param_macro(28,PIdx_6) ,param_macro(29,PIdx_6) );}  if(cBoolModel[PIdx_7 +0*SMemMat.N]){CuDerivModel_ca(dt, v_7,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*SMemMat.N]){CuDerivModel_cad(dt, v_7,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*SMemMat.N]){CuDerivModel_kca(dt, v_7,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*SMemMat.N]){CuDerivModel_km(dt, v_7,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*SMemMat.N]){CuDerivModel_kv(dt, v_7,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*SMemMat.N]){CuDerivModel_na(dt, v_7,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );} if(cBoolModel[PIdx_7 +6*SMemMat.N]){CuDerivModel_pas(dt, v_7,param_macro(28,PIdx_7) ,param_macro(29,PIdx_7) );}  if(cBoolModel[PIdx_8 +0*SMemMat.N]){CuDerivModel_ca(dt, v_8,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*SMemMat.N]){CuDerivModel_cad(dt, v_8,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*SMemMat.N]){CuDerivModel_kca(dt, v_8,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*SMemMat.N]){CuDerivModel_km(dt, v_8,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*SMemMat.N]){CuDerivModel_kv(dt, v_8,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*SMemMat.N]){CuDerivModel_na(dt, v_8,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );} if(cBoolModel[PIdx_8 +6*SMemMat.N]){CuDerivModel_pas(dt, v_8,param_macro(28,PIdx_8) ,param_macro(29,PIdx_8) );}  if(cBoolModel[PIdx_9 +0*SMemMat.N]){CuDerivModel_ca(dt, v_9,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*SMemMat.N]){CuDerivModel_cad(dt, v_9,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*SMemMat.N]){CuDerivModel_kca(dt, v_9,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*SMemMat.N]){CuDerivModel_km(dt, v_9,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*SMemMat.N]){CuDerivModel_kv(dt, v_9,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*SMemMat.N]){CuDerivModel_na(dt, v_9,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );} if(cBoolModel[PIdx_9 +6*SMemMat.N]){CuDerivModel_pas(dt, v_9,param_macro(28,PIdx_9) ,param_macro(29,PIdx_9) );}  if(cBoolModel[PIdx_10 +0*SMemMat.N]){CuDerivModel_ca(dt, v_10,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*SMemMat.N]){CuDerivModel_cad(dt, v_10,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*SMemMat.N]){CuDerivModel_kca(dt, v_10,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*SMemMat.N]){CuDerivModel_km(dt, v_10,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*SMemMat.N]){CuDerivModel_kv(dt, v_10,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*SMemMat.N]){CuDerivModel_na(dt, v_10,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );} if(cBoolModel[PIdx_10 +6*SMemMat.N]){CuDerivModel_pas(dt, v_10,param_macro(28,PIdx_10) ,param_macro(29,PIdx_10) );}  if(cBoolModel[PIdx_11 +0*SMemMat.N]){CuDerivModel_ca(dt, v_11,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*SMemMat.N]){CuDerivModel_cad(dt, v_11,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*SMemMat.N]){CuDerivModel_kca(dt, v_11,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*SMemMat.N]){CuDerivModel_km(dt, v_11,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*SMemMat.N]){CuDerivModel_kv(dt, v_11,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*SMemMat.N]){CuDerivModel_na(dt, v_11,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );} if(cBoolModel[PIdx_11 +6*SMemMat.N]){CuDerivModel_pas(dt, v_11,param_macro(28,PIdx_11) ,param_macro(29,PIdx_11) );}  if(cBoolModel[PIdx_12 +0*SMemMat.N]){CuDerivModel_ca(dt, v_12,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*SMemMat.N]){CuDerivModel_cad(dt, v_12,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*SMemMat.N]){CuDerivModel_kca(dt, v_12,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*SMemMat.N]){CuDerivModel_km(dt, v_12,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*SMemMat.N]){CuDerivModel_kv(dt, v_12,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*SMemMat.N]){CuDerivModel_na(dt, v_12,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );} if(cBoolModel[PIdx_12 +6*SMemMat.N]){CuDerivModel_pas(dt, v_12,param_macro(28,PIdx_12) ,param_macro(29,PIdx_12) );}  if(cBoolModel[PIdx_13 +0*SMemMat.N]){CuDerivModel_ca(dt, v_13,ModelStates_13[0],ModelStates_13[1],param_macro(0,PIdx_13) ,param_macro(1,PIdx_13) ,ModelStates_13[8],ModelStates_13[9]);} if(cBoolModel[PIdx_13 +1*SMemMat.N]){CuDerivModel_cad(dt, v_13,ModelStates_13[2],ModelStates_13[9],ModelStates_13[8]);} if(cBoolModel[PIdx_13 +2*SMemMat.N]){CuDerivModel_kca(dt, v_13,ModelStates_13[3],param_macro(2,PIdx_13) ,param_macro(3,PIdx_13) ,param_macro(4,PIdx_13) ,param_macro(5,PIdx_13) ,ModelStates_13[8]);} if(cBoolModel[PIdx_13 +3*SMemMat.N]){CuDerivModel_km(dt, v_13,ModelStates_13[4],param_macro(6,PIdx_13) ,param_macro(7,PIdx_13) ,param_macro(8,PIdx_13) ,param_macro(9,PIdx_13) ,param_macro(10,PIdx_13) );} if(cBoolModel[PIdx_13 +4*SMemMat.N]){CuDerivModel_kv(dt, v_13,ModelStates_13[5],param_macro(11,PIdx_13) ,param_macro(12,PIdx_13) ,param_macro(13,PIdx_13) ,param_macro(14,PIdx_13) ,param_macro(15,PIdx_13) );} if(cBoolModel[PIdx_13 +5*SMemMat.N]){CuDerivModel_na(dt, v_13,ModelStates_13[6],ModelStates_13[7],param_macro(16,PIdx_13) ,param_macro(17,PIdx_13) ,param_macro(18,PIdx_13) ,param_macro(19,PIdx_13) ,param_macro(20,PIdx_13) ,param_macro(21,PIdx_13) ,param_macro(22,PIdx_13) ,param_macro(23,PIdx_13) ,param_macro(24,PIdx_13) ,param_macro(25,PIdx_13) ,param_macro(26,PIdx_13) ,param_macro(27,PIdx_13) );} if(cBoolModel[PIdx_13 +6*SMemMat.N]){CuDerivModel_pas(dt, v_13,param_macro(28,PIdx_13) ,param_macro(29,PIdx_13) );}

 
 
    }
	//This one looks suspicious but leaving it and will check it later.
   for(int recInd =0;recInd<sim.NRecSites;recInd++){
		VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + Nt-WARPSIZE+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
	}
 
}

__global__ void NeuroGPUKernel(Stim stim,  Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal,MYDTYPE CompDepth,MYDTYPE CompFDepth,hipTextureObject_t tex)
{
	HMat SMemMat;
	MYFTYPE *Vs,*amps,*SMemVHot;
	MYDTYPE offset;
	
	
	offset = InitializeDeviceMemory(stim,sim,InMat,VHotGlobal,CompDepth,CompFDepth,&SMemMat,Vs,amps,SMemVHot);
	runSimulation(SMemMat,InMat,V,stim,sim,VHotGlobal,SMemVHot,amps,offset,tex);

}
void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}
MYFTYPE* ReadAllParams(const char* FN,MYDTYPE NParams,MYDTYPE Nx,MYDTYPE  &ntemp) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	//sprintf(FileName,"%sForC.mat",FN);
	MYFTYPE* ans;
	MYDTYPE tmp;
	FILE *fl = fopen(FN, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return nullptr;
	}
	fread(&tmp, sizeof(MYDTYPE),1, fl);
		ntemp = tmp;

	ans=(MYFTYPE *)malloc(Nx * NPARAMS * tmp * sizeof(MYFTYPE ));
	fread(ans, sizeof(MYFTYPE), tmp*Nx*NParams, fl);
	fclose(fl);
	return ans;
}
void ReadParamsMat(const char* FN,MYFTYPE** ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	for(int i=0;i<NParams;i++){
		ParamsM[i]=(MYFTYPE*)malloc(Nx * sizeof(MYFTYPE));
		fread(ParamsM[i], sizeof(MYFTYPE), Nx, fl);
	}
	fclose(fl);
}


hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) { 

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(1);
	cudaStatus = hipDeviceReset();
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=stim.Nt;
	hipHostMalloc((void**)&VHotsHost,stim.NStimuli*Nt*sim.NRecSites*NSets*sizeof(MYFTYPE));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.NComps=InMat.NComps;
	Mat_d.Depth=InMat.Depth;
	Mat_d.NModels=InMat.NModels;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	Mat_d.nLRel=InMat.nLRel;
	Mat_d.nFLRel=InMat.nFLRel;
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	stim_d.Nt = stim.Nt;
	Sim sim_d; 	 			
	sim_d.Celsius = sim.Celsius; 	 	
	sim_d.dt = sim.dt; 	 	
	sim_d.NRecSites = sim.NRecSites; 	 	
	sim_d.TFinal = sim.TFinal; 
#ifndef STIMFROMFILE
    cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE

    stim_d.numofdts = stim.numofdts;
    cudaStatus = hipMalloc((void**)&stim_d.dtInds, stim_d.numofdts * sizeof(MYDTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.numofdts * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE));
#endif // STIMFROMFILE
	cudaStatus = hipMalloc((void**)&sim_d.RecSites, sim_d.NRecSites * sizeof(MYDTYPE));
	


	//cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * InMat.NModels* sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*Nt *stim.NStimuli* sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*sim_d.NRecSites*Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N +1)* sizeof(MYDTYPE));
#endif
	/*cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (CompDepth)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (CompFDepth)*WARPSIZE* sizeof(MYDTYPE));

	cudaStatus = hipMalloc((void**)&Mat_d.LRelStarts, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.LRelEnds, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelStarts, Mat_d.nFLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelEnds, Mat_d.nFLRel* sizeof(MYDTYPE));
*/
	

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	#ifndef STIMFROMFILE
 
    cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
	
    cudaStatus = hipMemcpy(stim_d.dtInds, stim.dtInds, stim.numofdts * sizeof(MYDTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.numofdts * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // STIMFROMFILE
	cudaStatus = hipMemcpy(sim_d.RecSites, sim.RecSites, sim_d.NRecSites * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cE), InMat.e, 416 * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cF), InMat.f, 416 * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cKs), InMat.Ks, 416 * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(Mat_d.SegToComp, InMat.SegToComp, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegToComp), InMat.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cBoolModel), InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE));

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCm), InMat.Cms, 416 * sizeof(MYFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSonNoVec), InMat.SonNoVec, InMat.N * sizeof(MYDTYPE));
	
	
	//cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelStarts), InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelEnds), InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelVec), InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegStartI), InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegEndI), InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFathers), InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFIdxs), InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMemcpy(Mat_d.KsB, InMat.KsB,(InMat.N+1)*sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif

	//cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByLevel32), InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByFLevel32), InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE));


	//cudaStatus = hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelStarts), InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelEnds), InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelStarts), InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelEnds), InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE));


	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation

	

	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}
	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(NSets);

	MYDTYPE offset=0;
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
#ifdef BKSUB1
	//offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
#endif
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*InMat.NModels*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, SegToComp,boolModel
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	//offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	//offset+=WARPSIZE*(CompDepth)*sizeof(MYDTYPE);
	//offset+=WARPSIZE*(CompFDepth)*sizeof(MYDTYPE);
	//// 4 CompByLevel32 CompByFLevel32
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
#ifdef BKSUB2
	offset+=(Mat_d.N+1)*sizeof(MYDTYPE);
#endif
	// 5 LRelStarts LRelEnds FLRelStarts  FLRelEnds KsB
	//offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	MYDTYPE FrameworkMemSize=offset;
//PerStimulus

	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);  // uHP (Diag)
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE); // bHP (rhs)
	offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE)+1)*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE); // Vs

	offset+=WARPSIZE*sim.NRecSites*sizeof(MYFTYPE);//SMemVHot
#ifdef STIMFROMFILE
    offset+=WARPSIZE*sizeof(MYFTYPE);//amps
#endif // STIMFROMFILE
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+PerStimulus*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
	clock_t begin,end;
	begin=clock();
		MYDTYPE currKernelRun = NKERNEL;
	MYDTYPE prevRuns = 0;
	int memSizeForVHotGlobal = Nt*stim.NStimuli*sim.NRecSites;
	MYDTYPE memSizeForModelParams = NPARAMS * InMat.NComps;
	MYDTYPE streamID=0;
	hipStream_t stream0,stream1,stream2,stream3;
	hipStreamCreate (&stream0);
	hipStreamCreate (&stream1);
	hipStreamCreate (&stream2);
	hipStreamCreate (&stream3);
	begin=clock();
	while (prevRuns<NSets){
	if(currKernelRun>(NSets-prevRuns)){
		currKernelRun = (NSets-prevRuns);
	}
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.NComps *NSets * sizeof(MYFTYPE));
	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}


	// copying
	
	//cudaStatus = hipMemcpy(d_modelParams, ParamsM, NPARAMS * InMat.NComps * NSets *sizeof(MYFTYPE), hipMemcpyHostToDevice);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* paramsArr;
    hipMallocArray(&paramsArr, &channelDesc, NPARAMS * InMat.NComps, NKERNEL);



	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));

	resDesc.resType = hipResourceTypeArray; 
	resDesc.res.array.array = paramsArr;
	hipMemcpyToArray(paramsArr, 0, 0, ParamsM, NPARAMS * InMat.NComps *NKERNEL * sizeof(MYFTYPE),
                      hipMemcpyHostToDevice);

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	if(streamID==0){
		
	
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream0>>>(stim_d, sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth,tex); // RRR	
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream0); 
		
	}
	if(streamID==1){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream1>>>(stim_d, sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth,tex); // RRR

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream1); 

	}
	if(streamID==2){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream2>>>(stim_d,  sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth,tex); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt *sim.NRecSites *  stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream2); 

	}
	if(streamID==3){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream3>>>(stim_d, sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth,tex); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream3); 

	}
	prevRuns +=currKernelRun;
	if(streamID==3){
		streamID == 0;
	}	else {
		streamID +=1;
	}
	}
	hipDeviceSynchronize();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}
	//cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal,NSets * Nt * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost); 

	end=clock();
	double totalT = diffclock(end,begin);
	printf("stEfork2TimeLoopGPU took %g milliseconds\n",totalT);
	FILE *file = fopen(TIMES_FN, "wb");
	if ( file ) {
		MYDTYPE mul32 = MUL32;
		fwrite(&mul32,sizeof(MYDTYPE),1,file);
		fwrite(&totalT,sizeof(double),1,file);
	} else {
		printf("ERR SaveArrayToFile %s\n",TIMES_FN);
	}
	fclose(file);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,NSets*Nt*stim.NStimuli*sim.NRecSites,VHotsHost);
	
	return cudaStatus;
}


hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	
	  stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V,CompDepth,CompFDepth,NSets); //RRR sim
	  
	
	


	return cudaStatus;
}
