
#include "Util.h"
#include <direct.h>

int main()
{
		int nDevices;

		hipGetDeviceCount(&nDevices);
		for (int i = 0; i < nDevices; i++) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		}

	 //RunByModelSerial();// FOR ROY!!!!!!!!!!!!!!! // Run, output VHot and look
	 /*Add to first kernel
	 cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	*/
	
	//printf("we are in the %s directory\n",_getcwd( NULL, 0 ));
	RunByModelP();
	
    
    return 0;
}
