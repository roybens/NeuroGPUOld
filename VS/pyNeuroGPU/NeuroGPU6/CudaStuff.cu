#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
#include "AllModels.cu"
#include "AllModels.cuh"

__constant__ MYFTYPE cCm[NSEG];
__constant__ MYSECONDFTYPE cE[NSEG];
__constant__ MYSECONDFTYPE cF[NSEG];
__constant__ MYDTYPE cFIdxs[NSEG*LOG_N_DEPTH];
__constant__ MYDTYPE cKs[NSEG];
__constant__ MYDTYPE cSegToComp[NSEG];
__constant__ MYDTYPE cBoolModel[NSEG * N_MODELS];//One day change this to bool
__constant__ MYDTYPE cRelStarts[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelEnds[N_FATHERS];//nFathers
__constant__ MYDTYPE cFathers[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelVec[N_CALL_FOR_FATHER];//nCallForFather
__constant__ MYDTYPE cSegStartI[N_CALL_FOR_FATHER+1];//nCallForFather
__constant__ MYDTYPE cSegEndI[N_CALL_FOR_FATHER+1];//nCallForFather

__constant__ MYDTYPE cCompByLevel32[COMP_DEPTH*WARPSIZE];//CompDepth
__constant__ MYDTYPE cCompByFLevel32[COMP_DEPTH*WARPSIZE];//CompFDepth
__constant__ MYDTYPE cLRelStarts[N_L_REL];//nLRel
__constant__ MYDTYPE cLRelEnds[N_L_REL];//nLRel
__constant__ MYDTYPE cFLRelStarts[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cFLRelEnds[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cSonNoVec[NSEG];//InMat.N 


#ifdef ILP1
	#define SUPERILPMACRO(x) ; ## x ## (1);
#endif
#ifdef ILP2
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2);
#endif
#ifdef ILP3
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3);
#endif
#ifdef ILP4
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); 
#endif
#ifdef ILP5
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); 
#endif
#ifdef ILP6
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6);
#endif
#ifdef ILP7
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7);
#endif
#ifdef ILP8
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8);
#endif
#ifdef ILP9
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9);
#endif
#ifdef ILP10
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10);
#endif
#ifdef ILP11
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11);
#endif
#ifdef ILP12
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12);
#endif
#ifdef ILP13
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13);
#endif
#ifdef ILP14
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14);
#endif
#ifdef ILP15
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15);
#endif
#ifdef ILP16
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16);
#endif
#ifdef ILP17
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17);
#endif
#ifdef ILP18
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18);
#endif
#ifdef ILP19
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19);
#endif
#ifdef ILP20
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20);
#endif
#ifdef ILP21
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21);
#endif
#ifdef ILP22
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22);
#endif
#ifdef ILP23
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23);
#endif
#ifdef ILP24
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24);
#endif
#ifdef ILP25
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25);
#endif
#ifdef ILP26
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26);
#endif
#ifdef ILP27
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27);
#endif
#ifdef ILP28
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28);
#endif
#ifdef ILP29
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29);
#endif
#ifdef ILP30
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30);
#endif
#ifdef ILP31
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31);
#endif
#ifdef ILP32
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); 
#endif
#ifdef ILP33
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33);
#endif
#ifdef ILP34
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34);
#endif
#ifdef ILP35
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35);
#endif
#ifdef ILP36
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36);
#endif
#ifdef ILP37
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37);
#endif
#ifdef ILP38
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38);
#endif
#ifdef ILP39
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39);
#endif
#ifdef ILP40
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40);
#endif
#ifdef ILP41
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41);
#endif
#ifdef ILP42
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42);
#endif
#ifdef ILP43
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43);
#endif
#ifdef ILP44
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44);
#endif
#ifdef ILP45
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45);
#endif
#ifdef ILP46
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46);
#endif
#ifdef ILP47
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47);
#endif
#ifdef ILP48
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48);
#endif
#ifdef ILP49
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49);
#endif
#ifdef ILP50
	#define SUPERILPMACRO(x) ; ## x ## (1); ## x ## (2); ## x ## (3); ## x ## (4); ## x ## (5); ## x ## (6); ## x ## (7); ## x ## (8); ## x ## (9); ## x ## (10); ## x ## (11); ## x ## (12); ## x ## (13); ## x ## (14); ## x ## (15); ## x ## (16); ## x ## (17); ## x ## (18); ## x ## (19); ## x ## (20); ## x ## (21); ## x ## (22); ## x ## (23); ## x ## (24); ## x ## (25); ## x ## (26); ## x ## (27); ## x ## (28); ## x ## (29); ## x ## (30); ## x ## (31); ## x ## (32); ## x ## (33); ## x ## (34); ## x ## (35); ## x ## (36); ## x ## (37); ## x ## (38); ## x ## (39); ## x ## (40); ## x ## (41); ## x ## (42); ## x ## (43); ## x ## (44); ## x ## (45); ## x ## (46); ## x ## (47); ## x ## (48); ## x ## (49); ## x ## (50);
#endif

__device__ void BeforeLU(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx=threadIdx.x;
	MYDTYPE i,j,CurJ,CurB,t, CurLevel,LRelIndex;
	MYDTYPE JumctionI;
	LRelIndex=cLRelStarts[CurLevel];
	LRelIndex=LRelIndex+cLRelEnds[CurLevel];
	for(CurLevel=0;CurLevel<=Depth;CurLevel++) {
		
		for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
		//for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx]-1;
			for(i=cSegStartI[JumctionI]-1;i<cSegEndI[JumctionI];i++) {
				MYSECONDFTYPE uHPm1=uHP[i-1];
			
				uHP[i]=uHP[i]-cF[i-1]*(cE[i-1]/uHPm1); // So far same as paper parallel
				uHPm1=uHP[i-1];
				MYSECONDFTYPE bHPm1=bHP[i-1];
				bHP[i]=bHP[i]-bHPm1*cE[i-1]/uHPm1; // bH is y
			}
		}
		if(CurLevel<Depth) {
			for(LRelIndex=cFLRelStarts[CurLevel];LRelIndex<=cFLRelEnds[CurLevel];LRelIndex++){ 
				CurB=cCompByFLevel32[(LRelIndex)*WARPSIZE+PIdx]-1;//RB i inserted another  -1 into the index RB 2 i removed the-1 from the curlevel 
				CurJ=cFathers[CurB]-1;
				MYDTYPE St=cRelStarts[CurB];
				MYDTYPE En=cRelEnds[CurB];
				for(j=St;j<=En;j++) {
					t=cRelVec[j-1]-1;
					MYSECONDFTYPE uHPm1=uHP[t-1];
					uHP[CurJ]-=cF[t-1]*(cE[t-1]/uHPm1); 
					uHPm1=uHP[t-1];
					MYSECONDFTYPE bHPm1=bHP[t-1];
					bHP[CurJ]-=bHPm1*cE[t-1]/uHPm1; 
				}
			}
		}	
	}
}

#ifdef BKSUB1
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* PX, MYSECONDFTYPE* PF,MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE LognDepth)
{
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
#define THISCOMMANDHERE1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*(## VARILP -1)); MYDTYPE NextID_ ## VARILP ;
MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); MYDTYPE NextID_1 ;
MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); MYDTYPE NextID_2 ;
MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1)); MYDTYPE NextID_3 ;
MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1)); MYDTYPE NextID_4 ;
MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1)); MYDTYPE NextID_5 ;
MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1)); MYDTYPE NextID_6 ;
MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1)); MYDTYPE NextID_7 ;
MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1)); MYDTYPE NextID_8 ;
MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1)); MYDTYPE NextID_9 ;
MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1)); MYDTYPE NextID_10 ;
MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1)); MYDTYPE NextID_11 ;
MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1)); MYDTYPE NextID_12 ;
	MYDTYPE i;
	
	PX=bHP;
	PF=uHP;
	// PX[PIdx_1]=PX[PIdx_1]/PF[PIdx_1]; // PF[PIdx_1]=-cF[PIdx_1]/PF[PIdx_1];
#define THISCOMMANDHERE2(VARILP) PX[PIdx_ ## VARILP ]=PX[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ]; PF[PIdx_ ## VARILP ]=-cF[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ];
PX[PIdx_1 ]=PX[PIdx_1 ]/PF[PIdx_1 ]; PF[PIdx_1 ]=-cF[PIdx_1 ]/PF[PIdx_1 ];
PX[PIdx_2 ]=PX[PIdx_2 ]/PF[PIdx_2 ]; PF[PIdx_2 ]=-cF[PIdx_2 ]/PF[PIdx_2 ];
PX[PIdx_3 ]=PX[PIdx_3 ]/PF[PIdx_3 ]; PF[PIdx_3 ]=-cF[PIdx_3 ]/PF[PIdx_3 ];
PX[PIdx_4 ]=PX[PIdx_4 ]/PF[PIdx_4 ]; PF[PIdx_4 ]=-cF[PIdx_4 ]/PF[PIdx_4 ];
PX[PIdx_5 ]=PX[PIdx_5 ]/PF[PIdx_5 ]; PF[PIdx_5 ]=-cF[PIdx_5 ]/PF[PIdx_5 ];
PX[PIdx_6 ]=PX[PIdx_6 ]/PF[PIdx_6 ]; PF[PIdx_6 ]=-cF[PIdx_6 ]/PF[PIdx_6 ];
PX[PIdx_7 ]=PX[PIdx_7 ]/PF[PIdx_7 ]; PF[PIdx_7 ]=-cF[PIdx_7 ]/PF[PIdx_7 ];
PX[PIdx_8 ]=PX[PIdx_8 ]/PF[PIdx_8 ]; PF[PIdx_8 ]=-cF[PIdx_8 ]/PF[PIdx_8 ];
PX[PIdx_9 ]=PX[PIdx_9 ]/PF[PIdx_9 ]; PF[PIdx_9 ]=-cF[PIdx_9 ]/PF[PIdx_9 ];
PX[PIdx_10 ]=PX[PIdx_10 ]/PF[PIdx_10 ]; PF[PIdx_10 ]=-cF[PIdx_10 ]/PF[PIdx_10 ];
PX[PIdx_11 ]=PX[PIdx_11 ]/PF[PIdx_11 ]; PF[PIdx_11 ]=-cF[PIdx_11 ]/PF[PIdx_11 ];
PX[PIdx_12 ]=PX[PIdx_12 ]/PF[PIdx_12 ]; PF[PIdx_12 ]=-cF[PIdx_12 ]/PF[PIdx_12 ];

	PX[InMat.N]=0;
	PF[InMat.N]=1;
	for(i=0;i<LognDepth;i++) {	
		// NextID_1=cFIdxs[i*InMat.N+PIdx_1]-1; // MYFTYPE OldPXj_1=PX[PIdx_1]; // MYFTYPE OldPXNextID_1=PX[NextID_1]; // PX[PIdx_1]=OldPXj_1+OldPXNextID_1*PF[PIdx_1];
#define THISCOMMANDHERE3(VARILP) NextID_ ## VARILP =cFIdxs[i*InMat.N+PIdx_ ## VARILP ]-1; MYFTYPE OldPXj_ ## VARILP =PX[PIdx_ ## VARILP ]; MYFTYPE OldPXNextID_ ## VARILP =PX[NextID_ ## VARILP ]; PX[PIdx_ ## VARILP ]=OldPXj_ ## VARILP +OldPXNextID_ ## VARILP *PF[PIdx_ ## VARILP ];
NextID_1 =cFIdxs[i*InMat.N+PIdx_1 ]-1; MYFTYPE OldPXj_1 =PX[PIdx_1 ]; MYFTYPE OldPXNextID_1 =PX[NextID_1 ]; PX[PIdx_1 ]=OldPXj_1 +OldPXNextID_1 *PF[PIdx_1 ];
NextID_2 =cFIdxs[i*InMat.N+PIdx_2 ]-1; MYFTYPE OldPXj_2 =PX[PIdx_2 ]; MYFTYPE OldPXNextID_2 =PX[NextID_2 ]; PX[PIdx_2 ]=OldPXj_2 +OldPXNextID_2 *PF[PIdx_2 ];
NextID_3 =cFIdxs[i*InMat.N+PIdx_3 ]-1; MYFTYPE OldPXj_3 =PX[PIdx_3 ]; MYFTYPE OldPXNextID_3 =PX[NextID_3 ]; PX[PIdx_3 ]=OldPXj_3 +OldPXNextID_3 *PF[PIdx_3 ];
NextID_4 =cFIdxs[i*InMat.N+PIdx_4 ]-1; MYFTYPE OldPXj_4 =PX[PIdx_4 ]; MYFTYPE OldPXNextID_4 =PX[NextID_4 ]; PX[PIdx_4 ]=OldPXj_4 +OldPXNextID_4 *PF[PIdx_4 ];
NextID_5 =cFIdxs[i*InMat.N+PIdx_5 ]-1; MYFTYPE OldPXj_5 =PX[PIdx_5 ]; MYFTYPE OldPXNextID_5 =PX[NextID_5 ]; PX[PIdx_5 ]=OldPXj_5 +OldPXNextID_5 *PF[PIdx_5 ];
NextID_6 =cFIdxs[i*InMat.N+PIdx_6 ]-1; MYFTYPE OldPXj_6 =PX[PIdx_6 ]; MYFTYPE OldPXNextID_6 =PX[NextID_6 ]; PX[PIdx_6 ]=OldPXj_6 +OldPXNextID_6 *PF[PIdx_6 ];
NextID_7 =cFIdxs[i*InMat.N+PIdx_7 ]-1; MYFTYPE OldPXj_7 =PX[PIdx_7 ]; MYFTYPE OldPXNextID_7 =PX[NextID_7 ]; PX[PIdx_7 ]=OldPXj_7 +OldPXNextID_7 *PF[PIdx_7 ];
NextID_8 =cFIdxs[i*InMat.N+PIdx_8 ]-1; MYFTYPE OldPXj_8 =PX[PIdx_8 ]; MYFTYPE OldPXNextID_8 =PX[NextID_8 ]; PX[PIdx_8 ]=OldPXj_8 +OldPXNextID_8 *PF[PIdx_8 ];
NextID_9 =cFIdxs[i*InMat.N+PIdx_9 ]-1; MYFTYPE OldPXj_9 =PX[PIdx_9 ]; MYFTYPE OldPXNextID_9 =PX[NextID_9 ]; PX[PIdx_9 ]=OldPXj_9 +OldPXNextID_9 *PF[PIdx_9 ];
NextID_10 =cFIdxs[i*InMat.N+PIdx_10 ]-1; MYFTYPE OldPXj_10 =PX[PIdx_10 ]; MYFTYPE OldPXNextID_10 =PX[NextID_10 ]; PX[PIdx_10 ]=OldPXj_10 +OldPXNextID_10 *PF[PIdx_10 ];
NextID_11 =cFIdxs[i*InMat.N+PIdx_11 ]-1; MYFTYPE OldPXj_11 =PX[PIdx_11 ]; MYFTYPE OldPXNextID_11 =PX[NextID_11 ]; PX[PIdx_11 ]=OldPXj_11 +OldPXNextID_11 *PF[PIdx_11 ];
NextID_12 =cFIdxs[i*InMat.N+PIdx_12 ]-1; MYFTYPE OldPXj_12 =PX[PIdx_12 ]; MYFTYPE OldPXNextID_12 =PX[NextID_12 ]; PX[PIdx_12 ]=OldPXj_12 +OldPXNextID_12 *PF[PIdx_12 ];

		// PX[j]=PX[j]+PX[NextID]*PF[j];
		// PF[j]=PF[j]*PF[NextID];
		// MYFTYPE OldPFj_1=PF[PIdx_1]; // MYFTYPE OldPFNextID_1=PF[NextID_1]; // PF[PIdx_1]=OldPFj_1*OldPFNextID_1;
#define THISCOMMANDHERE4(VARILP) MYFTYPE OldPFj_ ## VARILP =PF[PIdx_ ## VARILP ]; MYFTYPE OldPFNextID_ ## VARILP =PF[NextID_ ## VARILP ]; PF[PIdx_ ## VARILP ]=OldPFj_ ## VARILP *OldPFNextID_ ## VARILP ;
MYFTYPE OldPFj_1 =PF[PIdx_1 ]; MYFTYPE OldPFNextID_1 =PF[NextID_1 ]; PF[PIdx_1 ]=OldPFj_1 *OldPFNextID_1 ;
MYFTYPE OldPFj_2 =PF[PIdx_2 ]; MYFTYPE OldPFNextID_2 =PF[NextID_2 ]; PF[PIdx_2 ]=OldPFj_2 *OldPFNextID_2 ;
MYFTYPE OldPFj_3 =PF[PIdx_3 ]; MYFTYPE OldPFNextID_3 =PF[NextID_3 ]; PF[PIdx_3 ]=OldPFj_3 *OldPFNextID_3 ;
MYFTYPE OldPFj_4 =PF[PIdx_4 ]; MYFTYPE OldPFNextID_4 =PF[NextID_4 ]; PF[PIdx_4 ]=OldPFj_4 *OldPFNextID_4 ;
MYFTYPE OldPFj_5 =PF[PIdx_5 ]; MYFTYPE OldPFNextID_5 =PF[NextID_5 ]; PF[PIdx_5 ]=OldPFj_5 *OldPFNextID_5 ;
MYFTYPE OldPFj_6 =PF[PIdx_6 ]; MYFTYPE OldPFNextID_6 =PF[NextID_6 ]; PF[PIdx_6 ]=OldPFj_6 *OldPFNextID_6 ;
MYFTYPE OldPFj_7 =PF[PIdx_7 ]; MYFTYPE OldPFNextID_7 =PF[NextID_7 ]; PF[PIdx_7 ]=OldPFj_7 *OldPFNextID_7 ;
MYFTYPE OldPFj_8 =PF[PIdx_8 ]; MYFTYPE OldPFNextID_8 =PF[NextID_8 ]; PF[PIdx_8 ]=OldPFj_8 *OldPFNextID_8 ;
MYFTYPE OldPFj_9 =PF[PIdx_9 ]; MYFTYPE OldPFNextID_9 =PF[NextID_9 ]; PF[PIdx_9 ]=OldPFj_9 *OldPFNextID_9 ;
MYFTYPE OldPFj_10 =PF[PIdx_10 ]; MYFTYPE OldPFNextID_10 =PF[NextID_10 ]; PF[PIdx_10 ]=OldPFj_10 *OldPFNextID_10 ;
MYFTYPE OldPFj_11 =PF[PIdx_11 ]; MYFTYPE OldPFNextID_11 =PF[NextID_11 ]; PF[PIdx_11 ]=OldPFj_11 *OldPFNextID_11 ;
MYFTYPE OldPFj_12 =PF[PIdx_12 ]; MYFTYPE OldPFNextID_12 =PF[NextID_12 ]; PF[PIdx_12 ]=OldPFj_12 *OldPFNextID_12 ;
	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP,MYSECONDFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1)); Out[PIdx_1 ]=0;
MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1)); Out[PIdx_2 ]=0;
MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1)); Out[PIdx_3 ]=0;
MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1)); Out[PIdx_4 ]=0;
MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1)); Out[PIdx_5 ]=0;
MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1)); Out[PIdx_6 ]=0;
MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1)); Out[PIdx_7 ]=0;
MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1)); Out[PIdx_8 ]=0;
MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1)); Out[PIdx_9 ]=0;
MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1)); Out[PIdx_10 ]=0;
MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1)); Out[PIdx_11 ]=0;
MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1)); Out[PIdx_12 ]=0;

	MYDTYPE j,CurJ,CurB,t;
	MYDTYPE JumctionI;
	short CurLevel,i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex,k;
	MYFTYPE temp;
 for(CurLevel=Depth;CurLevel>=0;CurLevel--) {
  //     Run all independent set for this level, in parallel
  // for JumctionI=find(Level==CurLevel) % in parallel
  for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=cLRelEnds[CurLevel];LRelIndex++){ 
   JumctionI=cCompByLevel32[LRelIndex*WARPSIZE+PIdx_1]-1;
   // for i=(cSegEndI(JumctionI)):-1:(cSegStartI(JumctionI)-1)
   for(i=cSegEndI[JumctionI]-1;i>=(cSegStartI[JumctionI]-2);i--) {
    // k=cKsB(i+1);
     k=cKsB[i+1];
	 
    // bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
   Out[i]=(bHP[i]-Out[k]*cF[i])/uHP[i];
   }
  }
 }
 } 
#endif

__device__ void runSimulation(HMat InMat,MYFTYPE* ParamsM, MYFTYPE* V,Stim stim,Sim sim,MYFTYPE* VHotGlobal,MYFTYPE* SMemVHot,MYFTYPE* amps,MYDTYPE offset)
{


	MYSECONDFTYPE *uHP,*bHP;
	MYDTYPE StimID=threadIdx.y;
		
#ifdef BKSUB2
    MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(InMat.N+2*WARPSIZE)*sizeof(MYFTYPE);
#endif
#ifdef BKSUB1

  MYDTYPE PerStimulus=((InMat.N+2)*2*sizeof(MYSECONDFTYPE))+(WARPSIZE + sim.NRecSites*(WARPSIZE))*sizeof(MYFTYPE);
  PerStimulus =ceilf(MYSECONDFTYPE(PerStimulus)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE); 
#endif
	//MYDTYPE PerStimulus;
	//PerStimulus = InMat.N+2;
	offset+=PerStimulus*threadIdx.y;
	uHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);
	//uHP[InMat.N+2] =0;
	bHP = (MYSECONDFTYPE*) &smem[offset];
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);
	//bHP[InMat.N+2] =0;
	offset = ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	//Vs = (MYFTYPE*) &smem[offset];
	//offset+=InMat.N*sizeof(MYFTYPE);
	// Vs[PIdx_3]=V[PIdx_3];
	SMemVHot = (MYFTYPE*) &smem[offset];
	offset+=(WARPSIZE)*sim.NRecSites*sizeof(MYFTYPE);
	SMemVHot[WARPSIZE]=0;
    amps = (MYFTYPE*) &smem[offset];
    offset+=WARPSIZE*sizeof(MYFTYPE);
	MYDTYPE NeuronID = blockIdx.x;
	int Nt=stim.Nt;
	MYFTYPE t=0;
	MYSECONDFTYPE *PX,*PF;
	PX=bHP;
	PF=uHP;
	#define THISCOMMANDHERE18(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1));
MYDTYPE PIdx_1 =threadIdx.x+(WARPSIZE*(1 -1));
MYDTYPE PIdx_2 =threadIdx.x+(WARPSIZE*(2 -1));
MYDTYPE PIdx_3 =threadIdx.x+(WARPSIZE*(3 -1));
MYDTYPE PIdx_4 =threadIdx.x+(WARPSIZE*(4 -1));
MYDTYPE PIdx_5 =threadIdx.x+(WARPSIZE*(5 -1));
MYDTYPE PIdx_6 =threadIdx.x+(WARPSIZE*(6 -1));
MYDTYPE PIdx_7 =threadIdx.x+(WARPSIZE*(7 -1));
MYDTYPE PIdx_8 =threadIdx.x+(WARPSIZE*(8 -1));
MYDTYPE PIdx_9 =threadIdx.x+(WARPSIZE*(9 -1));
MYDTYPE PIdx_10 =threadIdx.x+(WARPSIZE*(10 -1));
MYDTYPE PIdx_11 =threadIdx.x+(WARPSIZE*(11 -1));
MYDTYPE PIdx_12 =threadIdx.x+(WARPSIZE*(12 -1));

	 #define THISCOMMANDHERE19(VARILP) MYFTYPE isModel_ ## VARILP , dVec_ ## VARILP , Vmid_ ## VARILP ; MYFTYPE ModelStates_ ## VARILP [NSTATES]; MYFTYPE v_ ## VARILP =V[PIdx_ ## VARILP ]; MYFTYPE dv_ ## VARILP;
MYFTYPE isModel_1 , dVec_1 , Vmid_1 ; MYFTYPE ModelStates_1 [NSTATES]; MYFTYPE v_1 =V[PIdx_1 ]; MYFTYPE dv_1;
MYFTYPE isModel_2 , dVec_2 , Vmid_2 ; MYFTYPE ModelStates_2 [NSTATES]; MYFTYPE v_2 =V[PIdx_2 ]; MYFTYPE dv_2;
MYFTYPE isModel_3 , dVec_3 , Vmid_3 ; MYFTYPE ModelStates_3 [NSTATES]; MYFTYPE v_3 =V[PIdx_3 ]; MYFTYPE dv_3;
MYFTYPE isModel_4 , dVec_4 , Vmid_4 ; MYFTYPE ModelStates_4 [NSTATES]; MYFTYPE v_4 =V[PIdx_4 ]; MYFTYPE dv_4;
MYFTYPE isModel_5 , dVec_5 , Vmid_5 ; MYFTYPE ModelStates_5 [NSTATES]; MYFTYPE v_5 =V[PIdx_5 ]; MYFTYPE dv_5;
MYFTYPE isModel_6 , dVec_6 , Vmid_6 ; MYFTYPE ModelStates_6 [NSTATES]; MYFTYPE v_6 =V[PIdx_6 ]; MYFTYPE dv_6;
MYFTYPE isModel_7 , dVec_7 , Vmid_7 ; MYFTYPE ModelStates_7 [NSTATES]; MYFTYPE v_7 =V[PIdx_7 ]; MYFTYPE dv_7;
MYFTYPE isModel_8 , dVec_8 , Vmid_8 ; MYFTYPE ModelStates_8 [NSTATES]; MYFTYPE v_8 =V[PIdx_8 ]; MYFTYPE dv_8;
MYFTYPE isModel_9 , dVec_9 , Vmid_9 ; MYFTYPE ModelStates_9 [NSTATES]; MYFTYPE v_9 =V[PIdx_9 ]; MYFTYPE dv_9;
MYFTYPE isModel_10 , dVec_10 , Vmid_10 ; MYFTYPE ModelStates_10 [NSTATES]; MYFTYPE v_10 =V[PIdx_10 ]; MYFTYPE dv_10;
MYFTYPE isModel_11 , dVec_11 , Vmid_11 ; MYFTYPE ModelStates_11 [NSTATES]; MYFTYPE v_11 =V[PIdx_11 ]; MYFTYPE dv_11;
MYFTYPE isModel_12 , dVec_12 , Vmid_12 ; MYFTYPE ModelStates_12 [NSTATES]; MYFTYPE v_12 =V[PIdx_12 ]; MYFTYPE dv_12;

	 #define THISCOMMANDHERE20(VARILP) MYSECONDFTYPE sumCurrents_ ## VARILP=0 ,sumCurrentsDv_ ## VARILP=0; MYFTYPE  sumConductivity_ ## VARILP=0 ,sumConductivityDv_ ## VARILP=0 ;
MYSECONDFTYPE sumCurrents_1=0 ,sumCurrentsDv_1=0; MYFTYPE  sumConductivity_1=0 ,sumConductivityDv_1=0 ;
MYSECONDFTYPE sumCurrents_2=0 ,sumCurrentsDv_2=0; MYFTYPE  sumConductivity_2=0 ,sumConductivityDv_2=0 ;
MYSECONDFTYPE sumCurrents_3=0 ,sumCurrentsDv_3=0; MYFTYPE  sumConductivity_3=0 ,sumConductivityDv_3=0 ;
MYSECONDFTYPE sumCurrents_4=0 ,sumCurrentsDv_4=0; MYFTYPE  sumConductivity_4=0 ,sumConductivityDv_4=0 ;
MYSECONDFTYPE sumCurrents_5=0 ,sumCurrentsDv_5=0; MYFTYPE  sumConductivity_5=0 ,sumConductivityDv_5=0 ;
MYSECONDFTYPE sumCurrents_6=0 ,sumCurrentsDv_6=0; MYFTYPE  sumConductivity_6=0 ,sumConductivityDv_6=0 ;
MYSECONDFTYPE sumCurrents_7=0 ,sumCurrentsDv_7=0; MYFTYPE  sumConductivity_7=0 ,sumConductivityDv_7=0 ;
MYSECONDFTYPE sumCurrents_8=0 ,sumCurrentsDv_8=0; MYFTYPE  sumConductivity_8=0 ,sumConductivityDv_8=0 ;
MYSECONDFTYPE sumCurrents_9=0 ,sumCurrentsDv_9=0; MYFTYPE  sumConductivity_9=0 ,sumConductivityDv_9=0 ;
MYSECONDFTYPE sumCurrents_10=0 ,sumCurrentsDv_10=0; MYFTYPE  sumConductivity_10=0 ,sumConductivityDv_10=0 ;
MYSECONDFTYPE sumCurrents_11=0 ,sumCurrentsDv_11=0; MYFTYPE  sumConductivity_11=0 ,sumConductivityDv_11=0 ;
MYSECONDFTYPE sumCurrents_12=0 ,sumCurrentsDv_12=0; MYFTYPE  sumConductivity_12=0 ,sumConductivityDv_12=0 ;
	// end for model in init
ModelStates_1 [0]=0;ModelStates_1 [1]=0;ModelStates_1 [2]=0;ModelStates_1 [3]=0;ModelStates_1 [4]=0;ModelStates_1 [5]=0;ModelStates_1 [6]=0;ModelStates_1 [7]=0;ModelStates_1 [8]=0;ModelStates_1 [9]=0;;ModelStates_2 [0]=0;ModelStates_2 [1]=0;ModelStates_2 [2]=0;ModelStates_2 [3]=0;ModelStates_2 [4]=0;ModelStates_2 [5]=0;ModelStates_2 [6]=0;ModelStates_2 [7]=0;ModelStates_2 [8]=0;ModelStates_2 [9]=0;;ModelStates_3 [0]=0;ModelStates_3 [1]=0;ModelStates_3 [2]=0;ModelStates_3 [3]=0;ModelStates_3 [4]=0;ModelStates_3 [5]=0;ModelStates_3 [6]=0;ModelStates_3 [7]=0;ModelStates_3 [8]=0;ModelStates_3 [9]=0;;ModelStates_4 [0]=0;ModelStates_4 [1]=0;ModelStates_4 [2]=0;ModelStates_4 [3]=0;ModelStates_4 [4]=0;ModelStates_4 [5]=0;ModelStates_4 [6]=0;ModelStates_4 [7]=0;ModelStates_4 [8]=0;ModelStates_4 [9]=0;;ModelStates_5 [0]=0;ModelStates_5 [1]=0;ModelStates_5 [2]=0;ModelStates_5 [3]=0;ModelStates_5 [4]=0;ModelStates_5 [5]=0;ModelStates_5 [6]=0;ModelStates_5 [7]=0;ModelStates_5 [8]=0;ModelStates_5 [9]=0;;ModelStates_6 [0]=0;ModelStates_6 [1]=0;ModelStates_6 [2]=0;ModelStates_6 [3]=0;ModelStates_6 [4]=0;ModelStates_6 [5]=0;ModelStates_6 [6]=0;ModelStates_6 [7]=0;ModelStates_6 [8]=0;ModelStates_6 [9]=0;;ModelStates_7 [0]=0;ModelStates_7 [1]=0;ModelStates_7 [2]=0;ModelStates_7 [3]=0;ModelStates_7 [4]=0;ModelStates_7 [5]=0;ModelStates_7 [6]=0;ModelStates_7 [7]=0;ModelStates_7 [8]=0;ModelStates_7 [9]=0;;ModelStates_8 [0]=0;ModelStates_8 [1]=0;ModelStates_8 [2]=0;ModelStates_8 [3]=0;ModelStates_8 [4]=0;ModelStates_8 [5]=0;ModelStates_8 [6]=0;ModelStates_8 [7]=0;ModelStates_8 [8]=0;ModelStates_8 [9]=0;;ModelStates_9 [0]=0;ModelStates_9 [1]=0;ModelStates_9 [2]=0;ModelStates_9 [3]=0;ModelStates_9 [4]=0;ModelStates_9 [5]=0;ModelStates_9 [6]=0;ModelStates_9 [7]=0;ModelStates_9 [8]=0;ModelStates_9 [9]=0;;ModelStates_10 [0]=0;ModelStates_10 [1]=0;ModelStates_10 [2]=0;ModelStates_10 [3]=0;ModelStates_10 [4]=0;ModelStates_10 [5]=0;ModelStates_10 [6]=0;ModelStates_10 [7]=0;ModelStates_10 [8]=0;ModelStates_10 [9]=0;;ModelStates_11 [0]=0;ModelStates_11 [1]=0;ModelStates_11 [2]=0;ModelStates_11 [3]=0;ModelStates_11 [4]=0;ModelStates_11 [5]=0;ModelStates_11 [6]=0;ModelStates_11 [7]=0;ModelStates_11 [8]=0;ModelStates_11 [9]=0;;ModelStates_12 [0]=0;ModelStates_12 [1]=0;ModelStates_12 [2]=0;ModelStates_12 [3]=0;ModelStates_12 [4]=0;ModelStates_12 [5]=0;ModelStates_12 [6]=0;ModelStates_12 [7]=0;ModelStates_12 [8]=0;ModelStates_12 [9]=0;;
	#define THISCOMMANDHERE21(VARILP) MYDTYPE SonNo_ ## VARILP =cSonNoVec[PIdx_ ## VARILP ];
MYDTYPE SonNo_1 =cSonNoVec[PIdx_1 ];
MYDTYPE SonNo_2 =cSonNoVec[PIdx_2 ];
MYDTYPE SonNo_3 =cSonNoVec[PIdx_3 ];
MYDTYPE SonNo_4 =cSonNoVec[PIdx_4 ];
MYDTYPE SonNo_5 =cSonNoVec[PIdx_5 ];
MYDTYPE SonNo_6 =cSonNoVec[PIdx_6 ];
MYDTYPE SonNo_7 =cSonNoVec[PIdx_7 ];
MYDTYPE SonNo_8 =cSonNoVec[PIdx_8 ];
MYDTYPE SonNo_9 =cSonNoVec[PIdx_9 ];
MYDTYPE SonNo_10 =cSonNoVec[PIdx_10 ];
MYDTYPE SonNo_11 =cSonNoVec[PIdx_11 ];
MYDTYPE SonNo_12 =cSonNoVec[PIdx_12 ];
	
	#define THISCOMMANDHERE22(VARILP) bHP[PIdx_ ## VARILP]=0;dv_ ## VARILP =0;
bHP[PIdx_1]=0;dv_1 =0;
bHP[PIdx_2]=0;dv_2 =0;
bHP[PIdx_3]=0;dv_3 =0;
bHP[PIdx_4]=0;dv_4 =0;
bHP[PIdx_5]=0;dv_5 =0;
bHP[PIdx_6]=0;dv_6 =0;
bHP[PIdx_7]=0;dv_7 =0;
bHP[PIdx_8]=0;dv_8 =0;
bHP[PIdx_9]=0;dv_9 =0;
bHP[PIdx_10]=0;dv_10 =0;
bHP[PIdx_11]=0;dv_11 =0;
bHP[PIdx_12]=0;dv_12 =0;


	#define THISCOMMANDHERE23(VARILP) MYDTYPE parentIndex_ ## VARILP ; MYDTYPE Eidx_ ## VARILP ; Eidx_ ## VARILP = InMat.N-PIdx_ ## VARILP -1;
MYDTYPE parentIndex_1 ; MYDTYPE Eidx_1 ; Eidx_1 = InMat.N-PIdx_1 -1;
MYDTYPE parentIndex_2 ; MYDTYPE Eidx_2 ; Eidx_2 = InMat.N-PIdx_2 -1;
MYDTYPE parentIndex_3 ; MYDTYPE Eidx_3 ; Eidx_3 = InMat.N-PIdx_3 -1;
MYDTYPE parentIndex_4 ; MYDTYPE Eidx_4 ; Eidx_4 = InMat.N-PIdx_4 -1;
MYDTYPE parentIndex_5 ; MYDTYPE Eidx_5 ; Eidx_5 = InMat.N-PIdx_5 -1;
MYDTYPE parentIndex_6 ; MYDTYPE Eidx_6 ; Eidx_6 = InMat.N-PIdx_6 -1;
MYDTYPE parentIndex_7 ; MYDTYPE Eidx_7 ; Eidx_7 = InMat.N-PIdx_7 -1;
MYDTYPE parentIndex_8 ; MYDTYPE Eidx_8 ; Eidx_8 = InMat.N-PIdx_8 -1;
MYDTYPE parentIndex_9 ; MYDTYPE Eidx_9 ; Eidx_9 = InMat.N-PIdx_9 -1;
MYDTYPE parentIndex_10 ; MYDTYPE Eidx_10 ; Eidx_10 = InMat.N-PIdx_10 -1;
MYDTYPE parentIndex_11 ; MYDTYPE Eidx_11 ; Eidx_11 = InMat.N-PIdx_11 -1;
MYDTYPE parentIndex_12 ; MYDTYPE Eidx_12 ; Eidx_12 = InMat.N-PIdx_12 -1;
	// RRRXXX This probably should be in ILP too!
	if(Eidx_1>InMat.N-1){
		Eidx_1=InMat.N-1;
	}
	MYDTYPE perThreadParamMSize = InMat.NComps*NPARAMS;
#define param_macro(paramInd,segmentInd) ParamsM[NeuronID*perThreadParamMSize + paramInd*InMat.NComps+cSegToComp[segmentInd] ]
	
  if(cBoolModel[PIdx_1 +0*InMat.N]){CuInitModel_ca(v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*InMat.N]){CuInitModel_cad(v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*InMat.N]){CuInitModel_kca(v_1,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*InMat.N]){CuInitModel_km(v_1,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*InMat.N]){CuInitModel_kv(v_1,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*InMat.N]){CuInitModel_na(v_1,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*InMat.N]){CuInitModel_ca(v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*InMat.N]){CuInitModel_cad(v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*InMat.N]){CuInitModel_kca(v_2,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*InMat.N]){CuInitModel_km(v_2,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*InMat.N]){CuInitModel_kv(v_2,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*InMat.N]){CuInitModel_na(v_2,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );}  if(cBoolModel[PIdx_3 +0*InMat.N]){CuInitModel_ca(v_3,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*InMat.N]){CuInitModel_cad(v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*InMat.N]){CuInitModel_kca(v_3,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*InMat.N]){CuInitModel_km(v_3,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*InMat.N]){CuInitModel_kv(v_3,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*InMat.N]){CuInitModel_na(v_3,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );}  if(cBoolModel[PIdx_4 +0*InMat.N]){CuInitModel_ca(v_4,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*InMat.N]){CuInitModel_cad(v_4,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*InMat.N]){CuInitModel_kca(v_4,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*InMat.N]){CuInitModel_km(v_4,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*InMat.N]){CuInitModel_kv(v_4,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*InMat.N]){CuInitModel_na(v_4,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );}  if(cBoolModel[PIdx_5 +0*InMat.N]){CuInitModel_ca(v_5,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*InMat.N]){CuInitModel_cad(v_5,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*InMat.N]){CuInitModel_kca(v_5,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*InMat.N]){CuInitModel_km(v_5,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*InMat.N]){CuInitModel_kv(v_5,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*InMat.N]){CuInitModel_na(v_5,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );}  if(cBoolModel[PIdx_6 +0*InMat.N]){CuInitModel_ca(v_6,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*InMat.N]){CuInitModel_cad(v_6,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*InMat.N]){CuInitModel_kca(v_6,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*InMat.N]){CuInitModel_km(v_6,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*InMat.N]){CuInitModel_kv(v_6,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*InMat.N]){CuInitModel_na(v_6,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );}  if(cBoolModel[PIdx_7 +0*InMat.N]){CuInitModel_ca(v_7,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*InMat.N]){CuInitModel_cad(v_7,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*InMat.N]){CuInitModel_kca(v_7,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*InMat.N]){CuInitModel_km(v_7,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*InMat.N]){CuInitModel_kv(v_7,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*InMat.N]){CuInitModel_na(v_7,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );}  if(cBoolModel[PIdx_8 +0*InMat.N]){CuInitModel_ca(v_8,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*InMat.N]){CuInitModel_cad(v_8,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*InMat.N]){CuInitModel_kca(v_8,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*InMat.N]){CuInitModel_km(v_8,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*InMat.N]){CuInitModel_kv(v_8,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*InMat.N]){CuInitModel_na(v_8,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );}  if(cBoolModel[PIdx_9 +0*InMat.N]){CuInitModel_ca(v_9,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*InMat.N]){CuInitModel_cad(v_9,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*InMat.N]){CuInitModel_kca(v_9,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*InMat.N]){CuInitModel_km(v_9,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*InMat.N]){CuInitModel_kv(v_9,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*InMat.N]){CuInitModel_na(v_9,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );}  if(cBoolModel[PIdx_10 +0*InMat.N]){CuInitModel_ca(v_10,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*InMat.N]){CuInitModel_cad(v_10,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*InMat.N]){CuInitModel_kca(v_10,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*InMat.N]){CuInitModel_km(v_10,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*InMat.N]){CuInitModel_kv(v_10,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*InMat.N]){CuInitModel_na(v_10,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );}  if(cBoolModel[PIdx_11 +0*InMat.N]){CuInitModel_ca(v_11,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*InMat.N]){CuInitModel_cad(v_11,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*InMat.N]){CuInitModel_kca(v_11,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*InMat.N]){CuInitModel_km(v_11,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*InMat.N]){CuInitModel_kv(v_11,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*InMat.N]){CuInitModel_na(v_11,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );}  if(cBoolModel[PIdx_12 +0*InMat.N]){CuInitModel_ca(v_12,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*InMat.N]){CuInitModel_cad(v_12,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*InMat.N]){CuInitModel_kca(v_12,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*InMat.N]){CuInitModel_km(v_12,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*InMat.N]){CuInitModel_kv(v_12,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*InMat.N]){CuInitModel_na(v_12,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );}
	
		// parentIndex_3 =InMat.N-InMat.Ks[InMat.N-PIdx_3];
#define THISCOMMANDHERE24(VARILP) parentIndex_ ## VARILP =InMat.N-cKs[InMat.N-PIdx_ ## VARILP ];
parentIndex_1 =InMat.N-cKs[InMat.N-PIdx_1 ];
parentIndex_2 =InMat.N-cKs[InMat.N-PIdx_2 ];
parentIndex_3 =InMat.N-cKs[InMat.N-PIdx_3 ];
parentIndex_4 =InMat.N-cKs[InMat.N-PIdx_4 ];
parentIndex_5 =InMat.N-cKs[InMat.N-PIdx_5 ];
parentIndex_6 =InMat.N-cKs[InMat.N-PIdx_6 ];
parentIndex_7 =InMat.N-cKs[InMat.N-PIdx_7 ];
parentIndex_8 =InMat.N-cKs[InMat.N-PIdx_8 ];
parentIndex_9 =InMat.N-cKs[InMat.N-PIdx_9 ];
parentIndex_10 =InMat.N-cKs[InMat.N-PIdx_10 ];
parentIndex_11 =InMat.N-cKs[InMat.N-PIdx_11 ];
parentIndex_12 =InMat.N-cKs[InMat.N-PIdx_12 ];
	MYDTYPE stimLoc = stim.loc;
    MYFTYPE stimArea = stim.area;
    MYDTYPE dtCounter=0;
    MYFTYPE  dt = sim.dt;
#ifndef STIMFROMFILE
    MYFTYPE stimDel = stim.dels[StimID];//Should be constant
    MYFTYPE stimDur = stim.durs[StimID];//Should be constant
    MYFTYPE stimAmp = stim.amps[StimID];
#endif // !STIMFROMFILE
	// MYFTYPE rhs_3,D_3,gModel_3,StimCurrent_3,dv_3;
#define THISCOMMANDHERE25(VARILP) MYSECONDFTYPE rhs_ ## VARILP ,D_ ## VARILP; MYFTYPE gModel_ ## VARILP ,StimCurrent_ ## VARILP;
MYSECONDFTYPE rhs_1 ,D_1; MYFTYPE gModel_1 ,StimCurrent_1;
MYSECONDFTYPE rhs_2 ,D_2; MYFTYPE gModel_2 ,StimCurrent_2;
MYSECONDFTYPE rhs_3 ,D_3; MYFTYPE gModel_3 ,StimCurrent_3;
MYSECONDFTYPE rhs_4 ,D_4; MYFTYPE gModel_4 ,StimCurrent_4;
MYSECONDFTYPE rhs_5 ,D_5; MYFTYPE gModel_5 ,StimCurrent_5;
MYSECONDFTYPE rhs_6 ,D_6; MYFTYPE gModel_6 ,StimCurrent_6;
MYSECONDFTYPE rhs_7 ,D_7; MYFTYPE gModel_7 ,StimCurrent_7;
MYSECONDFTYPE rhs_8 ,D_8; MYFTYPE gModel_8 ,StimCurrent_8;
MYSECONDFTYPE rhs_9 ,D_9; MYFTYPE gModel_9 ,StimCurrent_9;
MYSECONDFTYPE rhs_10 ,D_10; MYFTYPE gModel_10 ,StimCurrent_10;
MYSECONDFTYPE rhs_11 ,D_11; MYFTYPE gModel_11 ,StimCurrent_11;
MYSECONDFTYPE rhs_12 ,D_12; MYFTYPE gModel_12 ,StimCurrent_12;

for(int i=0;i<Nt;i++) {
        if(i==stim.dtInds[dtCounter]){
                dt = stim.durs[dtCounter];
                if (dtCounter != stim.numofdts-1){
                    dtCounter++;
                }
        }
        t+=0.5*dt;
 
       if((i%(WARPSIZE)==0)){
            if(i>0){
               for(int recInd =0;recInd<sim.NRecSites;recInd++){
					VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + (i-WARPSIZE)+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
				}
            }
			
            amps[PIdx_1]=stim.amps[threadIdx.y*Nt+i+PIdx_1];
        }
        for(int recInd =0;recInd<sim.NRecSites;recInd++){
			if(sim.RecSites[recInd]%WARPSIZE == threadIdx.x)
				//This is done by all threads why???
				SMemVHot[recInd*WARPSIZE+i%(WARPSIZE)]=v_1;//This is going to be challenging to make it general but possible.
			
		}
		 // rhs_3=0; // D_3=0;// StimCurrent_3=0;
#define THISCOMMANDHERE26(VARILP) rhs_ ## VARILP =0; D_ ## VARILP =0; sumCurrents_ ## VARILP=0; sumConductivity_ ## VARILP=0; sumCurrentsDv_ ## VARILP=0; sumConductivityDv_ ## VARILP=0;StimCurrent_ ## VARILP =0;
rhs_1 =0; D_1 =0; sumCurrents_1=0; sumConductivity_1=0; sumCurrentsDv_1=0; sumConductivityDv_1=0;StimCurrent_1 =0;
rhs_2 =0; D_2 =0; sumCurrents_2=0; sumConductivity_2=0; sumCurrentsDv_2=0; sumConductivityDv_2=0;StimCurrent_2 =0;
rhs_3 =0; D_3 =0; sumCurrents_3=0; sumConductivity_3=0; sumCurrentsDv_3=0; sumConductivityDv_3=0;StimCurrent_3 =0;
rhs_4 =0; D_4 =0; sumCurrents_4=0; sumConductivity_4=0; sumCurrentsDv_4=0; sumConductivityDv_4=0;StimCurrent_4 =0;
rhs_5 =0; D_5 =0; sumCurrents_5=0; sumConductivity_5=0; sumCurrentsDv_5=0; sumConductivityDv_5=0;StimCurrent_5 =0;
rhs_6 =0; D_6 =0; sumCurrents_6=0; sumConductivity_6=0; sumCurrentsDv_6=0; sumConductivityDv_6=0;StimCurrent_6 =0;
rhs_7 =0; D_7 =0; sumCurrents_7=0; sumConductivity_7=0; sumCurrentsDv_7=0; sumConductivityDv_7=0;StimCurrent_7 =0;
rhs_8 =0; D_8 =0; sumCurrents_8=0; sumConductivity_8=0; sumCurrentsDv_8=0; sumConductivityDv_8=0;StimCurrent_8 =0;
rhs_9 =0; D_9 =0; sumCurrents_9=0; sumConductivity_9=0; sumCurrentsDv_9=0; sumConductivityDv_9=0;StimCurrent_9 =0;
rhs_10 =0; D_10 =0; sumCurrents_10=0; sumConductivity_10=0; sumCurrentsDv_10=0; sumConductivityDv_10=0;StimCurrent_10 =0;
rhs_11 =0; D_11 =0; sumCurrents_11=0; sumConductivity_11=0; sumCurrentsDv_11=0; sumConductivityDv_11=0;StimCurrent_11 =0;
rhs_12 =0; D_12 =0; sumCurrents_12=0; sumConductivity_12=0; sumCurrentsDv_12=0; sumConductivityDv_12=0;StimCurrent_12 =0;
		// Before matrix
		// if(t>=stimDel && t<stimDel+stimDur && PIdx_3 == stimLoc){StimCurrent_3 = 100*stimAmp/stimArea;}
#ifndef STIMFROMFILE
    #define THISCOMMANDHERE27(VARILP) if(t>=stimDel && t<stimDel+stimDur && PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_1  == stimLoc){StimCurrent_1  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_2  == stimLoc){StimCurrent_2  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_3  == stimLoc){StimCurrent_3  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_4  == stimLoc){StimCurrent_4  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_5  == stimLoc){StimCurrent_5  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_6  == stimLoc){StimCurrent_6  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_7  == stimLoc){StimCurrent_7  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_8  == stimLoc){StimCurrent_8  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_9  == stimLoc){StimCurrent_9  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_10  == stimLoc){StimCurrent_10  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_11  == stimLoc){StimCurrent_11  = 100*stimAmp/stimArea;}
if(t>=stimDel && t<stimDel+stimDur && PIdx_12  == stimLoc){StimCurrent_12  = 100*stimAmp/stimArea;}
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
    
    #define THISCOMMANDHERE28(VARILP) if(PIdx_ ## VARILP  == stimLoc){StimCurrent_ ## VARILP = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_1  == stimLoc){StimCurrent_1 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_2  == stimLoc){StimCurrent_2 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_3  == stimLoc){StimCurrent_3 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_4  == stimLoc){StimCurrent_4 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_5  == stimLoc){StimCurrent_5 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_6  == stimLoc){StimCurrent_6 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_7  == stimLoc){StimCurrent_7 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_8  == stimLoc){StimCurrent_8 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_9  == stimLoc){StimCurrent_9 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_10  == stimLoc){StimCurrent_10 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_11  == stimLoc){StimCurrent_11 = 100*amps[i%WARPSIZE]/stimArea;}
if(PIdx_12  == stimLoc){StimCurrent_12 = 100*amps[i%WARPSIZE]/stimArea;}
#endif // STIMFROMFILE
 if(cBoolModel[PIdx_1 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_1 , sumConductivityDv_1 ,v_1 +0.001,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );} if(cBoolModel[PIdx_2 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_2 , sumConductivityDv_2 ,v_2 +0.001,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );} if(cBoolModel[PIdx_3 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_3 , sumConductivityDv_3 ,v_3 +0.001,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );} if(cBoolModel[PIdx_4 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_4 , sumConductivityDv_4 ,v_4 +0.001,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );} if(cBoolModel[PIdx_5 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_5 , sumConductivityDv_5 ,v_5 +0.001,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );} if(cBoolModel[PIdx_6 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_6 , sumConductivityDv_6 ,v_6 +0.001,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );} if(cBoolModel[PIdx_7 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_7 , sumConductivityDv_7 ,v_7 +0.001,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );} if(cBoolModel[PIdx_8 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_8 , sumConductivityDv_8 ,v_8 +0.001,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );} if(cBoolModel[PIdx_9 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_9 , sumConductivityDv_9 ,v_9 +0.001,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );} if(cBoolModel[PIdx_10 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_10 , sumConductivityDv_10 ,v_10 +0.001,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );} if(cBoolModel[PIdx_11 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_11 , sumConductivityDv_11 ,v_11 +0.001,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );} if(cBoolModel[PIdx_12 +0*InMat.N]){CuBreakpointModel_ca(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*InMat.N]){CuBreakpointModel_cad(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*InMat.N]){CuBreakpointModel_kca(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*InMat.N]){CuBreakpointModel_km(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*InMat.N]){CuBreakpointModel_kv(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*InMat.N]){CuBreakpointModel_na(sumCurrentsDv_12 , sumConductivityDv_12 ,v_12 +0.001,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );}
 if(cBoolModel[PIdx_1 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_1 , sumConductivity_1,v_1 ,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );} if(cBoolModel[PIdx_2 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_2 , sumConductivity_2,v_2 ,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );} if(cBoolModel[PIdx_3 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_3 , sumConductivity_3,v_3 ,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );} if(cBoolModel[PIdx_4 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_4 , sumConductivity_4,v_4 ,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );} if(cBoolModel[PIdx_5 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_5 , sumConductivity_5,v_5 ,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );} if(cBoolModel[PIdx_6 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_6 , sumConductivity_6,v_6 ,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );} if(cBoolModel[PIdx_7 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_7 , sumConductivity_7,v_7 ,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );} if(cBoolModel[PIdx_8 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_8 , sumConductivity_8,v_8 ,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );} if(cBoolModel[PIdx_9 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_9 , sumConductivity_9,v_9 ,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );} if(cBoolModel[PIdx_10 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_10 , sumConductivity_10,v_10 ,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );} if(cBoolModel[PIdx_11 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_11 , sumConductivity_11,v_11 ,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );} if(cBoolModel[PIdx_12 +0*InMat.N]){CuBreakpointModel_ca(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*InMat.N]){CuBreakpointModel_cad(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*InMat.N]){CuBreakpointModel_kca(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*InMat.N]){CuBreakpointModel_km(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*InMat.N]){CuBreakpointModel_kv(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*InMat.N]){CuBreakpointModel_na(sumCurrents_12 , sumConductivity_12,v_12 ,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );}

	// gModel_3 = (sumCurrentsDv_3-sumCurrents_3)/0.001; 
#define THISCOMMANDHERE29(VARILP) gModel_ ## VARILP  = (sumCurrentsDv_ ## VARILP -sumCurrents_ ## VARILP )/EPS_V;  
gModel_1  = (sumCurrentsDv_1 -sumCurrents_1 )/EPS_V;  
gModel_2  = (sumCurrentsDv_2 -sumCurrents_2 )/EPS_V;  
gModel_3  = (sumCurrentsDv_3 -sumCurrents_3 )/EPS_V;  
gModel_4  = (sumCurrentsDv_4 -sumCurrents_4 )/EPS_V;  
gModel_5  = (sumCurrentsDv_5 -sumCurrents_5 )/EPS_V;  
gModel_6  = (sumCurrentsDv_6 -sumCurrents_6 )/EPS_V;  
gModel_7  = (sumCurrentsDv_7 -sumCurrents_7 )/EPS_V;  
gModel_8  = (sumCurrentsDv_8 -sumCurrents_8 )/EPS_V;  
gModel_9  = (sumCurrentsDv_9 -sumCurrents_9 )/EPS_V;  
gModel_10  = (sumCurrentsDv_10 -sumCurrents_10 )/EPS_V;  
gModel_11  = (sumCurrentsDv_11 -sumCurrents_11 )/EPS_V;  
gModel_12  = (sumCurrentsDv_12 -sumCurrents_12 )/EPS_V;  
	
#define THISCOMMANDHERE30(VARILP) rhs_ ## VARILP =StimCurrent_ ## VARILP -sumCurrents_ ## VARILP ;
rhs_1 =StimCurrent_1 -sumCurrents_1 ;
rhs_2 =StimCurrent_2 -sumCurrents_2 ;
rhs_3 =StimCurrent_3 -sumCurrents_3 ;
rhs_4 =StimCurrent_4 -sumCurrents_4 ;
rhs_5 =StimCurrent_5 -sumCurrents_5 ;
rhs_6 =StimCurrent_6 -sumCurrents_6 ;
rhs_7 =StimCurrent_7 -sumCurrents_7 ;
rhs_8 =StimCurrent_8 -sumCurrents_8 ;
rhs_9 =StimCurrent_9 -sumCurrents_9 ;
rhs_10 =StimCurrent_10 -sumCurrents_10 ;
rhs_11 =StimCurrent_11 -sumCurrents_11 ;
rhs_12 =StimCurrent_12 -sumCurrents_12 ;
#define THISCOMMANDHERE31(VARILP) D_ ## VARILP =gModel_ ## VARILP +cCm[PIdx_ ## VARILP ]/(dt*1000); D_ ## VARILP -=cF[InMat.N-PIdx_ ## VARILP -1];
D_1 =gModel_1 +cCm[PIdx_1 ]/(dt*1000); D_1 -=cF[InMat.N-PIdx_1 -1];
D_2 =gModel_2 +cCm[PIdx_2 ]/(dt*1000); D_2 -=cF[InMat.N-PIdx_2 -1];
D_3 =gModel_3 +cCm[PIdx_3 ]/(dt*1000); D_3 -=cF[InMat.N-PIdx_3 -1];
D_4 =gModel_4 +cCm[PIdx_4 ]/(dt*1000); D_4 -=cF[InMat.N-PIdx_4 -1];
D_5 =gModel_5 +cCm[PIdx_5 ]/(dt*1000); D_5 -=cF[InMat.N-PIdx_5 -1];
D_6 =gModel_6 +cCm[PIdx_6 ]/(dt*1000); D_6 -=cF[InMat.N-PIdx_6 -1];
D_7 =gModel_7 +cCm[PIdx_7 ]/(dt*1000); D_7 -=cF[InMat.N-PIdx_7 -1];
D_8 =gModel_8 +cCm[PIdx_8 ]/(dt*1000); D_8 -=cF[InMat.N-PIdx_8 -1];
D_9 =gModel_9 +cCm[PIdx_9 ]/(dt*1000); D_9 -=cF[InMat.N-PIdx_9 -1];
D_10 =gModel_10 +cCm[PIdx_10 ]/(dt*1000); D_10 -=cF[InMat.N-PIdx_10 -1];
D_11 =gModel_11 +cCm[PIdx_11 ]/(dt*1000); D_11 -=cF[InMat.N-PIdx_11 -1];
D_12 =gModel_12 +cCm[PIdx_12 ]/(dt*1000); D_12 -=cF[InMat.N-PIdx_12 -1];
   
    //this should be out and correct cKs
	if(PIdx_1==0) {
		parentIndex_1=0;
	};
#define THISCOMMANDHERE32(VARILP)  dv_ ## VARILP +=bHP[InMat.N-parentIndex_ ## VARILP -1]-bHP[InMat.N-PIdx_ ## VARILP -1]; rhs_ ## VARILP -=cF[InMat.N-PIdx_ ## VARILP -1]*dv_ ## VARILP ;
 dv_1 +=bHP[InMat.N-parentIndex_1 -1]-bHP[InMat.N-PIdx_1 -1]; rhs_1 -=cF[InMat.N-PIdx_1 -1]*dv_1 ;
 dv_2 +=bHP[InMat.N-parentIndex_2 -1]-bHP[InMat.N-PIdx_2 -1]; rhs_2 -=cF[InMat.N-PIdx_2 -1]*dv_2 ;
 dv_3 +=bHP[InMat.N-parentIndex_3 -1]-bHP[InMat.N-PIdx_3 -1]; rhs_3 -=cF[InMat.N-PIdx_3 -1]*dv_3 ;
 dv_4 +=bHP[InMat.N-parentIndex_4 -1]-bHP[InMat.N-PIdx_4 -1]; rhs_4 -=cF[InMat.N-PIdx_4 -1]*dv_4 ;
 dv_5 +=bHP[InMat.N-parentIndex_5 -1]-bHP[InMat.N-PIdx_5 -1]; rhs_5 -=cF[InMat.N-PIdx_5 -1]*dv_5 ;
 dv_6 +=bHP[InMat.N-parentIndex_6 -1]-bHP[InMat.N-PIdx_6 -1]; rhs_6 -=cF[InMat.N-PIdx_6 -1]*dv_6 ;
 dv_7 +=bHP[InMat.N-parentIndex_7 -1]-bHP[InMat.N-PIdx_7 -1]; rhs_7 -=cF[InMat.N-PIdx_7 -1]*dv_7 ;
 dv_8 +=bHP[InMat.N-parentIndex_8 -1]-bHP[InMat.N-PIdx_8 -1]; rhs_8 -=cF[InMat.N-PIdx_8 -1]*dv_8 ;
 dv_9 +=bHP[InMat.N-parentIndex_9 -1]-bHP[InMat.N-PIdx_9 -1]; rhs_9 -=cF[InMat.N-PIdx_9 -1]*dv_9 ;
 dv_10 +=bHP[InMat.N-parentIndex_10 -1]-bHP[InMat.N-PIdx_10 -1]; rhs_10 -=cF[InMat.N-PIdx_10 -1]*dv_10 ;
 dv_11 +=bHP[InMat.N-parentIndex_11 -1]-bHP[InMat.N-PIdx_11 -1]; rhs_11 -=cF[InMat.N-PIdx_11 -1]*dv_11 ;
 dv_12 +=bHP[InMat.N-parentIndex_12 -1]-bHP[InMat.N-PIdx_12 -1]; rhs_12 -=cF[InMat.N-PIdx_12 -1]*dv_12 ;
		// D_3=sumConductivity_3+InMat.Cms[PIdx_3]/(sim.dt*1000); // D_3-=InMat.f[InMat.N-PIdx_3-1];
		// bHP[InMat.N-PIdx_3-1]=rhs_3; // uHP[InMat.N-PIdx_3-1]=D_3;
#define THISCOMMANDHERE33(VARILP) bHP[InMat.N-PIdx_ ## VARILP -1]=rhs_ ## VARILP ; uHP[InMat.N-PIdx_ ## VARILP -1]=D_ ## VARILP ;
	bHP[InMat.N-PIdx_1 -1]=rhs_1 ; uHP[InMat.N-PIdx_1 -1]=D_1 ;
bHP[InMat.N-PIdx_2 -1]=rhs_2 ; uHP[InMat.N-PIdx_2 -1]=D_2 ;
bHP[InMat.N-PIdx_3 -1]=rhs_3 ; uHP[InMat.N-PIdx_3 -1]=D_3 ;
bHP[InMat.N-PIdx_4 -1]=rhs_4 ; uHP[InMat.N-PIdx_4 -1]=D_4 ;
bHP[InMat.N-PIdx_5 -1]=rhs_5 ; uHP[InMat.N-PIdx_5 -1]=D_5 ;
bHP[InMat.N-PIdx_6 -1]=rhs_6 ; uHP[InMat.N-PIdx_6 -1]=D_6 ;
bHP[InMat.N-PIdx_7 -1]=rhs_7 ; uHP[InMat.N-PIdx_7 -1]=D_7 ;
bHP[InMat.N-PIdx_8 -1]=rhs_8 ; uHP[InMat.N-PIdx_8 -1]=D_8 ;
bHP[InMat.N-PIdx_9 -1]=rhs_9 ; uHP[InMat.N-PIdx_9 -1]=D_9 ;
bHP[InMat.N-PIdx_10 -1]=rhs_10 ; uHP[InMat.N-PIdx_10 -1]=D_10 ;
bHP[InMat.N-PIdx_11 -1]=rhs_11 ; uHP[InMat.N-PIdx_11 -1]=D_11 ;
bHP[InMat.N-PIdx_12 -1]=rhs_12 ; uHP[InMat.N-PIdx_12 -1]=D_12 ;
		syncthreads();
		
		// if(SonNo_3==1) {bHP[InMat.N-parentIndex_3-1]+=cE[Eidx_3]*dv_3;uHP[InMat.N-parentIndex_3-1]-=cE[Eidx_3];}
		
#define THISCOMMANDHERE34(VARILP) if(cSonNoVec[PIdx_ ## VARILP] ==1) {bHP[InMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[InMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
	if(cSonNoVec[PIdx_1] ==1) {bHP[InMat.N-parentIndex_1 -1]+=cE[Eidx_1 ]*dv_1 ;uHP[InMat.N-parentIndex_1 -1]-=cE[Eidx_1 ];}
if(cSonNoVec[PIdx_2] ==1) {bHP[InMat.N-parentIndex_2 -1]+=cE[Eidx_2 ]*dv_2 ;uHP[InMat.N-parentIndex_2 -1]-=cE[Eidx_2 ];}
if(cSonNoVec[PIdx_3] ==1) {bHP[InMat.N-parentIndex_3 -1]+=cE[Eidx_3 ]*dv_3 ;uHP[InMat.N-parentIndex_3 -1]-=cE[Eidx_3 ];}
if(cSonNoVec[PIdx_4] ==1) {bHP[InMat.N-parentIndex_4 -1]+=cE[Eidx_4 ]*dv_4 ;uHP[InMat.N-parentIndex_4 -1]-=cE[Eidx_4 ];}
if(cSonNoVec[PIdx_5] ==1) {bHP[InMat.N-parentIndex_5 -1]+=cE[Eidx_5 ]*dv_5 ;uHP[InMat.N-parentIndex_5 -1]-=cE[Eidx_5 ];}
if(cSonNoVec[PIdx_6] ==1) {bHP[InMat.N-parentIndex_6 -1]+=cE[Eidx_6 ]*dv_6 ;uHP[InMat.N-parentIndex_6 -1]-=cE[Eidx_6 ];}
if(cSonNoVec[PIdx_7] ==1) {bHP[InMat.N-parentIndex_7 -1]+=cE[Eidx_7 ]*dv_7 ;uHP[InMat.N-parentIndex_7 -1]-=cE[Eidx_7 ];}
if(cSonNoVec[PIdx_8] ==1) {bHP[InMat.N-parentIndex_8 -1]+=cE[Eidx_8 ]*dv_8 ;uHP[InMat.N-parentIndex_8 -1]-=cE[Eidx_8 ];}
if(cSonNoVec[PIdx_9] ==1) {bHP[InMat.N-parentIndex_9 -1]+=cE[Eidx_9 ]*dv_9 ;uHP[InMat.N-parentIndex_9 -1]-=cE[Eidx_9 ];}
if(cSonNoVec[PIdx_10] ==1) {bHP[InMat.N-parentIndex_10 -1]+=cE[Eidx_10 ]*dv_10 ;uHP[InMat.N-parentIndex_10 -1]-=cE[Eidx_10 ];}
if(cSonNoVec[PIdx_11] ==1) {bHP[InMat.N-parentIndex_11 -1]+=cE[Eidx_11 ]*dv_11 ;uHP[InMat.N-parentIndex_11 -1]-=cE[Eidx_11 ];}
if(cSonNoVec[PIdx_12] ==1) {bHP[InMat.N-parentIndex_12 -1]+=cE[Eidx_12 ]*dv_12 ;uHP[InMat.N-parentIndex_12 -1]-=cE[Eidx_12 ];}
		// if(SonNo_3==2) {bHP[InMat.N-parentIndex_3-1]+=InMat.e[Eidx_3]*dv_3;uHP[InMat.N-parentIndex_3-1]-=InMat.e[Eidx_3];}
#define THISCOMMANDHERE35(VARILP) if(cSonNoVec[PIdx_ ## VARILP] ==2) {bHP[InMat.N-parentIndex_ ## VARILP -1]+=cE[Eidx_ ## VARILP ]*dv_ ## VARILP ;uHP[InMat.N-parentIndex_ ## VARILP -1]-=cE[Eidx_ ## VARILP ];}
	if(cSonNoVec[PIdx_1] ==2) {bHP[InMat.N-parentIndex_1 -1]+=cE[Eidx_1 ]*dv_1 ;uHP[InMat.N-parentIndex_1 -1]-=cE[Eidx_1 ];}
if(cSonNoVec[PIdx_2] ==2) {bHP[InMat.N-parentIndex_2 -1]+=cE[Eidx_2 ]*dv_2 ;uHP[InMat.N-parentIndex_2 -1]-=cE[Eidx_2 ];}
if(cSonNoVec[PIdx_3] ==2) {bHP[InMat.N-parentIndex_3 -1]+=cE[Eidx_3 ]*dv_3 ;uHP[InMat.N-parentIndex_3 -1]-=cE[Eidx_3 ];}
if(cSonNoVec[PIdx_4] ==2) {bHP[InMat.N-parentIndex_4 -1]+=cE[Eidx_4 ]*dv_4 ;uHP[InMat.N-parentIndex_4 -1]-=cE[Eidx_4 ];}
if(cSonNoVec[PIdx_5] ==2) {bHP[InMat.N-parentIndex_5 -1]+=cE[Eidx_5 ]*dv_5 ;uHP[InMat.N-parentIndex_5 -1]-=cE[Eidx_5 ];}
if(cSonNoVec[PIdx_6] ==2) {bHP[InMat.N-parentIndex_6 -1]+=cE[Eidx_6 ]*dv_6 ;uHP[InMat.N-parentIndex_6 -1]-=cE[Eidx_6 ];}
if(cSonNoVec[PIdx_7] ==2) {bHP[InMat.N-parentIndex_7 -1]+=cE[Eidx_7 ]*dv_7 ;uHP[InMat.N-parentIndex_7 -1]-=cE[Eidx_7 ];}
if(cSonNoVec[PIdx_8] ==2) {bHP[InMat.N-parentIndex_8 -1]+=cE[Eidx_8 ]*dv_8 ;uHP[InMat.N-parentIndex_8 -1]-=cE[Eidx_8 ];}
if(cSonNoVec[PIdx_9] ==2) {bHP[InMat.N-parentIndex_9 -1]+=cE[Eidx_9 ]*dv_9 ;uHP[InMat.N-parentIndex_9 -1]-=cE[Eidx_9 ];}
if(cSonNoVec[PIdx_10] ==2) {bHP[InMat.N-parentIndex_10 -1]+=cE[Eidx_10 ]*dv_10 ;uHP[InMat.N-parentIndex_10 -1]-=cE[Eidx_10 ];}
if(cSonNoVec[PIdx_11] ==2) {bHP[InMat.N-parentIndex_11 -1]+=cE[Eidx_11 ]*dv_11 ;uHP[InMat.N-parentIndex_11 -1]-=cE[Eidx_11 ];}
if(cSonNoVec[PIdx_12] ==2) {bHP[InMat.N-parentIndex_12 -1]+=cE[Eidx_12 ]*dv_12 ;uHP[InMat.N-parentIndex_12 -1]-=cE[Eidx_12 ];}
		syncthreads();
		BeforeLU(InMat,uHP,bHP,InMat.Depth);
#ifdef BKSUB1
		BkSub(InMat, PX,PF,uHP,bHP,InMat.LognDepth);
		#define THISCOMMANDHERE36(VARILP) Vmid_ ## VARILP =PX[InMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ; 
	Vmid_1 =PX[InMat.N-PIdx_1 -1]; v_1 +=Vmid_1 ; 
Vmid_2 =PX[InMat.N-PIdx_2 -1]; v_2 +=Vmid_2 ; 
Vmid_3 =PX[InMat.N-PIdx_3 -1]; v_3 +=Vmid_3 ; 
Vmid_4 =PX[InMat.N-PIdx_4 -1]; v_4 +=Vmid_4 ; 
Vmid_5 =PX[InMat.N-PIdx_5 -1]; v_5 +=Vmid_5 ; 
Vmid_6 =PX[InMat.N-PIdx_6 -1]; v_6 +=Vmid_6 ; 
Vmid_7 =PX[InMat.N-PIdx_7 -1]; v_7 +=Vmid_7 ; 
Vmid_8 =PX[InMat.N-PIdx_8 -1]; v_8 +=Vmid_8 ; 
Vmid_9 =PX[InMat.N-PIdx_9 -1]; v_9 +=Vmid_9 ; 
Vmid_10 =PX[InMat.N-PIdx_10 -1]; v_10 +=Vmid_10 ; 
Vmid_11 =PX[InMat.N-PIdx_11 -1]; v_11 +=Vmid_11 ; 
Vmid_12 =PX[InMat.N-PIdx_12 -1]; v_12 +=Vmid_12 ; 
#endif

		
#ifdef BKSUB2
		#define THISCOMMANDHERE37(VARILP) MYFTYPE vTemp_ ## VARILP=Vs[PIdx_ ## VARILP];
	MYFTYPE vTemp_1=Vs[PIdx_1];
MYFTYPE vTemp_2=Vs[PIdx_2];
MYFTYPE vTemp_3=Vs[PIdx_3];
MYFTYPE vTemp_4=Vs[PIdx_4];
MYFTYPE vTemp_5=Vs[PIdx_5];
MYFTYPE vTemp_6=Vs[PIdx_6];
MYFTYPE vTemp_7=Vs[PIdx_7];
MYFTYPE vTemp_8=Vs[PIdx_8];
MYFTYPE vTemp_9=Vs[PIdx_9];
MYFTYPE vTemp_10=Vs[PIdx_10];
MYFTYPE vTemp_11=Vs[PIdx_11];
MYFTYPE vTemp_12=Vs[PIdx_12];
		syncthreads();
		BkSub(InMat,uHP,bHP,Vs,InMat.Depth);
		syncthreads();
		PX=Vs;

#define THISCOMMANDHERE38(VARILP) Vmid_ ## VARILP =PX[InMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ;
	Vmid_1 =PX[InMat.N-PIdx_1 -1]; v_1 +=Vmid_1 ;
Vmid_2 =PX[InMat.N-PIdx_2 -1]; v_2 +=Vmid_2 ;
Vmid_3 =PX[InMat.N-PIdx_3 -1]; v_3 +=Vmid_3 ;
Vmid_4 =PX[InMat.N-PIdx_4 -1]; v_4 +=Vmid_4 ;
Vmid_5 =PX[InMat.N-PIdx_5 -1]; v_5 +=Vmid_5 ;
Vmid_6 =PX[InMat.N-PIdx_6 -1]; v_6 +=Vmid_6 ;
Vmid_7 =PX[InMat.N-PIdx_7 -1]; v_7 +=Vmid_7 ;
Vmid_8 =PX[InMat.N-PIdx_8 -1]; v_8 +=Vmid_8 ;
Vmid_9 =PX[InMat.N-PIdx_9 -1]; v_9 +=Vmid_9 ;
Vmid_10 =PX[InMat.N-PIdx_10 -1]; v_10 +=Vmid_10 ;
Vmid_11 =PX[InMat.N-PIdx_11 -1]; v_11 +=Vmid_11 ;
Vmid_12 =PX[InMat.N-PIdx_12 -1]; v_12 +=Vmid_12 ;
		syncthreads(); 
#define THISCOMMANDHERE39(VARILP) Vs[PIdx_ ## VARILP ]= vTemp_ ## VARILP +Vmid_ ## VARILP ;
	Vs[PIdx_1 ]= vTemp_1 +Vmid_1 ;
Vs[PIdx_2 ]= vTemp_2 +Vmid_2 ;
Vs[PIdx_3 ]= vTemp_3 +Vmid_3 ;
Vs[PIdx_4 ]= vTemp_4 +Vmid_4 ;
Vs[PIdx_5 ]= vTemp_5 +Vmid_5 ;
Vs[PIdx_6 ]= vTemp_6 +Vmid_6 ;
Vs[PIdx_7 ]= vTemp_7 +Vmid_7 ;
Vs[PIdx_8 ]= vTemp_8 +Vmid_8 ;
Vs[PIdx_9 ]= vTemp_9 +Vmid_9 ;
Vs[PIdx_10 ]= vTemp_10 +Vmid_10 ;
Vs[PIdx_11 ]= vTemp_11 +Vmid_11 ;
Vs[PIdx_12 ]= vTemp_12 +Vmid_12 ;
#endif		
            t+=0.5*dt;

        

//      if(InMat.boolModel[PIdx_1 +0*InMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],p0_1 ,p1_1 ,ModelStates_1[8],ModelStates_1[9]);} if(InMat.boolModel[PIdx_1 +1*InMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(InMat.boolModel[PIdx_1 +2*InMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],p2_1 ,p3_1 ,p4_1 ,p5_1 ,ModelStates_1[8]);} if(InMat.boolModel[PIdx_1 +3*InMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],p6_1 ,p7_1 ,p8_1 ,p9_1 ,p10_1 );} if(InMat.boolModel[PIdx_1 +4*InMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],p11_1 ,p12_1 ,p13_1 ,p14_1 ,p15_1 );} if(InMat.boolModel[PIdx_1 +5*InMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],p16_1 ,p17_1 ,p18_1 ,p19_1 ,p20_1 ,p21_1 ,p22_1 ,p23_1 ,p24_1 ,p25_1 ,p26_1 ,p27_1 );}  if(InMat.boolModel[PIdx_2 +0*InMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],p0_2 ,p1_2 ,ModelStates_2[8],ModelStates_2[9]);} if(InMat.boolModel[PIdx_2 +1*InMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(InMat.boolModel[PIdx_2 +2*InMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],p2_2 ,p3_2 ,p4_2 ,p5_2 ,ModelStates_2[8]);} if(InMat.boolModel[PIdx_2 +3*InMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],p6_2 ,p7_2 ,p8_2 ,p9_2 ,p10_2 );} if(InMat.boolModel[PIdx_2 +4*InMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],p11_2 ,p12_2 ,p13_2 ,p14_2 ,p15_2 );} if(InMat.boolModel[PIdx_2 +5*InMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],p16_2 ,p17_2 ,p18_2 ,p19_2 ,p20_2 ,p21_2 ,p22_2 ,p23_2 ,p24_2 ,p25_2 ,p26_2 ,p27_2 );}  if(InMat.boolModel[PIdx_3 +0*InMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],p0_3 ,p1_3 ,ModelStates_3[8],ModelStates_3[9]);} if(InMat.boolModel[PIdx_3 +1*InMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(InMat.boolModel[PIdx_3 +2*InMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],p2_3 ,p3_3 ,p4_3 ,p5_3 ,ModelStates_3[8]);} if(InMat.boolModel[PIdx_3 +3*InMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],p6_3 ,p7_3 ,p8_3 ,p9_3 ,p10_3 );} if(InMat.boolModel[PIdx_3 +4*InMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],p11_3 ,p12_3 ,p13_3 ,p14_3 ,p15_3 );} if(InMat.boolModel[PIdx_3 +5*InMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],p16_3 ,p17_3 ,p18_3 ,p19_3 ,p20_3 ,p21_3 ,p22_3 ,p23_3 ,p24_3 ,p25_3 ,p26_3 ,p27_3 );}
	  if(cBoolModel[PIdx_1 +0*InMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],param_macro(0,PIdx_1) ,param_macro(1,PIdx_1) ,ModelStates_1[8],ModelStates_1[9]);} if(cBoolModel[PIdx_1 +1*InMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(cBoolModel[PIdx_1 +2*InMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],param_macro(2,PIdx_1) ,param_macro(3,PIdx_1) ,param_macro(4,PIdx_1) ,param_macro(5,PIdx_1) ,ModelStates_1[8]);} if(cBoolModel[PIdx_1 +3*InMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],param_macro(6,PIdx_1) ,param_macro(7,PIdx_1) ,param_macro(8,PIdx_1) ,param_macro(9,PIdx_1) ,param_macro(10,PIdx_1) );} if(cBoolModel[PIdx_1 +4*InMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],param_macro(11,PIdx_1) ,param_macro(12,PIdx_1) ,param_macro(13,PIdx_1) ,param_macro(14,PIdx_1) ,param_macro(15,PIdx_1) );} if(cBoolModel[PIdx_1 +5*InMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],param_macro(16,PIdx_1) ,param_macro(17,PIdx_1) ,param_macro(18,PIdx_1) ,param_macro(19,PIdx_1) ,param_macro(20,PIdx_1) ,param_macro(21,PIdx_1) ,param_macro(22,PIdx_1) ,param_macro(23,PIdx_1) ,param_macro(24,PIdx_1) ,param_macro(25,PIdx_1) ,param_macro(26,PIdx_1) ,param_macro(27,PIdx_1) );}  if(cBoolModel[PIdx_2 +0*InMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],param_macro(0,PIdx_2) ,param_macro(1,PIdx_2) ,ModelStates_2[8],ModelStates_2[9]);} if(cBoolModel[PIdx_2 +1*InMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(cBoolModel[PIdx_2 +2*InMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],param_macro(2,PIdx_2) ,param_macro(3,PIdx_2) ,param_macro(4,PIdx_2) ,param_macro(5,PIdx_2) ,ModelStates_2[8]);} if(cBoolModel[PIdx_2 +3*InMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],param_macro(6,PIdx_2) ,param_macro(7,PIdx_2) ,param_macro(8,PIdx_2) ,param_macro(9,PIdx_2) ,param_macro(10,PIdx_2) );} if(cBoolModel[PIdx_2 +4*InMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],param_macro(11,PIdx_2) ,param_macro(12,PIdx_2) ,param_macro(13,PIdx_2) ,param_macro(14,PIdx_2) ,param_macro(15,PIdx_2) );} if(cBoolModel[PIdx_2 +5*InMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],param_macro(16,PIdx_2) ,param_macro(17,PIdx_2) ,param_macro(18,PIdx_2) ,param_macro(19,PIdx_2) ,param_macro(20,PIdx_2) ,param_macro(21,PIdx_2) ,param_macro(22,PIdx_2) ,param_macro(23,PIdx_2) ,param_macro(24,PIdx_2) ,param_macro(25,PIdx_2) ,param_macro(26,PIdx_2) ,param_macro(27,PIdx_2) );}  if(cBoolModel[PIdx_3 +0*InMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],param_macro(0,PIdx_3) ,param_macro(1,PIdx_3) ,ModelStates_3[8],ModelStates_3[9]);} if(cBoolModel[PIdx_3 +1*InMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(cBoolModel[PIdx_3 +2*InMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],param_macro(2,PIdx_3) ,param_macro(3,PIdx_3) ,param_macro(4,PIdx_3) ,param_macro(5,PIdx_3) ,ModelStates_3[8]);} if(cBoolModel[PIdx_3 +3*InMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],param_macro(6,PIdx_3) ,param_macro(7,PIdx_3) ,param_macro(8,PIdx_3) ,param_macro(9,PIdx_3) ,param_macro(10,PIdx_3) );} if(cBoolModel[PIdx_3 +4*InMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],param_macro(11,PIdx_3) ,param_macro(12,PIdx_3) ,param_macro(13,PIdx_3) ,param_macro(14,PIdx_3) ,param_macro(15,PIdx_3) );} if(cBoolModel[PIdx_3 +5*InMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],param_macro(16,PIdx_3) ,param_macro(17,PIdx_3) ,param_macro(18,PIdx_3) ,param_macro(19,PIdx_3) ,param_macro(20,PIdx_3) ,param_macro(21,PIdx_3) ,param_macro(22,PIdx_3) ,param_macro(23,PIdx_3) ,param_macro(24,PIdx_3) ,param_macro(25,PIdx_3) ,param_macro(26,PIdx_3) ,param_macro(27,PIdx_3) );}  if(cBoolModel[PIdx_4 +0*InMat.N]){CuDerivModel_ca(dt, v_4,ModelStates_4[0],ModelStates_4[1],param_macro(0,PIdx_4) ,param_macro(1,PIdx_4) ,ModelStates_4[8],ModelStates_4[9]);} if(cBoolModel[PIdx_4 +1*InMat.N]){CuDerivModel_cad(dt, v_4,ModelStates_4[2],ModelStates_4[9],ModelStates_4[8]);} if(cBoolModel[PIdx_4 +2*InMat.N]){CuDerivModel_kca(dt, v_4,ModelStates_4[3],param_macro(2,PIdx_4) ,param_macro(3,PIdx_4) ,param_macro(4,PIdx_4) ,param_macro(5,PIdx_4) ,ModelStates_4[8]);} if(cBoolModel[PIdx_4 +3*InMat.N]){CuDerivModel_km(dt, v_4,ModelStates_4[4],param_macro(6,PIdx_4) ,param_macro(7,PIdx_4) ,param_macro(8,PIdx_4) ,param_macro(9,PIdx_4) ,param_macro(10,PIdx_4) );} if(cBoolModel[PIdx_4 +4*InMat.N]){CuDerivModel_kv(dt, v_4,ModelStates_4[5],param_macro(11,PIdx_4) ,param_macro(12,PIdx_4) ,param_macro(13,PIdx_4) ,param_macro(14,PIdx_4) ,param_macro(15,PIdx_4) );} if(cBoolModel[PIdx_4 +5*InMat.N]){CuDerivModel_na(dt, v_4,ModelStates_4[6],ModelStates_4[7],param_macro(16,PIdx_4) ,param_macro(17,PIdx_4) ,param_macro(18,PIdx_4) ,param_macro(19,PIdx_4) ,param_macro(20,PIdx_4) ,param_macro(21,PIdx_4) ,param_macro(22,PIdx_4) ,param_macro(23,PIdx_4) ,param_macro(24,PIdx_4) ,param_macro(25,PIdx_4) ,param_macro(26,PIdx_4) ,param_macro(27,PIdx_4) );}  if(cBoolModel[PIdx_5 +0*InMat.N]){CuDerivModel_ca(dt, v_5,ModelStates_5[0],ModelStates_5[1],param_macro(0,PIdx_5) ,param_macro(1,PIdx_5) ,ModelStates_5[8],ModelStates_5[9]);} if(cBoolModel[PIdx_5 +1*InMat.N]){CuDerivModel_cad(dt, v_5,ModelStates_5[2],ModelStates_5[9],ModelStates_5[8]);} if(cBoolModel[PIdx_5 +2*InMat.N]){CuDerivModel_kca(dt, v_5,ModelStates_5[3],param_macro(2,PIdx_5) ,param_macro(3,PIdx_5) ,param_macro(4,PIdx_5) ,param_macro(5,PIdx_5) ,ModelStates_5[8]);} if(cBoolModel[PIdx_5 +3*InMat.N]){CuDerivModel_km(dt, v_5,ModelStates_5[4],param_macro(6,PIdx_5) ,param_macro(7,PIdx_5) ,param_macro(8,PIdx_5) ,param_macro(9,PIdx_5) ,param_macro(10,PIdx_5) );} if(cBoolModel[PIdx_5 +4*InMat.N]){CuDerivModel_kv(dt, v_5,ModelStates_5[5],param_macro(11,PIdx_5) ,param_macro(12,PIdx_5) ,param_macro(13,PIdx_5) ,param_macro(14,PIdx_5) ,param_macro(15,PIdx_5) );} if(cBoolModel[PIdx_5 +5*InMat.N]){CuDerivModel_na(dt, v_5,ModelStates_5[6],ModelStates_5[7],param_macro(16,PIdx_5) ,param_macro(17,PIdx_5) ,param_macro(18,PIdx_5) ,param_macro(19,PIdx_5) ,param_macro(20,PIdx_5) ,param_macro(21,PIdx_5) ,param_macro(22,PIdx_5) ,param_macro(23,PIdx_5) ,param_macro(24,PIdx_5) ,param_macro(25,PIdx_5) ,param_macro(26,PIdx_5) ,param_macro(27,PIdx_5) );}  if(cBoolModel[PIdx_6 +0*InMat.N]){CuDerivModel_ca(dt, v_6,ModelStates_6[0],ModelStates_6[1],param_macro(0,PIdx_6) ,param_macro(1,PIdx_6) ,ModelStates_6[8],ModelStates_6[9]);} if(cBoolModel[PIdx_6 +1*InMat.N]){CuDerivModel_cad(dt, v_6,ModelStates_6[2],ModelStates_6[9],ModelStates_6[8]);} if(cBoolModel[PIdx_6 +2*InMat.N]){CuDerivModel_kca(dt, v_6,ModelStates_6[3],param_macro(2,PIdx_6) ,param_macro(3,PIdx_6) ,param_macro(4,PIdx_6) ,param_macro(5,PIdx_6) ,ModelStates_6[8]);} if(cBoolModel[PIdx_6 +3*InMat.N]){CuDerivModel_km(dt, v_6,ModelStates_6[4],param_macro(6,PIdx_6) ,param_macro(7,PIdx_6) ,param_macro(8,PIdx_6) ,param_macro(9,PIdx_6) ,param_macro(10,PIdx_6) );} if(cBoolModel[PIdx_6 +4*InMat.N]){CuDerivModel_kv(dt, v_6,ModelStates_6[5],param_macro(11,PIdx_6) ,param_macro(12,PIdx_6) ,param_macro(13,PIdx_6) ,param_macro(14,PIdx_6) ,param_macro(15,PIdx_6) );} if(cBoolModel[PIdx_6 +5*InMat.N]){CuDerivModel_na(dt, v_6,ModelStates_6[6],ModelStates_6[7],param_macro(16,PIdx_6) ,param_macro(17,PIdx_6) ,param_macro(18,PIdx_6) ,param_macro(19,PIdx_6) ,param_macro(20,PIdx_6) ,param_macro(21,PIdx_6) ,param_macro(22,PIdx_6) ,param_macro(23,PIdx_6) ,param_macro(24,PIdx_6) ,param_macro(25,PIdx_6) ,param_macro(26,PIdx_6) ,param_macro(27,PIdx_6) );}  if(cBoolModel[PIdx_7 +0*InMat.N]){CuDerivModel_ca(dt, v_7,ModelStates_7[0],ModelStates_7[1],param_macro(0,PIdx_7) ,param_macro(1,PIdx_7) ,ModelStates_7[8],ModelStates_7[9]);} if(cBoolModel[PIdx_7 +1*InMat.N]){CuDerivModel_cad(dt, v_7,ModelStates_7[2],ModelStates_7[9],ModelStates_7[8]);} if(cBoolModel[PIdx_7 +2*InMat.N]){CuDerivModel_kca(dt, v_7,ModelStates_7[3],param_macro(2,PIdx_7) ,param_macro(3,PIdx_7) ,param_macro(4,PIdx_7) ,param_macro(5,PIdx_7) ,ModelStates_7[8]);} if(cBoolModel[PIdx_7 +3*InMat.N]){CuDerivModel_km(dt, v_7,ModelStates_7[4],param_macro(6,PIdx_7) ,param_macro(7,PIdx_7) ,param_macro(8,PIdx_7) ,param_macro(9,PIdx_7) ,param_macro(10,PIdx_7) );} if(cBoolModel[PIdx_7 +4*InMat.N]){CuDerivModel_kv(dt, v_7,ModelStates_7[5],param_macro(11,PIdx_7) ,param_macro(12,PIdx_7) ,param_macro(13,PIdx_7) ,param_macro(14,PIdx_7) ,param_macro(15,PIdx_7) );} if(cBoolModel[PIdx_7 +5*InMat.N]){CuDerivModel_na(dt, v_7,ModelStates_7[6],ModelStates_7[7],param_macro(16,PIdx_7) ,param_macro(17,PIdx_7) ,param_macro(18,PIdx_7) ,param_macro(19,PIdx_7) ,param_macro(20,PIdx_7) ,param_macro(21,PIdx_7) ,param_macro(22,PIdx_7) ,param_macro(23,PIdx_7) ,param_macro(24,PIdx_7) ,param_macro(25,PIdx_7) ,param_macro(26,PIdx_7) ,param_macro(27,PIdx_7) );}  if(cBoolModel[PIdx_8 +0*InMat.N]){CuDerivModel_ca(dt, v_8,ModelStates_8[0],ModelStates_8[1],param_macro(0,PIdx_8) ,param_macro(1,PIdx_8) ,ModelStates_8[8],ModelStates_8[9]);} if(cBoolModel[PIdx_8 +1*InMat.N]){CuDerivModel_cad(dt, v_8,ModelStates_8[2],ModelStates_8[9],ModelStates_8[8]);} if(cBoolModel[PIdx_8 +2*InMat.N]){CuDerivModel_kca(dt, v_8,ModelStates_8[3],param_macro(2,PIdx_8) ,param_macro(3,PIdx_8) ,param_macro(4,PIdx_8) ,param_macro(5,PIdx_8) ,ModelStates_8[8]);} if(cBoolModel[PIdx_8 +3*InMat.N]){CuDerivModel_km(dt, v_8,ModelStates_8[4],param_macro(6,PIdx_8) ,param_macro(7,PIdx_8) ,param_macro(8,PIdx_8) ,param_macro(9,PIdx_8) ,param_macro(10,PIdx_8) );} if(cBoolModel[PIdx_8 +4*InMat.N]){CuDerivModel_kv(dt, v_8,ModelStates_8[5],param_macro(11,PIdx_8) ,param_macro(12,PIdx_8) ,param_macro(13,PIdx_8) ,param_macro(14,PIdx_8) ,param_macro(15,PIdx_8) );} if(cBoolModel[PIdx_8 +5*InMat.N]){CuDerivModel_na(dt, v_8,ModelStates_8[6],ModelStates_8[7],param_macro(16,PIdx_8) ,param_macro(17,PIdx_8) ,param_macro(18,PIdx_8) ,param_macro(19,PIdx_8) ,param_macro(20,PIdx_8) ,param_macro(21,PIdx_8) ,param_macro(22,PIdx_8) ,param_macro(23,PIdx_8) ,param_macro(24,PIdx_8) ,param_macro(25,PIdx_8) ,param_macro(26,PIdx_8) ,param_macro(27,PIdx_8) );}  if(cBoolModel[PIdx_9 +0*InMat.N]){CuDerivModel_ca(dt, v_9,ModelStates_9[0],ModelStates_9[1],param_macro(0,PIdx_9) ,param_macro(1,PIdx_9) ,ModelStates_9[8],ModelStates_9[9]);} if(cBoolModel[PIdx_9 +1*InMat.N]){CuDerivModel_cad(dt, v_9,ModelStates_9[2],ModelStates_9[9],ModelStates_9[8]);} if(cBoolModel[PIdx_9 +2*InMat.N]){CuDerivModel_kca(dt, v_9,ModelStates_9[3],param_macro(2,PIdx_9) ,param_macro(3,PIdx_9) ,param_macro(4,PIdx_9) ,param_macro(5,PIdx_9) ,ModelStates_9[8]);} if(cBoolModel[PIdx_9 +3*InMat.N]){CuDerivModel_km(dt, v_9,ModelStates_9[4],param_macro(6,PIdx_9) ,param_macro(7,PIdx_9) ,param_macro(8,PIdx_9) ,param_macro(9,PIdx_9) ,param_macro(10,PIdx_9) );} if(cBoolModel[PIdx_9 +4*InMat.N]){CuDerivModel_kv(dt, v_9,ModelStates_9[5],param_macro(11,PIdx_9) ,param_macro(12,PIdx_9) ,param_macro(13,PIdx_9) ,param_macro(14,PIdx_9) ,param_macro(15,PIdx_9) );} if(cBoolModel[PIdx_9 +5*InMat.N]){CuDerivModel_na(dt, v_9,ModelStates_9[6],ModelStates_9[7],param_macro(16,PIdx_9) ,param_macro(17,PIdx_9) ,param_macro(18,PIdx_9) ,param_macro(19,PIdx_9) ,param_macro(20,PIdx_9) ,param_macro(21,PIdx_9) ,param_macro(22,PIdx_9) ,param_macro(23,PIdx_9) ,param_macro(24,PIdx_9) ,param_macro(25,PIdx_9) ,param_macro(26,PIdx_9) ,param_macro(27,PIdx_9) );}  if(cBoolModel[PIdx_10 +0*InMat.N]){CuDerivModel_ca(dt, v_10,ModelStates_10[0],ModelStates_10[1],param_macro(0,PIdx_10) ,param_macro(1,PIdx_10) ,ModelStates_10[8],ModelStates_10[9]);} if(cBoolModel[PIdx_10 +1*InMat.N]){CuDerivModel_cad(dt, v_10,ModelStates_10[2],ModelStates_10[9],ModelStates_10[8]);} if(cBoolModel[PIdx_10 +2*InMat.N]){CuDerivModel_kca(dt, v_10,ModelStates_10[3],param_macro(2,PIdx_10) ,param_macro(3,PIdx_10) ,param_macro(4,PIdx_10) ,param_macro(5,PIdx_10) ,ModelStates_10[8]);} if(cBoolModel[PIdx_10 +3*InMat.N]){CuDerivModel_km(dt, v_10,ModelStates_10[4],param_macro(6,PIdx_10) ,param_macro(7,PIdx_10) ,param_macro(8,PIdx_10) ,param_macro(9,PIdx_10) ,param_macro(10,PIdx_10) );} if(cBoolModel[PIdx_10 +4*InMat.N]){CuDerivModel_kv(dt, v_10,ModelStates_10[5],param_macro(11,PIdx_10) ,param_macro(12,PIdx_10) ,param_macro(13,PIdx_10) ,param_macro(14,PIdx_10) ,param_macro(15,PIdx_10) );} if(cBoolModel[PIdx_10 +5*InMat.N]){CuDerivModel_na(dt, v_10,ModelStates_10[6],ModelStates_10[7],param_macro(16,PIdx_10) ,param_macro(17,PIdx_10) ,param_macro(18,PIdx_10) ,param_macro(19,PIdx_10) ,param_macro(20,PIdx_10) ,param_macro(21,PIdx_10) ,param_macro(22,PIdx_10) ,param_macro(23,PIdx_10) ,param_macro(24,PIdx_10) ,param_macro(25,PIdx_10) ,param_macro(26,PIdx_10) ,param_macro(27,PIdx_10) );}  if(cBoolModel[PIdx_11 +0*InMat.N]){CuDerivModel_ca(dt, v_11,ModelStates_11[0],ModelStates_11[1],param_macro(0,PIdx_11) ,param_macro(1,PIdx_11) ,ModelStates_11[8],ModelStates_11[9]);} if(cBoolModel[PIdx_11 +1*InMat.N]){CuDerivModel_cad(dt, v_11,ModelStates_11[2],ModelStates_11[9],ModelStates_11[8]);} if(cBoolModel[PIdx_11 +2*InMat.N]){CuDerivModel_kca(dt, v_11,ModelStates_11[3],param_macro(2,PIdx_11) ,param_macro(3,PIdx_11) ,param_macro(4,PIdx_11) ,param_macro(5,PIdx_11) ,ModelStates_11[8]);} if(cBoolModel[PIdx_11 +3*InMat.N]){CuDerivModel_km(dt, v_11,ModelStates_11[4],param_macro(6,PIdx_11) ,param_macro(7,PIdx_11) ,param_macro(8,PIdx_11) ,param_macro(9,PIdx_11) ,param_macro(10,PIdx_11) );} if(cBoolModel[PIdx_11 +4*InMat.N]){CuDerivModel_kv(dt, v_11,ModelStates_11[5],param_macro(11,PIdx_11) ,param_macro(12,PIdx_11) ,param_macro(13,PIdx_11) ,param_macro(14,PIdx_11) ,param_macro(15,PIdx_11) );} if(cBoolModel[PIdx_11 +5*InMat.N]){CuDerivModel_na(dt, v_11,ModelStates_11[6],ModelStates_11[7],param_macro(16,PIdx_11) ,param_macro(17,PIdx_11) ,param_macro(18,PIdx_11) ,param_macro(19,PIdx_11) ,param_macro(20,PIdx_11) ,param_macro(21,PIdx_11) ,param_macro(22,PIdx_11) ,param_macro(23,PIdx_11) ,param_macro(24,PIdx_11) ,param_macro(25,PIdx_11) ,param_macro(26,PIdx_11) ,param_macro(27,PIdx_11) );}  if(cBoolModel[PIdx_12 +0*InMat.N]){CuDerivModel_ca(dt, v_12,ModelStates_12[0],ModelStates_12[1],param_macro(0,PIdx_12) ,param_macro(1,PIdx_12) ,ModelStates_12[8],ModelStates_12[9]);} if(cBoolModel[PIdx_12 +1*InMat.N]){CuDerivModel_cad(dt, v_12,ModelStates_12[2],ModelStates_12[9],ModelStates_12[8]);} if(cBoolModel[PIdx_12 +2*InMat.N]){CuDerivModel_kca(dt, v_12,ModelStates_12[3],param_macro(2,PIdx_12) ,param_macro(3,PIdx_12) ,param_macro(4,PIdx_12) ,param_macro(5,PIdx_12) ,ModelStates_12[8]);} if(cBoolModel[PIdx_12 +3*InMat.N]){CuDerivModel_km(dt, v_12,ModelStates_12[4],param_macro(6,PIdx_12) ,param_macro(7,PIdx_12) ,param_macro(8,PIdx_12) ,param_macro(9,PIdx_12) ,param_macro(10,PIdx_12) );} if(cBoolModel[PIdx_12 +4*InMat.N]){CuDerivModel_kv(dt, v_12,ModelStates_12[5],param_macro(11,PIdx_12) ,param_macro(12,PIdx_12) ,param_macro(13,PIdx_12) ,param_macro(14,PIdx_12) ,param_macro(15,PIdx_12) );} if(cBoolModel[PIdx_12 +5*InMat.N]){CuDerivModel_na(dt, v_12,ModelStates_12[6],ModelStates_12[7],param_macro(16,PIdx_12) ,param_macro(17,PIdx_12) ,param_macro(18,PIdx_12) ,param_macro(19,PIdx_12) ,param_macro(20,PIdx_12) ,param_macro(21,PIdx_12) ,param_macro(22,PIdx_12) ,param_macro(23,PIdx_12) ,param_macro(24,PIdx_12) ,param_macro(25,PIdx_12) ,param_macro(26,PIdx_12) ,param_macro(27,PIdx_12) );}

 
 
    }
	//This one looks suspicious but leaving it and will check it later.
   for(int recInd =0;recInd<sim.NRecSites;recInd++){
		VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) +threadIdx.y*Nt*sim.NRecSites+recInd*Nt + Nt-WARPSIZE+PIdx_1]=SMemVHot[WARPSIZE*recInd + PIdx_1];
	}
 
}
__global__ void NeuroGPUKernel(Stim stim, MYFTYPE* ParamsM, Sim sim, HMat InMat, MYFTYPE *V,MYFTYPE* VHotGlobal,MYDTYPE CompDepth,MYDTYPE CompFDepth)
{

	MYFTYPE *amps,*SMemVHot;
	MYDTYPE offset=0;
	
	
	
	runSimulation(InMat,ParamsM,V,stim,sim,VHotGlobal,SMemVHot,amps,offset);

}
void ReadParamsMatX(const char* FN,MYFTYPE* ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}
MYFTYPE* ReadAllParams(const char* FN,MYDTYPE NParams,MYDTYPE Nx,MYDTYPE  &ntemp) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	//sprintf(FileName,"%sForC.mat",FN);
	MYFTYPE* ans;
	MYDTYPE tmp;
	FILE *fl = fopen(FN, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return nullptr;
	}
	fread(&tmp, sizeof(MYDTYPE),1, fl);
		ntemp = tmp;

	ans=(MYFTYPE *)malloc(Nx * NPARAMS * tmp * sizeof(MYFTYPE ));
	fread(ans, sizeof(MYFTYPE), tmp*Nx*NParams, fl);
	fclose(fl);
	return ans;
}
void ReadParamsMat(const char* FN,MYFTYPE** ParamsM,MYDTYPE NParams,MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName,"%sForC.mat",FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	for(int i=0;i<NParams;i++){
		ParamsM[i]=(MYFTYPE*)malloc(Nx * sizeof(MYFTYPE));
		fread(ParamsM[i], sizeof(MYFTYPE), Nx, fl);
	}
	fclose(fl);
}


hipError_t stEfork2TimeLoopGPU(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) { 

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipDeviceReset();
	MYFTYPE *VHotsGlobal,*VHotsHost;
	MYDTYPE Nt=stim.Nt;
	hipHostMalloc((void**)&VHotsHost,stim.NStimuli*Nt*sim.NRecSites*NSets*sizeof(MYFTYPE));
	
	int i,j,t;
	// For matrix -
	MYFTYPE *PXOut_d,*PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	HMat Mat_d;
	Mat_d.N=InMat.N;
	Mat_d.NComps=InMat.NComps;
	Mat_d.Depth=InMat.Depth;
	Mat_d.NModels=InMat.NModels;
	Mat_d.LognDepth=InMat.LognDepth;
	Mat_d.nFathers=InMat.nFathers;
	Mat_d.nCallForFather=InMat.nCallForFather;
	Mat_d.nLRel=InMat.nLRel;
	Mat_d.nFLRel=InMat.nFLRel;
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	stim_d.Nt = stim.Nt;
	Sim sim_d; 	 			
	sim_d.Celsius = sim.Celsius; 	 	
	sim_d.dt = sim.dt; 	 	
	sim_d.NRecSites = sim.NRecSites; 	 	
	sim_d.TFinal = sim.TFinal; 
#ifndef STIMFROMFILE
    cudaStatus = hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE

    stim_d.numofdts = stim.numofdts;
    cudaStatus = hipMalloc((void**)&stim_d.dtInds, stim_d.numofdts * sizeof(MYDTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.durs, stim_d.numofdts * sizeof(MYFTYPE));
    cudaStatus = hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE));
#endif // STIMFROMFILE
	cudaStatus = hipMalloc((void**)&sim_d.RecSites, sim_d.NRecSites * sizeof(MYDTYPE));
	


	//cudaStatus = hipMalloc((void**)&Mat_d.e, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.f, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Ks, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.boolModel, InMat.N * InMat.NModels* sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.Cms, InMat.N * sizeof(MYFTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SonNoVec, InMat.N* sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	//cudaStatus = hipMalloc((void**)&Mat_d.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelEnds, (InMat.nFathers) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.RelVec, InMat.nCallForFather * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMalloc((void**)&Mat_d.SegEndI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*Nt *stim.NStimuli* sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHotsGlobal, NSets*sim_d.NRecSites*Nt *stim.NStimuli* sizeof(MYFTYPE));
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N +1)* sizeof(MYDTYPE));
#endif
	/*cudaStatus = hipMalloc((void**)&Mat_d.CompByLevel32, (CompDepth)*WARPSIZE* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.CompByFLevel32, (CompFDepth)*WARPSIZE* sizeof(MYDTYPE));

	cudaStatus = hipMalloc((void**)&Mat_d.LRelStarts, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.LRelEnds, Mat_d.nLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelStarts, Mat_d.nFLRel* sizeof(MYDTYPE));
	cudaStatus = hipMalloc((void**)&Mat_d.FLRelEnds, Mat_d.nFLRel* sizeof(MYDTYPE));
*/
	

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}

	#ifndef STIMFROMFILE
 
    cudaStatus = hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // !STIMFROMFILE
#ifdef STIMFROMFILE
	
    cudaStatus = hipMemcpy(stim_d.dtInds, stim.dtInds, stim.numofdts * sizeof(MYDTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.durs, stim.durs, stim.numofdts * sizeof(MYFTYPE), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // STIMFROMFILE
	cudaStatus = hipMemcpy(sim_d.RecSites, sim.RecSites, sim_d.NRecSites * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMalloc((void**)&Mat_d.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.e, InMat.e, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cE), InMat.e, InMat.N * sizeof(MYSECONDFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.f, InMat.f, InMat.N * sizeof(MYSECONDFTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cF), InMat.f, InMat.N * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cKs), InMat.Ks, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Ks, InMat.Ks, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(Mat_d.SegToComp, InMat.SegToComp, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegToComp), InMat.SegToComp, InMat.N * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.boolModel, InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cBoolModel), InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE));

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCm), InMat.Cms, InMat.N * sizeof(MYFTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SonNoVec, InMat.SonNoVec, InMat.N * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSonNoVec), InMat.SonNoVec, InMat.N * sizeof(MYDTYPE));
	
	
	//cudaStatus = hipMemcpy(Mat_d.RelStarts, InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelStarts), InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelEnds, InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelEnds), InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.RelVec, InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cRelVec), InMat.RelVec, InMat.nCallForFather* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegStartI, InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegStartI), InMat.SegStartI, (InMat.nCallForFather+1) * sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cSegEndI), InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFathers), InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE));
	
	
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFIdxs), InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMemcpy(Mat_d.KsB, InMat.KsB,(InMat.N+1)*sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif

	//cudaStatus = hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByLevel32), InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCompByFLevel32), InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE));


	//cudaStatus = hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelStarts), InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cLRelEnds), InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelStarts), InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE));
	//cudaStatus = hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cFLRelEnds), InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE));


	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	cudaStatus = hipMalloc((void**)&PXOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipMalloc((void**)&PFOut_d, (InMat.N+1) * sizeof(MYSECONDFTYPE));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPU!\n", cudaStatus);
		return cudaStatus;
	}

	// A bit more
	/*MYFTYPE *d_d,*Iapp_d,*VHots_d,*dOrig_d,*V_d;

	cudaStatus = hipMalloc((void**)&d_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&VHots_d, sim.Nt*NSTIMULI * sizeof(MYFTYPE*));
	cudaStatus = hipMalloc((void**)&dOrig_d, InMat.N * sizeof(MYFTYPE));
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}*/
	MYFTYPE *V_d;
	cudaStatus = hipMalloc((void**)&V_d, InMat.N * sizeof(MYFTYPE));

	//RRR  For model
	// allocation
	MYFTYPE *d_modelParams;
	cudaStatus = hipMalloc((void**)&d_modelParams, NPARAMS * InMat.NComps *NSets * sizeof(MYFTYPE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;  
	}
	// copying
	
	cudaStatus = hipMemcpy(d_modelParams, ParamsM, NPARAMS * InMat.NComps * NSets *sizeof(MYFTYPE), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(V_d, V, InMat.N * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyToDevice failed!");
		return cudaStatus;  
	}

	dim3 blockDim(WARPSIZE,stim.NStimuli);
	dim3 gridDim(NSets);

	MYDTYPE offset=0;
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE);
	// 1 - e,f,Cms
#ifdef BKSUB1
	//offset+=InMat.N*InMat.LognDepth*sizeof(MYDTYPE);
#endif
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*sizeof(MYDTYPE);
	//offset+=InMat.N*InMat.NModels*sizeof(MYDTYPE);
	// 2 FIdxs, Ks, SegToComp,boolModel
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelStarts
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // RelEnds
	//offset+=InMat.nFathers*sizeof(MYDTYPE); // Fathers
	//offset+=InMat.nCallForFather*sizeof(MYDTYPE); // RelVec
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegStartI
	//offset+=(InMat.nCallForFather+1)*sizeof(MYDTYPE); // SegEndI
	
	// 3 RelStarts RelEnds RelVec SegStartI SegEndI Fathers
	//offset+=WARPSIZE*(CompDepth)*sizeof(MYDTYPE);
	//offset+=WARPSIZE*(CompFDepth)*sizeof(MYDTYPE);
	//// 4 CompByLevel32 CompByFLevel32
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
	//offset+=Mat_d.nFLRel*sizeof(MYDTYPE);
#ifdef BKSUB2
	offset+=(Mat_d.N+1)*sizeof(MYDTYPE);
#endif
	// 5 LRelStarts LRelEnds FLRelStarts  FLRelEnds KsB
	//offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	MYDTYPE FrameworkMemSize=offset;
//PerStimulus
	printf("offset is%d\n",offset);

	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE);  // uHP (Diag)
	offset+=(InMat.N+2)*sizeof(MYSECONDFTYPE); // bHP (rhs)
	offset=ceilf(MYSECONDFTYPE(offset)/sizeof(MYSECONDFTYPE))*sizeof(MYSECONDFTYPE);
	//offset+=InMat.N*sizeof(MYFTYPE); // Vs

	offset+=((WARPSIZE)*sim.NRecSites)*sizeof(MYFTYPE);//SMemVHot
#ifdef STIMFROMFILE
    offset+=WARPSIZE*sizeof(MYFTYPE);//amps
#endif // STIMFROMFILE
	MYDTYPE PerStimulus=offset-FrameworkMemSize;
	
	MYDTYPE TotalSMem=FrameworkMemSize+(PerStimulus)*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n",TotalSMem,FrameworkMemSize,PerStimulus,stim.NStimuli);
		hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
		MYDTYPE currKernelRun = NKERNEL;
	MYDTYPE prevRuns = 0;
	int memSizeForVHotGlobal = Nt*stim.NStimuli*sim.NRecSites;
	MYDTYPE memSizeForModelParams = NPARAMS * InMat.NComps;
	MYDTYPE streamID=0;
	hipStream_t stream0,stream1,stream2,stream3;
	hipStreamCreate (&stream0);
	hipStreamCreate (&stream1);
	hipStreamCreate (&stream2);
	hipStreamCreate (&stream3);
	hipEventRecord(start, 0);
	while (prevRuns<NSets){
	if(currKernelRun>(NSets-prevRuns)){
		currKernelRun = (NSets-prevRuns);
	}
	if(streamID==0){
		NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream0>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR	

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream0); 
		
	}
	if(streamID==1){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream1>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR

	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream1); 

	}
	if(streamID==2){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream2>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt *sim.NRecSites *  stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream2); 

	}
	if(streamID==3){
	NeuroGPUKernel<<<currKernelRun, blockDim,TotalSMem,stream3>>>(stim_d, &d_modelParams[prevRuns*memSizeForModelParams], sim_d, Mat_d, V_d,&VHotsGlobal[prevRuns*memSizeForVHotGlobal],CompDepth,CompFDepth); // RRR
	
	cudaStatus = hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal],currKernelRun * Nt * sim.NRecSites * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost,stream3); 

	}
	prevRuns +=currKernelRun;
	if(streamID==3){
		streamID == 0;
	}	else {
		streamID +=1;
	}
	}
	//hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time required : %f", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching stEfork2TimeLoopGPUKernel!\n", cudaStatus);
		return cudaStatus;
	}
	//cudaStatus = hipMemcpy(VHotsHost, VHotsGlobal,NSets * Nt * stim.NStimuli* sizeof(MYFTYPE), hipMemcpyDeviceToHost); 
	printf("stEfork2TimeLoopGPU took %f milliseconds\n",elapsedTime);
	FILE *file = fopen(TIMES_FN, "wb");
	if ( file ) {
		MYDTYPE mul32 = MUL32;
		fwrite(&mul32,sizeof(MYDTYPE),1,file);
		fwrite(&elapsedTime,sizeof(float),1,file);
	} else {
		printf("ERR SaveArrayToFile %s\n",TIMES_FN);
	}
	fclose(file);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		return cudaStatus;  
	}
	SaveArrayToFile(VHOT_OUT_FN_P,NSets*Nt*stim.NStimuli*sim.NRecSites,VHotsHost);
	
	return cudaStatus;
}


hipError_t stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, HMat& InMat, MYFTYPE* V,MYDTYPE CompDepth,MYDTYPE CompFDepth,MYDTYPE NSets) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
	  stEfork2TimeLoopGPU(stim, sim, ParamsM, InMat, V,CompDepth,CompFDepth,NSets); //RRR sim
	  
	
	


	return cudaStatus;
}
